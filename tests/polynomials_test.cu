#include "hip/hip_runtime.h"
#include <catch2/catch.hpp>
#include <iostream>
#include <cmath>
#include <array>
#include "entities/NDG_t.cuh"
#include "polynomials/ChebyshevPolynomial_t.cuh"
#include "polynomials/LegendrePolynomial_t.cuh"
#include "helpers/float_types.h"

constexpr double pi = 3.14159265358979323846;

TEST_CASE("ChebyshevPolynomials", "Checks the Chebyshev polynomials"){
    const int N_max = 16;
    const int N_test = 16;
    const size_t n_interpolation_points = N_max * 8;
    const size_t offset_1D = N_test * (N_test + 1) /2;
    const size_t offset_2D = N_test * (N_test + 1) * (2 * N_test + 1) /6;
    const double error = 1e-6;

    REQUIRE(N_test <= N_max);

    hipStream_t stream;
    hipStreamCreate(&stream); 
    
    SEM::Device::Entities::NDG_t<SEM::Device::Polynomials::ChebyshevPolynomial_t> NDG(N_max, n_interpolation_points, stream);

    std::vector<deviceFloat> host_nodes(NDG.vector_length_);
    std::vector<deviceFloat> host_weights(NDG.vector_length_);
    std::vector<deviceFloat> host_barycentric_weights(NDG.vector_length_);
    std::vector<deviceFloat> host_lagrange_interpolant_left(NDG.vector_length_);
    std::vector<deviceFloat> host_lagrange_interpolant_right(NDG.vector_length_);
    std::vector<deviceFloat> host_lagrange_interpolant_derivative_left(NDG.vector_length_);
    std::vector<deviceFloat> host_lagrange_interpolant_derivative_right(NDG.vector_length_);
    std::vector<deviceFloat> host_derivative_matrices(NDG.matrix_length_);
    std::vector<deviceFloat> host_g_hat_derivative_matrices(NDG.matrix_length_);
    std::vector<deviceFloat> host_derivative_matrices_hat(NDG.matrix_length_);
    std::vector<deviceFloat> host_interpolation_matrices(NDG.interpolation_length_);

    NDG.nodes_.copy_to(host_nodes, stream);
    NDG.weights_.copy_to(host_weights, stream);
    NDG.barycentric_weights_.copy_to(host_barycentric_weights, stream);
    NDG.lagrange_interpolant_left_.copy_to(host_lagrange_interpolant_left, stream);
    NDG.lagrange_interpolant_right_.copy_to(host_lagrange_interpolant_right, stream);
    NDG.lagrange_interpolant_derivative_left_.copy_to(host_lagrange_interpolant_derivative_left, stream);
    NDG.lagrange_interpolant_derivative_right_.copy_to(host_lagrange_interpolant_derivative_right, stream);
    NDG.derivative_matrices_.copy_to(host_derivative_matrices, stream);
    NDG.g_hat_derivative_matrices_.copy_to(host_g_hat_derivative_matrices, stream);
    NDG.derivative_matrices_hat_.copy_to(host_derivative_matrices_hat, stream);
    NDG.interpolation_matrices_.copy_to(host_interpolation_matrices, stream);
    hipStreamSynchronize(stream);

    const std::array<double, N_test+1> nodes {-0.9957341762950345218712,
                                                -0.9618256431728190704088,
                                                -0.895163291355062322067,
                                                -0.7980172272802395033328,
                                                -0.673695643646557211713,
                                                -0.5264321628773558002446,
                                                -0.3612416661871529487447,
                                                -0.1837495178165703315744,
                                                0,
                                                0.183749517816570331574,
                                                0.3612416661871529487447,
                                                0.5264321628773558002446,
                                                0.6736956436465572117127,
                                                0.7980172272802395033328,
                                                0.895163291355062322067,
                                                0.9618256431728190704088,
                                                0.9957341762950345218712};

    for (int i = 0; i <= N_test; ++i) {
        REQUIRE(std::abs(nodes[i] - host_nodes[offset_1D + i]) < error);
    }

    const std::array<double, N_test+1> weights {0.1847995678582231316743,
                                                0.1847995678582231316743,
                                                0.1847995678582231316743,
                                                0.1847995678582231316743,
                                                0.1847995678582231316743,
                                                0.1847995678582231316743,
                                                0.1847995678582231316743,
                                                0.1847995678582231316743,
                                                0.1847995678582231316743,
                                                0.1847995678582231316743,
                                                0.1847995678582231316743,
                                                0.1847995678582231316743,
                                                0.1847995678582231316743,
                                                0.1847995678582231316743,
                                                0.1847995678582231316743,
                                                0.1847995678582231316743,
                                                0.1847995678582231316743};
        
    for (int i = 0; i <= N_test; ++i) {
        REQUIRE(std::abs(weights[i] - host_weights[offset_1D + i]) < error);
    }

    std::array<double, N_test+1> phi;
    for (int i = 0; i <= N_test; ++i) {
        phi[i] = std::sin(pi * host_nodes[offset_1D + i]);
    }

    std::array<double, N_test+1> phi_prime_expected;
    for (int i = 0; i <= N_test; ++i) {
        phi_prime_expected[i] = pi * std::cos(pi * host_nodes[offset_1D + i]);
    }

    std::array<double, N_test+1> phi_prime;
    for (int i = 0; i <= N_test; ++i) {
        phi_prime[i] = 0.0;
        for (int j = 0; j <= N_test; ++j) {
            phi_prime[i] += host_derivative_matrices[offset_2D + i * (N_test + 1) + j] * phi[j];
        }
    }
    
    for (int i = 0; i <= N_test; ++i) {
        REQUIRE(std::abs(phi_prime[i] - phi_prime_expected[i]) < error*100);
    }

    double phi_L_expected = std::sin(pi * -1.0);
    double phi_R_expected = std::sin(pi * 1.0);
    double phi_L = 0.0;
    double phi_R = 0.0;

    for (int j = 0; j <= N_test; ++j) {
        phi_L += host_lagrange_interpolant_left[offset_1D + j] * phi[j];
        phi_R += host_lagrange_interpolant_right[offset_1D + j] * phi[j];
    }
    
    REQUIRE(std::abs(phi_L - phi_L_expected) < error);
    REQUIRE(std::abs(phi_R - phi_R_expected) < error);

    double phi_prime_L = 0.0;
    double phi_prime_R = 0.0;
    double phi_prime_L_expected = pi * std::cos(pi * -1.0);
    double phi_prime_R_expected = pi * std::cos(pi * 1.0);

    for (int j = 0; j <= N_test; ++j) {
        phi_prime_L += host_lagrange_interpolant_derivative_left[offset_1D + j] * (phi_L - phi[j]);
        phi_prime_R += host_lagrange_interpolant_derivative_right[offset_1D + j] * (phi_R - phi[j]);
    }
    
    REQUIRE(std::abs(phi_prime_L - phi_prime_L_expected) < error);
    REQUIRE(std::abs(phi_prime_R - phi_prime_R_expected) < error);

    hipStreamDestroy(stream);
}

TEST_CASE("LegendrePolynomials", "Checks the Legendre polynomials"){
    const int N_max = 16;
    const int N_test = 16;
    const size_t n_interpolation_points = N_max * 8;
    const size_t offset_1D = N_test * (N_test + 1) /2;
    const size_t offset_2D = N_test * (N_test + 1) * (2 * N_test + 1) /6;
    const double error = 1e-6;

    REQUIRE(N_test <= N_max);

    hipStream_t stream;
    hipStreamCreate(&stream); 
    
    SEM::Device::Entities::NDG_t<SEM::Device::Polynomials::LegendrePolynomial_t> NDG(N_max, n_interpolation_points, stream);

    std::vector<deviceFloat> host_nodes(NDG.vector_length_);
    std::vector<deviceFloat> host_weights(NDG.vector_length_);
    std::vector<deviceFloat> host_barycentric_weights(NDG.vector_length_);
    std::vector<deviceFloat> host_lagrange_interpolant_left(NDG.vector_length_);
    std::vector<deviceFloat> host_lagrange_interpolant_right(NDG.vector_length_);
    std::vector<deviceFloat> host_lagrange_interpolant_derivative_left(NDG.vector_length_);
    std::vector<deviceFloat> host_lagrange_interpolant_derivative_right(NDG.vector_length_);
    std::vector<deviceFloat> host_derivative_matrices(NDG.matrix_length_);
    std::vector<deviceFloat> host_g_hat_derivative_matrices(NDG.matrix_length_);
    std::vector<deviceFloat> host_derivative_matrices_hat(NDG.matrix_length_);
    std::vector<deviceFloat> host_interpolation_matrices(NDG.interpolation_length_);

    NDG.nodes_.copy_to(host_nodes, stream);
    NDG.weights_.copy_to(host_weights, stream);
    NDG.barycentric_weights_.copy_to(host_barycentric_weights, stream);
    NDG.lagrange_interpolant_left_.copy_to(host_lagrange_interpolant_left, stream);
    NDG.lagrange_interpolant_right_.copy_to(host_lagrange_interpolant_right, stream);
    NDG.lagrange_interpolant_derivative_left_.copy_to(host_lagrange_interpolant_derivative_left, stream);
    NDG.lagrange_interpolant_derivative_right_.copy_to(host_lagrange_interpolant_derivative_right, stream);
    NDG.derivative_matrices_.copy_to(host_derivative_matrices, stream);
    NDG.g_hat_derivative_matrices_.copy_to(host_g_hat_derivative_matrices, stream);
    NDG.derivative_matrices_hat_.copy_to(host_derivative_matrices_hat, stream);
    NDG.interpolation_matrices_.copy_to(host_interpolation_matrices, stream);
    hipStreamSynchronize(stream);

    const std::array<double, N_test+1> nodes {-0.9905754753144173356754,
                                                -0.9506755217687677612227,
                                                -0.880239153726985902123,
                                                -0.7815140038968014069252,
                                                -0.6576711592166907658503,
                                                -0.5126905370864769678863,
                                                -0.3512317634538763152972,
                                                -0.1784841814958478558507,
                                                0,
                                                0.1784841814958478558507,
                                                0.3512317634538763152972,
                                                0.5126905370864769678863,
                                                0.6576711592166907658503,
                                                0.7815140038968014069252,
                                                0.880239153726985902123,
                                                0.9506755217687677612227,
                                                0.9905754753144173356754};

    for (int i = 0; i <= N_test; ++i) {
        REQUIRE(std::abs(nodes[i] - host_nodes[offset_1D + i]) < error);
    }

    const std::array<double, N_test+1> weights {0.0241483028685479319601,
                                                0.0554595293739872011294,
                                                0.0850361483171791808835,
                                                0.111883847193403971095,
                                                0.1351363684685254732863,
                                                0.1540457610768102880814,
                                                0.16800410215645004451,
                                                0.1765627053669926463253,
                                                0.1794464703562065254583,
                                                0.1765627053669926463253,
                                                0.16800410215645004451,
                                                0.1540457610768102880814,
                                                0.1351363684685254732863,
                                                0.111883847193403971095,
                                                0.0850361483171791808835,
                                                0.055459529373987201129,
                                                0.0241483028685479319601};
        
    for (int i = 0; i <= N_test; ++i) {
        REQUIRE(std::abs(weights[i] - host_weights[offset_1D + i]) < error);
    }

    std::array<double, N_test+1> phi;
    for (int i = 0; i <= N_test; ++i) {
        phi[i] = std::sin(pi * host_nodes[offset_1D + i]);
    }

    std::array<double, N_test+1> phi_prime_expected;
    for (int i = 0; i <= N_test; ++i) {
        phi_prime_expected[i] = pi * std::cos(pi * host_nodes[offset_1D + i]);
    }

    std::array<double, N_test+1> phi_prime;
    for (int i = 0; i <= N_test; ++i) {
        phi_prime[i] = 0.0;
        for (int j = 0; j <= N_test; ++j) {
            phi_prime[i] += host_derivative_matrices[offset_2D + i * (N_test + 1) + j] * phi[j];
        }
    }
    
    for (int i = 0; i <= N_test; ++i) {
        REQUIRE(std::abs(phi_prime[i] - phi_prime_expected[i]) < error*100);
    }

    double phi_L_expected = std::sin(pi * -1.0);
    double phi_R_expected = std::sin(pi * 1.0);
    double phi_L = 0.0;
    double phi_R = 0.0;

    for (int j = 0; j <= N_test; ++j) {
        phi_L += host_lagrange_interpolant_left[offset_1D + j] * phi[j];
        phi_R += host_lagrange_interpolant_right[offset_1D + j] * phi[j];
    }
    
    REQUIRE(std::abs(phi_L - phi_L_expected) < error);
    REQUIRE(std::abs(phi_R - phi_R_expected) < error);

    double phi_prime_L = 0.0;
    double phi_prime_R = 0.0;
    double phi_prime_L_expected = pi * std::cos(pi * -1.0);
    double phi_prime_R_expected = pi * std::cos(pi * 1.0);

    for (int j = 0; j <= N_test; ++j) {
        phi_prime_L += host_lagrange_interpolant_derivative_left[offset_1D + j] * (phi_L - phi[j]);
        phi_prime_R += host_lagrange_interpolant_derivative_right[offset_1D + j] * (phi_R - phi[j]);
    }
    
    REQUIRE(std::abs(phi_prime_L - phi_prime_L_expected) < error);
    REQUIRE(std::abs(phi_prime_R - phi_prime_R_expected) < error);

    hipStreamDestroy(stream);
}