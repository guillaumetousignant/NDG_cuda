#include "hip/hip_runtime.h"
#include <catch2/catch.hpp>
#include <iostream>
#include <cmath>
#include <array>
#include "NDG_t.cuh"
#include "ChebyshevPolynomial_t.cuh"
#include "LegendrePolynomial_t.cuh"
#include "float_types.h"

constexpr double pi = 3.14159265358979323846;

TEST_CASE("ChebyshevPolynomials", "Checks the Chebyshev polynomials"){
    const int N_max = 16;
    const int N_test = 16;
    const size_t N_interpolation_points = N_max * 8;
    const size_t offset_1D = N_test * (N_test + 1) /2;
    const size_t offset_2D = N_test * (N_test + 1) * (2 * N_test + 1) /6;
    const double error = 1e-6;
    
    NDG_t<ChebyshevPolynomial_t> NDG(N_max, N_interpolation_points);

    deviceFloat* host_nodes = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_weights = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_barycentric_weights = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_lagrange_interpolant_left = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_lagrange_interpolant_right = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_lagrange_interpolant_derivative_left = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_lagrange_interpolant_derivative_right = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_derivative_matrices = new deviceFloat[NDG.matrix_length_];
    deviceFloat* host_g_hat_derivative_matrices = new deviceFloat[NDG.matrix_length_];
    deviceFloat* host_derivative_matrices_hat = new deviceFloat[NDG.matrix_length_];
    deviceFloat* host_interpolation_matrices = new deviceFloat[NDG.interpolation_length_];

    hipMemcpy(host_nodes, NDG.nodes_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_weights, NDG.weights_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_barycentric_weights, NDG.barycentric_weights_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_lagrange_interpolant_left, NDG.lagrange_interpolant_left_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_lagrange_interpolant_right, NDG.lagrange_interpolant_right_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_lagrange_interpolant_derivative_left, NDG.lagrange_interpolant_derivative_left_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_lagrange_interpolant_derivative_right, NDG.lagrange_interpolant_derivative_right_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_derivative_matrices, NDG.derivative_matrices_, NDG.matrix_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_g_hat_derivative_matrices, NDG.g_hat_derivative_matrices_, NDG.matrix_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_derivative_matrices_hat, NDG.derivative_matrices_hat_, NDG.matrix_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_interpolation_matrices, NDG.interpolation_matrices_, NDG.interpolation_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);

    REQUIRE(N_test <= N_max);

    SECTION("Polynomial nodes") {
        const std::array<double, N_test+1> nodes {-0.9957341762950345218712,
                                                  -0.9618256431728190704088,
                                                  -0.895163291355062322067,
                                                  -0.7980172272802395033328,
                                                  -0.673695643646557211713,
                                                  -0.5264321628773558002446,
                                                  -0.3612416661871529487447,
                                                  -0.1837495178165703315744,
                                                   0,
                                                   0.183749517816570331574,
                                                   0.3612416661871529487447,
                                                   0.5264321628773558002446,
                                                   0.6736956436465572117127,
                                                   0.7980172272802395033328,
                                                   0.895163291355062322067,
                                                   0.9618256431728190704088,
                                                   0.9957341762950345218712};

        for (int i = 0; i <= N_test; ++i) {
            REQUIRE(std::abs(nodes[i] - host_nodes[offset_1D + i]) < error);
        }
    }

    SECTION("Polynomial weights") {
        const std::array<double, N_test+1> weights {0.1847995678582231316743,
                                                    0.1847995678582231316743,
                                                    0.1847995678582231316743,
                                                    0.1847995678582231316743,
                                                    0.1847995678582231316743,
                                                    0.1847995678582231316743,
                                                    0.1847995678582231316743,
                                                    0.1847995678582231316743,
                                                    0.1847995678582231316743,
                                                    0.1847995678582231316743,
                                                    0.1847995678582231316743,
                                                    0.1847995678582231316743,
                                                    0.1847995678582231316743,
                                                    0.1847995678582231316743,
                                                    0.1847995678582231316743,
                                                    0.1847995678582231316743,
                                                    0.1847995678582231316743};
            
        for (int i = 0; i <= N_test; ++i) {
            REQUIRE(std::abs(weights[i] - host_weights[offset_1D + i]) < error);
        }
    }

    SECTION("Polynomial derivative") {
        std::array<double, N_test+1> phi;
        std::array<double, N_test+1> phi_prime_expected;
        for (int i = 0; i <= N_test; ++i) {
            phi[i] = std::sin(pi * host_nodes[offset_1D + i]);
            phi_prime_expected[i] = pi * std::cos(pi * host_nodes[offset_1D + i]);
        }

        std::array<double, N_test+1> phi_prime;
        for (int i = 0; i <= N_test; ++i) {
            phi_prime[i] = 0.0;
            for (int j = 0; j <= N_test; ++j) {
                phi_prime[i] += host_derivative_matrices[offset_2D + i * (N_test + 1) + j] * phi[j];
            }
        }
        
        for (int i = 0; i <= N_test; ++i) {
            REQUIRE(std::abs(phi_prime[i] - phi_prime_expected[i]) < error*100);
        }
    }

    SECTION("Polynomial second derivative") {
        std::array<double, N_test+1> phi;
        std::array<double, N_test+1> phi_prime_prime_expected;
        for (int i = 0; i <= N_test; ++i) {
            phi[i] = std::sin(pi * host_nodes[offset_1D + i]);
            phi_prime_prime_expected[i] = -std::pow(pi, 2) * std::sin(pi * host_nodes[offset_1D + i]);
        }

        std::array<double, N_test+1> phi_prime_prime;
        for (int i = 0; i <= N_test; ++i) {
            phi_prime_prime[i] = 0.0;
            for (int j = 0; j <= N_test; ++j) {
                phi_prime_prime[i] += host_g_hat_derivative_matrices[offset_2D + i * (N_test + 1) + j] * phi[j] * host_weights[offset_1D + i];
            }
        }
        
        for (int i = 0; i <= N_test; ++i) {
            REQUIRE(std::abs(phi_prime_prime[i] - phi_prime_prime_expected[i]) < error*100);
        }
    }

    delete[] host_nodes;
    delete[] host_weights;
    delete[] host_barycentric_weights;
    delete[] host_lagrange_interpolant_left;
    delete[] host_lagrange_interpolant_right;
    delete[] host_lagrange_interpolant_derivative_left;
    delete[] host_lagrange_interpolant_derivative_right;
    delete[] host_derivative_matrices;
    delete[] host_g_hat_derivative_matrices;
    delete[] host_derivative_matrices_hat;
    delete[] host_interpolation_matrices;
}

TEST_CASE("LegendrePolynomials", "Checks the Legendre polynomials"){
    const int N_max = 16;
    const int N_test = 16;
    const size_t N_interpolation_points = N_max * 8;
    const size_t offset_1D = N_test * (N_test + 1) /2;
    const size_t offset_2D = N_test * (N_test + 1) * (2 * N_test + 1) /6;
    const double error = 1e-6;
    
    NDG_t<LegendrePolynomial_t> NDG(N_max, N_interpolation_points);

    deviceFloat* host_nodes = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_weights = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_barycentric_weights = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_lagrange_interpolant_left = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_lagrange_interpolant_right = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_lagrange_interpolant_derivative_left = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_lagrange_interpolant_derivative_right = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_derivative_matrices = new deviceFloat[NDG.matrix_length_];
    deviceFloat* host_g_hat_derivative_matrices = new deviceFloat[NDG.matrix_length_];
    deviceFloat* host_derivative_matrices_hat = new deviceFloat[NDG.matrix_length_];
    deviceFloat* host_interpolation_matrices = new deviceFloat[NDG.interpolation_length_];

    hipMemcpy(host_nodes, NDG.nodes_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_weights, NDG.weights_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_barycentric_weights, NDG.barycentric_weights_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_lagrange_interpolant_left, NDG.lagrange_interpolant_left_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_lagrange_interpolant_right, NDG.lagrange_interpolant_right_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_lagrange_interpolant_derivative_left, NDG.lagrange_interpolant_derivative_left_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_lagrange_interpolant_derivative_right, NDG.lagrange_interpolant_derivative_right_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_derivative_matrices, NDG.derivative_matrices_, NDG.matrix_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_g_hat_derivative_matrices, NDG.g_hat_derivative_matrices_, NDG.matrix_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_derivative_matrices_hat, NDG.derivative_matrices_hat_, NDG.matrix_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_interpolation_matrices, NDG.interpolation_matrices_, NDG.interpolation_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);

    REQUIRE(N_test <= N_max);

    SECTION("Polynomial nodes") {
        const std::array<double, N_test+1> nodes {-0.9905754753144173356754,
                                                  -0.9506755217687677612227,
                                                  -0.880239153726985902123,
                                                  -0.7815140038968014069252,
                                                  -0.6576711592166907658503,
                                                  -0.5126905370864769678863,
                                                  -0.3512317634538763152972,
                                                  -0.1784841814958478558507,
                                                   0,
                                                   0.1784841814958478558507,
                                                   0.3512317634538763152972,
                                                   0.5126905370864769678863,
                                                   0.6576711592166907658503,
                                                   0.7815140038968014069252,
                                                   0.880239153726985902123,
                                                   0.9506755217687677612227,
                                                   0.9905754753144173356754};

        for (int i = 0; i <= N_test; ++i) {
            REQUIRE(std::abs(nodes[i] - host_nodes[offset_1D + i]) < error);
        }
    }

    SECTION("Polynomial weights") {
        const std::array<double, N_test+1> weights {0.0241483028685479319601,
                                                    0.0554595293739872011294,
                                                    0.0850361483171791808835,
                                                    0.111883847193403971095,
                                                    0.1351363684685254732863,
                                                    0.1540457610768102880814,
                                                    0.16800410215645004451,
                                                    0.1765627053669926463253,
                                                    0.1794464703562065254583,
                                                    0.1765627053669926463253,
                                                    0.16800410215645004451,
                                                    0.1540457610768102880814,
                                                    0.1351363684685254732863,
                                                    0.111883847193403971095,
                                                    0.0850361483171791808835,
                                                    0.055459529373987201129,
                                                    0.0241483028685479319601};
            
        for (int i = 0; i <= N_test; ++i) {
            REQUIRE(std::abs(weights[i] - host_weights[offset_1D + i]) < error);
        }
    }

    SECTION("Polynomial derivative") {
        std::array<double, N_test+1> phi;
        std::array<double, N_test+1> phi_prime_expected;
        for (int i = 0; i <= N_test; ++i) {
            phi[i] = std::sin(pi * host_nodes[offset_1D + i]);
            phi_prime_expected[i] = pi * std::cos(pi * host_nodes[offset_1D + i]);
        }

        std::array<double, N_test+1> phi_prime;
        for (int i = 0; i <= N_test; ++i) {
            phi_prime[i] = 0.0;
            for (int j = 0; j <= N_test; ++j) {
                phi_prime[i] += host_derivative_matrices[offset_2D + i * (N_test + 1) + j] * phi[j];
            }
        }
        
        for (int i = 0; i <= N_test; ++i) {
            REQUIRE(std::abs(phi_prime[i] - phi_prime_expected[i]) < error*100);
        }
    }

    SECTION("Polynomial second derivative") {
        std::array<double, N_test+1> phi;
        std::array<double, N_test+1> phi_prime_prime_expected;
        for (int i = 0; i <= N_test; ++i) {
            phi[i] = std::sin(pi * host_nodes[offset_1D + i]);
            phi_prime_prime_expected[i] = -std::pow(pi, 2) * std::sin(pi * host_nodes[offset_1D + i]);
        }

        std::array<double, N_test+1> phi_prime_prime;
        for (int i = 0; i <= N_test; ++i) {
            phi_prime_prime[i] = 0.0;
            for (int j = 0; j <= N_test; ++j) {
                phi_prime_prime[i] += host_g_hat_derivative_matrices[offset_2D + i * (N_test + 1) + j] * phi[j] * host_weights[offset_1D + i];
            }
        }
        
        for (int i = 0; i <= N_test; ++i) {
            REQUIRE(std::abs(phi_prime_prime[i] - phi_prime_prime_expected[i]) < error*100);
        }

        for (int i = 0; i <= N_test; ++i) {
            std::cout << phi_prime_prime[i] << "    " << phi_prime_prime_expected[i] << std::endl;;
        }
    }

    delete[] host_nodes;
    delete[] host_weights;
    delete[] host_barycentric_weights;
    delete[] host_lagrange_interpolant_left;
    delete[] host_lagrange_interpolant_right;
    delete[] host_lagrange_interpolant_derivative_left;
    delete[] host_lagrange_interpolant_derivative_right;
    delete[] host_derivative_matrices;
    delete[] host_g_hat_derivative_matrices;
    delete[] host_derivative_matrices_hat;
    delete[] host_interpolation_matrices;
}