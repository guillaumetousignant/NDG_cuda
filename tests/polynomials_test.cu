#include "hip/hip_runtime.h"
#include <catch2/catch.hpp>
#include <iostream>
#include <cmath>
#include "NDG_t.cuh"
#include "ChebyshevPolynomial_t.cuh"
#include "LegendrePolynomial_t.cuh"
#include "float_types.h"

TEST_CASE("ChebyshevPolynomials", "Checks the Chebyshev polynomials"){
    const int N_max = 16;
    const int N_test = 16;
    const int N_interpolation_points = N_max * 8;
    const int offset_1D = N_test * (N_test + 1) /2;
    const int offset_2D = N_test * (N_test + 1) * (2 * N_test + 1) /6;
    const int offset_interp = N_test * (N_test + 1) * N_interpolation_points /2;
    const double error = 1e-6;
    
    NDG_t<ChebyshevPolynomial_t> NDG(N_max, N_interpolation_points);

    deviceFloat* host_nodes = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_weights = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_barycentric_weights = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_lagrange_interpolant_left = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_lagrange_interpolant_right = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_derivative_matrices = new deviceFloat[NDG.matrix_length_];
    deviceFloat* host_derivative_matrices_hat = new deviceFloat[NDG.matrix_length_];
    deviceFloat* host_interpolation_matrices = new deviceFloat[NDG.interpolation_length_];

    hipDeviceSynchronize();

    hipMemcpy(host_nodes, NDG.nodes_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_weights, NDG.weights_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_barycentric_weights, NDG.barycentric_weights_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_lagrange_interpolant_left, NDG.lagrange_interpolant_left_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_lagrange_interpolant_right, NDG.lagrange_interpolant_right_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_derivative_matrices, NDG.derivative_matrices_, NDG.matrix_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_derivative_matrices_hat, NDG.derivative_matrices_hat_, NDG.matrix_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_interpolation_matrices, NDG.interpolation_matrices_, NDG.interpolation_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);

    REQUIRE(N_test <= N_max);

    SECTION("Polynomial nodes") {
        const double nodes[] = {-0.9957341762950345218712,
                                -0.9618256431728190704088,
                                -0.895163291355062322067,
                                -0.7980172272802395033328,
                                -0.673695643646557211713,
                                -0.5264321628773558002446,
                                -0.3612416661871529487447,
                                -0.1837495178165703315744,
                                 0,
                                 0.183749517816570331574,
                                 0.3612416661871529487447,
                                 0.5264321628773558002446,
                                 0.6736956436465572117127,
                                 0.7980172272802395033328,
                                 0.895163291355062322067,
                                 0.9618256431728190704088,
                                 0.9957341762950345218712};

        for (int i = 0; i <= N_test; ++i) {
            REQUIRE(std::abs(nodes[i] - host_nodes[offset_1D + i]) < error);
        }
    }

    SECTION("Polynomial weights") {
        const double weights[] = {0.1847995678582231316743,
                                  0.1847995678582231316743,
                                  0.1847995678582231316743,
                                  0.1847995678582231316743,
                                  0.1847995678582231316743,
                                  0.1847995678582231316743,
                                  0.1847995678582231316743,
                                  0.1847995678582231316743,
                                  0.1847995678582231316743,
                                  0.1847995678582231316743,
                                  0.1847995678582231316743,
                                  0.1847995678582231316743,
                                  0.1847995678582231316743,
                                  0.1847995678582231316743,
                                  0.1847995678582231316743,
                                  0.1847995678582231316743,
                                  0.1847995678582231316743};
            
        for (int i = 0; i <= N_test; ++i) {
            REQUIRE(std::abs(weights[i] - host_weights[offset_1D + i]) < error);
        }
    }

    delete[] host_nodes;
    delete[] host_weights;
    delete[] host_barycentric_weights;
    delete[] host_lagrange_interpolant_left;
    delete[] host_lagrange_interpolant_right;
    delete[] host_derivative_matrices;
    delete[] host_derivative_matrices_hat;
    delete[] host_interpolation_matrices;
}

TEST_CASE("LegendrePolynomials", "Checks the Legendre polynomials"){
    const int N_max = 16;
    const int N_test = 16;
    const int N_interpolation_points = N_max * 8;
    const int offset_1D = N_test * (N_test + 1) /2;
    const int offset_2D = N_test * (N_test + 1) * (2 * N_test + 1) /6;
    const int offset_interp = N_test * (N_test + 1) * N_interpolation_points /2;
    const double error = 1e-6;
    
    NDG_t<LegendrePolynomial_t> NDG(N_max, N_interpolation_points);

    deviceFloat* host_nodes = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_weights = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_barycentric_weights = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_lagrange_interpolant_left = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_lagrange_interpolant_right = new deviceFloat[NDG.vector_length_];
    deviceFloat* host_derivative_matrices = new deviceFloat[NDG.matrix_length_];
    deviceFloat* host_derivative_matrices_hat = new deviceFloat[NDG.matrix_length_];
    deviceFloat* host_interpolation_matrices = new deviceFloat[NDG.interpolation_length_];

    hipDeviceSynchronize();

    hipMemcpy(host_nodes, NDG.nodes_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_weights, NDG.weights_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_barycentric_weights, NDG.barycentric_weights_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_lagrange_interpolant_left, NDG.lagrange_interpolant_left_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_lagrange_interpolant_right, NDG.lagrange_interpolant_right_, NDG.vector_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_derivative_matrices, NDG.derivative_matrices_, NDG.matrix_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_derivative_matrices_hat, NDG.derivative_matrices_hat_, NDG.matrix_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_interpolation_matrices, NDG.interpolation_matrices_, NDG.interpolation_length_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);

    REQUIRE(N_test <= N_max);

    SECTION("Polynomial nodes") {
        const double nodes[] = {-0.9905754753144173356754,
                                -0.9506755217687677612227,
                                -0.880239153726985902123,
                                -0.7815140038968014069252,
                                -0.6576711592166907658503,
                                -0.5126905370864769678863,
                                -0.3512317634538763152972,
                                -0.1784841814958478558507,
                                 0,
                                 0.1784841814958478558507,
                                 0.3512317634538763152972,
                                 0.5126905370864769678863,
                                 0.6576711592166907658503,
                                 0.7815140038968014069252,
                                 0.880239153726985902123,
                                 0.9506755217687677612227,
                                 0.9905754753144173356754};

        for (int i = 0; i <= N_test; ++i) {
            REQUIRE(std::abs(nodes[i] - host_nodes[offset_1D + i]) < error);
        }
    }

    SECTION("Polynomial weights") {
        const double weights[] = {0.0241483028685479319601,
                                  0.0554595293739872011294,
                                  0.0850361483171791808835,
                                  0.111883847193403971095,
                                  0.1351363684685254732863,
                                  0.1540457610768102880814,
                                  0.16800410215645004451,
                                  0.1765627053669926463253,
                                  0.1794464703562065254583,
                                  0.1765627053669926463253,
                                  0.16800410215645004451,
                                  0.1540457610768102880814,
                                  0.1351363684685254732863,
                                  0.111883847193403971095,
                                  0.0850361483171791808835,
                                  0.055459529373987201129,
                                  0.0241483028685479319601};
            
        for (int i = 0; i <= N_test; ++i) {
            REQUIRE(std::abs(weights[i] - host_weights[offset_1D + i]) < error);
        }
    }

    delete[] host_nodes;
    delete[] host_weights;
    delete[] host_barycentric_weights;
    delete[] host_lagrange_interpolant_left;
    delete[] host_lagrange_interpolant_right;
    delete[] host_derivative_matrices;
    delete[] host_derivative_matrices_hat;
    delete[] host_interpolation_matrices;
}