#include "hip/hip_runtime.h"
#include <catch2/catch.hpp>
#include <cmath>
#include <array>
#include <vector>
#include "helpers/float_types.h"
#include "polynomials/LegendrePolynomial_t.cuh"
#include "entities/NDG_t.cuh"
#include "entities/Face2D_t.cuh"
#include "entities/Element2D_t.cuh"
#include "meshes/Mesh2D_t.cuh"
#include "entities/device_vector.cuh"
#include "functions/Hilbert_splitting.cuh"

using SEM::Entities::Vec2;

__global__
auto face_to_element_projection_init(int N_test_elements, int N_test_faces, size_t n_faces, SEM::Entities::Element2D_t* elements, SEM::Entities::Face2D_t* faces, const Vec2<deviceFloat>* nodes, const deviceFloat* polynomial_nodes) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < n_faces; i += stride) {
        std::array<SEM::Entities::cuda_vector<size_t>, 4> element_faces {SEM::Entities::cuda_vector<size_t>(1),
                                                                         SEM::Entities::cuda_vector<size_t>(1),
                                                                         SEM::Entities::cuda_vector<size_t>(1),
                                                                         SEM::Entities::cuda_vector<size_t>(1)};

        element_faces[0][0] = i;
        element_faces[1][0] = i;
        element_faces[2][0] = i;
        element_faces[3][0] = i;

        elements[2 * i]     = SEM::Entities::Element2D_t(N_test_elements, 0, SEM::Hilbert::Status::H, 0, element_faces, std::array<size_t, 4>{6 * i, 6 * i + 1, 6 * i + 2, 6 * i + 3});
        elements[2 * i + 1] = SEM::Entities::Element2D_t(N_test_elements, 0, SEM::Hilbert::Status::R, 0, element_faces, std::array<size_t, 4>{6 * i + 4, 6 * i + 5, 6 * i + 1, 6 * i});

        faces[i] = SEM::Entities::Face2D_t(N_test_faces, std::array<size_t, 2>{6 * i, 6 * i + 1}, std::array<size_t, 2>{2 * i, 2 * i + 1}, std::array<size_t, 2>{0, 2});

        faces[i].offset_     = {0.0, 0.0};
        faces[i].scale_      = {1.0, 1.0};

        SEM::Entities::Face2D_t& face = faces[i];
        const size_t offset_1D = face.N_ * (face.N_ + 1) /2;

        for (int i = 0; i <= face.N_; ++i) {
            const deviceFloat interp = (polynomial_nodes[offset_1D + i] + 1)/2;
            const Vec2<deviceFloat> global_coordinates = nodes[face.nodes_[1]] * interp + nodes[face.nodes_[0]] * (1 - interp);

            face.p_flux_[i] = std::cos(global_coordinates.y());
            face.u_flux_[i] = global_coordinates.y() * global_coordinates.y();
            face.v_flux_[i] = global_coordinates.y();
        }
    }
}

__global__
auto faces_to_element_projection_init(int N, size_t n_faces, SEM::Entities::Element2D_t* elements, SEM::Entities::Face2D_t* faces, const Vec2<deviceFloat>* nodes, const deviceFloat* polynomial_nodes) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < n_faces/2; i += stride) {
        std::array<SEM::Entities::cuda_vector<size_t>, 4> element_faces {SEM::Entities::cuda_vector<size_t>(2),
                                                                         SEM::Entities::cuda_vector<size_t>(2),
                                                                         SEM::Entities::cuda_vector<size_t>(2),
                                                                         SEM::Entities::cuda_vector<size_t>(2)};

        element_faces[0][0] = 2 * i;
        element_faces[0][1] = 2 * i + 1;
        element_faces[1][0] = 2 * i;
        element_faces[1][1] = 2 * i + 1;
        element_faces[2][0] = 2 * i;
        element_faces[2][1] = 2 * i + 1;
        element_faces[3][0] = 2 * i;
        element_faces[3][1] = 2 * i + 1;

        elements[2 * i]     = SEM::Entities::Element2D_t(N, 0, SEM::Hilbert::Status::H, 0, element_faces, std::array<size_t, 4>{6 * i, 6 * i + 1, 6 * i + 2, 6 * i + 3});
        elements[2 * i + 1] = SEM::Entities::Element2D_t(N, 0, SEM::Hilbert::Status::B, 0, element_faces, std::array<size_t, 4>{6 * i + 4, 6 * i + 5, 6 * i + 1, 6 * i});

        faces[2 * i]     = SEM::Entities::Face2D_t(N, std::array<size_t, 2>{8 * i, 8 * i + 1},     std::array<size_t, 2>{2 * i, 2 * i + 1}, std::array<size_t, 2>{0, 2});
        faces[2 * i + 1] = SEM::Entities::Face2D_t(N, std::array<size_t, 2>{8 * i + 1, 8 * i + 2}, std::array<size_t, 2>{2 * i, 2 * i + 1}, std::array<size_t, 2>{0, 2});

        faces[2 * i].offset_     = {0.0, 0.5};
        faces[2 * i].scale_      = {0.5, 0.5};
        faces[2 * i + 1].offset_ = {0.5, 0.0};
        faces[2 * i + 1].scale_  = {0.5, 0.5};

        SEM::Entities::Face2D_t& face_0 = faces[2 * i];
        SEM::Entities::Face2D_t& face_1 = faces[2 * i + 1];
        const size_t offset_1D_0 = face_0.N_ * (face_0.N_ + 1) /2;
        const size_t offset_1D_1 = face_1.N_ * (face_1.N_ + 1) /2;

        for (int i = 0; i <= face_0.N_; ++i) {
            const deviceFloat interp = (polynomial_nodes[offset_1D_0 + i] + 1)/2;
            const Vec2<deviceFloat> global_coordinates = nodes[face_0.nodes_[1]] * interp + nodes[face_0.nodes_[0]] * (1 - interp);

            face_0.p_flux_[i] = std::cos(global_coordinates.y());
            face_0.u_flux_[i] = global_coordinates.y() * global_coordinates.y();
            face_0.v_flux_[i] = global_coordinates.y();
        }
        for (int i = 0; i <= face_1.N_; ++i) {
            const deviceFloat interp = (polynomial_nodes[offset_1D_1 + i] + 1)/2;
            const Vec2<deviceFloat> global_coordinates = nodes[face_1.nodes_[1]] * interp + nodes[face_1.nodes_[0]] * (1 - interp);

            face_1.p_flux_[i] = std::cos(global_coordinates.y());
            face_1.u_flux_[i] = global_coordinates.y() * global_coordinates.y();
            face_1.v_flux_[i] = global_coordinates.y();
        }
    }
}

__global__
auto retrieve_element_projected_solution(int N, size_t n_faces, const SEM::Entities::Element2D_t* elements, const SEM::Entities::Face2D_t* faces, deviceFloat* p, deviceFloat* u, deviceFloat* v) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < n_faces; i += stride) {
        const size_t offset_results = 2 * i * (N + 1);
        const SEM::Entities::Face2D_t& face = faces[i];
        const SEM::Entities::Element2D_t& element_left = elements[face.elements_[0]];
        const SEM::Entities::Element2D_t& element_right = elements[face.elements_[1]];
        const size_t element_side_left = face.elements_side_[0];
        const size_t element_side_right = face.elements_side_[1];
        
        for (int j = 0; j <= N; ++j) {
            p[offset_results + j]               = element_left.p_flux_extrapolated_[element_side_left][j];
            p[offset_results + (N + 1) + j]     = element_right.p_flux_extrapolated_[element_side_right][j];

            u[offset_results + j]               = element_left.u_flux_extrapolated_[element_side_left][j];
            u[offset_results + (N + 1) + j]     = element_right.u_flux_extrapolated_[element_side_right][j];

            v[offset_results + j]               = element_left.v_flux_extrapolated_[element_side_left][j];
            v[offset_results + (N + 1) + j]     = element_right.v_flux_extrapolated_[element_side_right][j];
        }
    }
}

__global__
auto retrieve_element_projected_solution_2(int N, size_t n_faces, const SEM::Entities::Element2D_t* elements, const SEM::Entities::Face2D_t* faces, deviceFloat* p, deviceFloat* u, deviceFloat* v) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < n_faces/2; i += stride) {
        const size_t offset_results = 2 * i * (N + 1);
        const SEM::Entities::Face2D_t& face = faces[2 * i];
        const SEM::Entities::Element2D_t& element_left = elements[face.elements_[0]];
        const SEM::Entities::Element2D_t& element_right = elements[face.elements_[1]];
        const size_t element_side_left = face.elements_side_[0];
        const size_t element_side_right = face.elements_side_[1];
        
        for (int j = 0; j <= N; ++j) {
            p[offset_results + j]               = element_left.p_flux_extrapolated_[element_side_left][j];
            p[offset_results + (N + 1) + j]     = element_right.p_flux_extrapolated_[element_side_right][j];

            u[offset_results + j]               = element_left.u_flux_extrapolated_[element_side_left][j];
            u[offset_results + (N + 1) + j]     = element_right.u_flux_extrapolated_[element_side_right][j];

            v[offset_results + j]               = element_left.v_flux_extrapolated_[element_side_left][j];
            v[offset_results + (N + 1) + j]     = element_right.v_flux_extrapolated_[element_side_right][j];
        }
    }
}

TEST_CASE("Face to element projection test", "Projects the face flux solution of a face to its elements and checks the values match.") {   
    const int N_max = 16;
    const int N_test_faces = 16;
    const int N_test_elements = N_test_faces;
    const size_t n_interpolation_points = N_max;
    constexpr size_t n_faces = 1;
    constexpr size_t n_elements = 2 * n_faces;
    const double max_error = 1e-9;

    REQUIRE(N_test_elements <= N_max);
    REQUIRE(N_test_faces <= N_max);

    hipStream_t stream;
    hipStreamCreate(&stream); 

    const std::vector<Vec2<deviceFloat>> host_nodes {Vec2<deviceFloat>{0, -1},
                                                     Vec2<deviceFloat>{0, 1},
                                                     Vec2<deviceFloat>{-1, 1},
                                                     Vec2<deviceFloat>{-1, -1},
                                                     Vec2<deviceFloat>{1, -1},
                                                     Vec2<deviceFloat>{1, 1}};

    SEM::Entities::NDG_t<SEM::Polynomials::LegendrePolynomial_t> NDG(N_max, n_interpolation_points, stream);
    SEM::Entities::device_vector<SEM::Entities::Element2D_t> elements(n_elements, stream);
    SEM::Entities::device_vector<SEM::Entities::Face2D_t> faces(n_faces, stream);
    SEM::Entities::device_vector<Vec2<deviceFloat>> nodes(host_nodes, stream);

    constexpr int faces_blockSize = 32;
    constexpr int faces_numBlocks = (n_faces + faces_blockSize - 1) / faces_blockSize;
    constexpr int elements_blockSize = 32;
    constexpr int elements_numBlocks = (n_elements + elements_blockSize - 1) / elements_blockSize;

    face_to_element_projection_init<<<faces_blockSize, faces_numBlocks, 0, stream>>>(N_test_elements, N_test_faces, n_faces, elements.data(), faces.data(), nodes.data(), NDG.nodes_.data());

    SEM::Meshes::compute_element_geometry<<<elements_blockSize, elements_numBlocks, 0, stream>>>(n_elements, elements.data(), nodes.data(), NDG.nodes_.data());

    SEM::Meshes::project_to_elements<<<elements_blockSize, elements_numBlocks, 0, stream>>>(n_elements, faces.data(), elements.data(), NDG.nodes_.data(), NDG.weights_.data(), NDG.barycentric_weights_.data());
    
    std::vector<deviceFloat> polynomial_nodes_host(NDG.nodes_.size());

    NDG.nodes_.copy_to(polynomial_nodes_host, stream);

    std::vector<std::array<std::vector<deviceFloat>, 2>> p_expected(n_faces);
    std::vector<std::array<std::vector<deviceFloat>, 2>> u_expected(n_faces);
    std::vector<std::array<std::vector<deviceFloat>, 2>> v_expected(n_faces);
    
    hipStreamSynchronize(stream);
    constexpr size_t offset_1D = N_test_elements * (N_test_elements + 1) /2;
    for (size_t i = 0; i < n_faces; ++i) {
        p_expected[i] = {std::vector<deviceFloat>(N_test_elements + 1), std::vector<deviceFloat>(N_test_elements + 1)};
        u_expected[i] = {std::vector<deviceFloat>(N_test_elements + 1), std::vector<deviceFloat>(N_test_elements + 1)};
        v_expected[i] = {std::vector<deviceFloat>(N_test_elements + 1), std::vector<deviceFloat>(N_test_elements + 1)};
    
        for (int j = 0; j <= N_test_elements; ++j) {
            const deviceFloat interp = (polynomial_nodes_host[offset_1D + j] + 1)/2;
            const Vec2<deviceFloat> global_coordinates = host_nodes[6 * i + 1] * interp + host_nodes[6 * i] * (1 - interp);
            
            p_expected[i][0][j] = std::cos(global_coordinates.y());
            u_expected[i][0][j] = global_coordinates.y() * global_coordinates.y();
            v_expected[i][0][j] = global_coordinates.y();

            p_expected[i][1][N_test_elements - j] = -std::cos(global_coordinates.y());
            u_expected[i][1][N_test_elements - j] = -global_coordinates.y() * global_coordinates.y();
            v_expected[i][1][N_test_elements - j] = -global_coordinates.y();
        }
    }

    SEM::Entities::device_vector<deviceFloat> p(n_elements * (N_test_elements + 1), stream);
    SEM::Entities::device_vector<deviceFloat> u(n_elements * (N_test_elements + 1), stream);
    SEM::Entities::device_vector<deviceFloat> v(n_elements * (N_test_elements + 1), stream);

    retrieve_element_projected_solution<<<elements_numBlocks, elements_blockSize, 0, stream>>>(N_test_elements, n_faces, elements.data(), faces.data(), p.data(), u.data(), v.data());

    std::vector<deviceFloat> p_host(n_elements * (N_test_elements + 1));
    std::vector<deviceFloat> u_host(n_elements * (N_test_elements + 1));
    std::vector<deviceFloat> v_host(n_elements * (N_test_elements + 1));

    p.copy_to(p_host, stream);
    u.copy_to(u_host, stream);
    v.copy_to(v_host, stream);
    hipStreamSynchronize(stream);

    for (size_t i = 0; i < n_faces; ++i) {
        const size_t offset_results = 2 * i * (N_test_elements + 1);

        for (int j = 0; j <= N_test_elements; ++j) {
            REQUIRE(std::abs(p_expected[i][0][j] - p_host[offset_results + j]) < max_error);
            REQUIRE(std::abs(u_expected[i][0][j] - u_host[offset_results + j]) < max_error);
            REQUIRE(std::abs(v_expected[i][0][j] - v_host[offset_results + j]) < max_error);

            REQUIRE(std::abs(p_expected[i][1][j] - p_host[offset_results + N_test_elements + 1 + j]) < max_error);
            REQUIRE(std::abs(u_expected[i][1][j] - u_host[offset_results + N_test_elements + 1 + j]) < max_error);
            REQUIRE(std::abs(v_expected[i][1][j] - v_host[offset_results + N_test_elements + 1 + j]) < max_error);
        }
    }

    elements.clear(stream);
    faces.clear(stream);
    nodes.clear(stream);
    p.clear(stream);
    u.clear(stream);
    v.clear(stream);

    hipStreamDestroy(stream);
}

TEST_CASE("Face to lower order element projection test", "Projects the face flux solution of a face to its lower order elements and checks the values match.") {   
    const int N_max = 16;
    const int N_test_faces = 16;
    const int N_test_elements = N_test_faces - 2;
    const size_t n_interpolation_points = N_max;
    constexpr size_t n_faces = 1;
    constexpr size_t n_elements = 2 * n_faces;
    const double max_error = 1e-9;

    REQUIRE(N_test_elements <= N_max);
    REQUIRE(N_test_faces <= N_max);

    hipStream_t stream;
    hipStreamCreate(&stream); 

    const std::vector<Vec2<deviceFloat>> host_nodes {Vec2<deviceFloat>{0, -1},
                                                     Vec2<deviceFloat>{0, 1},
                                                     Vec2<deviceFloat>{-1, 1},
                                                     Vec2<deviceFloat>{-1, -1},
                                                     Vec2<deviceFloat>{1, -1},
                                                     Vec2<deviceFloat>{1, 1}};

    SEM::Entities::NDG_t<SEM::Polynomials::LegendrePolynomial_t> NDG(N_max, n_interpolation_points, stream);
    SEM::Entities::device_vector<SEM::Entities::Element2D_t> elements(n_elements, stream);
    SEM::Entities::device_vector<SEM::Entities::Face2D_t> faces(n_faces, stream);
    SEM::Entities::device_vector<Vec2<deviceFloat>> nodes(host_nodes, stream);

    constexpr int faces_blockSize = 32;
    constexpr int faces_numBlocks = (n_faces + faces_blockSize - 1) / faces_blockSize;
    constexpr int elements_blockSize = 32;
    constexpr int elements_numBlocks = (n_elements + elements_blockSize - 1) / elements_blockSize;

    face_to_element_projection_init<<<faces_blockSize, faces_numBlocks, 0, stream>>>(N_test_elements, N_test_faces, n_faces, elements.data(), faces.data(), nodes.data(), NDG.nodes_.data());

    SEM::Meshes::compute_element_geometry<<<elements_blockSize, elements_numBlocks, 0, stream>>>(n_elements, elements.data(), nodes.data(), NDG.nodes_.data());

    SEM::Meshes::project_to_elements<<<elements_blockSize, elements_numBlocks, 0, stream>>>(n_elements, faces.data(), elements.data(), NDG.nodes_.data(), NDG.weights_.data(), NDG.barycentric_weights_.data());
    
    std::vector<deviceFloat> polynomial_nodes_host(NDG.nodes_.size());

    NDG.nodes_.copy_to(polynomial_nodes_host, stream);

    std::vector<std::array<std::vector<deviceFloat>, 2>> p_expected(n_faces);
    std::vector<std::array<std::vector<deviceFloat>, 2>> u_expected(n_faces);
    std::vector<std::array<std::vector<deviceFloat>, 2>> v_expected(n_faces);
    
    hipStreamSynchronize(stream);
    constexpr size_t offset_1D = N_test_elements * (N_test_elements + 1) /2;
    for (size_t i = 0; i < n_faces; ++i) {
        p_expected[i] = {std::vector<deviceFloat>(N_test_elements + 1), std::vector<deviceFloat>(N_test_elements + 1)};
        u_expected[i] = {std::vector<deviceFloat>(N_test_elements + 1), std::vector<deviceFloat>(N_test_elements + 1)};
        v_expected[i] = {std::vector<deviceFloat>(N_test_elements + 1), std::vector<deviceFloat>(N_test_elements + 1)};
    
        for (int j = 0; j <= N_test_elements; ++j) {
            const deviceFloat interp = (polynomial_nodes_host[offset_1D + j] + 1)/2;
            const Vec2<deviceFloat> global_coordinates = host_nodes[6 * i + 1] * interp + host_nodes[6 * i] * (1 - interp);
            
            p_expected[i][0][j] = std::cos(global_coordinates.y());
            u_expected[i][0][j] = global_coordinates.y() * global_coordinates.y();
            v_expected[i][0][j] = global_coordinates.y();

            p_expected[i][1][N_test_elements - j] = -std::cos(global_coordinates.y());
            u_expected[i][1][N_test_elements - j] = -global_coordinates.y() * global_coordinates.y();
            v_expected[i][1][N_test_elements - j] = -global_coordinates.y();
        }
    }

    SEM::Entities::device_vector<deviceFloat> p(n_elements * (N_test_elements + 1), stream);
    SEM::Entities::device_vector<deviceFloat> u(n_elements * (N_test_elements + 1), stream);
    SEM::Entities::device_vector<deviceFloat> v(n_elements * (N_test_elements + 1), stream);

    retrieve_element_projected_solution<<<elements_numBlocks, elements_blockSize, 0, stream>>>(N_test_elements, n_faces, elements.data(), faces.data(), p.data(), u.data(), v.data());

    std::vector<deviceFloat> p_host(n_elements * (N_test_elements + 1));
    std::vector<deviceFloat> u_host(n_elements * (N_test_elements + 1));
    std::vector<deviceFloat> v_host(n_elements * (N_test_elements + 1));

    p.copy_to(p_host, stream);
    u.copy_to(u_host, stream);
    v.copy_to(v_host, stream);
    hipStreamSynchronize(stream);

    for (size_t i = 0; i < n_faces; ++i) {
        const size_t offset_results = 2 * i * (N_test_elements + 1);

        for (int j = 0; j <= N_test_elements; ++j) {
            REQUIRE(std::abs(p_expected[i][0][j] - p_host[offset_results + j]) < max_error);
            REQUIRE(std::abs(u_expected[i][0][j] - u_host[offset_results + j]) < max_error);
            REQUIRE(std::abs(v_expected[i][0][j] - v_host[offset_results + j]) < max_error);

            REQUIRE(std::abs(p_expected[i][1][j] - p_host[offset_results + N_test_elements + 1 + j]) < max_error);
            REQUIRE(std::abs(u_expected[i][1][j] - u_host[offset_results + N_test_elements + 1 + j]) < max_error);
            REQUIRE(std::abs(v_expected[i][1][j] - v_host[offset_results + N_test_elements + 1 + j]) < max_error);
        }
    }

    elements.clear(stream);
    faces.clear(stream);
    nodes.clear(stream);
    p.clear(stream);
    u.clear(stream);
    v.clear(stream);

    hipStreamDestroy(stream);
}

TEST_CASE("Faces to element projection test", "Projects the face flux solution of two faces to their elements and checks the values match.") {   
    const int N_max = 16;
    const int N_test = 16;
    const size_t n_interpolation_points = N_max;
    constexpr size_t n_faces = 2;
    constexpr size_t n_elements = n_faces;
    const double max_error = 1e-9;

    REQUIRE(N_test <= N_max);

    hipStream_t stream;
    hipStreamCreate(&stream); 

    const std::vector<Vec2<deviceFloat>> host_nodes {Vec2<deviceFloat>{0, -1},
                                                     Vec2<deviceFloat>{0, 0},
                                                     Vec2<deviceFloat>{0, 1},
                                                     Vec2<deviceFloat>{-1, 1},
                                                     Vec2<deviceFloat>{-1, -1},
                                                     Vec2<deviceFloat>{1, -1},
                                                     Vec2<deviceFloat>{1, 1}};

    SEM::Entities::NDG_t<SEM::Polynomials::LegendrePolynomial_t> NDG(N_max, n_interpolation_points, stream);
    SEM::Entities::device_vector<SEM::Entities::Element2D_t> elements(n_elements, stream);
    SEM::Entities::device_vector<SEM::Entities::Face2D_t> faces(n_faces, stream);
    SEM::Entities::device_vector<Vec2<deviceFloat>> nodes(host_nodes, stream);

    constexpr int faces_blockSize = 32;
    constexpr int faces_numBlocks = (n_faces + faces_blockSize - 1) / faces_blockSize;
    constexpr int elements_blockSize = 32;
    constexpr int elements_numBlocks = (n_elements + elements_blockSize - 1) / elements_blockSize;

    faces_to_element_projection_init<<<faces_blockSize, (faces_numBlocks + 1)/2, 0, stream>>>(N_test, n_faces, elements.data(), faces.data(), nodes.data(), NDG.nodes_.data());

    SEM::Meshes::compute_element_geometry<<<elements_blockSize, elements_numBlocks, 0, stream>>>(n_elements, elements.data(), nodes.data(), NDG.nodes_.data());

    SEM::Meshes::project_to_elements<<<elements_blockSize, elements_numBlocks, 0, stream>>>(n_elements, faces.data(), elements.data(), NDG.nodes_.data(), NDG.weights_.data(), NDG.barycentric_weights_.data());
    
    std::vector<deviceFloat> polynomial_nodes_host(NDG.nodes_.size());

    NDG.nodes_.copy_to(polynomial_nodes_host, stream);

    std::vector<std::array<std::vector<deviceFloat>, 2>> p_expected(n_faces/2);
    std::vector<std::array<std::vector<deviceFloat>, 2>> u_expected(n_faces/2);
    std::vector<std::array<std::vector<deviceFloat>, 2>> v_expected(n_faces/2);
    
    hipStreamSynchronize(stream);
    constexpr size_t offset_1D = N_test * (N_test + 1) /2;
    for (size_t i = 0; i < n_faces/2; ++i) {
        p_expected[i] = {std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1)};
        u_expected[i] = {std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1)};
        v_expected[i] = {std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1)};
    
        for (int j = 0; j <= N_test; ++j) {
            const deviceFloat interp = (polynomial_nodes_host[offset_1D + j] + 1)/2;
            const Vec2<deviceFloat> global_coordinates = host_nodes[8 * i + 2] * interp + host_nodes[8 * i] * (1 - interp);
            
            p_expected[i][0][j] = std::cos(global_coordinates.y());
            u_expected[i][0][j] = global_coordinates.y() * global_coordinates.y();
            v_expected[i][0][j] = global_coordinates.y();

            p_expected[i][1][N_test - j] = -std::cos(global_coordinates.y());
            u_expected[i][1][N_test - j] = -global_coordinates.y() * global_coordinates.y();
            v_expected[i][1][N_test - j] = -global_coordinates.y();
        }
    }

    SEM::Entities::device_vector<deviceFloat> p(n_faces * (N_test + 1), stream);
    SEM::Entities::device_vector<deviceFloat> u(n_faces * (N_test + 1), stream);
    SEM::Entities::device_vector<deviceFloat> v(n_faces * (N_test + 1), stream);

    retrieve_element_projected_solution_2<<<faces_numBlocks, (faces_blockSize + 1)/2, 0, stream>>>(N_test, n_faces, elements.data(), faces.data(), p.data(), u.data(), v.data());

    std::vector<deviceFloat> p_host(n_faces * (N_test + 1));
    std::vector<deviceFloat> u_host(n_faces * (N_test + 1));
    std::vector<deviceFloat> v_host(n_faces * (N_test + 1));

    p.copy_to(p_host, stream);
    u.copy_to(u_host, stream);
    v.copy_to(v_host, stream);
    hipStreamSynchronize(stream);
   
    for (size_t i = 0; i < n_faces/2; ++i) {
        const size_t offset_results = 2 * i * (N_test + 1);

        for (int j = 0; j <= N_test; ++j) {
            REQUIRE(std::abs(p_expected[i][0][j] - p_host[offset_results + j]) < max_error);
            REQUIRE(std::abs(u_expected[i][0][j] - u_host[offset_results + j]) < max_error);
            REQUIRE(std::abs(v_expected[i][0][j] - v_host[offset_results + j]) < max_error);

            REQUIRE(std::abs(p_expected[i][1][j] - p_host[offset_results + N_test + 1 + j]) < max_error);
            REQUIRE(std::abs(u_expected[i][1][j] - u_host[offset_results + N_test + 1 + j]) < max_error);
            REQUIRE(std::abs(v_expected[i][1][j] - v_host[offset_results + N_test + 1 + j]) < max_error);
        }
    }

    elements.clear(stream);
    faces.clear(stream);
    nodes.clear(stream);
    p.clear(stream);
    u.clear(stream);
    v.clear(stream);

    hipStreamDestroy(stream);
}