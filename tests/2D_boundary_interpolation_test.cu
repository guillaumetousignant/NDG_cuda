#include "hip/hip_runtime.h"
#include <catch2/catch.hpp>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <array>
#include <vector>
#include "helpers/float_types.h"
#include "functions/quad_map.cuh"
#include "entities/NDG_t.cuh"
#include "entities/Element2D_t.cuh"
#include "entities/device_vector.cuh"
#include "meshes/Mesh2D_t.cuh"
#include "polynomials/ChebyshevPolynomial_t.cuh"

using SEM::Device::Entities::Vec2;
using SEM::Device::Entities::device_vector;
using SEM::Device::Entities::cuda_vector;

__device__ const std::array<Vec2<deviceFloat>, 4> points {Vec2<deviceFloat>{1, -1},
                                                              Vec2<deviceFloat>{1, 1},
                                                              Vec2<deviceFloat>{-1, 1},
                                                              Vec2<deviceFloat>{-1, -1}};

__global__
auto elements_init(size_t n_elements, SEM::Device::Entities::Element2D_t* elements, const deviceFloat* polynomial_nodes) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < n_elements; i += stride) {
        SEM::Device::Entities::Element2D_t& element = elements[i];
        const size_t offset_1D = element.N_ * (element.N_ + 1) /2;

        const int N = element.N_;
        element.p_ = cuda_vector<deviceFloat>((N + 1) * (N + 1));
        element.u_ = cuda_vector<deviceFloat>((N + 1) * (N + 1));
        element.v_ = cuda_vector<deviceFloat>((N + 1) * (N + 1));
        element.p_extrapolated_ = {cuda_vector<deviceFloat>(N + 1), cuda_vector<deviceFloat>(N + 1), cuda_vector<deviceFloat>(N + 1), cuda_vector<deviceFloat>(N + 1)};
        element.u_extrapolated_ = {cuda_vector<deviceFloat>(N + 1), cuda_vector<deviceFloat>(N + 1), cuda_vector<deviceFloat>(N + 1), cuda_vector<deviceFloat>(N + 1)};
        element.v_extrapolated_ = {cuda_vector<deviceFloat>(N + 1), cuda_vector<deviceFloat>(N + 1), cuda_vector<deviceFloat>(N + 1), cuda_vector<deviceFloat>(N + 1)};

        for (int i = 0; i <= element.N_; ++i) {
            for (int j = 0; j <= element.N_; ++j) {
                const Vec2<deviceFloat> coordinates {polynomial_nodes[offset_1D + i], polynomial_nodes[offset_1D + j]};
                const Vec2<deviceFloat> global_coordinates = SEM::Device::quad_map(coordinates, points);

                element.p_[i * (element.N_ + 1) + j] = std::sin(global_coordinates[0]) * std::cos(global_coordinates[1]);
                element.u_[i * (element.N_ + 1) + j] = global_coordinates[0];
                element.v_[i * (element.N_ + 1) + j] = global_coordinates[1];
            }
        }
    }
}

__global__
auto get_boundary_solution(size_t n_elements, const SEM::Device::Entities::Element2D_t* elements, std::array<deviceFloat*, 4> p, std::array<deviceFloat*, 4> u, std::array<deviceFloat*, 4> v) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < n_elements; i += stride) {
        for (size_t k = 0; k < elements[i].p_extrapolated_.size(); ++k){
            for (int j = 0; j <= elements[i].N_; ++j) {
                p[k][j] = elements[i].p_extrapolated_[k][j];
                u[k][j] = elements[i].u_extrapolated_[k][j];
                v[k][j] = elements[i].v_extrapolated_[k][j];
            }
        }
    }
}

TEST_CASE("2D boundary interpolation test", "Checks the interpolated value of the solution at the element's edges.") {   
    const int N_max = 16;
    const int N_test = 16;
    const size_t n_interpolation_points = std::pow(N_max, 2);
    const double max_error = 1e-6;

    REQUIRE(N_test <= N_max);

    hipStream_t stream;
    hipStreamCreate(&stream); 
    
    SEM::Device::Entities::NDG_t<SEM::Device::Polynomials::ChebyshevPolynomial_t> NDG(N_max, n_interpolation_points, stream);

    std::vector<SEM::Device::Entities::Element2D_t> host_elements(1);
    host_elements[0].N_ = N_test;

    device_vector<SEM::Device::Entities::Element2D_t> device_elements(host_elements, stream);

    elements_init<<<1, 1, 0, stream>>>(1, device_elements.data(), NDG.nodes_.data());

    SEM::Device::Meshes::interpolate_to_boundaries<<<1, 1, 0, stream>>>(1, device_elements.data(), NDG.lagrange_interpolant_left_.data(), NDG.lagrange_interpolant_right_.data());

    std::array<device_vector<deviceFloat>, 4> p {device_vector<deviceFloat>(N_test + 1, stream), 
                                                 device_vector<deviceFloat>(N_test + 1, stream), 
                                                 device_vector<deviceFloat>(N_test + 1, stream), 
                                                 device_vector<deviceFloat>(N_test + 1, stream)};
    std::array<device_vector<deviceFloat>, 4> u {device_vector<deviceFloat>(N_test + 1, stream), 
                                                 device_vector<deviceFloat>(N_test + 1, stream), 
                                                 device_vector<deviceFloat>(N_test + 1, stream), 
                                                 device_vector<deviceFloat>(N_test + 1, stream)};
    std::array<device_vector<deviceFloat>, 4> v {device_vector<deviceFloat>(N_test + 1, stream), 
                                                 device_vector<deviceFloat>(N_test + 1, stream), 
                                                 device_vector<deviceFloat>(N_test + 1, stream), 
                                                 device_vector<deviceFloat>(N_test + 1, stream)};
    get_boundary_solution<<<1, 1, 0, stream>>>(1, device_elements.data(), {p[0].data(), p[1].data(), p[2].data(), p[3].data()}, 
                                                                          {u[0].data(), u[1].data(), u[2].data(), u[3].data()}, 
                                                                          {v[0].data(), v[1].data(), v[2].data(), v[3].data()});

    // Getting computed values
    std::array<std::vector<deviceFloat>, 4> p_host {std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1)};
    std::array<std::vector<deviceFloat>, 4> u_host {std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1)};
    std::array<std::vector<deviceFloat>, 4> v_host {std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1)};
    for (size_t k = 0; k < p_host.size(); ++k) {
        p[k].copy_to(p_host[k], stream);
        u[k].copy_to(u_host[k], stream);
        v[k].copy_to(v_host[k], stream);
    }

    // Generating target values
    std::vector<deviceFloat> polynomial_nodes_host(NDG.nodes_.size());
    NDG.nodes_.copy_to(polynomial_nodes_host, stream);
    const size_t offset_1D = N_test * (N_test + 1) /2;

    std::array<std::vector<deviceFloat>, 4> p_target {std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1)};
    std::array<std::vector<deviceFloat>, 4> u_target {std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1)};
    std::array<std::vector<deviceFloat>, 4> v_target {std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1), std::vector<deviceFloat>(N_test + 1)};
    
    hipStreamSynchronize(stream);
    for (int i = 0; i <= N_test; ++i) {
        const std::array<Vec2<deviceFloat>, 4> coordinates {Vec2<deviceFloat>{polynomial_nodes_host[offset_1D + i], -1},
                                                            Vec2<deviceFloat>{1, polynomial_nodes_host[offset_1D + i]},
                                                            Vec2<deviceFloat>{polynomial_nodes_host[offset_1D + i], 1},
                                                            Vec2<deviceFloat>{-1, polynomial_nodes_host[offset_1D + i]}};
        const std::array<Vec2<deviceFloat>, 4> global_coordinates {SEM::Device::quad_map(coordinates[0], points),
                                                                   SEM::Device::quad_map(coordinates[1], points),
                                                                   SEM::Device::quad_map(coordinates[2], points),
                                                                   SEM::Device::quad_map(coordinates[3], points)};
                                                                
        p_target[0][i] = std::sin(global_coordinates[0][0]) * std::cos(global_coordinates[0][1]);
        u_target[0][i] = global_coordinates[0][0];
        v_target[0][i] = global_coordinates[0][1];

        p_target[1][i] = std::sin(global_coordinates[1][0]) * std::cos(global_coordinates[1][1]);
        u_target[1][i] = global_coordinates[1][0];
        v_target[1][i] = global_coordinates[1][1];

        p_target[2][N_test - i] = std::sin(global_coordinates[2][0]) * std::cos(global_coordinates[2][1]);
        u_target[2][N_test - i] = global_coordinates[2][0];
        v_target[2][N_test - i] = global_coordinates[2][1];

        p_target[3][N_test - i] = std::sin(global_coordinates[3][0]) * std::cos(global_coordinates[3][1]);
        u_target[3][N_test - i] = global_coordinates[3][0];
        v_target[3][N_test - i] = global_coordinates[3][1];
    }

    // Verifying values
    for (int i = 0; i <= N_test; ++i) {
        for (size_t k = 0; k < p_target.size(); ++k) {
            REQUIRE(std::abs(p_target[k][i] - p_host[k][i]) < max_error);
            REQUIRE(std::abs(u_target[k][i] - u_host[k][i]) < max_error);
            REQUIRE(std::abs(v_target[k][i] - v_host[k][i]) < max_error);
        }
    }

    device_elements.clear(stream);
    p[0].clear(stream);
    p[1].clear(stream);
    p[2].clear(stream);
    p[3].clear(stream);
    u[0].clear(stream);
    u[1].clear(stream);
    u[2].clear(stream);
    u[3].clear(stream);
    v[0].clear(stream);
    v[1].clear(stream);
    v[2].clear(stream);
    v[3].clear(stream);

    hipStreamDestroy(stream);
}