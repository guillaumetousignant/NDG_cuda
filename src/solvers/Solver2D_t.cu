#include "hip/hip_runtime.h"
#include "solvers/Solver2D_t.cuh"
#include "polynomials/ChebyshevPolynomial_t.cuh"
#include "polynomials/LegendrePolynomial_t.cuh"
#include "helpers/ProgressBar_t.h"
#include "helpers/constants.h"
#include <mpi.h>
#include <limits>

using SEM::Entities::device_vector;
using SEM::Entities::Vec2;
using SEM::Entities::Element2D_t;
using SEM::Entities::Face2D_t;

SEM::Solvers::Solver2D_t::Solver2D_t(deviceFloat CFL, std::vector<deviceFloat> output_times, deviceFloat viscosity) :
        CFL_{CFL},
        output_times_{output_times},
        viscosity_{viscosity} {}

template auto SEM::Solvers::Solver2D_t::solve(const SEM::Entities::NDG_t<SEM::Polynomials::ChebyshevPolynomial_t> &NDG, SEM::Meshes::Mesh2D_t& mesh, const SEM::Helpers::DataWriter_t& data_writer) -> void; // Get with the times c++, it's crazy I have to do this
template auto SEM::Solvers::Solver2D_t::solve(const SEM::Entities::NDG_t<SEM::Polynomials::LegendrePolynomial_t> &NDG, SEM::Meshes::Mesh2D_t& mesh, const SEM::Helpers::DataWriter_t& data_writer) -> void;

template<typename Polynomial>
auto SEM::Solvers::Solver2D_t::solve(const SEM::Entities::NDG_t<Polynomial> &NDG, SEM::Meshes::Mesh2D_t& mesh, const SEM::Helpers::DataWriter_t& data_writer) -> void {
    int global_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &global_rank);
    deviceFloat time = 0.0;
    const deviceFloat t_end = output_times_.back();
    SEM::Helpers::ProgressBar_t bar;
    size_t timestep = 0;

    deviceFloat delta_t = get_delta_t(mesh);
    if (global_rank == 0) {
        bar.set_status_text("Writing solution");
        bar.update(0.0);
    }
    mesh.write_data(time, NDG.N_interpolation_points_, NDG.interpolation_matrices_.data(), data_writer);
    if (global_rank == 0) {
        bar.set_status_text("Iteration 0");
        bar.update(0.0);
    }
    
    while (time < t_end) {
        ++timestep;
        delta_t = get_delta_t(mesh);
        if (time + delta_t > t_end) {
            delta_t = t_end - time;
        }

        // Kinda algorithm 62
        deviceFloat t = time;
        mesh.interpolate_to_boundaries(NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        mesh.boundary_conditions();
        mesh.project_to_boundaries();
        SEM::Solvers::calculate_wave_fluxes<<<mesh.faces_numBlocks_, mesh.faces_blockSize_, 0, mesh.stream_>>>(mesh.faces_.size(), mesh.faces_.data(), mesh.elements_.data());
        mesh.project_to_elements();
        SEM::Solvers::compute_dg_wave_derivative<<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), mesh.faces_.data(), NDG.weights_.data(), NDG.derivative_matrices_hat_.data(), NDG.lagrange_interpolant_left_.data(), NDG.lagrange_interpolant_right_.data());
        SEM::Solvers::rk3_first_step<<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), delta_t, 1.0/3.0);

        t = time + 0.33333333333f * delta_t;
        mesh.interpolate_to_boundaries(NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        mesh.boundary_conditions();
        mesh.project_to_boundaries();
        SEM::Solvers::calculate_wave_fluxes<<<mesh.faces_numBlocks_, mesh.faces_blockSize_, 0, mesh.stream_>>>(mesh.faces_.size(), mesh.faces_.data(), mesh.elements_.data());
        mesh.project_to_elements();
        SEM::Solvers::compute_dg_wave_derivative<<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), mesh.faces_.data(), NDG.weights_.data(), NDG.derivative_matrices_hat_.data(), NDG.lagrange_interpolant_left_.data(), NDG.lagrange_interpolant_right_.data());
        SEM::Solvers::rk3_step<<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), delta_t, -5.0/9.0, 15.0/16.0);

        t = time + 0.75f * delta_t;
        mesh.interpolate_to_boundaries(NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        mesh.boundary_conditions();
        mesh.project_to_boundaries();
        SEM::Solvers::calculate_wave_fluxes<<<mesh.faces_numBlocks_, mesh.faces_blockSize_, 0, mesh.stream_>>>(mesh.faces_.size(), mesh.faces_.data(), mesh.elements_.data());
        mesh.project_to_elements();
        SEM::Solvers::compute_dg_wave_derivative<<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), mesh.faces_.data(), NDG.weights_.data(), NDG.derivative_matrices_hat_.data(), NDG.lagrange_interpolant_left_.data(), NDG.lagrange_interpolant_right_.data());
        SEM::Solvers::rk3_step<<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), delta_t, -153.0/128.0, 8.0/15.0);
        
        time += delta_t;
        for (auto const& e : std::as_const(output_times_)) {
            if ((time >= e) && (time < e + delta_t)) {
                //SEM::Meshes::estimate_error<Polynomial><<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), NDG.nodes_.data(), NDG.weights_.data());
                if (global_rank == 0) {
                    bar.set_status_text("Writing solution");
                    bar.update(time/t_end);
                }
                mesh.write_data(time, NDG.N_interpolation_points_, NDG.interpolation_matrices_.data(), data_writer);
                break;
            }
        }
        if (global_rank == 0) {
            std::stringstream ss;
            ss << "Iteration " << timestep;
            bar.set_status_text(ss.str());
            bar.update(time/t_end);
        }

        if (timestep % mesh.adaptivity_interval_ == 0) {
            //SEM::Meshes::estimate_error<Polynomial><<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), NDG.nodes_.data(), NDG.weights_.data());
            mesh.adapt(NDG.N_max_, NDG.nodes_.data(), NDG.barycentric_weights_.data());
        }
    }

    bool did_write = false;
    for (auto const& e : std::as_const(output_times_)) {
        if ((time >= e) && (time < e + delta_t)) {
            did_write = true;
            break;
        }
    }

    if (!did_write) {
        //SEM::Meshes::estimate_error<Polynomial><<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), NDG.nodes_.data(), NDG.weights_.data());
        if (global_rank == 0) {
            bar.set_status_text("Writing solution");
            bar.update(1.0);
        }
        mesh.write_data(time, NDG.N_interpolation_points_, NDG.interpolation_matrices_.data(), data_writer);
    }
    if (global_rank == 0) {
        bar.set_status_text("Done");
        bar.update(1.0);
    }
    if (global_rank == 0) {
        std::cout << std::endl;
    }
}

auto SEM::Solvers::Solver2D_t::get_delta_t(SEM::Meshes::Mesh2D_t& mesh) -> deviceFloat {   
    SEM::Solvers::reduce_wave_delta_t<mesh.elements_blockSize_/2><<<mesh.elements_numBlocks_, mesh.elements_blockSize_/2, 0, mesh.stream_>>>(CFL_, mesh.N_elements_, mesh.elements_.data(), mesh.device_delta_t_array_.data());
    mesh.device_delta_t_array_.copy_to(mesh.host_delta_t_array_);

    deviceFloat delta_t_min_local = std::numeric_limits<deviceFloat>::infinity();
    for (int i = 0; i < mesh.elements_numBlocks_; ++i) {
        delta_t_min_local = min(delta_t_min_local, mesh.host_delta_t_array_[i]);
    }

    deviceFloat delta_t_min;
    constexpr MPI_Datatype data_type = (sizeof(deviceFloat) == sizeof(float)) ? MPI_FLOAT : MPI_DOUBLE;
    MPI_Allreduce(&delta_t_min_local, &delta_t_min, 1, data_type, MPI_MIN, MPI_COMM_WORLD);
    return delta_t_min;
}

__host__ __device__
auto SEM::Solvers::Solver2D_t::x_flux(deviceFloat p, deviceFloat u, deviceFloat v) -> std::array<deviceFloat, 3> {
    return {SEM::Constants::c * u, p, 0};
}

__host__ __device__
auto SEM::Solvers::Solver2D_t::y_flux(deviceFloat p, deviceFloat u, deviceFloat v) -> std::array<deviceFloat, 3> {
    return {SEM::Constants::c * v, 0, p};
}

__device__
void SEM::Solvers::Solver2D_t::matrix_vector_multiply(int N, const deviceFloat* matrix, const deviceFloat* vector, deviceFloat* result) {
    for (int i = 0; i <= N; ++i) {
        result[i] = 0.0;
        for (int j = 0; j <= N; ++j) {
            result[i] +=  matrix[i * (N + 1) + j] * vector[j];
        }
    }
}

__global__
auto SEM::Solvers::calculate_wave_fluxes(size_t N_faces, Face2D_t* faces, const Element2D_t* elements) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t face_index = index; face_index < N_faces; face_index += stride) {
        Face2D_t& face = faces[face_index];

        // Computing fluxes
        for (int i = 0; i <= face.N_; ++i) {
            const Vec2<deviceFloat> u_L {face.u_[0][i], face.v_[0][i]};
            const Vec2<deviceFloat> u_R {face.u_[1][i], face.v_[1][i]};
            const Vec2<deviceFloat> u_prime_L {u_L.dot(face.normal_), u_L.dot(face.tangent_)};
            const Vec2<deviceFloat> u_prime_R {u_R.dot(face.normal_), u_R.dot(face.tangent_)};

            const deviceFloat w_L = (face.p_[0][i] + SEM::Constants::c * u_prime_L.x()) / 2;
            const deviceFloat w_R = (face.p_[1][i] - SEM::Constants::c * u_prime_R.x()) / 2;

            const Vec2<deviceFloat> normal_inv {face.normal_[0], face.tangent_[0]};
            const Vec2<deviceFloat> tangent_inv {face.normal_[1], face.tangent_[1]};

            const Vec2<deviceFloat> velocity_flux {w_L + w_R, 0};

            face.p_flux_[i] = SEM::Constants::c * (w_L - w_R);
            face.u_flux_[i] = velocity_flux.dot(normal_inv);
            face.v_flux_[i] = velocity_flux.dot(tangent_inv);
        }
    }
}

// Algorithm 114
__global__
auto SEM::Solvers::compute_dg_wave_derivative(size_t N_elements, Element2D_t* elements, const Face2D_t* faces, const deviceFloat* weights, const deviceFloat* derivative_matrices_hat, const deviceFloat* lagrange_interpolant_left, const deviceFloat* lagrange_interpolant_right) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index; element_index < N_elements; element_index += stride) {
        Element2D_t& element = elements[element_index];
        const size_t offset_1D = element.N_ * (element.N_ + 1) /2; // CHECK cache?
        const size_t offset_2D = element.N_ * (element.N_ + 1) * (2 * element.N_ + 1) /6;

        for (int i = 0; i <= element.N_; ++i) {
            for (int j = 0; j <= element.N_; ++j) {
                const std::array<deviceFloat, 3> flux_x = SEM::Solvers::Solver2D_t::x_flux(element.p_[i * (element.N_ + 1) + j], element.u_[i * (element.N_ + 1) + j], element.v_[i * (element.N_ + 1) + j]);
                const std::array<deviceFloat, 3> flux_y = SEM::Solvers::Solver2D_t::y_flux(element.p_[i * (element.N_ + 1) + j], element.u_[i * (element.N_ + 1) + j], element.v_[i * (element.N_ + 1) + j]);
            
                element.p_flux_[j] = element.deta_dy_[i * (element.N_ + 1) + j] * flux_x[0] - element.deta_dx_[i * (element.N_ + 1) + j] * flux_y[0];
                element.u_flux_[j] = element.deta_dy_[i * (element.N_ + 1) + j] * flux_x[1] - element.deta_dx_[i * (element.N_ + 1) + j] * flux_y[1];
                element.v_flux_[j] = element.deta_dy_[i * (element.N_ + 1) + j] * flux_x[2] - element.deta_dx_[i * (element.N_ + 1) + j] * flux_y[2];
            }

            SEM::Solvers::Solver2D_t::matrix_vector_multiply(element.N_, derivative_matrices_hat + offset_2D, element.p_flux_.data(), element.p_flux_derivative_.data());
            SEM::Solvers::Solver2D_t::matrix_vector_multiply(element.N_, derivative_matrices_hat + offset_2D, element.u_flux_.data(), element.u_flux_derivative_.data());
            SEM::Solvers::Solver2D_t::matrix_vector_multiply(element.N_, derivative_matrices_hat + offset_2D, element.v_flux_.data(), element.v_flux_derivative_.data());

            for (int j = 0; j <= element.N_; ++j) {
                element.p_flux_derivative_[j] += (element.p_flux_extrapolated_[1][j] * lagrange_interpolant_right[offset_1D + j] + element.p_flux_extrapolated_[3][j] * lagrange_interpolant_left[offset_1D + j]) / weights[offset_1D + j];
                element.u_flux_derivative_[j] += (element.u_flux_extrapolated_[1][j] * lagrange_interpolant_right[offset_1D + j] + element.u_flux_extrapolated_[3][j] * lagrange_interpolant_left[offset_1D + j]) / weights[offset_1D + j];
                element.v_flux_derivative_[j] += (element.v_flux_extrapolated_[1][j] * lagrange_interpolant_right[offset_1D + j] + element.v_flux_extrapolated_[3][j] * lagrange_interpolant_left[offset_1D + j]) / weights[offset_1D + j];
            }

            for (int j = 0; j <= element.N_; ++j) {
                element.G_p_[i * (element.N_ + 1) + j] = -element.p_flux_derivative_[j];
                element.G_u_[i * (element.N_ + 1) + j] = -element.u_flux_derivative_[j];
                element.G_v_[i * (element.N_ + 1) + j] = -element.v_flux_derivative_[j];
            }
        }

        for (int j = 0; j <= element.N_; ++j) {
            for (int i = 0; i <= element.N_; ++i) {
                const std::array<deviceFloat, 3> flux_x = SEM::Solvers::Solver2D_t::x_flux(element.p_[i * (element.N_ + 1) + j], element.u_[i * (element.N_ + 1) + j], element.v_[i * (element.N_ + 1) + j]);
                const std::array<deviceFloat, 3> flux_y = SEM::Solvers::Solver2D_t::y_flux(element.p_[i * (element.N_ + 1) + j], element.u_[i * (element.N_ + 1) + j], element.v_[i * (element.N_ + 1) + j]);

                element.p_flux_[i] = -element.dxi_dy_[i * (element.N_ + 1) + j] * flux_x[0] + element.dxi_dx_[i * (element.N_ + 1) + j] * flux_y[0];
                element.u_flux_[i] = -element.dxi_dy_[i * (element.N_ + 1) + j] * flux_x[1] + element.dxi_dx_[i * (element.N_ + 1) + j] * flux_y[1];
                element.v_flux_[i] = -element.dxi_dy_[i * (element.N_ + 1) + j] * flux_x[2] + element.dxi_dx_[i * (element.N_ + 1) + j] * flux_y[2];
            }

            SEM::Solvers::Solver2D_t::matrix_vector_multiply(element.N_, derivative_matrices_hat + offset_2D, element.p_flux_.data(), element.p_flux_derivative_.data());
            SEM::Solvers::Solver2D_t::matrix_vector_multiply(element.N_, derivative_matrices_hat + offset_2D, element.u_flux_.data(), element.u_flux_derivative_.data());
            SEM::Solvers::Solver2D_t::matrix_vector_multiply(element.N_, derivative_matrices_hat + offset_2D, element.v_flux_.data(), element.v_flux_derivative_.data());

            for (int i = 0; i <= element.N_; ++i) {
                element.p_flux_derivative_[i] += (element.p_flux_extrapolated_[2][i] * lagrange_interpolant_right[offset_1D + i] + element.p_flux_extrapolated_[0][i] * lagrange_interpolant_left[offset_1D + i]) / weights[offset_1D + i];
                element.u_flux_derivative_[i] += (element.u_flux_extrapolated_[2][i] * lagrange_interpolant_right[offset_1D + i] + element.u_flux_extrapolated_[0][i] * lagrange_interpolant_left[offset_1D + i]) / weights[offset_1D + i];
                element.v_flux_derivative_[i] += (element.v_flux_extrapolated_[2][i] * lagrange_interpolant_right[offset_1D + i] + element.v_flux_extrapolated_[0][i] * lagrange_interpolant_left[offset_1D + i]) / weights[offset_1D + i];
            }

            for (int i = 0; i <= element.N_; ++i) {
                element.G_p_[i * (element.N_ + 1) + j] = element.G_p_[i * (element.N_ + 1) + j] - element.p_flux_derivative_[i];
                element.G_u_[i * (element.N_ + 1) + j] = element.G_u_[i * (element.N_ + 1) + j] - element.u_flux_derivative_[i];
                element.G_v_[i * (element.N_ + 1) + j] = element.G_v_[i * (element.N_ + 1) + j] - element.v_flux_derivative_[i];
            }
        }
    }
}

__global__
auto SEM::Solvers::rk3_first_step(size_t N_elements, SEM::Entities::Element2D_t* elements, deviceFloat delta_t, deviceFloat g) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index; element_index < N_elements; element_index += stride) {
        Element2D_t& element = elements[element_index];

        for (int i = 0; i <= element.N_; ++i){
            for (int j = 0; j <= element.N_; ++j){
                element.p_intermediate_[i * (element.N_ + 1) + j] = element.G_p_[i * (element.N_ + 1) + j];
                element.u_intermediate_[i * (element.N_ + 1) + j] = element.G_u_[i * (element.N_ + 1) + j];
                element.v_intermediate_[i * (element.N_ + 1) + j] = element.G_v_[i * (element.N_ + 1) + j];

                element.p_[i * (element.N_ + 1) + j] += g * delta_t * element.p_intermediate_[i * (element.N_ + 1) + j];
                element.u_[i * (element.N_ + 1) + j] += g * delta_t * element.u_intermediate_[i * (element.N_ + 1) + j];
                element.v_[i * (element.N_ + 1) + j] += g * delta_t * element.v_intermediate_[i * (element.N_ + 1) + j];
            }
        }
    }
}

__global__
auto SEM::Solvers::rk3_step(size_t N_elements, SEM::Entities::Element2D_t* elements, deviceFloat delta_t, deviceFloat a, deviceFloat g) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index; element_index < N_elements; element_index += stride) {
        Element2D_t& element = elements[element_index];

        for (int i = 0; i <= element.N_; ++i){
            for (int j = 0; j <= element.N_; ++j){
                element.p_intermediate_[i * (element.N_ + 1) + j] = a * element.p_intermediate_[i * (element.N_ + 1) + j] + element.G_p_[i * (element.N_ + 1) + j];
                element.u_intermediate_[i * (element.N_ + 1) + j] = a * element.u_intermediate_[i * (element.N_ + 1) + j] + element.G_u_[i * (element.N_ + 1) + j];
                element.v_intermediate_[i * (element.N_ + 1) + j] = a * element.v_intermediate_[i * (element.N_ + 1) + j] + element.G_v_[i * (element.N_ + 1) + j];

                element.p_[i * (element.N_ + 1) + j] += g * delta_t * element.p_intermediate_[i * (element.N_ + 1) + j];
                element.u_[i * (element.N_ + 1) + j] += g * delta_t * element.u_intermediate_[i * (element.N_ + 1) + j];
                element.v_[i * (element.N_ + 1) + j] += g * delta_t * element.v_intermediate_[i * (element.N_ + 1) + j];
            }
        }
    }
}
