#include "hip/hip_runtime.h"
#include "solvers/Solver2D_t.cuh"
#include "polynomials/ChebyshevPolynomial_t.cuh"
#include "polynomials/LegendrePolynomial_t.cuh"
#include "helpers/ProgressBar_t.h"
#include "helpers/constants.h"
#include <mpi.h>
#include <limits>

using SEM::Entities::device_vector;
using SEM::Entities::Vec2;
using SEM::Entities::Element2D_t;
using SEM::Entities::Face2D_t;

SEM::Solvers::Solver2D_t::Solver2D_t(deviceFloat CFL, std::vector<deviceFloat> output_times, deviceFloat viscosity) :
        CFL_{CFL},
        output_times_{output_times},
        viscosity_{viscosity} {}

template auto SEM::Solvers::Solver2D_t::solve(const SEM::Entities::NDG_t<SEM::Polynomials::ChebyshevPolynomial_t> &NDG, SEM::Meshes::Mesh2D_t& mesh, const SEM::Helpers::DataWriter_t& data_writer) const -> void; // Get with the times c++, it's crazy I have to do this
template auto SEM::Solvers::Solver2D_t::solve(const SEM::Entities::NDG_t<SEM::Polynomials::LegendrePolynomial_t> &NDG, SEM::Meshes::Mesh2D_t& mesh, const SEM::Helpers::DataWriter_t& data_writer) const -> void;

template<typename Polynomial>
auto SEM::Solvers::Solver2D_t::solve(const SEM::Entities::NDG_t<Polynomial> &NDG, SEM::Meshes::Mesh2D_t& mesh, const SEM::Helpers::DataWriter_t& data_writer) const -> void {
    int global_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &global_rank);
    deviceFloat time = 0.0;
    const deviceFloat t_end = output_times_.back();
    SEM::Helpers::ProgressBar_t bar;
    size_t timestep = 0;

    deviceFloat delta_t = get_delta_t(mesh);
    if (global_rank == 0) {
        bar.set_status_text("Writing solution");
        bar.update(0.0);
    }
    mesh.write_data(time, NDG.N_interpolation_points_, NDG.interpolation_matrices_.data(), data_writer);
    if (global_rank == 0) {
        bar.set_status_text("Iteration 0");
        bar.update(0.0);
    }
    
    while (time < t_end) {
        ++timestep;
        delta_t = get_delta_t(mesh);
        if (time + delta_t > t_end) {
            delta_t = t_end - time;
        }

        // Kinda algorithm 62
        deviceFloat t = time;
        mesh.interpolate_to_boundaries(NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        mesh.boundary_conditions();
        mesh.project_to_faces();
        SEM::Solvers::calculate_wave_fluxes<<<mesh.faces_numBlocks_, mesh.faces_blockSize_, 0, mesh.stream_>>>(mesh.faces_.size(), mesh.faces_.data(), mesh.elements_.data());
        mesh.project_to_elements();
        SEM::Solvers::compute_dg_wave_derivative<<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), mesh.faces_.data(), NDG.weights_.data(), NDG.derivative_matrices_hat_.data(), NDG.lagrange_interpolant_left_.data(), NDG.lagrange_interpolant_right_.data());
        SEM::Solvers::rk3_first_step<<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), delta_t, 1.0/3.0);

        t = time + 0.33333333333f * delta_t;
        mesh.interpolate_to_boundaries(NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        mesh.boundary_conditions();
        mesh.project_to_faces();
        SEM::Solvers::calculate_wave_fluxes<<<mesh.faces_numBlocks_, mesh.faces_blockSize_, 0, mesh.stream_>>>(mesh.faces_.size(), mesh.faces_.data(), mesh.elements_.data());
        mesh.project_to_elements();
        SEM::Solvers::compute_dg_wave_derivative<<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), mesh.faces_.data(), NDG.weights_.data(), NDG.derivative_matrices_hat_.data(), NDG.lagrange_interpolant_left_.data(), NDG.lagrange_interpolant_right_.data());
        SEM::Solvers::rk3_step<<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), delta_t, -5.0/9.0, 15.0/16.0);

        t = time + 0.75f * delta_t;
        mesh.interpolate_to_boundaries(NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        mesh.boundary_conditions();
        mesh.project_to_faces();
        SEM::Solvers::calculate_wave_fluxes<<<mesh.faces_numBlocks_, mesh.faces_blockSize_, 0, mesh.stream_>>>(mesh.faces_.size(), mesh.faces_.data(), mesh.elements_.data());
        mesh.project_to_elements();
        SEM::Solvers::compute_dg_wave_derivative<<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), mesh.faces_.data(), NDG.weights_.data(), NDG.derivative_matrices_hat_.data(), NDG.lagrange_interpolant_left_.data(), NDG.lagrange_interpolant_right_.data());
        SEM::Solvers::rk3_step<<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), delta_t, -153.0/128.0, 8.0/15.0);
        
        time += delta_t;
        for (auto const& e : std::as_const(output_times_)) {
            if ((time >= e) && (time < e + delta_t)) {
                //SEM::Meshes::estimate_error<Polynomial><<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), NDG.nodes_.data(), NDG.weights_.data());
                if (global_rank == 0) {
                    bar.set_status_text("Writing solution");
                    bar.update(time/t_end);
                }
                mesh.write_data(time, NDG.N_interpolation_points_, NDG.interpolation_matrices_.data(), data_writer);
                break;
            }
        }
        if (global_rank == 0) {
            std::stringstream ss;
            ss << "Iteration " << timestep;
            bar.set_status_text(ss.str());
            bar.update(time/t_end);
        }

        if (timestep % mesh.adaptivity_interval_ == 0) {
            //SEM::Meshes::estimate_error<Polynomial><<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), NDG.nodes_.data(), NDG.weights_.data());
            mesh.adapt(NDG.N_max_, NDG.nodes_.data(), NDG.barycentric_weights_.data());
        }
    }

    bool did_write = false;
    for (auto const& e : std::as_const(output_times_)) {
        if ((time >= e) && (time < e + delta_t)) {
            did_write = true;
            break;
        }
    }

    if (!did_write) {
        //SEM::Meshes::estimate_error<Polynomial><<<mesh.elements_numBlocks_, mesh.elements_blockSize_, 0, mesh.stream_>>>(mesh.N_elements_, mesh.elements_.data(), NDG.nodes_.data(), NDG.weights_.data());
        if (global_rank == 0) {
            bar.set_status_text("Writing solution");
            bar.update(1.0);
        }
        mesh.write_data(time, NDG.N_interpolation_points_, NDG.interpolation_matrices_.data(), data_writer);
    }
    if (global_rank == 0) {
        bar.set_status_text("Done");
        bar.update(1.0);
    }
    if (global_rank == 0) {
        std::cout << std::endl;
    }
}

auto SEM::Solvers::Solver2D_t::get_delta_t(SEM::Meshes::Mesh2D_t& mesh) const -> deviceFloat {   
    SEM::Solvers::reduce_wave_delta_t<mesh.elements_blockSize_/2><<<mesh.elements_numBlocks_, mesh.elements_blockSize_/2, 0, mesh.stream_>>>(CFL_, mesh.N_elements_, mesh.elements_.data(), mesh.device_delta_t_array_.data());
    mesh.device_delta_t_array_.copy_to(mesh.host_delta_t_array_);

    deviceFloat delta_t_min_local = std::numeric_limits<deviceFloat>::infinity();
    for (int i = 0; i < mesh.elements_numBlocks_; ++i) {
        delta_t_min_local = min(delta_t_min_local, mesh.host_delta_t_array_[i]);
    }

    deviceFloat delta_t_min;
    constexpr MPI_Datatype data_type = (sizeof(deviceFloat) == sizeof(float)) ? MPI_FLOAT : MPI_DOUBLE;
    MPI_Allreduce(&delta_t_min_local, &delta_t_min, 1, data_type, MPI_MIN, MPI_COMM_WORLD);
    return delta_t_min;
}

__host__ __device__
auto SEM::Solvers::Solver2D_t::x_flux(deviceFloat p, deviceFloat u, deviceFloat v) -> std::array<deviceFloat, 3> {
    return {SEM::Constants::c * u, p, 0};
}

__host__ __device__
auto SEM::Solvers::Solver2D_t::y_flux(deviceFloat p, deviceFloat u, deviceFloat v) -> std::array<deviceFloat, 3> {
    return {SEM::Constants::c * v, 0, p};
}

__device__
void SEM::Solvers::Solver2D_t::matrix_vector_multiply(int N, const deviceFloat* matrix, const deviceFloat* vector, deviceFloat* result) {
    for (int i = 0; i <= N; ++i) {
        result[i] = 0.0;
        for (int j = 0; j <= N; ++j) {
            result[i] +=  matrix[i * (N + 1) + j] * vector[j];
        }
    }
}

__global__
auto SEM::Solvers::calculate_wave_fluxes(size_t N_faces, Face2D_t* faces, const Element2D_t* elements) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t face_index = index; face_index < N_faces; face_index += stride) {
        Face2D_t& face = faces[face_index];

        // Computing fluxes
        for (int i = 0; i <= face.N_; ++i) {
            const Vec2<deviceFloat> u_L {face.u_[0][i], face.v_[0][i]};
            const Vec2<deviceFloat> u_R {face.u_[1][i], face.v_[1][i]};

            const deviceFloat w_L = face.p_[0][i] + SEM::Constants::c * u_L.dot(face.normal_);
            const deviceFloat w_R = face.p_[1][i] - SEM::Constants::c * u_R.dot(face.normal_);

            face.p_flux_[i] = SEM::Constants::c * (w_L - w_R) / 2;
            face.u_flux_[i] = face.normal_.x() * (w_L + w_R) / 2;
            face.v_flux_[i] = face.normal_.y() * (w_L + w_R) / 2;
        }
    }
}

// Algorithm 114
__global__
auto SEM::Solvers::compute_dg_wave_derivative(size_t N_elements, Element2D_t* elements, const Face2D_t* faces, const deviceFloat* weights, const deviceFloat* derivative_matrices_hat, const deviceFloat* lagrange_interpolant_left, const deviceFloat* lagrange_interpolant_right) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index; element_index < N_elements; element_index += stride) {
        Element2D_t& element = elements[element_index];
        const size_t offset_1D = element.N_ * (element.N_ + 1) /2; // CHECK cache?
        const size_t offset_2D = element.N_ * (element.N_ + 1) * (2 * element.N_ + 1) /6;

        // Horizontal direction
        for (int j = 0; j <= element.N_; ++j) {
            for (int i = 0; i <= element.N_; ++i) {
                const std::array<deviceFloat, 3> flux_x = SEM::Solvers::Solver2D_t::x_flux(element.p_[i * (element.N_ + 1) + j], element.u_[i * (element.N_ + 1) + j], element.v_[i * (element.N_ + 1) + j]);
                const std::array<deviceFloat, 3> flux_y = SEM::Solvers::Solver2D_t::y_flux(element.p_[i * (element.N_ + 1) + j], element.u_[i * (element.N_ + 1) + j], element.v_[i * (element.N_ + 1) + j]);
            
                element.p_flux_[i] = element.deta_dy_[i * (element.N_ + 1) + j] * flux_x[0] - element.deta_dx_[i * (element.N_ + 1) + j] * flux_y[0];
                element.u_flux_[i] = element.deta_dy_[i * (element.N_ + 1) + j] * flux_x[1] - element.deta_dx_[i * (element.N_ + 1) + j] * flux_y[1];
                element.v_flux_[i] = element.deta_dy_[i * (element.N_ + 1) + j] * flux_x[2] - element.deta_dx_[i * (element.N_ + 1) + j] * flux_y[2];
            }

            SEM::Solvers::Solver2D_t::matrix_vector_multiply(element.N_, derivative_matrices_hat + offset_2D, element.p_flux_.data(), element.p_flux_derivative_.data());
            SEM::Solvers::Solver2D_t::matrix_vector_multiply(element.N_, derivative_matrices_hat + offset_2D, element.u_flux_.data(), element.u_flux_derivative_.data());
            SEM::Solvers::Solver2D_t::matrix_vector_multiply(element.N_, derivative_matrices_hat + offset_2D, element.v_flux_.data(), element.v_flux_derivative_.data());

            // For the boundaries, the numbering increases from the first node to the second. 
            // Inside the element, the ksi and eta coordinates increase from left to right, bottom to top.
            // This means that there is an inconsistency on the top and left edges, and the numbering has to be reversed.
            // This way, the projection from the element edge to the face(s) can always be done in the same way.
            // The same process has to be done when interpolating to the boundaries.
            for (int i = 0; i <= element.N_; ++i) {
                element.p_flux_derivative_[i] += (element.p_flux_extrapolated_[1][j] * lagrange_interpolant_right[offset_1D + i] + element.p_flux_extrapolated_[3][element.N_ - j] * lagrange_interpolant_left[offset_1D + i]) / weights[offset_1D + i];
                element.u_flux_derivative_[i] += (element.u_flux_extrapolated_[1][j] * lagrange_interpolant_right[offset_1D + i] + element.u_flux_extrapolated_[3][element.N_ - j] * lagrange_interpolant_left[offset_1D + i]) / weights[offset_1D + i];
                element.v_flux_derivative_[i] += (element.v_flux_extrapolated_[1][j] * lagrange_interpolant_right[offset_1D + i] + element.v_flux_extrapolated_[3][element.N_ - j] * lagrange_interpolant_left[offset_1D + i]) / weights[offset_1D + i];
            }

            for (int i = 0; i <= element.N_; ++i) {
                element.G_p_[i * (element.N_ + 1) + j] = -element.p_flux_derivative_[i];
                element.G_u_[i * (element.N_ + 1) + j] = -element.u_flux_derivative_[i];
                element.G_v_[i * (element.N_ + 1) + j] = -element.v_flux_derivative_[i];
            }
        }

        // Vertical direction
        for (int i = 0; i <= element.N_; ++i) {
            for (int j = 0; j <= element.N_; ++j) {
                const std::array<deviceFloat, 3> flux_x = SEM::Solvers::Solver2D_t::x_flux(element.p_[i * (element.N_ + 1) + j], element.u_[i * (element.N_ + 1) + j], element.v_[i * (element.N_ + 1) + j]);
                const std::array<deviceFloat, 3> flux_y = SEM::Solvers::Solver2D_t::y_flux(element.p_[i * (element.N_ + 1) + j], element.u_[i * (element.N_ + 1) + j], element.v_[i * (element.N_ + 1) + j]);

                element.p_flux_[j] = -element.dxi_dy_[i * (element.N_ + 1) + j] * flux_x[0] + element.dxi_dx_[i * (element.N_ + 1) + j] * flux_y[0];
                element.u_flux_[j] = -element.dxi_dy_[i * (element.N_ + 1) + j] * flux_x[1] + element.dxi_dx_[i * (element.N_ + 1) + j] * flux_y[1];
                element.v_flux_[j] = -element.dxi_dy_[i * (element.N_ + 1) + j] * flux_x[2] + element.dxi_dx_[i * (element.N_ + 1) + j] * flux_y[2];
            }

            SEM::Solvers::Solver2D_t::matrix_vector_multiply(element.N_, derivative_matrices_hat + offset_2D, element.p_flux_.data(), element.p_flux_derivative_.data());
            SEM::Solvers::Solver2D_t::matrix_vector_multiply(element.N_, derivative_matrices_hat + offset_2D, element.u_flux_.data(), element.u_flux_derivative_.data());
            SEM::Solvers::Solver2D_t::matrix_vector_multiply(element.N_, derivative_matrices_hat + offset_2D, element.v_flux_.data(), element.v_flux_derivative_.data());

            // For the boundaries, the numbering increases from the first node to the second. 
            // Inside the element, the ksi and eta coordinates increase from left to right, bottom to top.
            // This means that there is an inconsistency on the top and left edges, and the numbering has to be reversed.
            // This way, the projection from the element edge to the face(s) can always be done in the same way.
            // The same process has to be done when interpolating to the boundaries.
            for (int j = 0; j <= element.N_; ++j) {
                element.p_flux_derivative_[j] += (element.p_flux_extrapolated_[2][element.N_ - i] * lagrange_interpolant_right[offset_1D + j] + element.p_flux_extrapolated_[0][i] * lagrange_interpolant_left[offset_1D + j]) / weights[offset_1D + j];
                element.u_flux_derivative_[j] += (element.u_flux_extrapolated_[2][element.N_ - i] * lagrange_interpolant_right[offset_1D + j] + element.u_flux_extrapolated_[0][i] * lagrange_interpolant_left[offset_1D + j]) / weights[offset_1D + j];
                element.v_flux_derivative_[j] += (element.v_flux_extrapolated_[2][element.N_ - i] * lagrange_interpolant_right[offset_1D + j] + element.v_flux_extrapolated_[0][i] * lagrange_interpolant_left[offset_1D + j]) / weights[offset_1D + j];
            }

            for (int j = 0; j <= element.N_; ++j) {
                element.G_p_[i * (element.N_ + 1) + j] = (element.G_p_[i * (element.N_ + 1) + j] - element.p_flux_derivative_[j]) / element.jacobian_[i * (element.N_ + 1) + j];
                element.G_u_[i * (element.N_ + 1) + j] = (element.G_u_[i * (element.N_ + 1) + j] - element.u_flux_derivative_[j]) / element.jacobian_[i * (element.N_ + 1) + j];
                element.G_v_[i * (element.N_ + 1) + j] = (element.G_v_[i * (element.N_ + 1) + j] - element.v_flux_derivative_[j]) / element.jacobian_[i * (element.N_ + 1) + j];
            }
        }
    }
}

__global__
auto SEM::Solvers::rk3_first_step(size_t N_elements, SEM::Entities::Element2D_t* elements, deviceFloat delta_t, deviceFloat g) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index; element_index < N_elements; element_index += stride) {
        Element2D_t& element = elements[element_index];

        for (int i = 0; i <= element.N_; ++i){
            for (int j = 0; j <= element.N_; ++j){
                element.p_intermediate_[i * (element.N_ + 1) + j] = element.G_p_[i * (element.N_ + 1) + j];
                element.u_intermediate_[i * (element.N_ + 1) + j] = element.G_u_[i * (element.N_ + 1) + j];
                element.v_intermediate_[i * (element.N_ + 1) + j] = element.G_v_[i * (element.N_ + 1) + j];

                element.p_[i * (element.N_ + 1) + j] += g * delta_t * element.p_intermediate_[i * (element.N_ + 1) + j];
                element.u_[i * (element.N_ + 1) + j] += g * delta_t * element.u_intermediate_[i * (element.N_ + 1) + j];
                element.v_[i * (element.N_ + 1) + j] += g * delta_t * element.v_intermediate_[i * (element.N_ + 1) + j];
            }
        }
    }
}

__global__
auto SEM::Solvers::rk3_step(size_t N_elements, SEM::Entities::Element2D_t* elements, deviceFloat delta_t, deviceFloat a, deviceFloat g) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index; element_index < N_elements; element_index += stride) {
        Element2D_t& element = elements[element_index];

        for (int i = 0; i <= element.N_; ++i){
            for (int j = 0; j <= element.N_; ++j){
                element.p_intermediate_[i * (element.N_ + 1) + j] = a * element.p_intermediate_[i * (element.N_ + 1) + j] + element.G_p_[i * (element.N_ + 1) + j];
                element.u_intermediate_[i * (element.N_ + 1) + j] = a * element.u_intermediate_[i * (element.N_ + 1) + j] + element.G_u_[i * (element.N_ + 1) + j];
                element.v_intermediate_[i * (element.N_ + 1) + j] = a * element.v_intermediate_[i * (element.N_ + 1) + j] + element.G_v_[i * (element.N_ + 1) + j];

                element.p_[i * (element.N_ + 1) + j] += g * delta_t * element.p_intermediate_[i * (element.N_ + 1) + j];
                element.u_[i * (element.N_ + 1) + j] += g * delta_t * element.u_intermediate_[i * (element.N_ + 1) + j];
                element.v_[i * (element.N_ + 1) + j] += g * delta_t * element.v_intermediate_[i * (element.N_ + 1) + j];
            }
        }
    }
}
