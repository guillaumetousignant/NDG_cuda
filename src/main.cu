
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream> 
#include <iomanip>
#include <cmath>
#include <limits>
#include <chrono>
#include <cfloat>
#include <filesystem>
#include <vector>

namespace fs = std::filesystem;

constexpr float pi = 3.14159265358979323846f;
constexpr float c = 5.0f;
constexpr int poly_blockSize = 16; // Small number of threads per block because N will never be huge
constexpr int elements_blockSize = 32; // For when we'll have multiple elements
constexpr int faces_blockSize = 32; // Same number of faces as elements for periodic BC
constexpr int interpolation_blockSize = 32;
const dim3 matrix_blockSize(16, 16); // Small number of threads per block because N will never be huge

// Algorithm 26
__global__
void chebyshev_gauss_nodes_and_weights(int N, float* nodes, float* weights) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int offset = N * (N + 1) /2;

    for (int i = index; i <= N; i += stride) {
        nodes[offset + i] = -cos(pi * (2 * i + 1) / (2 * N + 2));
        weights[offset + i] = pi / (N + 1);
    }
}

// Algorithm 22
__device__
void legendre_polynomial_and_derivative(int N, float x, float &L_N, float &L_N_prime) {
    if (N == 0) {
        L_N = 1.0f;
        L_N_prime = 0.0f;
    }
    else if (N == 1) {
        L_N = x;
        L_N_prime = 1.0f;
    }
    else {
        float L_N_2 = 1.0f;
        float L_N_1 = x;
        float L_N_2_prime = 0.0f;
        float L_N_1_prime = 1.0f;

        for (int k = 2; k <= N; ++k) {
            L_N = (2 * k - 1) * x * L_N_1/k - (k - 1) * L_N_2/k; // L_N_1(x) ??
            L_N_prime = L_N_2_prime + (2 * k - 1) * L_N_1;
            L_N_2 = L_N_1;
            L_N_1 = L_N;
            L_N_2_prime = L_N_1_prime;
            L_N_1_prime = L_N_prime;
        }
    }
}

// Algorithm 23
__global__
void legendre_gauss_nodes_and_weights(int N, float* nodes, float* weights) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int offset = N * (N + 1) /2;

    if (index == 0) {
        if (N == 0) {
            nodes[offset] = 0.0f;
            weights[offset] = 2.0f;
        }
        else if (N == 1) {
            nodes[offset] = -std::sqrt(1.0f/3.0f);
            weights[offset] = 1.0f;
            nodes[offset + 1] = -nodes[offset];
            weights[offset + 1] = weights[offset];
        }
        else {
            for (int j = 0; j < (N + 1)/2; ++j) {
                nodes[offset + j] = -std::cos(pi * (2 * j + 1)/(2 * N + 2));
                
                for (int k = 0; k < 1000; ++k) {
                    float L_N_plus1, L_N_plus1_prime;
                    legendre_polynomial_and_derivative(N + 1, nodes[offset + j], L_N_plus1, L_N_plus1_prime);
                    float delta = -L_N_plus1/L_N_plus1_prime;
                    nodes[offset + j] += delta;
                    if (std::abs(delta) <= 0.00000001f * std::abs(nodes[offset + j])) {
                        break;
                    }

                }

                float dummy, L_N_plus1_prime_final;
                legendre_polynomial_and_derivative(N + 1, nodes[offset + j], dummy, L_N_plus1_prime_final);
                nodes[offset + N - j] = -nodes[offset + j];
                weights[offset + j] = 2.0f/((1.0f - std::pow(nodes[offset + j], 2)) * std::pow(L_N_plus1_prime_final, 2));
                weights[offset + N - j] = weights[offset + j];
            }
        }
    }

    if (N % 2 == 0) {
        float dummy, L_N_plus1_prime_final;
        legendre_polynomial_and_derivative(N + 1, 0.0f, dummy, L_N_plus1_prime_final);
        nodes[offset + N/2] = 0.0f;
        weights[offset + N/2] = 2/std::pow(L_N_plus1_prime_final, 2);
    }
}

// Algorithm 30
__global__
void calculate_barycentric_weights(int N, const float* nodes, float* barycentric_weights) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int offset = N * (N + 1) /2;

    for (int j = index; j <= N; j += stride) {
        float xjxi = 1.0f;
        for (int i = 0; i < j; ++i) {
            xjxi *= nodes[offset + j] - nodes[offset + i];
        }
        for (int i = j + 1; i <= N; ++i) {
            xjxi *= nodes[offset + j] - nodes[offset + i];
        }

        barycentric_weights[offset + j] = 1.0f/xjxi;
    }
}

/*__device__
bool almost_equal(float a, float b) {
    return (std::abs(a) > std::numeric_limits<float>::min()) * (std::abs(b) > std::numeric_limits<float>::min()) * ((std::abs(a - b) <= std::numeric_limits<float>::epsilon() * a) * (std::abs(a - b) <= std::numeric_limits<float>::epsilon() * b)) 
    + (1 - (std::abs(a) > std::numeric_limits<float>::min()) * (std::abs(b) > std::numeric_limits<float>::min())) * (std::abs(a - b) <= std::numeric_limits<float>::epsilon() * 2);
}*/

// From cppreference.com
__device__
bool almost_equal(float x, float y) {
    constexpr int ulp = 2; // ULP
    // the machine epsilon has to be scaled to the magnitude of the values used
    // and multiplied by the desired precision in ULPs (units in the last place)
    return std::abs(x-y) <= FLT_EPSILON * std::abs(x+y) * ulp // CHECK change this to double equivalent if using double instead of float
        // unless the result is subnormal
        || std::abs(x-y) < FLT_MIN; // CHECK change this to 64F if using double instead of float
}

// This will not work if we are on a node, or at least be pretty inefficient
// Algorithm 34
__global__
void lagrange_interpolating_polynomials(float x, int N, const float* nodes, const float* barycentric_weights, float* lagrange_interpolant) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int offset = N * (N + 1) /2;

    for (int i = index; i <= N; i += stride) {
        lagrange_interpolant[offset + i] = barycentric_weights[offset + i] / (x - nodes[offset + i]);
    }
}

// Algorithm 34
__global__
void normalize_lagrange_interpolating_polynomials(int N_max, float* lagrange_interpolant) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int N = index; N <= N_max; N += stride) {
        const int offset = N * (N + 1) /2;
        float sum = 0.0f;
        for (int i = 0; i <= N; ++i) {
            sum += lagrange_interpolant[offset + i];
        }
        for (int i = 0; i <= N; ++i) {
            lagrange_interpolant[offset + i] /= sum;
        }
    }
}

// Be sure to compute the diagonal afterwards
// Algorithm 37
__global__
void polynomial_derivative_matrices(int N, const float* nodes, const float* barycentric_weights, float* derivative_matrices) {
    const int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    const int stride_x = blockDim.x * gridDim.x;
    const int stride_y = blockDim.y * gridDim.y;
    const int offset_1D = N * (N + 1) /2;
    const int offset_2D = N * (N + 1) * (2 * N + 1) /6;

    for (int i = index_x; i <= N; i += stride_x) {
        for (int j = index_y; j <= N; j += stride_y) {
            if (i != j) { // CHECK remove for branchless, i == j will be overwritten anyway
                derivative_matrices[offset_2D + i * (N + 1) + j] = barycentric_weights[offset_1D + j] / (barycentric_weights[offset_1D + i] * (nodes[offset_1D + i] - nodes[offset_1D + j]));
            }
        }
    }
}

// Algorithm 37
__global__
void polynomial_derivative_matrices_diagonal(int N, float* derivative_matrices) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int offset_2D = N * (N + 1) * (2 * N + 1) /6;

    for (int i = index; i <= N; i += stride) {
        derivative_matrices[offset_2D + i * (N + 2)] = 0.0f;
        for (int j = 0; j < i; ++j) {
            derivative_matrices[offset_2D + i * (N + 2)] -= derivative_matrices[offset_2D + i * (N + 1) + j];
        }
        for (int j = i + 1; j <= N; ++j) {
            derivative_matrices[offset_2D + i * (N + 2)] -= derivative_matrices[offset_2D + i * (N + 1) + j];
        }
    }
}

__global__
void polynomial_derivative_matrices_hat(int N, const float* weights, const float* derivative_matrices, float* derivative_matrices_hat) {
    const int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    const int stride_x = blockDim.x * gridDim.x;
    const int stride_y = blockDim.y * gridDim.y;
    const int offset_1D = N * (N + 1) /2;
    const int offset_2D = N * (N + 1) * (2 * N + 1) /6;

    for (int i = index_x; i <= N; i += stride_x) {
        for (int j = index_y; j <= N; j += stride_y) {
            derivative_matrices_hat[offset_2D + i * (N + 1) + j] = -derivative_matrices[offset_2D + j * (N + 1) + i] * weights[offset_1D + j] / weights[offset_1D + i];
        }
    }
}

// Will interpolate N_interpolation_points between -1 and 1
__global__
void create_interpolation_matrices(int N, int N_interpolation_points, const float* nodes, const float* barycentric_weights, float* interpolation_matrices) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int offset_1D = N * (N + 1) /2;
    const int offset_interp = N * (N + 1) * N_interpolation_points/2;

    for (int j = index; j < N_interpolation_points; j += stride) {
        bool row_has_match = false;
        const float x_coord = 2.0f * j / (N_interpolation_points - 1) - 1.0f;

        for (int k = 0; k <= N; ++k) {
            interpolation_matrices[offset_interp + j * (N + 1) + k] = 0.0f;
            if (almost_equal(x_coord, nodes[offset_1D + k])) {
                interpolation_matrices[offset_interp + j * (N + 1) + k] = 1.0f;
                row_has_match = true;
            }
        }

        if (!row_has_match) {
            float total = 0.0f;
            for (int k = 0; k <= N; ++k) {
                interpolation_matrices[offset_interp + j * (N + 1) + k] = barycentric_weights[offset_1D + k] / (x_coord - nodes[offset_1D + k]);
                total += interpolation_matrices[offset_interp + j * (N + 1) + k];
            }
            for (int k = 0; k <= N; ++k) {
                interpolation_matrices[offset_interp + j * (N + 1) + k] /= total;
            }
        }
    }
}

// Algorithm 19
__device__
void matrix_vector_derivative(int N, const float* derivative_matrices_hat, const float* phi, float* phi_prime) {
    // s = 0, e = N (p.55 says N - 1)
    const int offset_2D = N * (N + 1) * (2 * N + 1) /6;

    for (int i = 0; i <= N; ++i) {
        phi_prime[i] = 0.0f;
        for (int j = 0; j <= N; ++j) {
            phi_prime[i] += derivative_matrices_hat[offset_2D + i * (N + 1) + j] * phi[j]; // phi not squared in textbook, squared for Burger's
        }
    }
}

class NDG_t { 
public: 
    NDG_t(int N_max, int N_interpolation_points) : 
            N_max_(N_max), 
            N_interpolation_points_(N_interpolation_points),
            vector_length_((N_max_ + 1) * (N_max_ + 2)/2), 
            matrix_length_((N_max_ + 1) * (N_max_ + 2) * (2 * N_max_ + 3)/6),
            interpolation_length_((N_max_ + 1) * (N_max_ + 2) * N_interpolation_points_/2) {

        hipMalloc(&nodes_, vector_length_ * sizeof(float));
        hipMalloc(&weights_, vector_length_ * sizeof(float));
        hipMalloc(&barycentric_weights_, vector_length_ * sizeof(float));
        hipMalloc(&lagrange_interpolant_left_, vector_length_ * sizeof(float));
        hipMalloc(&lagrange_interpolant_right_, vector_length_ * sizeof(float));
        hipMalloc(&derivative_matrices_, matrix_length_ * sizeof(float));
        hipMalloc(&derivative_matrices_hat_, matrix_length_ * sizeof(float));
        hipMalloc(&interpolation_matrices_, interpolation_length_ * sizeof(float));

        for (int N = 0; N <= N_max_; ++N) {
            const int vector_numBlocks = (N + poly_blockSize) / poly_blockSize; // Should be (N + poly_blockSize - 1) if N is not inclusive
            legendre_gauss_nodes_and_weights<<<vector_numBlocks, poly_blockSize>>>(N, nodes_, weights_);
        }

        // Nodes are needed to compute barycentric weights
        hipDeviceSynchronize();
        for (int N = 0; N <= N_max_; ++N) {
            const int vector_numBlocks = (N + poly_blockSize) / poly_blockSize; // Should be (N + poly_blockSize - 1) if N is not inclusive
            calculate_barycentric_weights<<<vector_numBlocks, poly_blockSize>>>(N, nodes_, barycentric_weights_);
        }

        // We need the barycentric weights for derivative matrix, interpolation matrices and Lagrange interpolants
        hipDeviceSynchronize();
        const int interpolation_numBlocks = (N_interpolation_points_ + interpolation_blockSize) / interpolation_blockSize;
        for (int N = 0; N <= N_max_; ++N) {
            const dim3 matrix_numBlocks((N +  matrix_blockSize.x) / matrix_blockSize.x, (N +  matrix_blockSize.y) / matrix_blockSize.y); // Should be (N + poly_blockSize - 1) if N is not inclusive
            const int vector_numBlocks = (N + poly_blockSize) / poly_blockSize; // Should be (N + poly_blockSize - 1) if N is not inclusive
            polynomial_derivative_matrices<<<matrix_numBlocks, matrix_blockSize>>>(N, nodes_, barycentric_weights_, derivative_matrices_);
            create_interpolation_matrices<<<interpolation_numBlocks, interpolation_blockSize>>>(N, N_interpolation_points_, nodes_, barycentric_weights_, interpolation_matrices_);
            lagrange_interpolating_polynomials<<<vector_numBlocks, poly_blockSize>>>(-1.0f, N, nodes_, barycentric_weights_, lagrange_interpolant_left_);
            lagrange_interpolating_polynomials<<<vector_numBlocks, poly_blockSize>>>(1.0f, N, nodes_, barycentric_weights_, lagrange_interpolant_right_);
        }

        // Then we calculate the derivative matrix diagonal and normalize the Lagrange interpolants
        hipDeviceSynchronize();
        const int poly_numBlocks = (N_max_ + poly_blockSize) / poly_blockSize;
        normalize_lagrange_interpolating_polynomials<<<poly_numBlocks, poly_blockSize>>>(N_max_, lagrange_interpolant_left_);
        normalize_lagrange_interpolating_polynomials<<<poly_numBlocks, poly_blockSize>>>(N_max_, lagrange_interpolant_right_);
        for (int N = 0; N <= N_max_; ++N) {
            const int vector_numBlocks = (N + poly_blockSize) / poly_blockSize; // Should be (N + poly_blockSize - 1) if N is not inclusive
            polynomial_derivative_matrices_diagonal<<<vector_numBlocks, poly_blockSize>>>(N, derivative_matrices_);
        }

        // All the derivative matrix has to be computed before D^
        hipDeviceSynchronize();
        for (int N = 0; N <= N_max_; ++N) {
            const dim3 matrix_numBlocks((N +  matrix_blockSize.x) / matrix_blockSize.x, (N +  matrix_blockSize.y) / matrix_blockSize.y); // Should be (N + poly_blockSize - 1) if N is not inclusive
            polynomial_derivative_matrices_hat<<<matrix_numBlocks, matrix_blockSize>>>(N, weights_, derivative_matrices_, derivative_matrices_hat_);
        }
    }

    ~NDG_t() {
        // Not sure if null checks are needed
        if (nodes_ != nullptr){
            hipFree(nodes_);
        }
        if (weights_ != nullptr){
            hipFree(weights_);
        }
        if (barycentric_weights_ != nullptr){
            hipFree(barycentric_weights_);
        }
        if (lagrange_interpolant_left_ != nullptr){
            hipFree(lagrange_interpolant_left_);
        }
        if (lagrange_interpolant_right_ != nullptr){
            hipFree(lagrange_interpolant_right_);
        }
        if (derivative_matrices_ != nullptr){
            hipFree(derivative_matrices_);
        }
        if (derivative_matrices_hat_ != nullptr){
            hipFree(derivative_matrices_hat_);
        }
        if (interpolation_matrices_ != nullptr){
            hipFree(interpolation_matrices_);
        }
    }

    int N_max_;
    int N_interpolation_points_;
    int vector_length_; // Flattened length of all N one after the other
    int matrix_length_; // Flattened length of all N² one after the other
    int interpolation_length_;
    float* nodes_;
    float* weights_;
    float* barycentric_weights_;
    float* lagrange_interpolant_left_;
    float* lagrange_interpolant_right_;
    float* derivative_matrices_;
    float* derivative_matrices_hat_;
    float* interpolation_matrices_;

    void print() {
        // Copy vectors from device memory to host memory
        float* host_nodes = new float[vector_length_];
        float* host_weights = new float[vector_length_];
        float* host_barycentric_weights = new float[vector_length_];
        float* host_lagrange_interpolant_left = new float[vector_length_];
        float* host_lagrange_interpolant_right = new float[vector_length_];
        float* host_derivative_matrices = new float[matrix_length_];
        float* host_derivative_matrices_hat = new float[matrix_length_];
        float* host_interpolation_matrices = new float[interpolation_length_];

        hipDeviceSynchronize();

        hipMemcpy(host_nodes, nodes_, vector_length_ * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(host_weights, weights_, vector_length_ * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(host_barycentric_weights, barycentric_weights_, vector_length_ * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(host_lagrange_interpolant_left, lagrange_interpolant_left_, vector_length_ * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(host_lagrange_interpolant_right, lagrange_interpolant_right_, vector_length_ * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(host_derivative_matrices, derivative_matrices_, matrix_length_ * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(host_derivative_matrices_hat, derivative_matrices_hat_, matrix_length_ * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(host_interpolation_matrices, interpolation_matrices_, interpolation_length_ * sizeof(float), hipMemcpyDeviceToHost);

        std::cout << "Nodes: " << std::endl;
        for (int N = 0; N <= N_max_; ++N) {
            const int offset = N * (N + 1) /2;

            std::cout << '\t' << "N = " << N << ": ";
            std::cout << '\t' << '\t';
            for (int i = 0; i <= N; ++i) {
                std::cout << host_nodes[offset + i] << " ";
            }
            std::cout << std::endl;
        }

        std::cout << std::endl << "Weights: " << std::endl;
        for (int N = 0; N <= N_max_; ++N) {
            const int offset = N * (N + 1) /2;

            std::cout << '\t' << "N = " << N << ": ";
            std::cout << '\t' << '\t';
            for (int i = 0; i <= N; ++i) {
                std::cout << host_weights[offset + i] << " ";
            }
            std::cout << std::endl;
        }
        
        std::cout << std::endl << "Barycentric weights: " << std::endl;
        for (int N = 0; N <= N_max_; ++N) {
            const int offset = N * (N + 1) /2;

            std::cout << '\t' << "N = " << N << ": ";
            std::cout << '\t' << '\t';
            for (int i = 0; i <= N; ++i) {
                std::cout << host_barycentric_weights[offset + i] << " ";
            }
            std::cout << std::endl;
        }

        std::cout << std::endl << "Lagrange interpolants -1: " << std::endl;
        for (int N = 0; N <= N_max_; ++N) {
            const int offset = N * (N + 1) /2;

            std::cout << '\t' << "N = " << N << ": ";
            std::cout << '\t' << '\t';
            for (int i = 0; i <= N; ++i) {
                std::cout << host_lagrange_interpolant_left[offset + i] << " ";
            }
            std::cout << std::endl;
        }

        std::cout << std::endl << "Lagrange interpolants +1: " << std::endl;
        for (int N = 0; N <= N_max_; ++N) {
            const int offset = N * (N + 1) /2;

            std::cout << '\t' << "N = " << N << ": ";
            std::cout << '\t' << '\t';
            for (int i = 0; i <= N; ++i) {
                std::cout << host_lagrange_interpolant_right[offset + i] << " ";
            }
            std::cout << std::endl;
        }

        std::cout << std::endl << "Derivative matrices: " << std::endl;
        for (int N = 0; N <= N_max_; ++N) {
            const int offset_2D = N * (N + 1) * (2 * N + 1) /6;

            std::cout << '\t' << "N = " << N << ": " << std::endl;
            for (int i = 0; i <= N; ++i) {
                std::cout << '\t' << '\t';
                for (int j = 0; j <= N; ++j) {
                    std::cout << host_derivative_matrices[offset_2D + i * (N + 1) + j] << " ";
                }
                std::cout << std::endl;
            }
        }

        std::cout << std::endl << "Derivative matrices hat: " << std::endl;
        for (int N = 0; N <= N_max_; ++N) {
            const int offset_2D = N * (N + 1) * (2 * N + 1) /6;

            std::cout << '\t' << "N = " << N << ": " << std::endl;
            for (int i = 0; i <= N; ++i) {
                std::cout << '\t' << '\t';
                for (int j = 0; j <= N; ++j) {
                    std::cout << host_derivative_matrices_hat[offset_2D + i * (N + 1) + j] << " ";
                }
                std::cout << std::endl;
            }
        }

        /*std::cout << std::endl << "Interpolation matrices: " << std::endl;
        for (int N = 0; N <= N_max_; ++N) {
            const int offset_interp = N * (N + 1) * N_interpolation_points_/2;

            std::cout << '\t' << "N = " << N << ": " << std::endl;
            for (int i = 0; i < N_interpolation_points_; ++i) {
                std::cout << '\t' << '\t';
                for (int j = 0; j <= N; ++j) {
                    std::cout << host_interpolation_matrices[offset_interp + i * (N + 1) + j] << " ";
                }
                std::cout << std::endl;
            }
        }*/

        const int N = 8;
        const int offset = N * (N + 1) /2;
        float* u = new float[N + 1];
        float* u_prime = new float[N + 1];
        float* u_prime_expected = new float[N + 1];
        float u_L = 0.0f;
        float u_R = 0.0f;

        for (int i = 0; i <= N; ++i) {
            u[i] = -sin(pi * host_nodes[offset + i]);
        }

        const int offset_2D = N * (N + 1) * (2 * N + 1) /6;
        for (int i = 0; i <= N; ++i) {
            u_prime[i] = 0.0f;
            for (int j = 0; j <= N; ++j) {
                u_prime[i] += host_derivative_matrices[offset_2D + i * (N + 1) + j] * u[j];
            }
        }

        for (int i = 0; i <= N; ++i) {
            u_prime_expected[i] = -pi * cos(pi * host_nodes[offset + i]);
        }
        
        for (int i = 0; i <= N; ++i) {
            u_L += host_lagrange_interpolant_left[offset + i] * u[i];
        }

        for (int i = 0; i <= N; ++i) {
            u_R += host_lagrange_interpolant_right[offset + i] * u[i];
        }

        std::cout << "x:" << std::endl;
        std::cout << '\t';
        for (int i = 0; i <= N; ++i) {
            std::cout << std::setw(12) << host_nodes[offset + i] << "    ";
        }
        std::cout << std::endl;

        std::cout << "u:" << std::endl;
        std::cout << '\t';
        for (int i = 0; i <= N; ++i) {
            std::cout << std::setw(12) << u[i] << "    ";
        }
        std::cout << std::endl;

        std::cout << "u prime:" << std::endl;
        std::cout << '\t';
        for (int i = 0; i <= N; ++i) {
            std::cout << std::setw(12) << u_prime[i] << "    ";
        }
        std::cout << std::endl;

        std::cout << "Expected u prime:" << std::endl;
        std::cout << '\t';
        for (int i = 0; i <= N; ++i) {
            std::cout << std::setw(12) << u_prime_expected[i] << "    ";
        }
        std::cout << std::endl;

        std::cout << "Interpolated u:" << std::endl;
        std::cout << '\t';
        std::cout << std::setw(12) << u_L << "    ";
        std::cout << std::setw(12) << u_R;
        std::cout << std::endl;
        

        delete[] u;
        delete[] u_prime;
        delete[] u_prime_expected;

        delete[] host_nodes;
        delete[] host_weights;
        delete[] host_barycentric_weights;
        delete[] host_lagrange_interpolant_left;
        delete[] host_lagrange_interpolant_right;
        delete[] host_derivative_matrices;
        delete[] host_derivative_matrices_hat;
        delete[] host_interpolation_matrices;
    }
};

class Element_t { // Turn this into separate vectors, because cache exists
public:
    __device__ 
    Element_t(int N, int neighbour_L, int neighbour_R, int face_L, int face_R, float x_L, float x_R) : 
            N_(N),
            neighbours_{neighbour_L, neighbour_R},
            faces_{face_L, face_R},
            x_{x_L, x_R},
            delta_x_(x_R - x_L) {
        phi_ = new float[N_ + 1];
        phi_prime_ = new float[N_ + 1];
        intermediate_ = new float[N_ + 1];
        for (int i = 0; i <= N_; ++i) {
            intermediate_[i] = 0.0f;
        }
    }

    __host__ 
    Element_t() {};

    __host__ __device__
    ~Element_t() {
        if (phi_ != nullptr){
            delete[] phi_;
        }
        if (phi_prime_ != nullptr) {
            delete[] phi_prime_;
        }
        if (intermediate_ != nullptr) {
            delete[] intermediate_;
        }
    }

    int N_;
    int neighbours_[2]; // Could also be pointers
    int faces_[2]; // Could also be pointers. left, right
    float x_[2];
    float delta_x_;
    float phi_L_;
    float phi_R_;
    float* phi_; // Solution
    float* phi_prime_;
    float* intermediate_;
};

__global__
void build_elements(int N_elements, int N, Element_t* elements, float x_min, float x_max) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N_elements; i += stride) {
        const int neighbour_L = (i > 0) ? i - 1 : N_elements - 1; // First cell has last cell as left neighbour
        const int neighbour_R = (i < N_elements - 1) ? i + 1 : 0; // Last cell has first cell as right neighbour
        const int face_L = (i > 0) ? i - 1 : N_elements - 1;
        const int face_R = i;
        const float delta_x = (x_max - x_min)/N_elements;
        const float element_x_min = x_min + i * delta_x;
        const float element_y_min = x_min + (i + 1) * delta_x;
        elements[i] = Element_t(N, neighbour_L, neighbour_R, face_L, face_R, element_x_min, element_y_min);
    }
}

__device__
float g(float t, float x) {
    constexpr float sigma = 0.2;
    return std::exp(-std::log(2.0f) * std::pow(x - t * c, 2) / std::pow(sigma, 2));
}


__global__
void initial_conditions(int N_elements, Element_t* elements, const float* nodes) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N_elements; i += stride) {
        const int offset = elements[i].N_ * (elements[i].N_ + 1) /2;
        for (int j = 0; j <= elements[i].N_; ++j) {
            const float x = (0.5 + nodes[offset + j]/2.0f) * (elements[i].x_[1] - elements[i].x_[0]) + elements[i].x_[0];
            //elements[i].phi_[j] = -sin(pi * x);
            elements[i].phi_[j] = g(0.0f, x);
        }
    }
}

// Basically useless, find better solution when multiple elements.
__global__
void get_elements_data(int N_elements, const Element_t* elements, float* phi, float* phi_prime) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N_elements; i += stride) {
        const int element_offset = i * (elements[i].N_ + 1);
        for (int j = 0; j <= elements[i].N_; ++j) {
            phi[element_offset + j] = elements[i].phi_[j];
            phi_prime[element_offset + j] = elements[i].phi_prime_[j];
        }
    }
}

// Basically useless, find better solution when multiple elements.
__global__
void get_phi(int N_elements, const Element_t* elements, float* phi) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N_elements; i += stride) {
        for (int j = 0; j <= elements[i].N_; ++j) {
            phi[j] = elements[i].phi_[j];
        }
    }
}

__global__
void get_solution(int N_elements, int N_interpolation_points, const Element_t* elements, const float* interpolation_matrices, float* phi, float* x) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N_elements; i += stride) {
        const int offset_interp_1D = i * N_interpolation_points;
        const int offset_interp = elements[i].N_ * (elements[i].N_ + 1) * N_interpolation_points/2;

        for (int j = 0; j < N_interpolation_points; ++j) {
            phi[offset_interp_1D + j] = 0.0f;
            for (int k = 0; k <= elements[i].N_; ++k) {
                phi[offset_interp_1D + j] += interpolation_matrices[offset_interp + j * (elements[i].N_ + 1) + k] * elements[i].phi_[k];
            }
            x[offset_interp_1D + j] = j * (elements[i].x_[1] - elements[i].x_[0]) / (N_interpolation_points - 1) + elements[i].x_[0];
        }
    }
}

// Algorithm 61
__device__
float interpolate_to_boundary(int N, const float* phi, const float* lagrange_interpolant) {
    const int offset_1D = N * (N + 1) /2;
    float result = 0.0f;

    for (int j = 0; j <= N; ++j) {
        result += lagrange_interpolant[offset_1D + j] * phi[j];
    }

    return result;
}

__global__
void rk3_step(int N_elements, Element_t* elements, float delta_t, float a, float g) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N_elements; i += stride) {
        for (int j = 0; j <= elements[i].N_; ++j){
            elements[i].intermediate_[j] = a * elements[i].intermediate_[j] + elements[i].phi_prime_[j];
            elements[i].phi_[j] += g * delta_t * elements[i].intermediate_[j];
        }
    }
}

__global__
void interpolate_to_boundaries(int N_elements, Element_t* elements, const float* lagrange_interpolant_left, const float* lagrange_interpolant_right) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N_elements; i += stride) {
        elements[i].phi_L_ = interpolate_to_boundary(elements[i].N_, elements[i].phi_, lagrange_interpolant_left);
        elements[i].phi_R_ = interpolate_to_boundary(elements[i].N_, elements[i].phi_, lagrange_interpolant_right);
    }
}

class Face_t {
public:
    __device__ 
    Face_t(int element_L, int element_R) : elements_{element_L, element_R} {}

    __host__
    Face_t() {}

    __host__ __device__
    ~Face_t() {}

    int elements_[2]; // left, right
    float flux_;
};

__global__
void build_faces(int N_faces, Face_t* faces) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N_faces; i += stride) {
        const int neighbour_L = i;
        const int neighbour_R = (i < N_faces - 1) ? i + 1 : 0; // Last face links last element to first element
        faces[i] = Face_t(neighbour_L, neighbour_R);
    }
}

__global__
void calculate_fluxes(int N_faces, Face_t* faces, const Element_t* elements) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N_faces; i += stride) {
        const float u_left = elements[faces[i].elements_[0]].phi_R_;
        const float u_right = elements[faces[i].elements_[1]].phi_L_;

        if (c >= 0.0f) {
            faces[i].flux_ = u_left;
        }
        else  {
            faces[i].flux_ = u_right;
        }
    }
}

// Algorithm 60 (not really anymore)
__global__
void compute_dg_derivative(float t, int N_elements, Element_t* elements, const Face_t* faces, const float* weights, const float* derivative_matrices_hat, const float* lagrange_interpolant_left, const float* lagrange_interpolant_right) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N_elements; i += stride) {
        const int offset_1D = elements[i].N_ * (elements[i].N_ + 1) /2; // CHECK cache?

        float flux_L = faces[elements[i].faces_[0]].flux_;
        float flux_R = faces[elements[i].faces_[1]].flux_;

        if (c > 0.0f) {
            flux_L = g(t, -1.0f);
            flux_R = elements[i].phi_R_;
        } else {
            flux_L = elements[i].phi_L_;
            flux_R =  g(t, 1.0f);
        }

        matrix_vector_derivative(elements[i].N_, derivative_matrices_hat, elements[i].phi_, elements[i].phi_prime_);

        for (int j = 0; j <= elements[i].N_; ++j) {
            elements[i].phi_prime_[j] += (flux_R * lagrange_interpolant_right[offset_1D + j] - flux_L * lagrange_interpolant_left[offset_1D + j]) / weights[offset_1D + j];
            elements[i].phi_prime_[j] *= -c * 2.0f/elements[i].delta_x_;
        }
    }
}

class Mesh_t {
public:
    Mesh_t(int N_elements, int initial_N, float x_min, float x_max) : N_elements_(N_elements), N_faces_(N_elements), initial_N_(initial_N) {
        // CHECK N_faces = N_elements only for periodic BC.
        hipMalloc(&elements_, N_elements_ * sizeof(Element_t));
        hipMalloc(&faces_, N_faces_ * sizeof(Face_t));

        const int elements_numBlocks = (N_elements_ + elements_blockSize - 1) / elements_blockSize;
        const int faces_numBlocks = (N_faces_ + faces_blockSize - 1) / faces_blockSize;
        build_elements<<<elements_numBlocks, elements_blockSize>>>(N_elements_, initial_N_, elements_, x_min, x_max);
        build_faces<<<faces_numBlocks, faces_blockSize>>>(N_faces_, faces_); // CHECK
    }

    ~Mesh_t() {
        if (elements_ != nullptr){
            hipFree(elements_);
        }

        if (faces_ != nullptr){
            hipFree(faces_);
        }
    }

    int N_elements_;
    int N_faces_;
    int initial_N_;
    Element_t* elements_;
    Face_t* faces_;

    void set_initial_conditions(const float* nodes) {
        const int elements_numBlocks = (N_elements_ + elements_blockSize - 1) / elements_blockSize;
        initial_conditions<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, nodes);
    }

    void print() {
        // CHECK find better solution for multiple elements. This only works if all elements have the same N.
        float* phi;
        float* phi_prime;
        float* host_phi = new float[(initial_N_ + 1) * N_elements_];
        float* host_phi_prime = new float[(initial_N_ + 1) * N_elements_];
        Face_t* host_faces = new Face_t[N_faces_];
        Element_t* host_elements = new Element_t[N_elements_];
        hipMalloc(&phi, (initial_N_ + 1) * N_elements_ * sizeof(float));
        hipMalloc(&phi_prime, (initial_N_ + 1) * N_elements_ * sizeof(float));

        const int elements_numBlocks = (N_elements_ + elements_blockSize - 1) / elements_blockSize;
        get_elements_data<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, phi, phi_prime);
        
        hipDeviceSynchronize();
        hipMemcpy(host_phi, phi, (initial_N_ + 1) * N_elements_ * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(host_phi_prime, phi_prime, (initial_N_ + 1) * N_elements_ * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(host_faces, faces_, N_faces_ * sizeof(Face_t), hipMemcpyDeviceToHost);
        hipMemcpy(host_elements, elements_, N_elements_ * sizeof(Element_t), hipMemcpyDeviceToHost);

        // Invalidate GPU pointers, or else they will be deleted on the CPU, where they point to random stuff
        for (int i = 0; i < N_elements_; ++i) {
            host_elements[i].phi_ = nullptr;
            host_elements[i].phi_prime_ = nullptr;
            host_elements[i].intermediate_ = nullptr;
        }

        std::cout << std::endl << "Phi: " << std::endl;
        for (int i = 0; i < N_elements_; ++i) {
            const int element_offset = i * (initial_N_ + 1);
            std::cout << '\t' << "Element " << i << ": ";
            std::cout << '\t' << '\t';
            for (int j = 0; j <= initial_N_; ++j) {
                std::cout << host_phi[element_offset + j] << " ";
            }
            std::cout << std::endl;
        }

        std::cout << std::endl << "Phi prime: " << std::endl;
        for (int i = 0; i < N_elements_; ++i) {
            const int element_offset = i * (initial_N_ + 1);
            std::cout << '\t' << "Element " << i << ": ";
            std::cout << '\t' << '\t';
            for (int j = 0; j <= initial_N_; ++j) {
                std::cout << host_phi_prime[element_offset + j] << " ";
            }
            std::cout << std::endl;
        }

        std::cout << std::endl << "Phi interpolated: " << std::endl;
        for (int i = 0; i < N_elements_; ++i) {
            std::cout << '\t' << "Element " << i << ": ";
            std::cout << '\t' << '\t';
            std::cout << host_elements[i].phi_L_ << " ";
            std::cout << host_elements[i].phi_R_;
            std::cout << std::endl;
        }

        std::cout << std::endl << "x: " << std::endl;
        for (int i = 0; i < N_elements_; ++i) {
            std::cout << '\t' << "Element " << i << ": ";
            std::cout << '\t' << '\t';
            std::cout << host_elements[i].x_[0] << " ";
            std::cout << host_elements[i].x_[1];
            std::cout << std::endl;
        }

        std::cout << std::endl << "Neighbouring elements: " << std::endl;
        for (int i = 0; i < N_elements_; ++i) {
            std::cout << '\t' << "Element " << i << ": ";
            std::cout << '\t' << '\t';
            std::cout << host_elements[i].neighbours_[0] << " ";
            std::cout << host_elements[i].neighbours_[1];
            std::cout << std::endl;
        }

        std::cout << std::endl << "Neighbouring faces: " << std::endl;
        for (int i = 0; i < N_elements_; ++i) {
            std::cout << '\t' << "Element " << i << ": ";
            std::cout << '\t' << '\t';
            std::cout << host_elements[i].faces_[0] << " ";
            std::cout << host_elements[i].faces_[1];
            std::cout << std::endl;
        }

        std::cout << std::endl << "N: " << std::endl;
        for (int i = 0; i < N_elements_; ++i) {
            std::cout << '\t' << "Element " << i << ": ";
            std::cout << '\t' << '\t';
            std::cout << host_elements[i].N_;
            std::cout << std::endl;
        }

        std::cout << std::endl << "delta x: " << std::endl;
        for (int i = 0; i < N_elements_; ++i) {
            std::cout << '\t' << "Element " << i << ": ";
            std::cout << '\t' << '\t';
            std::cout << host_elements[i].delta_x_;
            std::cout << std::endl;
        }

        std::cout << std::endl << "Fluxes: " << std::endl;
        for (int i = 0; i < N_faces_; ++i) {
            std::cout << '\t' << "Face " << i << ": ";
            std::cout << '\t' << '\t';
            std::cout << host_faces[i].flux_ << std::endl;
        }

        std::cout << std::endl << "Elements: " << std::endl;
        for (int i = 0; i < N_faces_; ++i) {
            std::cout << '\t' << "Face " << i << ": ";
            std::cout << '\t' << '\t';
            std::cout << host_faces[i].elements_[0] << " ";
            std::cout << host_faces[i].elements_[1] << std::endl;
        }

        delete[] host_phi;
        delete[] host_phi_prime;
        delete[] host_faces;
        delete[] host_elements;

        hipFree(phi);
        hipFree(phi_prime);
    }

    void write_file_data(int N_points, float time, const float* velocity, const float* coordinates) {
        std::stringstream ss;
        std::ofstream file;
    
        fs::path save_dir = fs::current_path() / "data";
        fs::create_directory(save_dir);
    
        ss << "output_t" << std::setprecision(4) << std::fixed << time << ".dat";
        file.open(save_dir / ss.str());
    
        file << "TITLE = \"Velocity  at t= " << time << "\"" << std::endl;
        file << "VARIABLES = \"X\", \"U_x\"" << std::endl;
        file << "ZONE T= \"Zone     1\",  I= " << N_points << ",  J= 1,  DATAPACKING = POINT, SOLUTIONTIME = " << time << std::endl;
    
        for (int i = 0; i < N_points; ++i) {
            file << std::setw(12) << coordinates[i] << " " << std::setw(12) << velocity[i] << std::endl;
        }
    
        file.close();
    }

    void write_data(float time, int N_interpolation_points, const float* interpolation_matrices) {
        // CHECK find better solution for multiple elements
        float* phi;
        float* x;
        float* host_phi = new float[N_elements_ * N_interpolation_points];
        float* host_x = new float[N_elements_ * N_interpolation_points];
        hipMalloc(&phi, N_elements_ * N_interpolation_points * sizeof(float));
        hipMalloc(&x, N_elements_ * N_interpolation_points * sizeof(float));

        const int elements_numBlocks = (N_elements_ + elements_blockSize - 1) / elements_blockSize;
        get_solution<<<elements_numBlocks, elements_blockSize>>>(N_elements_, N_interpolation_points, elements_, interpolation_matrices, phi, x);
        
        hipDeviceSynchronize();
        hipMemcpy(host_phi, phi, N_elements_ * N_interpolation_points * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(host_x, x , N_elements_ * N_interpolation_points * sizeof(float), hipMemcpyDeviceToHost);

        write_file_data(N_elements_ * N_interpolation_points, time, host_phi, host_x);

        delete[] host_phi;
        delete[] host_x;
        hipFree(phi);
        hipFree(x);
    }

    void solve(const NDG_t &NDG) {
        const float delta_t = 0.00015f;
        const int elements_numBlocks = (N_elements_ + elements_blockSize - 1) / elements_blockSize;
        const int faces_numBlocks = (N_faces_ + faces_blockSize - 1) / faces_blockSize;
        float time = 0.0;
        std::vector<float> output_times{0.5f, 1.0f, 1.5f};
        const float t_end = output_times.back();

        write_data(time, NDG.N_interpolation_points_, NDG.interpolation_matrices_);

        while (time < (t_end + delta_t)) {
            // Kinda algorithm 62
            float t = time;
            interpolate_to_boundaries<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
            calculate_fluxes<<<faces_numBlocks, faces_blockSize>>>(N_faces_, faces_, elements_);
            compute_dg_derivative<<<elements_numBlocks, elements_blockSize>>>(t, N_elements_, elements_, faces_, NDG.weights_, NDG.derivative_matrices_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
            rk3_step<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, delta_t, 0.0f, 1.0f/3.0f);

            t = time + 0.33333333333f * delta_t;
            interpolate_to_boundaries<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
            calculate_fluxes<<<faces_numBlocks, faces_blockSize>>>(N_faces_, faces_, elements_);
            compute_dg_derivative<<<elements_numBlocks, elements_blockSize>>>(t, N_elements_, elements_, faces_, NDG.weights_, NDG.derivative_matrices_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
            rk3_step<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, delta_t, -5.0f/9.0f, 15.0f/16.0f);

            t = time + 0.75f * delta_t;
            interpolate_to_boundaries<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
            calculate_fluxes<<<faces_numBlocks, faces_blockSize>>>(N_faces_, faces_, elements_);
            compute_dg_derivative<<<elements_numBlocks, elements_blockSize>>>(t, N_elements_, elements_, faces_, NDG.weights_, NDG.derivative_matrices_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
            rk3_step<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, delta_t, -153.0f/128.0f, 8.0f/15.0f);
                  
            time += delta_t;
            for (auto const& e : std::as_const(output_times)) {
                if ((time >= e) && (time < e + delta_t)) {
                    write_data(time, NDG.N_interpolation_points_, NDG.interpolation_matrices_);
                    break;
                }
            }
        }

        bool did_write = false;
        for (auto const& e : std::as_const(output_times)) {
            if ((time >= e) && (time < e + delta_t)) {
                did_write = true;
                break;
            }
        }

        if (!did_write) {
            write_data(time, NDG.N_interpolation_points_, NDG.interpolation_matrices_);
        }
    }
};

int main(void) {
    const int N_elements = 1;
    const int N_max = 8;
    const int initial_N = N_max;
    const int N_interpolation_points = 100;
    
    NDG_t NDG(N_max, N_interpolation_points);
    Mesh_t Mesh(N_elements, initial_N, -1.0f, 1.0f);
    Mesh.set_initial_conditions(NDG.nodes_);

    // Starting actual computation
    hipDeviceSynchronize();
    auto t_start = std::chrono::high_resolution_clock::now();
    Mesh.solve(NDG);
    // Wait for GPU to finish before copying to host
    hipDeviceSynchronize();
    auto t_end = std::chrono::high_resolution_clock::now();

    std::cout << "GPU computation time: " 
            << std::chrono::duration<double, std::milli>(t_end-t_start).count()/1000.0 
            << "s." << std::endl;

    NDG.print();
    Mesh.print();
    
    return 0;
}