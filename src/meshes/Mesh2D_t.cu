#include "hip/hip_runtime.h"
#include "meshes/Mesh2D_t.cuh"
#include "polynomials/ChebyshevPolynomial_t.cuh"
#include "polynomials/LegendrePolynomial_t.cuh"
#include "helpers/constants.h"
#include "functions/Utilities.h"
#include "functions/quad_map.cuh"
#include "cgnslib.h"
#include <iostream>
#include <fstream>
#include <sstream> 
#include <iomanip>
#include <filesystem>
#include <algorithm>
#include <limits>

namespace fs = std::filesystem;

using SEM::Entities::device_vector;
using SEM::Entities::host_vector;
using SEM::Entities::Vec2;
using SEM::Entities::Element2D_t;
using SEM::Entities::Face2D_t;

constexpr int CGIO_MAX_NAME_LENGTH = 33; // Includes the null terminator

SEM::Meshes::Mesh2D_t::Mesh2D_t(std::filesystem::path filename, int initial_N, int maximum_N, size_t n_interpolation_points, int max_split_level, int adaptivity_interval, deviceFloat tolerance_min, deviceFloat tolerance_max, const SEM::Entities::device_vector<deviceFloat>& polynomial_nodes, const hipStream_t &stream) :       
        initial_N_{initial_N},  
        maximum_N_{maximum_N},
        n_interpolation_points_{n_interpolation_points},
        max_split_level_{max_split_level},
        adaptivity_interval_{adaptivity_interval},
        tolerance_min_{tolerance_min},
        tolerance_max_{tolerance_max},
        stream_{stream} {

    std::string extension = filename.extension().string();
    SEM::to_lower(extension);

    if (extension == ".cgns") {
        read_cgns(filename);
    }
    else if (extension == ".su2") {
        read_su2(filename);
    }
    else {
        std::cerr << "Error: extension '" << extension << "' not recognized. Exiting." << std::endl;
        exit(14);
    }

    compute_element_geometry<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(n_elements_, elements_.data(), nodes_.data(), polynomial_nodes.data());
    compute_boundary_geometry<<<ghosts_numBlocks_, boundaries_blockSize_, 0, stream_>>>(n_elements_, elements_.size(), elements_.data(), nodes_.data(), polynomial_nodes.data());
    compute_face_geometry<<<faces_numBlocks_, faces_blockSize_, 0, stream_>>>(faces_.size(), faces_.data(), elements_.data(), nodes_.data());
}

auto SEM::Meshes::Mesh2D_t::read_su2(std::filesystem::path filename) -> void {
    std::cerr << "Error: SU2 meshes not implemented yet. Exiting." << std::endl;
    exit(15);
}

auto SEM::Meshes::Mesh2D_t::read_cgns(std::filesystem::path filename) -> void {
    int index_file = 0;
    const int open_error = cg_open(filename.string().c_str(), CG_MODE_READ, &index_file);
    if (open_error != CG_OK) {
        std::cerr << "Error: file '" << filename << "' could not be opened with error '" << cg_get_error() << "'. Exiting." << std::endl;
        exit(16);
    }

    // Getting base information
    int n_bases = 0;
    cg_nbases(index_file, &n_bases);
    if (n_bases != 1) {
        std::cerr << "Error: CGNS mesh has " << n_bases << " base(s), but for now only a single base is supported. Exiting." << std::endl;
        exit(17);
    }
    constexpr int index_base = 1;

    std::array<char, CGIO_MAX_NAME_LENGTH> base_name; // Oh yeah cause it's the 80s still
    int dim = 0;
    int physDim = 0;
    cg_base_read(index_file, index_base, base_name.data(), &dim, &physDim);
    if (dim != 2) {
        std::cerr << "Error: CGNS mesh, base " << index_base << " has " << dim << " dimensions, but the program only supports 2 dimensions. Exiting." << std::endl;
        exit(18);
    }
    if (physDim != 2) {
        std::cerr << "Error: CGNS mesh, base " << index_base << " has " << physDim << " physical dimensions, but the program only supports 2 physical dimensions. Exiting." << std::endl;
        exit(19);
    }

    // Getting zone information
    int n_zones = 0;
    cg_nzones(index_file, index_base, &n_zones);

    int global_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &global_rank);
    int global_size;
    MPI_Comm_size(MPI_COMM_WORLD, &global_size);

    if (n_zones != global_size) {
        std::cerr << "Error: CGNS mesh, base " << index_base << " has " << n_zones << " zone(s), but the program has been run with " << global_size << " process(es). For now only a single zone per process is supported. Exiting." << std::endl;
        exit(48);
    }

    // Getting all zone names
    std::vector<std::array<char, CGIO_MAX_NAME_LENGTH>> zone_names(n_zones);
    for (int i = 0; i < n_zones; ++i) {
        std::array<cgsize_t, 3> temp{0, 0, 0};
        cg_zone_read(index_file, index_base, i + 1, zone_names[i].data(), temp.data());
    }

    const int index_zone = global_rank + 1;

    ZoneType_t zone_type = ZoneType_t::ZoneTypeNull;
    cg_zone_type(index_file, index_base, index_zone, &zone_type);
    if (zone_type != ZoneType_t::Unstructured) {
        std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << " is not an unstructured zone. For now only unstructured zones are supported. Exiting." << std::endl;
        exit(21);
    }

    std::array<char, CGIO_MAX_NAME_LENGTH> zone_name; // Oh yeah cause it's the 80s still
    std::array<cgsize_t, 3> isize{0, 0, 0};
    cg_zone_read(index_file, index_base, index_zone, zone_name.data(), isize.data());
    if (isize[2] != 0) {
        std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << " has " << isize[2] << " boundary vertices, but to be honest I'm not sure how to deal with them. Exiting." << std::endl;
        exit(22);
    }
    const cgsize_t n_nodes = isize[0];
    const cgsize_t n_elements = isize[1];

    // Getting nodes
    int n_coords = 0;
    cg_ncoords(index_file, index_base, index_zone, &n_coords);
    if (n_coords != 2) {
        std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << " has " << n_coords << " sets of coordinates, but for now only two are supported. Exiting." << std::endl;
        exit(23);
    }

    std::array<std::array<char, CGIO_MAX_NAME_LENGTH>, 2> coord_names; // Oh yeah cause it's the 80s still
    std::array<DataType_t, 2> coord_data_types {DataType_t::DataTypeNull, DataType_t::DataTypeNull};
    for (int index_coord = 1; index_coord <= n_coords; ++index_coord) {
        cg_coord_info(index_file, index_base, index_zone, index_coord, &coord_data_types[index_coord - 1], coord_names[index_coord - 1].data());
    }

    std::array<std::vector<double>, 2> xy{std::vector<double>(n_nodes), std::vector<double>(n_nodes)};

    for (int index_coord = 1; index_coord <= n_coords; ++index_coord) {
        const cgsize_t index_coord_start = 1;
        cg_coord_read(index_file, index_base, index_zone, coord_names[index_coord - 1].data(), DataType_t::RealDouble, &index_coord_start, &n_nodes, xy[index_coord - 1].data());
    }
    
    // Getting connectivity
    int n_sections = 0;
    cg_nsections(index_file, index_base, index_zone, &n_sections);

    std::vector<cgsize_t> section_data_size(n_sections);
    std::vector<std::array<char, CGIO_MAX_NAME_LENGTH>> section_names(n_sections); // Oh yeah cause it's the 80s still
    std::vector<ElementType_t> section_type(n_sections);
    std::vector<std::array<cgsize_t, 2>> section_ranges(n_sections);
    std::vector<int> section_n_boundaries(n_sections);
    std::vector<int> section_parent_flags(n_sections);
    for (int index_section = 1; index_section <= n_sections; ++index_section) {
        cg_ElementDataSize(index_file, index_base, index_zone, index_section, &section_data_size[index_section - 1]);
        cg_section_read(index_file, index_base, index_zone, index_section, section_names[index_section - 1].data(), &section_type[index_section - 1], &section_ranges[index_section - 1][0], &section_ranges[index_section - 1][1], &section_n_boundaries[index_section - 1], &section_parent_flags[index_section - 1]);
        if (section_n_boundaries[index_section - 1] != 0) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", section " << index_section << " has " << section_n_boundaries[index_section - 1] << " boundary elements, but to be honest I'm not sure how to deal with them. Exiting." << std::endl;
            exit(24);
        }
    }

    std::vector<std::vector<cgsize_t>> connectivity(n_sections);
    std::vector<std::vector<cgsize_t>> parent_data(n_sections);
    for (int index_section = 1; index_section <= n_sections; ++index_section) {
        connectivity[index_section - 1] = std::vector<cgsize_t>(section_data_size[index_section - 1]);
        parent_data[index_section - 1] = std::vector<cgsize_t>(section_ranges[index_section - 1][1] - section_ranges[index_section - 1][0] + 1);

        cg_elements_read(index_file, index_base, index_zone, index_section, connectivity[index_section - 1].data(), parent_data[index_section - 1].data());
    }

    // Interfaces
    int n_connectivity = 0;
    cg_nconns(index_file, index_base, index_zone, &n_connectivity);

    std::vector<std::array<char, CGIO_MAX_NAME_LENGTH>> connectivity_names(n_connectivity); // Oh yeah cause it's the 80s still
    std::vector<GridLocation_t> connectivity_grid_locations(n_connectivity);
    std::vector<GridConnectivityType_t> connectivity_types(n_connectivity);
    std::vector<PointSetType_t> connectivity_point_set_types(n_connectivity);
    std::vector<cgsize_t> connectivity_sizes(n_connectivity);
    std::vector<std::array<char, CGIO_MAX_NAME_LENGTH>> connectivity_donor_names(n_connectivity); // Oh yeah cause it's the 80s still
    std::vector<ZoneType_t> connectivity_donor_zone_types(n_connectivity);
    std::vector<PointSetType_t> connectivity_donor_point_set_types(n_connectivity);
    std::vector<DataType_t> connectivity_donor_data_types(n_connectivity);
    std::vector<cgsize_t> connectivity_donor_sizes(n_connectivity);
    for (int index_connectivity = 1; index_connectivity <= n_connectivity; ++index_connectivity) {
        cg_conn_info(index_file, index_base, index_zone, index_connectivity, connectivity_names[index_connectivity - 1].data(),
            &connectivity_grid_locations[index_connectivity - 1], &connectivity_types[index_connectivity - 1],
            &connectivity_point_set_types[index_connectivity - 1], &connectivity_sizes[index_connectivity - 1], connectivity_donor_names[index_connectivity - 1].data(),
            &connectivity_donor_zone_types[index_connectivity - 1], &connectivity_donor_point_set_types[index_connectivity - 1],
            &connectivity_donor_data_types[index_connectivity - 1], &connectivity_donor_sizes[index_connectivity - 1]);

        if (connectivity_donor_zone_types[index_connectivity - 1] != ZoneType_t::Unstructured) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", connectivity " << index_connectivity << " has a donor zone type that isn't unstructured. For now only unstructured zones are supported. Exiting." << std::endl;
            exit(25);
        }
        if (connectivity_point_set_types[index_connectivity - 1] != PointSetType_t::PointList) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", connectivity " << index_connectivity << " has a point set type that isn't PointList. For now only PointList point set types are supported. Exiting." << std::endl;
            exit(26);
        }
        if (connectivity_donor_point_set_types[index_connectivity - 1] != PointSetType_t::PointListDonor) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", connectivity " << index_connectivity << " has a donor point set type that isn't PointListDonor. For now only PointListDonor point set types are supported. Exiting." << std::endl;
            exit(27);
        }

        if (connectivity_grid_locations[index_connectivity - 1] != GridLocation_t::FaceCenter) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", connectivity " << index_connectivity << " has a grid location that isn't FaceCenter. For now only FaceCenter grid locations are supported. Exiting." << std::endl;
            exit(28);
        }

        if (connectivity_types[index_connectivity - 1] != GridConnectivityType_t::Abutting1to1) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", connectivity " << index_connectivity << " has a grid connectivity type that isn't Abutting1to1. For now only Abutting1to1 grid connectivity types are supported. Exiting." << std::endl;
            exit(29);
        }

        if (connectivity_sizes[index_connectivity - 1] != connectivity_donor_sizes[index_connectivity - 1]) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", connectivity " << index_connectivity << " has a different number of elements in the origin and destination zones. Exiting." << std::endl;
            exit(30);
        }
    }

    std::vector<std::vector<cgsize_t>> interface_elements(n_connectivity);
    std::vector<std::vector<cgsize_t>> interface_donor_elements(n_connectivity);
    for (int index_connectivity = 1; index_connectivity <= n_connectivity; ++index_connectivity) {
        interface_elements[index_connectivity - 1] = std::vector<cgsize_t>(connectivity_sizes[index_connectivity - 1]);
        interface_donor_elements[index_connectivity - 1] = std::vector<cgsize_t>(connectivity_donor_sizes[index_connectivity - 1]);
        cg_conn_read(index_file, index_base, index_zone, index_connectivity, interface_elements[index_connectivity - 1].data(),
            DataType_t::Integer, interface_donor_elements[index_connectivity - 1].data());
    }

    // Boundary conditions
    int n_boundaries = 0;
    cg_nbocos(index_file, index_base, index_zone, &n_boundaries);

    std::vector<std::array<char, CGIO_MAX_NAME_LENGTH>> boundary_names(n_boundaries); // Oh yeah cause it's the 80s still
    std::vector<BCType_t> boundary_types(n_boundaries);
    std::vector<PointSetType_t> boundary_point_set_types(n_boundaries);
    std::vector<cgsize_t> boundary_sizes(n_boundaries);
    std::vector<int> boundary_normal_indices(n_boundaries);
    std::vector<cgsize_t> boundary_normal_list_sizes(n_boundaries);
    std::vector<DataType_t> boundary_normal_data_types(n_boundaries);
    std::vector<int> boundary_n_datasets(n_boundaries);
    std::vector<GridLocation_t> boundary_grid_locations(n_boundaries);
    for (int index_boundary = 1; index_boundary <= n_boundaries; ++index_boundary) {
        cg_boco_info(index_file, index_base, index_zone, index_boundary, boundary_names[index_boundary - 1].data(),
            &boundary_types[index_boundary - 1], &boundary_point_set_types[index_boundary - 1], &boundary_sizes[index_boundary - 1],
            &boundary_normal_indices[index_boundary - 1], &boundary_normal_list_sizes[index_boundary - 1],
            &boundary_normal_data_types[index_boundary - 1], &boundary_n_datasets[index_boundary - 1]);

        if (boundary_point_set_types[index_boundary - 1] != PointSetType_t::PointList) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", boundary " << index_boundary << " has a point set type that isn't PointList. For now only PointList point set types are supported. Exiting." << std::endl;
            exit(31);
        }

        cg_boco_gridlocation_read(index_file, index_base, index_zone, index_boundary, &boundary_grid_locations[index_boundary - 1]);

        if (boundary_grid_locations[index_boundary - 1] != GridLocation_t::EdgeCenter) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", boundary " << index_boundary << " has a grid location that isn't EdgeCenter. For now only EdgeCenter grid locations are supported. Exiting." << std::endl;
            exit(32);
        }
    }

    std::vector<std::vector<cgsize_t>> boundary_elements(n_boundaries);
    std::vector<std::vector<cgsize_t>> boundary_normals(n_boundaries);
    for (int index_boundary = 1; index_boundary <= n_boundaries; ++index_boundary) {
        boundary_elements[index_boundary - 1] = std::vector<cgsize_t>(boundary_sizes[index_boundary - 1]);
        boundary_normals[index_boundary - 1] = std::vector<cgsize_t>(boundary_normal_list_sizes[index_boundary - 1]);
        cg_boco_read(index_file, index_base, index_zone, index_boundary, boundary_elements[index_boundary - 1].data(), boundary_normals[index_boundary - 1].data());
    }

    const int close_error = cg_close(index_file);
    if (close_error != CG_OK) {
        std::cerr << "Error: file '" << filename << "' could not be closed with error '" << cg_get_error() << "'. Exiting." << std::endl;
        exit(44);
    }

    // Putting nodes in the format used by the mesh
    std::vector<Vec2<deviceFloat>> host_nodes(n_nodes);
    for (cgsize_t i = 0; i < n_nodes; ++i) {
        host_nodes[i].x() = xy[0][i];
        host_nodes[i].y() = xy[1][i];
    }

    // Transferring onto the GPU
    nodes_ = device_vector<Vec2<deviceFloat>>(host_nodes, stream_);

    // Figuring out which sections are the domain and which are ghost cells
    std::vector<bool> section_is_domain(n_sections);
    cgsize_t n_elements_domain = 0;
    cgsize_t n_elements_ghost = 0;
    for (int i = 0; i < n_sections; ++i) {
        switch (section_type[i]) {
            case ElementType_t::BAR_2:
                section_is_domain[i] = false;
                n_elements_ghost += section_ranges[i][1] - section_ranges[i][0] + 1;
                break;

            case ElementType_t::QUAD_4:
                section_is_domain[i] = true;
                n_elements_domain += section_ranges[i][1] - section_ranges[i][0] + 1;
                break;

            default:
                std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", section " << i << " has an unknown element type. For now only BAR_2 and QUAD_4 are implemented, for boundaries and domain respectively. Exiting." << std::endl;
                exit(33);
        }
    }

    if (n_elements_domain + n_elements_ghost != n_elements) {
        std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << " has " << n_elements << " elements but the sum of its sections is " << n_elements_domain + n_elements_ghost << " elements. Exiting." << std::endl;
        exit(34);
    }

    // Putting connectivity data in the format used by the mesh
    std::vector<Element2D_t> host_elements(n_elements);
    std::vector<size_t> section_start_indices(n_sections);
    size_t element_domain_index = 0;
    size_t element_ghost_index = n_elements_domain;
    for (int i = 0; i < n_sections; ++i) {
        if (section_is_domain[i]) {
            section_start_indices[i] = element_domain_index;
            for (cgsize_t j = 0; j < section_ranges[i][1] - section_ranges[i][0] + 1; ++j) {
                Element2D_t& element = host_elements[section_start_indices[i] + j];
                element.N_ = initial_N_;
                element.nodes_ = {static_cast<size_t>(connectivity[i][4 * j] - 1),
                                  static_cast<size_t>(connectivity[i][4 * j + 1] - 1),
                                  static_cast<size_t>(connectivity[i][4 * j + 2] - 1),
                                  static_cast<size_t>(connectivity[i][4 * j + 3] - 1)};       
            }
            element_domain_index += section_ranges[i][1] - section_ranges[i][0] + 1;
        }
        else {
            section_start_indices[i] = element_ghost_index;
            for (cgsize_t j = 0; j < section_ranges[i][1] - section_ranges[i][0] + 1; ++j) {
                Element2D_t& element = host_elements[section_start_indices[i] + j];
                element.N_ = initial_N_;
                element.nodes_ = {static_cast<size_t>(connectivity[i][2 * j] - 1),
                                  static_cast<size_t>(connectivity[i][2 * j + 1] - 1),
                                  static_cast<size_t>(connectivity[i][2 * j + 1] - 1),
                                  static_cast<size_t>(connectivity[i][2 * j] - 1)};
            }
            element_ghost_index += section_ranges[i][1] - section_ranges[i][0] + 1;
        }
    }

    // Computing nodes to elements
    const std::vector<std::vector<size_t>> node_to_element = build_node_to_element(n_nodes, host_elements);

    // Computing element to elements
    const std::vector<std::vector<size_t>> element_to_element = build_element_to_element(host_elements, node_to_element);

    // Computing faces and filling element faces
    auto [host_faces, node_to_face, element_to_face] = build_faces(n_elements_domain, n_nodes, initial_N_, host_elements);

    // Transferring onto the GPU
    elements_ = device_vector<Element2D_t>(host_elements, stream_);
    faces_ = device_vector<Face2D_t>(host_faces, stream_);

    // Building boundaries
    std::vector<size_t> wall_boundaries;
    std::vector<size_t> symmetry_boundaries;
    std::vector<size_t> inflow_boundaries;
    std::vector<size_t> outflow_boundaries;

    for (int i = 0; i < n_boundaries; ++i) {
        switch (boundary_types[i]) {
            case BCType_t::BCWall:
                wall_boundaries.reserve(wall_boundaries.size() + boundary_sizes[i]);
                for (cgsize_t j = 0; j < boundary_sizes[i]; ++j) {
                    int section_index = -1;
                    for (int k = 0; k < n_sections; ++k) {
                        if ((boundary_elements[i][j] >= section_ranges[k][0]) && (boundary_elements[i][j] <= section_ranges[k][1])) {
                            section_index = k;
                            break;
                        }
                    }

                    if (section_index == -1) {
                        std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", boundary " << i << " is a wall boundary and contains element " << boundary_elements[i][j] << " but it is not found in any mesh section. Exiting." << std::endl;
                        exit(36);
                    }

                    wall_boundaries.push_back(section_start_indices[section_index] + boundary_elements[i][j] - section_ranges[section_index][0]);
                }
                break;

            case BCType_t::BCSymmetryPlane:
                symmetry_boundaries.reserve(symmetry_boundaries.size() + boundary_sizes[i]);
                for (cgsize_t j = 0; j < boundary_sizes[i]; ++j) {
                    int section_index = -1;
                    for (int k = 0; k < n_sections; ++k) {
                        if ((boundary_elements[i][j] >= section_ranges[k][0]) && (boundary_elements[i][j] <= section_ranges[k][1])) {
                            section_index = k;
                            break;
                        }
                    }

                    if (section_index == -1) {
                        std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", boundary " << i << " is a symmetry boundary and contains element " << boundary_elements[i][j] << " but it is not found in any mesh section. Exiting." << std::endl;
                        exit(37);
                    }

                    symmetry_boundaries.push_back(section_start_indices[section_index] + boundary_elements[i][j] - section_ranges[section_index][0]);
                }
                
                break;

            case BCType_t::BCInflow:
                inflow_boundaries.reserve(inflow_boundaries.size() + boundary_sizes[i]);
                for (cgsize_t j = 0; j < boundary_sizes[i]; ++j) {
                    int section_index = -1;
                    for (int k = 0; k < n_sections; ++k) {
                        if ((boundary_elements[i][j] >= section_ranges[k][0]) && (boundary_elements[i][j] <= section_ranges[k][1])) {
                            section_index = k;
                            break;
                        }
                    }

                    if (section_index == -1) {
                        std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", boundary " << i << " is an inflow boundary and contains element " << boundary_elements[i][j] << " but it is not found in any mesh section. Exiting." << std::endl;
                        exit(36);
                    }

                    inflow_boundaries.push_back(section_start_indices[section_index] + boundary_elements[i][j] - section_ranges[section_index][0]);
                }

                break;
            
            case BCType_t::BCOutflow:
                outflow_boundaries.reserve(outflow_boundaries.size() + boundary_sizes[i]);
                for (cgsize_t j = 0; j < boundary_sizes[i]; ++j) {
                    int section_index = -1;
                    for (int k = 0; k < n_sections; ++k) {
                        if ((boundary_elements[i][j] >= section_ranges[k][0]) && (boundary_elements[i][j] <= section_ranges[k][1])) {
                            section_index = k;
                            break;
                        }
                    }

                    if (section_index == -1) {
                        std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", boundary " << i << " is an outflow boundary and contains element " << boundary_elements[i][j] << " but it is not found in any mesh section. Exiting." << std::endl;
                        exit(36);
                    }

                    outflow_boundaries.push_back(section_start_indices[section_index] + boundary_elements[i][j] - section_ranges[section_index][0]);
                }
            
                break;

            case BCType_t::BCTypeNull:
                break;

            default:
                std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", boundary " << i << " has an unknown boundary type. For now only BCWall, BCSymmetryPlane and BCTypeNull are implemented. Exiting." << std::endl;
                exit(35);
        }
    }

    // Transferring onto the GPU
    if (!wall_boundaries.empty()) {
        wall_boundaries_ = device_vector<size_t>(wall_boundaries, stream_);
    }
    if (!symmetry_boundaries.empty()) {
        symmetry_boundaries_ = device_vector<size_t>(symmetry_boundaries, stream_);
    }
    if (!inflow_boundaries.empty()) {
        inflow_boundaries_ = device_vector<size_t>(inflow_boundaries, stream_);
    }
    if (!outflow_boundaries.empty()) {
        outflow_boundaries_ = device_vector<size_t>(outflow_boundaries, stream_);
    }

    // Building self interfaces
    size_t n_interface_elements = 0;
    std::vector<size_t> interface_start_index(n_connectivity);
    for (int i = 0; i < n_connectivity; ++i) {
        if (strncmp(zone_name.data(), connectivity_donor_names[i].data(), CGIO_MAX_NAME_LENGTH) == 0) {
            interface_start_index[i] = n_interface_elements;
            n_interface_elements += connectivity_sizes[i];
        }
    }

    if (n_interface_elements > 0) {
        std::vector<size_t> interfaces_origin(n_interface_elements);
        std::vector<size_t> interfaces_origin_side(n_interface_elements);
        std::vector<size_t> interfaces_destination(n_interface_elements);

        for (int i = 0; i < n_connectivity; ++i) {
            if (strncmp(zone_name.data(), connectivity_donor_names[i].data(), CGIO_MAX_NAME_LENGTH) == 0) {
                for (cgsize_t j = 0; j < connectivity_sizes[i]; ++j) {
                    int origin_section_index = -1;
                    for (int k = 0; k < n_sections; ++k) {
                        if ((interface_elements[i][j] >= section_ranges[k][0]) && (interface_elements[i][j] <= section_ranges[k][1])) {
                            origin_section_index = k;
                            break;
                        }
                    }

                    if (origin_section_index == -1) {
                        std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", connectivity " << i << " contains element " << interface_elements[i][j] << " but it is not found in any mesh section. Exiting." << std::endl;
                        exit(38);
                    }

                    int donor_section_index = -1;
                    for (int k = 0; k < n_sections; ++k) {
                        if ((interface_donor_elements[i][j] >= section_ranges[k][0]) && (interface_donor_elements[i][j] <= section_ranges[k][1])) {
                            donor_section_index = k;
                            break;
                        }
                    }

                    if (donor_section_index == -1) {
                        std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", connectivity " << i << " contains donor element " << interface_donor_elements[i][j] << " but it is not found in any mesh section. Exiting." << std::endl;
                        exit(39);
                    }

                    const size_t donor_boundary_element_index = section_start_indices[donor_section_index] + interface_donor_elements[i][j] - section_ranges[donor_section_index][0];
                    const size_t face_index = element_to_face[donor_boundary_element_index][0];
                    const size_t face_side_index = host_faces[face_index].elements_[0] == donor_boundary_element_index;
                    const size_t donor_domain_element_index = host_faces[face_index].elements_[face_side_index];
                    
                    interfaces_origin[interface_start_index[i] + j] = donor_domain_element_index;
                    interfaces_origin_side[interface_start_index[i] + j] = host_faces[face_index].elements_side_[face_side_index];
                    interfaces_destination[interface_start_index[i] + j] = section_start_indices[origin_section_index] + interface_elements[i][j] - section_ranges[origin_section_index][0];
                }
            }
        }

        // Transferring onto the GPU
        interfaces_origin_ = device_vector<size_t>(interfaces_origin, stream_);
        interfaces_origin_side_ = device_vector<size_t>(interfaces_origin_side, stream_);
        interfaces_destination_ = device_vector<size_t>(interfaces_destination, stream_);
    }

    // Building MPI interfaces
    // These will be backwards due to how I did the element_side thing. Shouldn't affect much. If it does, just MPI transmit 
    std::vector<size_t> mpi_interface_process(n_connectivity, global_rank);
    std::vector<bool> process_used_in_interface(n_zones);
    size_t n_mpi_interface_elements = 0;
    for (int i = 0; i < n_connectivity; ++i) {
        if (strncmp(zone_name.data(), connectivity_donor_names[i].data(), CGIO_MAX_NAME_LENGTH) != 0) {
            mpi_interface_process[i] = global_rank;
            for (int j = 0; j < n_zones; ++j) {
                if (strncmp(connectivity_donor_names[i].data(), zone_names[j].data(), CGIO_MAX_NAME_LENGTH) == 0) {
                    mpi_interface_process[i] = j;
                    process_used_in_interface[j] = true;
                    break;
                }
            }
            if (mpi_interface_process[i] == global_rank) {
                std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", connectivity " << i << " links to zone \"" << connectivity_donor_names[i].data() << "\" but it is not found in any mesh section. Exiting." << std::endl;
                exit(50);
            }
            n_mpi_interface_elements += connectivity_sizes[i];
        }
    }

    if (n_mpi_interface_elements > 0) {
        size_t n_mpi_interfaces = 0;
        for (int j = 0; j < n_zones; ++j) {
            n_mpi_interfaces += process_used_in_interface[j];
        }

        mpi_interfaces_size_ = std::vector<size_t>(n_mpi_interfaces);
        mpi_interfaces_offset_ = std::vector<size_t>(n_mpi_interfaces);
        mpi_interfaces_process_ = std::vector<size_t>(n_mpi_interfaces);
        std::vector<size_t> mpi_interfaces_origin(n_mpi_interface_elements);
        std::vector<size_t> mpi_interfaces_origin_side(n_mpi_interface_elements);
        std::vector<size_t> mpi_interfaces_destination(n_mpi_interface_elements);
        std::vector<size_t> mpi_interfaces_destination_in_this_proc(n_mpi_interface_elements);

        size_t mpi_interface_offset = 0;
        size_t mpi_interface_index = 0;
        for (int j = 0; j < n_zones; ++j) {
            if (process_used_in_interface[j]) {
                mpi_interfaces_offset_[mpi_interface_index] = mpi_interface_offset;
                mpi_interfaces_process_[mpi_interface_index] = j;
                for (int i = 0; i < n_connectivity; ++i) {
                    if (mpi_interface_process[i] == j) {
                        mpi_interfaces_size_[mpi_interface_index] += connectivity_sizes[i];
                        for (size_t k = 0; k < connectivity_sizes[i]; ++k) {
                            int origin_section_index = -1;
                            for (int m = 0; m < n_sections; ++m) {
                                if ((interface_elements[i][k] >= section_ranges[m][0]) && (interface_elements[i][k] <= section_ranges[m][1])) {
                                    origin_section_index = m;
                                    break;
                                }
                            }

                            if (origin_section_index == -1) {
                                std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", connectivity " << i << " contains element " << interface_elements[i][k] << " but it is not found in any mesh section. Exiting." << std::endl;
                                exit(38);
                            }

                            // Starts to be backwards here
                            const size_t boundary_element_index = section_start_indices[origin_section_index] + interface_elements[i][k] - section_ranges[origin_section_index][0];
                            const size_t face_index = element_to_face[boundary_element_index][0];
                            const size_t face_side_index = host_faces[face_index].elements_[0] == boundary_element_index;

                            mpi_interfaces_origin[mpi_interface_offset + k]      = host_faces[face_index].elements_[face_side_index];;
                            mpi_interfaces_origin_side[mpi_interface_offset + k] = host_faces[face_index].elements_side_[face_side_index];
                            mpi_interfaces_destination[mpi_interface_offset + k] = interface_donor_elements[i][k]; // Still in local referential, will have to exchange info to know.
                        }

                        mpi_interface_offset += connectivity_sizes[i];
                    }
                }

                ++mpi_interface_index;
            }
        }

        // Exchanging mpi interfaces destination
        std::vector<MPI_Request> adaptivity_requests(2 * n_mpi_interfaces);
        std::vector<MPI_Status> adaptivity_statuses(2 * n_mpi_interfaces);
        constexpr MPI_Datatype data_type = (sizeof(size_t) == sizeof(unsigned long long)) ? MPI_UNSIGNED_LONG_LONG : (sizeof(size_t) == sizeof(unsigned long)) ? MPI_UNSIGNED_LONG : MPI_UNSIGNED; // CHECK this is a bad way of doing this

        for (size_t i = 0; i < n_mpi_interfaces; ++i) {
            MPI_Isend(mpi_interfaces_destination.data() + mpi_interfaces_offset_[i], mpi_interfaces_size_[i], data_type, mpi_interfaces_process_[i], global_size * global_rank + mpi_interfaces_process_[i], MPI_COMM_WORLD, &adaptivity_requests[n_mpi_interfaces + i]);
            MPI_Irecv(mpi_interfaces_destination_in_this_proc.data() + mpi_interfaces_offset_[i], mpi_interfaces_size_[i], data_type, mpi_interfaces_process_[i],  global_size * mpi_interfaces_process_[i] + global_rank, MPI_COMM_WORLD, &adaptivity_requests[i]);
        }

        MPI_Waitall(n_mpi_interfaces, adaptivity_requests.data(), adaptivity_statuses.data());

        for (size_t i = 0; i < n_mpi_interfaces; ++i) {
            for (size_t j = 0; j < mpi_interfaces_size_[i]; ++j) {
                int donor_section_index = -1;
                for (int k = 0; k < n_sections; ++k) {
                    if ((mpi_interfaces_destination_in_this_proc[mpi_interfaces_offset_[i] + j] >= section_ranges[k][0]) && (mpi_interfaces_destination_in_this_proc[mpi_interfaces_offset_[i] + j] <= section_ranges[k][1])) {
                        donor_section_index = k;
                        break;
                    }
                }

                if (donor_section_index == -1) {
                    std::cerr << "Error: Process " << mpi_interfaces_process_[i] << " sent element " << mpi_interfaces_destination_in_this_proc[mpi_interfaces_offset_[i] + j] << " to process " << global_rank << " but it is not found in any mesh section. Exiting." << std::endl;
                    exit(51);
                }

                mpi_interfaces_destination_in_this_proc[mpi_interfaces_offset_[i] + j] = section_start_indices[donor_section_index] + mpi_interfaces_destination_in_this_proc[mpi_interfaces_offset_[i] + j] - section_ranges[donor_section_index][0];
            }
        }

        MPI_Waitall(n_mpi_interfaces, adaptivity_requests.data() + n_mpi_interfaces, adaptivity_statuses.data() + n_mpi_interfaces);

        // Transferring onto the GPU
        mpi_interfaces_origin_ = device_vector<size_t>(mpi_interfaces_origin, stream_);
        mpi_interfaces_origin_side_ = device_vector<size_t>(mpi_interfaces_origin_side, stream_);
        mpi_interfaces_destination_ = device_vector<size_t>(mpi_interfaces_destination_in_this_proc, stream_);

        device_interfaces_p_ = device_vector<deviceFloat>(mpi_interfaces_origin.size() * (maximum_N_ + 1), stream_);
        device_interfaces_u_ = device_vector<deviceFloat>(mpi_interfaces_origin.size() * (maximum_N_ + 1), stream_);
        device_interfaces_v_ = device_vector<deviceFloat>(mpi_interfaces_origin.size() * (maximum_N_ + 1), stream_);
        device_interfaces_N_ = device_vector<int>(mpi_interfaces_origin.size(), stream_);
        host_interfaces_p_ = host_vector<deviceFloat>(mpi_interfaces_origin.size() * (maximum_N_ + 1));
        host_interfaces_u_ = host_vector<deviceFloat>(mpi_interfaces_origin.size() * (maximum_N_ + 1));
        host_interfaces_v_ = host_vector<deviceFloat>(mpi_interfaces_origin.size() * (maximum_N_ + 1));
        host_interfaces_N_ = std::vector<int>(mpi_interfaces_origin.size());
        host_receiving_interfaces_p_ = host_vector<deviceFloat>(mpi_interfaces_origin.size() * (maximum_N_ + 1));
        host_receiving_interfaces_u_ = host_vector<deviceFloat>(mpi_interfaces_origin.size() * (maximum_N_ + 1));
        host_receiving_interfaces_v_ = host_vector<deviceFloat>(mpi_interfaces_origin.size() * (maximum_N_ + 1));
        host_receiving_interfaces_N_ = std::vector<int>(mpi_interfaces_origin.size());

        requests_ = std::vector<MPI_Request>(n_mpi_interfaces * 6);
        statuses_ = std::vector<MPI_Status>(n_mpi_interfaces * 6);
        requests_N_ = std::vector<MPI_Request>(n_mpi_interfaces * 2);
        statuses_N_ = std::vector<MPI_Status>(n_mpi_interfaces * 2);
    }

    // Setting sizes
    n_elements_ = n_elements_domain;
    elements_numBlocks_ = (n_elements_ + elements_blockSize_ - 1) / elements_blockSize_;
    faces_numBlocks_ = (faces_.size() + faces_blockSize_ - 1) / faces_blockSize_;
    wall_boundaries_numBlocks_ = (wall_boundaries_.size() + boundaries_blockSize_ - 1) / boundaries_blockSize_;
    symmetry_boundaries_numBlocks_ = (symmetry_boundaries_.size() + boundaries_blockSize_ - 1) / boundaries_blockSize_;
    inflow_boundaries_numBlocks_ = (inflow_boundaries_.size() + boundaries_blockSize_ - 1) / boundaries_blockSize_;
    outflow_boundaries_numBlocks_ = (outflow_boundaries_.size() + boundaries_blockSize_ - 1) / boundaries_blockSize_;
    ghosts_numBlocks_ = (n_elements_ghost + boundaries_blockSize_ - 1) / boundaries_blockSize_;
    interfaces_numBlocks_ = (interfaces_origin_.size() + boundaries_blockSize_ - 1) / boundaries_blockSize_;
    mpi_interfaces_numBlocks_ = (mpi_interfaces_origin_.size() + boundaries_blockSize_ - 1) / boundaries_blockSize_;

    // Sharing number of elements to calculate offset
    std::vector<size_t> n_elements_per_process(global_size);
    constexpr MPI_Datatype size_t_data_type = (sizeof(size_t) == sizeof(unsigned long long)) ? MPI_UNSIGNED_LONG_LONG : (sizeof(size_t) == sizeof(unsigned long)) ? MPI_UNSIGNED_LONG : MPI_UNSIGNED; // CHECK this is a bad way of doing this
    MPI_Allgather(&n_elements_, 1, size_t_data_type, n_elements_per_process.data(), 1, size_t_data_type, MPI_COMM_WORLD);

    size_t n_elements_global = 0;
    for (int i = 0; i < global_rank; ++i) {
        n_elements_global += n_elements_per_process[i];
    }
    global_element_offset_ = n_elements_global;
    for (int i = global_rank; i < global_size; ++i) {
        n_elements_global += n_elements_per_process[i];
    }
    n_elements_global_ = n_elements_global;

    // Transfer arrays
    host_delta_t_array_ = host_vector<deviceFloat>(elements_numBlocks_);
    device_delta_t_array_ = device_vector<deviceFloat>(elements_numBlocks_, stream_);
    host_refine_array_ = std::vector<size_t>(elements_numBlocks_);
    device_refine_array_ = device_vector<size_t>(elements_numBlocks_, stream_);
    host_nodes_refine_array_ = std::vector<size_t>(elements_numBlocks_);
    device_nodes_refine_array_ = device_vector<size_t>(elements_numBlocks_, stream_);

    allocate_element_storage<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(n_elements_, elements_.data());
    allocate_boundary_storage<<<ghosts_numBlocks_, boundaries_blockSize_, 0, stream_>>>(n_elements_, elements_.size(), elements_.data());
    allocate_face_storage<<<faces_numBlocks_, faces_blockSize_, 0, stream_>>>(faces_.size(), faces_.data());

    device_vector<std::array<size_t, 4>> device_element_to_face(element_to_face, stream_);
    fill_element_faces<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(elements_.size(), elements_.data(), device_element_to_face.data());
    device_element_to_face.clear(stream_);

    // Allocating output arrays
    x_output_host_ = std::vector<deviceFloat>(n_elements_ * std::pow(n_interpolation_points_, 2));
    y_output_host_ = std::vector<deviceFloat>(n_elements_ * std::pow(n_interpolation_points_, 2));
    p_output_host_ = std::vector<deviceFloat>(n_elements_ * std::pow(n_interpolation_points_, 2));
    u_output_host_ = std::vector<deviceFloat>(n_elements_ * std::pow(n_interpolation_points_, 2));
    v_output_host_ = std::vector<deviceFloat>(n_elements_ * std::pow(n_interpolation_points_, 2));
    x_output_device_ = device_vector<deviceFloat>(n_elements_ * std::pow(n_interpolation_points_, 2), stream_);
    y_output_device_ = device_vector<deviceFloat>(n_elements_ * std::pow(n_interpolation_points_, 2), stream_);
    p_output_device_ = device_vector<deviceFloat>(n_elements_ * std::pow(n_interpolation_points_, 2), stream_);
    u_output_device_ = device_vector<deviceFloat>(n_elements_ * std::pow(n_interpolation_points_, 2), stream_);
    v_output_device_ = device_vector<deviceFloat>(n_elements_ * std::pow(n_interpolation_points_, 2), stream_);
}

auto SEM::Meshes::Mesh2D_t::build_node_to_element(size_t n_nodes, const std::vector<Element2D_t>& elements) -> std::vector<std::vector<size_t>> {
    std::vector<std::vector<size_t>> node_to_element(n_nodes);

    for (size_t j = 0; j < elements.size(); ++j) {
        for (auto node_index: elements[j].nodes_) {
            if (std::find(node_to_element[node_index].begin(), node_to_element[node_index].end(), j) == node_to_element[node_index].end()) { // This will be slower, but is needed because boundaries have 4 sides and not 2. Remove when variable geometry elements are added.
                node_to_element[node_index].push_back(j);
            }
        }
    }

    return node_to_element;
}

auto SEM::Meshes::Mesh2D_t::build_element_to_element(const std::vector<Element2D_t>& elements, const std::vector<std::vector<size_t>>& node_to_element) -> std::vector<std::vector<size_t>> {
    std::vector<std::vector<size_t>> element_to_element(elements.size());

    for (size_t i = 0; i < elements.size(); ++i) {
        const Element2D_t& element = elements[i];
        element_to_element[i] = std::vector<size_t>(element.nodes_.size());

        for (size_t j = 0; j < element.nodes_.size(); ++j) {
            const size_t node_index = element.nodes_[j];
            const size_t node_index_next = (j < element.nodes_.size() - 1) ? element.nodes_[j + 1] : element.nodes_[0];

            for (auto element_index : node_to_element[node_index]) {
                if (element_index != i) {
                    const Element2D_t& element_neighbor = elements[element_index];

                    auto it = std::find(element_neighbor.nodes_.begin(), element_neighbor.nodes_.end(), node_index);
                    if (it != element_neighbor.nodes_.end()) {
                        const size_t node_element_index = it - element_neighbor.nodes_.begin();
                        
                        for (size_t node_element_index_next = 0; node_element_index_next < node_element_index; ++node_element_index_next) {
                            if (element_neighbor.nodes_[node_element_index_next] == node_index_next) {
                                element_to_element[i][j] = element_index;
                                goto endloop; // I hate this too don't worry
                            }
                        }

                        for (size_t node_element_index_next = node_element_index + 1; node_element_index_next < element_neighbor.nodes_.size(); ++node_element_index_next) {
                            if (element_neighbor.nodes_[node_element_index_next] == node_index_next) {
                                element_to_element[i][j] = element_index;
                                goto endloop; // I hate this too don't worry
                            }
                        }
                    }
                }
            }
            endloop: ;
        }
    }
 
    return element_to_element;
}

auto SEM::Meshes::Mesh2D_t::build_faces(size_t n_elements_domain, size_t n_nodes, int initial_N, const std::vector<Element2D_t>& elements) -> std::tuple<std::vector<Face2D_t>, std::vector<std::vector<size_t>>, std::vector<std::array<size_t, 4>>> {
    size_t total_edges = 0;
    for (const auto& element: elements) {
        total_edges += element.nodes_.size();
    }

    std::vector<Face2D_t> faces;
    faces.reserve(total_edges/2); // This is not exact

    std::vector<std::vector<size_t>> node_to_face(n_nodes);
    std::vector<std::array<size_t, 4>> element_to_face(elements.size());

    for (size_t i = 0; i < n_elements_domain; ++i) {
        for (size_t j = 0; j < elements[i].nodes_.size(); ++j) {
            const std::array<size_t, 2> nodes{elements[i].nodes_[j], (j < elements[i].nodes_.size() - 1) ? elements[i].nodes_[j + 1] : elements[i].nodes_[0]};

            bool found = false;
            for (auto face_index: node_to_face[nodes[0]]) {
                if ((faces[face_index].nodes_[0] == nodes[1]) && (faces[face_index].nodes_[1] == nodes[0])) {
                    found = true;
                    faces[face_index].elements_[1] = i;
                    faces[face_index].elements_side_[1] = j;
                    element_to_face[i][j] = face_index;
                    break;
                }
            }

            if (!found) {
                element_to_face[i][j] = faces.size();
                node_to_face[nodes[0]].push_back(faces.size());
                if (nodes[1] != nodes[0]) {
                    node_to_face[nodes[1]].push_back(faces.size());
                }
                faces.emplace_back();
                faces.back().N_ = initial_N;
                faces.back().nodes_ = {nodes[0], nodes[1]};
                faces.back().elements_ = {i, static_cast<size_t>(-1)};
                faces.back().elements_side_ = {j, static_cast<size_t>(-1)};
            }
        }
    }

    for (size_t i = n_elements_domain; i < elements.size(); ++i) {
        const std::array<size_t, 2> nodes{elements[i].nodes_[0], elements[i].nodes_[1]};

        for (auto face_index: node_to_face[nodes[0]]) {
            if ((faces[face_index].nodes_[0] == nodes[1]) && (faces[face_index].nodes_[1] == nodes[0])) {
                faces[face_index].elements_[1] = i;
                faces[face_index].elements_side_[1] = 0;
                element_to_face[i][0] = face_index;
                for (size_t j = 1; j < element_to_face[i].size(); ++j) {
                    element_to_face[i][j] = static_cast<size_t>(-1);
                }
                break;
            }
        }
    }

    // Faces have to be moved, or else this copies the vector, and the device (???) vector copy for face vectors is used, which bad allocs for some reason.
    // 1) Why doesn't this move the vector, as it would be if it was plain returned?
    // 2) Why is the device copy used, it shouldn't be able to be called from that's like the whole point.
    // 3) Why does it bad alloc, the copied face should have its size default-constructed to 0.
    return {std::move(faces), std::move(node_to_face), std::move(element_to_face)};
}

auto SEM::Meshes::Mesh2D_t::initial_conditions(const device_vector<deviceFloat>& polynomial_nodes) -> void {
    initial_conditions_2D<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(n_elements_, elements_.data(), nodes_.data(), polynomial_nodes.data());
}

auto SEM::Meshes::Mesh2D_t::print() const -> void {
    std::vector<Face2D_t> host_faces(faces_.size());
    std::vector<Element2D_t> host_elements(elements_.size());
    std::vector<Vec2<deviceFloat>> host_nodes(nodes_.size());
    std::vector<size_t> host_wall_boundaries(wall_boundaries_.size());
    std::vector<size_t> host_symmetry_boundaries(symmetry_boundaries_.size());
    std::vector<size_t> host_inflow_boundaries(inflow_boundaries_.size());
    std::vector<size_t> host_outflow_boundaries(outflow_boundaries_.size());
    std::vector<size_t> host_interfaces_origin(interfaces_origin_.size());
    std::vector<size_t> host_interfaces_origin_side(interfaces_origin_side_.size());
    std::vector<size_t> host_interfaces_destination(interfaces_destination_.size());
    
    faces_.copy_to(host_faces, stream_);
    elements_.copy_to(host_elements, stream_);
    nodes_.copy_to(host_nodes, stream_);
    wall_boundaries_.copy_to(host_wall_boundaries, stream_);
    symmetry_boundaries_.copy_to(host_symmetry_boundaries, stream_);
    inflow_boundaries_.copy_to(host_inflow_boundaries, stream_);
    outflow_boundaries_.copy_to(host_outflow_boundaries, stream_);
    interfaces_origin_.copy_to(host_interfaces_origin, stream_);
    interfaces_origin_side_.copy_to(host_interfaces_origin_side, stream_);
    interfaces_destination_.copy_to(host_interfaces_destination, stream_);
    hipStreamSynchronize(stream_);
    
    std::cout << "N elements: " << n_elements_ << std::endl;
    std::cout << "N elements and ghosts: " << elements_.size() << std::endl;
    std::cout << "N faces: " << faces_.size() << std::endl;
    std::cout << "N nodes: " << nodes_.size() << std::endl;
    std::cout << "N wall boundaries: " << wall_boundaries_.size() << std::endl;
    std::cout << "N symmetry boundaries: " << symmetry_boundaries_.size() << std::endl;
    std::cout << "N inflow boundaries: " << inflow_boundaries_.size() << std::endl;
    std::cout << "N outflow boundaries: " << outflow_boundaries_.size() << std::endl;
    std::cout << "N interfaces: " << interfaces_origin_.size() << std::endl;
    std::cout << "Initial N: " << initial_N_ << std::endl;

    std::cout << std::endl <<  "Connectivity" << std::endl;
    std::cout << '\t' <<  "Nodes:" << std::endl;
    for (size_t i = 0; i < host_nodes.size(); ++i) {
        std::cout << '\t' << '\t' << "node " << i << " : " << host_nodes[i] << std::endl;
    }

    std::cout << std::endl << '\t' <<  "Element nodes:" << std::endl;
    for (size_t i = 0; i < host_elements.size(); ++i) {
        std::cout << '\t' << '\t' << "element " << i << " : ";
        for (auto node_index : host_elements[i].nodes_) {
            std::cout << node_index << " ";
        }
        std::cout << std::endl;
    }

    std::cout << std::endl << '\t' <<  "Face nodes:" << std::endl;
    for (size_t i = 0; i < host_faces.size(); ++i) {
        std::cout << '\t' << '\t' << "face " << i << " : ";
        for (auto node_index : host_faces[i].nodes_) {
            std::cout << node_index << " ";
        }
        std::cout << std::endl;
    }

    std::cout << std::endl << '\t' <<  "Face elements:" << std::endl;
    for (size_t i = 0; i < host_faces.size(); ++i) {
        std::cout << '\t' << '\t' << "face " << i << " : ";
        for (auto element_index : host_faces[i].elements_) {
            std::cout << element_index << " ";
        }
        std::cout << std::endl;
    }

    std::cout << std::endl << '\t' <<  "Face elements side:" << std::endl;
    for (size_t i = 0; i < host_faces.size(); ++i) {
        std::cout << '\t' << '\t' << "face " << i << " : ";
        for (auto side_index : host_faces[i].elements_side_) {
            std::cout << side_index << " ";
        }
        std::cout << std::endl;
    }

    std::cout << std::endl <<  "Geometry" << std::endl;
    std::cout << '\t' <<  "Element min length:" << std::endl;
    for (size_t i = 0; i < host_elements.size(); ++i) {
        std::cout << '\t' << '\t' << "element " << i << " : " << host_elements[i].delta_xy_min_ << std::endl;
    }

    std::cout << std::endl << '\t' <<  "Element N:" << std::endl;
    for (size_t i = 0; i < host_elements.size(); ++i) {
        std::cout << '\t' << '\t' << "element " << i << " : " << host_elements[i].N_ << std::endl;
    }
    
    std::cout << std::endl << '\t' <<  "Face N:" << std::endl;
    for (size_t i = 0; i < host_faces.size(); ++i) {
        std::cout << '\t' << '\t' << "face " << i << " : " << host_faces[i].N_ << std::endl;
    }

    std::cout << std::endl << '\t' <<  "Face length:" << std::endl;
    for (size_t i = 0; i < host_faces.size(); ++i) {
        std::cout << '\t' << '\t' << "face " << i << " : " << host_faces[i].length_ << std::endl;
    }

    std::cout << std::endl << '\t' <<  "Face normal:" << std::endl;
    for (size_t i = 0; i < host_faces.size(); ++i) {
        std::cout << '\t' << '\t' << "face " << i << " : " << host_faces[i].normal_ << std::endl;
    }

    std::cout << std::endl << '\t' <<  "Face tangent:" << std::endl;
    for (size_t i = 0; i < host_faces.size(); ++i) {
        std::cout << '\t' << '\t' << "face " << i << " : " << host_faces[i].tangent_ << std::endl;
    }

    std::cout << std::endl <<  "Interfaces" << std::endl;
    std::cout << '\t' <<  "Interface destination, origin and origin side:" << std::endl;
    for (size_t i = 0; i < host_interfaces_origin.size(); ++i) {
        std::cout << '\t' << '\t' << "interface " << i << " : " << host_interfaces_destination[i] << " " << host_interfaces_origin[i] << " " << host_interfaces_origin_side[i] << std::endl;
    }

    std::cout << std::endl << '\t' <<  "Wall boundaries:" << std::endl;
    for (size_t i = 0; i < host_wall_boundaries.size(); ++i) {
        std::cout << '\t' << '\t' << "wall " << i << " : " << host_wall_boundaries[i] << std::endl;
    }

    std::cout << std::endl << '\t' <<  "Symmetry boundaries:" << std::endl;
    for (size_t i = 0; i < host_symmetry_boundaries.size(); ++i) {
        std::cout << '\t' << '\t' << "symmetry " << i << " : " << host_symmetry_boundaries[i] << std::endl;
    }

    std::cout << std::endl << '\t' <<  "Inflow boundaries:" << std::endl;
    for (size_t i = 0; i < host_inflow_boundaries.size(); ++i) {
        std::cout << '\t' << '\t' << "inflow " << i << " : " << host_inflow_boundaries[i] << std::endl;
    }

    std::cout << std::endl << '\t' <<  "Outflow boundaries:" << std::endl;
    for (size_t i = 0; i < host_outflow_boundaries.size(); ++i) {
        std::cout << '\t' << '\t' << "outflow " << i << " : " << host_outflow_boundaries[i] << std::endl;
    }

    std::cout << std::endl;
}

auto SEM::Meshes::Mesh2D_t::write_data(deviceFloat time, const device_vector<deviceFloat>& interpolation_matrices, const SEM::Helpers::DataWriter_t& data_writer) -> void {
    SEM::Meshes::get_solution<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(n_elements_, n_interpolation_points_, elements_.data(), nodes_.data(), interpolation_matrices.data(), x_output_device_.data(), y_output_device_.data(), p_output_device_.data(), u_output_device_.data(), v_output_device_.data());
    
    x_output_device_.copy_to(x_output_host_, stream_);
    y_output_device_.copy_to(y_output_host_, stream_);
    p_output_device_.copy_to(p_output_host_, stream_);
    u_output_device_.copy_to(u_output_host_, stream_);
    v_output_device_.copy_to(v_output_host_, stream_);
    hipStreamSynchronize(stream_);

    data_writer.write_data(n_interpolation_points_, n_elements_, time, x_output_host_, y_output_host_, p_output_host_, u_output_host_, v_output_host_);
}

auto SEM::Meshes::Mesh2D_t::write_complete_data(deviceFloat time, const device_vector<deviceFloat>& interpolation_matrices, const SEM::Helpers::DataWriter_t& data_writer) -> void {
    device_vector<deviceFloat> dp_dt(n_elements_ * n_interpolation_points_ * n_interpolation_points_, stream_);
    device_vector<deviceFloat> du_dt(n_elements_ * n_interpolation_points_ * n_interpolation_points_, stream_);
    device_vector<deviceFloat> dv_dt(n_elements_ * n_interpolation_points_ * n_interpolation_points_, stream_);
    device_vector<int> N(n_elements_, stream_);
    device_vector<deviceFloat> p_error(n_elements_, stream_);
    device_vector<deviceFloat> u_error(n_elements_, stream_);
    device_vector<deviceFloat> v_error(n_elements_, stream_);
    device_vector<deviceFloat> p_sigma(n_elements_, stream_);
    device_vector<deviceFloat> u_sigma(n_elements_, stream_);
    device_vector<deviceFloat> v_sigma(n_elements_, stream_);
    device_vector<int> refine(n_elements_, stream_);
    device_vector<int> coarsen(n_elements_, stream_);
    device_vector<int> split_level(n_elements_, stream_);

    SEM::Meshes::get_complete_solution<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(n_elements_, n_interpolation_points_, elements_.data(), nodes_.data(), interpolation_matrices.data(), x_output_device_.data(), y_output_device_.data(), p_output_device_.data(), u_output_device_.data(), v_output_device_.data(), N.data(), dp_dt.data(), du_dt.data(), dv_dt.data(), p_error.data(), u_error.data(), v_error.data(), p_sigma.data(), u_sigma.data(), v_sigma.data(), refine.data(), coarsen.data(), split_level.data());
    
    std::vector<deviceFloat> dp_dt_host(n_elements_ * n_interpolation_points_ * n_interpolation_points_);
    std::vector<deviceFloat> du_dt_host(n_elements_ * n_interpolation_points_ * n_interpolation_points_);
    std::vector<deviceFloat> dv_dt_host(n_elements_ * n_interpolation_points_ * n_interpolation_points_);
    std::vector<int> N_host(n_elements_);
    std::vector<deviceFloat> p_error_host(n_elements_);
    std::vector<deviceFloat> u_error_host(n_elements_);
    std::vector<deviceFloat> v_error_host(n_elements_);
    std::vector<deviceFloat> p_sigma_host(n_elements_);
    std::vector<deviceFloat> u_sigma_host(n_elements_);
    std::vector<deviceFloat> v_sigma_host(n_elements_);
    std::vector<int> refine_host(n_elements_);
    std::vector<int> coarsen_host(n_elements_);
    std::vector<int> split_level_host(n_elements_);

    x_output_device_.copy_to(x_output_host_, stream_);
    y_output_device_.copy_to(y_output_host_, stream_);
    p_output_device_.copy_to(p_output_host_, stream_);
    u_output_device_.copy_to(u_output_host_, stream_);
    v_output_device_.copy_to(v_output_host_, stream_);
    dp_dt.copy_to(dp_dt_host, stream_);
    du_dt.copy_to(du_dt_host, stream_);
    dv_dt.copy_to(dv_dt_host, stream_);
    N.copy_to(N_host, stream_);
    p_error.copy_to(p_error_host, stream_);
    u_error.copy_to(u_error_host, stream_);
    v_error.copy_to(v_error_host, stream_);
    p_sigma.copy_to(p_sigma_host, stream_);
    u_sigma.copy_to(u_sigma_host, stream_);
    v_sigma.copy_to(v_sigma_host, stream_);
    refine.copy_to(refine_host, stream_);
    coarsen.copy_to(coarsen_host, stream_);
    split_level.copy_to(split_level_host, stream_);
    hipStreamSynchronize(stream_);

    data_writer.write_complete_data(n_interpolation_points_, n_elements_, time, x_output_host_, y_output_host_, p_output_host_, u_output_host_, v_output_host_, N_host, dp_dt_host, du_dt_host, dv_dt_host, p_error_host, u_error_host, v_error_host, p_sigma_host, u_sigma_host, v_sigma_host, refine_host, coarsen_host, split_level_host);

    dp_dt.clear(stream_);
    du_dt.clear(stream_);
    dv_dt.clear(stream_);
    N.clear(stream_);
    p_error.clear(stream_);
    u_error.clear(stream_);
    v_error.clear(stream_);
    p_sigma.clear(stream_);
    u_sigma.clear(stream_);
    v_sigma.clear(stream_);
    refine.clear(stream_);
    coarsen.clear(stream_);
    split_level.clear(stream_);
}

__host__ __device__
auto SEM::Meshes::Mesh2D_t::g(Vec2<deviceFloat> xy, deviceFloat t) -> std::array<deviceFloat, 3> {    
    const deviceFloat p = std::exp(-(SEM::Constants::k.x() * (xy.x() - SEM::Constants::xy0.x()) + SEM::Constants::k.y() * (xy.y() - SEM::Constants::xy0.y()) - SEM::Constants::c * t) * (SEM::Constants::k.x() * (xy.x() - SEM::Constants::xy0.x()) + SEM::Constants::k.y() * (xy.y() - SEM::Constants::xy0.y()) - SEM::Constants::c * t) / (SEM::Constants::d * SEM::Constants::d));

    return {p,
            p * SEM::Constants::k.x() / SEM::Constants::c,
            p * SEM::Constants::k.y() / SEM::Constants::c};
}

auto SEM::Meshes::Mesh2D_t::adapt(int N_max, const device_vector<deviceFloat>& polynomial_nodes, const device_vector<deviceFloat>& barycentric_weights) -> void {
    SEM::Meshes::reduce_refine_2D<elements_blockSize_/2><<<elements_numBlocks_, elements_blockSize_/2, 0, stream_>>>(n_elements_, max_split_level_, elements_.data(), device_refine_array_.data());
    device_refine_array_.copy_to(host_refine_array_, stream_);
    hipStreamSynchronize(stream_);

    size_t splitting_elements = 0;
    for (int i = 0; i < elements_numBlocks_; ++i) {
        host_refine_array_[i] = 3 * splitting_elements; // Current block offset
        splitting_elements += host_refine_array_[i];
    }

    int global_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &global_rank);
    int global_size;
    MPI_Comm_size(MPI_COMM_WORLD, &global_size);

    std::vector<size_t> splitting_elements_global(global_size);
    constexpr MPI_Datatype data_type = (sizeof(size_t) == sizeof(unsigned long long)) ? MPI_UNSIGNED_LONG_LONG : (sizeof(size_t) == sizeof(unsigned long)) ? MPI_UNSIGNED_LONG : MPI_UNSIGNED; // CHECK this is a bad way of doing this

    MPI_Allgather(&splitting_elements, 1, data_type, splitting_elements_global.data(), 1, data_type, MPI_COMM_WORLD);

    size_t N_splitting_elements_previous = 0;
    for (int i = 0; i < global_rank; ++i) {
        N_splitting_elements_previous += splitting_elements_global[i];
    }
    const size_t global_element_offset_current = global_element_offset_ + 3 * N_splitting_elements_previous;
    size_t N_splitting_elements_global = 0;
    for (int i = 0; i < global_size; ++i) {
        N_splitting_elements_global += splitting_elements_global[i];
    }
    n_elements_global_ += 3 * N_splitting_elements_global;
    const size_t global_element_offset_end_current = global_element_offset_current + n_elements_ + 3 * splitting_elements - 1;

    const size_t n_elements_per_process = (n_elements_global_ + global_size - 1)/global_size;
    global_element_offset_ = global_rank * n_elements_per_process;
    const size_t global_element_offset_end = std::min(global_element_offset_ + n_elements_per_process - 1, n_elements_global_ - 1);

    if ((splitting_elements == 0) && (global_element_offset_ == global_element_offset_current) && (global_element_offset_end == global_element_offset_end_current)) {
        SEM::Meshes::p_adapt<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(n_elements_, elements_.data(), N_max, nodes_.data(), polynomial_nodes.data(), barycentric_weights.data());

        // We need to adjust the boundaries in all cases, or check if of our neighbours have to change
        if (!wall_boundaries_.empty()) {
            SEM::Meshes::adjust_boundaries<<<wall_boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(wall_boundaries_.size(), elements_.data(), wall_boundaries_.data(), faces_.data());
        }
        if (!symmetry_boundaries_.empty()) {
            SEM::Meshes::adjust_boundaries<<<symmetry_boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(symmetry_boundaries_.size(), elements_.data(), symmetry_boundaries_.data(), faces_.data());
        }
        if (!inflow_boundaries_.empty()) {
            SEM::Meshes::adjust_boundaries<<<inflow_boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(inflow_boundaries_.size(), elements_.data(), inflow_boundaries_.data(), faces_.data());
        }
        if (!outflow_boundaries_.empty()) {
            SEM::Meshes::adjust_boundaries<<<outflow_boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(outflow_boundaries_.size(), elements_.data(), outflow_boundaries_.data(), faces_.data());
        }
        if (!interfaces_origin_.empty()) {
            SEM::Meshes::adjust_interfaces<<<interfaces_numBlocks_, boundaries_blockSize_, 0, stream_>>>(interfaces_origin_.size(), elements_.data(), interfaces_origin_.data(), interfaces_destination_.data());
        }
        if (!mpi_interfaces_origin_.empty()) {
            SEM::Meshes::get_MPI_interfaces_N<<<mpi_interfaces_numBlocks_, boundaries_blockSize_, 0, stream_>>>(mpi_interfaces_origin_.size(), elements_.data(), mpi_interfaces_origin_.data(), device_interfaces_N_.data());

            device_interfaces_N_.copy_to(host_interfaces_N_, stream_);
            hipStreamSynchronize(stream_);

            for (size_t i = 0; i < mpi_interfaces_size_.size(); ++i) {
                MPI_Isend(host_interfaces_N_.data() + mpi_interfaces_offset_[i], mpi_interfaces_size_[i], MPI_INT, mpi_interfaces_process_[i], 3 * global_size * global_size + global_size * global_rank + mpi_interfaces_process_[i], MPI_COMM_WORLD, &requests_N_[mpi_interfaces_size_.size() + i]);
                MPI_Irecv(host_receiving_interfaces_N_.data() + mpi_interfaces_offset_[i], mpi_interfaces_size_[i], MPI_INT, mpi_interfaces_process_[i], 3 * global_size * global_size + global_size * mpi_interfaces_process_[i] + global_rank, MPI_COMM_WORLD, &requests_N_[i]);
            }

            MPI_Waitall(mpi_interfaces_size_.size(), requests_N_.data(), statuses_N_.data());

            device_interfaces_N_.copy_from(host_receiving_interfaces_N_, stream_);

            SEM::Meshes::put_MPI_interfaces_N<<<mpi_interfaces_numBlocks_, boundaries_blockSize_, 0, stream_>>>(mpi_interfaces_destination_.size(), elements_.data(), mpi_interfaces_destination_.data(), device_interfaces_N_.data());

            MPI_Waitall(mpi_interfaces_size_.size(), requests_N_.data() + mpi_interfaces_size_.size(), statuses_N_.data() + mpi_interfaces_size_.size());
        }

        SEM::Meshes::adjust_faces<<<faces_numBlocks_, faces_blockSize_, 0, stream_>>>(faces_.size(), faces_.data(), elements_.data());

        return;
    }

    SEM::Meshes::reduce_nodes_2D<elements_blockSize_/2><<<elements_numBlocks_, elements_blockSize_/2, 0, stream_>>>(n_elements_, max_split_level_, elements_.data(), faces_.data(), nodes_.data(), device_nodes_refine_array_.data());
    device_nodes_refine_array_.copy_to(host_nodes_refine_array_, stream_);
    hipStreamSynchronize(stream_);
    
    size_t n_additional_nodes = 0;
    size_t n_additional_faces = 0;
    for (int i = 0; i < elements_numBlocks_; ++i) {
        host_nodes_refine_array_[i] = n_additional_nodes; // Current block offset
        n_additional_nodes += host_nodes_refine_array_[i];
        n_additional_faces += host_nodes_refine_array_[i] + 3 * (host_nodes_refine_array_[i] > 0);
    }

    device_nodes_refine_array_.copy_from(host_nodes_refine_array_, stream_);
    device_refine_array_.copy_from(host_refine_array_, stream_);

    device_vector<Vec2<deviceFloat>> new_nodes(nodes_.size() + n_additional_nodes, stream_);
    hipMemcpyAsync(new_nodes.data(), nodes_.data(), nodes_.size() * sizeof(Vec2<deviceFloat>), hipMemcpyDeviceToDevice, stream_); // Apparently slower than using a kernel
    
    device_vector<Face2D_t> new_faces(faces_.size() + n_additional_faces, stream_);

    device_vector<Element2D_t> new_elements(elements_.size() + 3 * splitting_elements, stream_);
    SEM::Meshes::hp_adapt<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(n_elements_, faces_.size(), nodes_.size(), elements_.data(), new_elements.data(), faces_.data(), new_faces.data(), device_refine_array_.data(), device_nodes_refine_array_.data(), max_split_level_, N_max, new_nodes.data(), polynomial_nodes.data(), barycentric_weights.data());

    SEM::Meshes::move_faces<<<faces_numBlocks_, faces_blockSize_, 0, stream_>>>(faces_.size(), faces_.data(), new_faces.data(), new_elements.data());

    if (!wall_boundaries_.empty()) {
        SEM::Meshes::rebuild_boundaries<<<wall_boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(wall_boundaries_.size(), elements_.data(), elements_.data(), wall_boundaries_.data(), faces_.data());
    }
    if (!symmetry_boundaries_.empty()) {
        SEM::Meshes::rebuild_boundaries<<<symmetry_boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(symmetry_boundaries_.size(), elements_.data(), elements_.data(), symmetry_boundaries_.data(), faces_.data());
    }
    if (!inflow_boundaries_.empty()) {
        SEM::Meshes::rebuild_boundaries<<<inflow_boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(inflow_boundaries_.size(), elements_.data(), elements_.data(), inflow_boundaries_.data(), faces_.data());
    }
    if (!outflow_boundaries_.empty()) {
        SEM::Meshes::rebuild_boundaries<<<outflow_boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(outflow_boundaries_.size(), elements_.data(), elements_.data(), outflow_boundaries_.data(), faces_.data());
    }
    if (!interfaces_origin_.empty()) {
        SEM::Meshes::rebuild_interfaces<<<interfaces_numBlocks_, boundaries_blockSize_, 0, stream_>>>(interfaces_origin_.size(), elements_.data(), new_elements.data(), interfaces_origin_.data(), interfaces_destination_.data());
    }
    if (!mpi_interfaces_origin_.empty()) {
        SEM::Meshes::get_MPI_interfaces_N<<<mpi_interfaces_numBlocks_, boundaries_blockSize_, 0, stream_>>>(mpi_interfaces_origin_.size(), new_elements.data(), mpi_interfaces_origin_.data(), device_interfaces_N_.data());

        device_interfaces_N_.copy_to(host_interfaces_N_, stream_);
        hipStreamSynchronize(stream_);

        for (size_t i = 0; i < mpi_interfaces_size_.size(); ++i) {
            MPI_Isend(host_interfaces_N_.data() + mpi_interfaces_offset_[i], mpi_interfaces_size_[i], MPI_INT, mpi_interfaces_process_[i], 3 * global_size * global_size + global_size * global_rank + mpi_interfaces_process_[i], MPI_COMM_WORLD, &requests_N_[mpi_interfaces_size_.size() + i]);
            MPI_Irecv(host_receiving_interfaces_N_.data() + mpi_interfaces_offset_[i], mpi_interfaces_size_[i], MPI_INT, mpi_interfaces_process_[i], 3 * global_size * global_size + global_size * mpi_interfaces_process_[i] + global_rank, MPI_COMM_WORLD, &requests_N_[i]);
        }

        MPI_Waitall(mpi_interfaces_size_.size(), requests_N_.data(), statuses_N_.data());

        device_interfaces_N_.copy_from(host_receiving_interfaces_N_, stream_);

        SEM::Meshes::put_MPI_interfaces_N_and_rebuild<<<mpi_interfaces_numBlocks_, boundaries_blockSize_, 0, stream_>>>(mpi_interfaces_destination_.size(), elements_.data(), new_elements.data(), mpi_interfaces_destination_.data(), device_interfaces_N_.data());

        MPI_Waitall(mpi_interfaces_size_.size(), requests_N_.data() + mpi_interfaces_size_.size(), statuses_N_.data() + mpi_interfaces_size_.size());
    }

    elements_ = std::move(new_elements);
    faces_ = std::move(new_faces);
    nodes_ = std::move(new_nodes);



    SEM::Meshes::adjust_faces<<<faces_numBlocks_, faces_blockSize_, 0, stream_>>>(faces_.size(), faces_.data(), elements_.data());
}

auto SEM::Meshes::Mesh2D_t::boundary_conditions(deviceFloat t, const device_vector<deviceFloat>& polynomial_nodes, const device_vector<deviceFloat>& weights, const device_vector<deviceFloat>& barycentric_weights) -> void {
    // Boundary conditions
    if (!wall_boundaries_.empty()) {
        SEM::Meshes::compute_wall_boundaries<<<wall_boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(wall_boundaries_.size(), elements_.data(), wall_boundaries_.data(), faces_.data(), polynomial_nodes.data(), weights.data(), barycentric_weights.data());
    }
    if (!symmetry_boundaries_.empty()) {
        SEM::Meshes::compute_symmetry_boundaries<<<symmetry_boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(symmetry_boundaries_.size(), elements_.data(), symmetry_boundaries_.data(), faces_.data(), polynomial_nodes.data(), weights.data(), barycentric_weights.data());
    }
    if (!inflow_boundaries_.empty()) {
        SEM::Meshes::compute_inflow_boundaries<<<inflow_boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(inflow_boundaries_.size(), elements_.data(), inflow_boundaries_.data(), faces_.data(), t, nodes_.data(), polynomial_nodes.data());
    }
    if (!outflow_boundaries_.empty()) {
        SEM::Meshes::compute_outflow_boundaries<<<outflow_boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(outflow_boundaries_.size(), elements_.data(), outflow_boundaries_.data(), faces_.data(), polynomial_nodes.data(), weights.data(), barycentric_weights.data());
    }

    // Interfaces
    if (!interfaces_origin_.empty()) {
        SEM::Meshes::local_interfaces<<<interfaces_numBlocks_, boundaries_blockSize_, 0, stream_>>>(interfaces_origin_.size(), elements_.data(), interfaces_origin_.data(), interfaces_origin_side_.data(), interfaces_destination_.data());
    }

    if (!mpi_interfaces_origin_.empty()) {
        int global_rank;
        MPI_Comm_rank(MPI_COMM_WORLD, &global_rank);
        int global_size;
        MPI_Comm_size(MPI_COMM_WORLD, &global_size);

        SEM::Meshes::get_MPI_interfaces<<<mpi_interfaces_numBlocks_, boundaries_blockSize_, 0, stream_>>>(mpi_interfaces_origin_.size(), elements_.data(), mpi_interfaces_origin_.data(), mpi_interfaces_origin_side_.data(), maximum_N_, device_interfaces_p_.data(), device_interfaces_u_.data(), device_interfaces_v_.data());

        device_interfaces_p_.copy_to(host_interfaces_p_, stream_);
        device_interfaces_u_.copy_to(host_interfaces_u_, stream_);
        device_interfaces_v_.copy_to(host_interfaces_v_, stream_);
        hipStreamSynchronize(stream_);
        
        constexpr MPI_Datatype data_type = (sizeof(deviceFloat) == sizeof(float)) ? MPI_FLOAT : MPI_DOUBLE;
        for (size_t i = 0; i < mpi_interfaces_size_.size(); ++i) {
            MPI_Isend(host_interfaces_p_.data() + mpi_interfaces_offset_[i] * (maximum_N_ + 1), mpi_interfaces_size_[i] * (maximum_N_ + 1), data_type, mpi_interfaces_process_[i], 3 * (global_size * global_rank + mpi_interfaces_process_[i]), MPI_COMM_WORLD, &requests_[3 * (mpi_interfaces_size_.size() + i)]);
            MPI_Irecv(host_receiving_interfaces_p_.data() + mpi_interfaces_offset_[i] * (maximum_N_ + 1), mpi_interfaces_size_[i] * (maximum_N_ + 1), data_type, mpi_interfaces_process_[i], 3 * (global_size * mpi_interfaces_process_[i] + global_rank), MPI_COMM_WORLD, &requests_[3 * i]);

            MPI_Isend(host_interfaces_u_.data() + mpi_interfaces_offset_[i] * (maximum_N_ + 1), mpi_interfaces_size_[i] * (maximum_N_ + 1), data_type, mpi_interfaces_process_[i], 3 * (global_size * global_rank + mpi_interfaces_process_[i]) + 1, MPI_COMM_WORLD, &requests_[3 * (mpi_interfaces_size_.size() + i) + 1]);
            MPI_Irecv(host_receiving_interfaces_u_.data() + mpi_interfaces_offset_[i] * (maximum_N_ + 1), mpi_interfaces_size_[i] * (maximum_N_ + 1), data_type, mpi_interfaces_process_[i], 3 * (global_size * mpi_interfaces_process_[i] + global_rank) + 1, MPI_COMM_WORLD, &requests_[3 * i + 1]);

            MPI_Isend(host_interfaces_v_.data() + mpi_interfaces_offset_[i] * (maximum_N_ + 1), mpi_interfaces_size_[i] * (maximum_N_ + 1), data_type, mpi_interfaces_process_[i], 3 * (global_size * global_rank + mpi_interfaces_process_[i]) + 2, MPI_COMM_WORLD, &requests_[3 * (mpi_interfaces_size_.size() + i) + 2]);
            MPI_Irecv(host_receiving_interfaces_v_.data() + mpi_interfaces_offset_[i] * (maximum_N_ + 1), mpi_interfaces_size_[i] * (maximum_N_ + 1), data_type, mpi_interfaces_process_[i], 3 * (global_size * mpi_interfaces_process_[i] + global_rank) + 2, MPI_COMM_WORLD, &requests_[3 * i + 2]);
        }

        MPI_Waitall(3 * mpi_interfaces_size_.size(), requests_.data(), statuses_.data());

        device_interfaces_p_.copy_from(host_receiving_interfaces_p_, stream_);
        device_interfaces_u_.copy_from(host_receiving_interfaces_u_, stream_);
        device_interfaces_v_.copy_from(host_receiving_interfaces_v_, stream_);

        SEM::Meshes::put_MPI_interfaces<<<mpi_interfaces_numBlocks_, boundaries_blockSize_, 0, stream_>>>(mpi_interfaces_destination_.size(), elements_.data(), mpi_interfaces_destination_.data(), maximum_N_, device_interfaces_p_.data(), device_interfaces_u_.data(), device_interfaces_v_.data());

        MPI_Waitall(3 * mpi_interfaces_size_.size(), requests_.data() + 3 * mpi_interfaces_size_.size(), statuses_.data() + 3 * mpi_interfaces_size_.size());
    }
}

// From cppreference.com
__device__
auto SEM::Meshes::Mesh2D_t::almost_equal(deviceFloat x, deviceFloat y) -> bool {
    constexpr int ulp = 2; // ULP
    // the machine epsilon has to be scaled to the magnitude of the values used
    // and multiplied by the desired precision in ULPs (units in the last place)
    return std::abs(x-y) <= std::numeric_limits<deviceFloat>::epsilon() * std::abs(x+y) * ulp
        // unless the result is subnormal
        || std::abs(x-y) < std::numeric_limits<deviceFloat>::min();
}

auto SEM::Meshes::Mesh2D_t::interpolate_to_boundaries(const device_vector<deviceFloat>& lagrange_interpolant_left, const device_vector<deviceFloat>& lagrange_interpolant_right) -> void {
    SEM::Meshes::interpolate_to_boundaries<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(n_elements_, elements_.data(), lagrange_interpolant_left.data(), lagrange_interpolant_right.data());
}

auto SEM::Meshes::Mesh2D_t::project_to_faces(const device_vector<deviceFloat>& polynomial_nodes, const device_vector<deviceFloat>& barycentric_weights) -> void {
    SEM::Meshes::project_to_faces<<<faces_numBlocks_, faces_blockSize_, 0, stream_>>>(faces_.size(), faces_.data(), elements_.data(), polynomial_nodes.data(), barycentric_weights.data());
}

auto SEM::Meshes::Mesh2D_t::project_to_elements(const device_vector<deviceFloat>& polynomial_nodes, const device_vector<deviceFloat>& weights, const device_vector<deviceFloat>& barycentric_weights) -> void {
    SEM::Meshes::project_to_elements<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(n_elements_, faces_.data(), elements_.data(), polynomial_nodes.data(), weights.data(), barycentric_weights.data());
}

template auto SEM::Meshes::Mesh2D_t::estimate_error<SEM::Polynomials::ChebyshevPolynomial_t>(const device_vector<deviceFloat>& polynomial_nodes, const device_vector<deviceFloat>& weights) -> void;
template auto SEM::Meshes::Mesh2D_t::estimate_error<SEM::Polynomials::LegendrePolynomial_t>(const device_vector<deviceFloat>& polynomial_nodes, const device_vector<deviceFloat>& weights) -> void;

template<typename Polynomial>
auto SEM::Meshes::Mesh2D_t::estimate_error<Polynomial>(const device_vector<deviceFloat>& polynomial_nodes, const device_vector<deviceFloat>& weights) -> void {
    SEM::Meshes::estimate_error<Polynomial><<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(n_elements_, elements_.data(), tolerance_min_, tolerance_max_, polynomial_nodes.data(), weights.data());
}

__global__
auto SEM::Meshes::allocate_element_storage(size_t n_elements, Element2D_t* elements) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index; element_index < n_elements; element_index += stride) {
        elements[element_index].allocate_storage();
    }
}

__global__
auto SEM::Meshes::allocate_boundary_storage(size_t n_domain_elements, size_t n_total_elements, Element2D_t* elements) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index + n_domain_elements; element_index < n_total_elements; element_index += stride) {
        elements[element_index].allocate_boundary_storage();
    }
}

__global__
auto SEM::Meshes::compute_element_geometry(size_t n_elements, Element2D_t* elements, const Vec2<deviceFloat>* nodes, const deviceFloat* polynomial_nodes) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index; element_index < n_elements; element_index += stride) {
        const std::array<Vec2<deviceFloat>, 4> points {nodes[elements[element_index].nodes_[0]],
                                                       nodes[elements[element_index].nodes_[1]],
                                                       nodes[elements[element_index].nodes_[2]],
                                                       nodes[elements[element_index].nodes_[3]]};
        elements[element_index].compute_geometry(points, polynomial_nodes);  
    }
}

__global__
auto SEM::Meshes::compute_boundary_geometry(size_t n_domain_elements, size_t n_total_elements, Element2D_t* elements, const Vec2<deviceFloat>* nodes, const deviceFloat* polynomial_nodes) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index + n_domain_elements; element_index < n_total_elements; element_index += stride) {
        const std::array<Vec2<deviceFloat>, 4> points {nodes[elements[element_index].nodes_[0]],
                                                       nodes[elements[element_index].nodes_[1]],
                                                       nodes[elements[element_index].nodes_[2]],
                                                       nodes[elements[element_index].nodes_[3]]};
        elements[element_index].compute_boundary_geometry(points, polynomial_nodes);  
    }
}

__global__
auto SEM::Meshes::allocate_face_storage(size_t n_faces, Face2D_t* faces) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t face_index = index; face_index < n_faces; face_index += stride) {
        faces[face_index].allocate_storage();
    }
}

__global__
auto SEM::Meshes::fill_element_faces(size_t n_elements, Element2D_t* elements, const std::array<size_t, 4>* element_to_face) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index; element_index < n_elements; element_index += stride) {
        for (size_t j = 0; j < elements[element_index].faces_.size(); ++j) {
            elements[element_index].faces_[j][0] = element_to_face[element_index][j];
        }
    }
}

__global__
auto SEM::Meshes::compute_face_geometry(size_t n_faces, Face2D_t* faces, const Element2D_t* elements, const Vec2<deviceFloat>* nodes) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t face_index = index; face_index < n_faces; face_index += stride) {
        faces[face_index].compute_geometry(elements, nodes);
    }
}

__global__
auto SEM::Meshes::initial_conditions_2D(size_t n_elements, Element2D_t* elements, const Vec2<deviceFloat>* nodes, const deviceFloat* polynomial_nodes) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index; element_index < n_elements; element_index += stride) {
        Element2D_t& element = elements[element_index];
        const size_t offset_1D = element.N_ * (element.N_ + 1) /2;
        const std::array<Vec2<deviceFloat>, 4> points {nodes[element.nodes_[0]],
                                                       nodes[element.nodes_[1]],
                                                       nodes[element.nodes_[2]],
                                                       nodes[element.nodes_[3]]};
        
        for (int i = 0; i <= element.N_; ++i) {
            for (int j = 0; j <= element.N_; ++j) {
                const Vec2<deviceFloat> coordinates {polynomial_nodes[offset_1D + i], polynomial_nodes[offset_1D + j]};
                const Vec2<deviceFloat> global_coordinates = SEM::quad_map(coordinates, points);

                const std::array<deviceFloat, 3> state = SEM::Meshes::Mesh2D_t::g(global_coordinates, 0);
                element.p_[i * (element.N_ + 1) + j] = state[0];
                element.u_[i * (element.N_ + 1) + j] = state[1];
                element.v_[i * (element.N_ + 1) + j] = state[2];
            }
        }
    }
}

__global__
auto SEM::Meshes::get_solution(size_t n_elements, size_t n_interpolation_points, const Element2D_t* elements, const Vec2<deviceFloat>* nodes, const deviceFloat* interpolation_matrices, deviceFloat* x, deviceFloat* y, deviceFloat* p, deviceFloat* u, deviceFloat* v) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index; element_index < n_elements; element_index += stride) {
        const Element2D_t& element = elements[element_index];
        const size_t offset_interp_2D = element_index * n_interpolation_points * n_interpolation_points;
        const size_t offset_interp = element.N_ * (element.N_ + 1) * n_interpolation_points/2;

        const std::array<Vec2<deviceFloat>, 4> points {nodes[element.nodes_[0]],
                                                       nodes[element.nodes_[1]],
                                                       nodes[element.nodes_[2]],
                                                       nodes[element.nodes_[3]]};

        element.interpolate_solution(n_interpolation_points, points, interpolation_matrices + offset_interp, x + offset_interp_2D, y + offset_interp_2D, p + offset_interp_2D, u + offset_interp_2D, v + offset_interp_2D);
    }
}

__global__
auto SEM::Meshes::get_complete_solution(size_t n_elements, size_t n_interpolation_points, const Element2D_t* elements, const Vec2<deviceFloat>* nodes, const deviceFloat* interpolation_matrices, deviceFloat* x, deviceFloat* y, deviceFloat* p, deviceFloat* u, deviceFloat* v, int* N, deviceFloat* dp_dt, deviceFloat* du_dt, deviceFloat* dv_dt, deviceFloat* p_error, deviceFloat* u_error, deviceFloat* v_error, deviceFloat* p_sigma, deviceFloat* u_sigma, deviceFloat* v_sigma, int* refine, int* coarsen, int* split_level) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index; element_index < n_elements; element_index += stride) {
        const Element2D_t& element = elements[element_index];
        const size_t offset_interp_2D = element_index * n_interpolation_points * n_interpolation_points;
        const size_t offset_interp = element.N_ * (element.N_ + 1) * n_interpolation_points/2;

        N[element_index] = element.N_;
        p_error[element_index] = element.p_error_;
        u_error[element_index] = element.u_error_;
        v_error[element_index] = element.v_error_;
        p_sigma[element_index] = element.p_sigma_;
        u_sigma[element_index] = element.u_sigma_;
        v_sigma[element_index] = element.v_sigma_;
        refine[element_index] = element.refine_;
        coarsen[element_index] = element.coarsen_;
        split_level[element_index] = element.split_level_;
        const std::array<Vec2<deviceFloat>, 4> points {nodes[element.nodes_[0]],
                                                       nodes[element.nodes_[1]],
                                                       nodes[element.nodes_[2]],
                                                       nodes[element.nodes_[3]]};

        element.interpolate_complete_solution(n_interpolation_points, points, interpolation_matrices + offset_interp, x + offset_interp_2D, y + offset_interp_2D, p + offset_interp_2D, u + offset_interp_2D, v + offset_interp_2D, dp_dt + offset_interp_2D, du_dt + offset_interp_2D, dv_dt + offset_interp_2D);
    }
}

template __global__ auto SEM::Meshes::estimate_error<SEM::Polynomials::ChebyshevPolynomial_t>(size_t n_elements, Element2D_t* elements, deviceFloat tolerance_min, deviceFloat tolerance_max, const deviceFloat* polynomial_nodes, const deviceFloat* weights) -> void;
template __global__ auto SEM::Meshes::estimate_error<SEM::Polynomials::LegendrePolynomial_t>(size_t n_elements, Element2D_t* elements, deviceFloat tolerance_min, deviceFloat tolerance_max, const deviceFloat* polynomial_nodes, const deviceFloat* weights) -> void;

template<typename Polynomial>
__global__
auto SEM::Meshes::estimate_error<Polynomial>(size_t n_elements, Element2D_t* elements, deviceFloat tolerance_min, deviceFloat tolerance_max, const deviceFloat* polynomial_nodes, const deviceFloat* weights) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index; element_index < n_elements; element_index += stride) {
        elements[element_index].estimate_error<Polynomial>(tolerance_min, tolerance_max, polynomial_nodes, weights);
    }
}

__global__
auto SEM::Meshes::interpolate_to_boundaries(size_t n_elements, Element2D_t* elements, const deviceFloat* lagrange_interpolant_minus, const deviceFloat* lagrange_interpolant_plus) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index; element_index < n_elements; element_index += stride) {
        elements[element_index].interpolate_to_boundaries(lagrange_interpolant_minus, lagrange_interpolant_plus);
    }
}

__global__
auto SEM::Meshes::project_to_faces(size_t n_faces, Face2D_t* faces, const Element2D_t* elements, const deviceFloat* polynomial_nodes, const deviceFloat* barycentric_weights) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t face_index = index; face_index < n_faces; face_index += stride) {
        Face2D_t& face = faces[face_index];

        // Getting element solution
        const Element2D_t& element_L = elements[face.elements_[0]];
        // Conforming
        if ((face.N_ == element_L.N_) 
                && (face.nodes_[0] == element_L.nodes_[face.elements_side_[0]]) 
                && (face.nodes_[1] == element_L.nodes_[(face.elements_side_[0] + 1) * (!(face.elements_side_[0] == (element_L.nodes_.size() - 1)))])) {
            for (int i = 0; i <= face.N_; ++i) {
                face.p_[0][i] = element_L.p_extrapolated_[face.elements_side_[0]][i];
                face.u_[0][i] = element_L.u_extrapolated_[face.elements_side_[0]][i];
                face.v_[0][i] = element_L.v_extrapolated_[face.elements_side_[0]][i];
            }
        }
        else { // We need to interpolate
            const size_t offset_1D = face.N_ * (face.N_ + 1) /2;
            const size_t offset_1D_other = element_L.N_ * (element_L.N_ + 1) /2;

            for (int i = 0; i <= face.N_; ++i) {
                const deviceFloat coordinate = (polynomial_nodes[offset_1D + i] + 1) * face.scale_[0] + 2 * face.offset_[0] - 1;

                deviceFloat p_numerator = 0.0;
                deviceFloat u_numerator = 0.0;
                deviceFloat v_numerator = 0.0;
                deviceFloat denominator = 0.0;

                for (int j = 0; j <= element_L.N_; ++j) {
                    if (SEM::Meshes::Mesh2D_t::almost_equal(coordinate, polynomial_nodes[offset_1D_other + j])) {
                        p_numerator = element_L.p_extrapolated_[face.elements_side_[0]][j];
                        u_numerator = element_L.u_extrapolated_[face.elements_side_[0]][j];
                        v_numerator = element_L.v_extrapolated_[face.elements_side_[0]][j];
                        denominator = 1.0;
                        break;
                    }

                    const deviceFloat t = barycentric_weights[offset_1D_other + j]/(coordinate - polynomial_nodes[offset_1D_other + j]);
                    p_numerator += t * element_L.p_extrapolated_[face.elements_side_[0]][j];
                    u_numerator += t * element_L.u_extrapolated_[face.elements_side_[0]][j];
                    v_numerator += t * element_L.v_extrapolated_[face.elements_side_[0]][j];
                    denominator += t;
                }
                face.p_[0][i] = p_numerator/denominator;
                face.u_[0][i] = u_numerator/denominator;
                face.v_[0][i] = v_numerator/denominator;
            }
        }

        const Element2D_t& element_R = elements[face.elements_[1]];
        // Conforming, but reversed
        if ((face.N_ == element_R.N_) 
                && (face.nodes_[1] == element_R.nodes_[face.elements_side_[1]]) 
                && (face.nodes_[0] == element_R.nodes_[(face.elements_side_[1] + 1) * (!(face.elements_side_[1] == (element_R.nodes_.size() - 1)))])) {
            for (int i = 0; i <= face.N_; ++i) {
                face.p_[1][face.N_ - i] = element_R.p_extrapolated_[face.elements_side_[1]][i];
                face.u_[1][face.N_ - i] = element_R.u_extrapolated_[face.elements_side_[1]][i];
                face.v_[1][face.N_ - i] = element_R.v_extrapolated_[face.elements_side_[1]][i];
            }
        }
        else { // We need to interpolate
            const size_t offset_1D = face.N_ * (face.N_ + 1) /2;
            const size_t offset_1D_other = element_R.N_ * (element_R.N_ + 1) /2;

            for (int i = 0; i <= face.N_; ++i) {
                const deviceFloat coordinate = (polynomial_nodes[offset_1D + face.N_ - i] + 1) * face.scale_[1] + 2 * face.offset_[1] - 1;

                deviceFloat p_numerator = 0.0;
                deviceFloat u_numerator = 0.0;
                deviceFloat v_numerator = 0.0;
                deviceFloat denominator = 0.0;

                for (int j = 0; j <= element_R.N_; ++j) {
                    if (SEM::Meshes::Mesh2D_t::almost_equal(coordinate, polynomial_nodes[offset_1D_other + j])) {
                        p_numerator = element_R.p_extrapolated_[face.elements_side_[1]][j];
                        u_numerator = element_R.u_extrapolated_[face.elements_side_[1]][j];
                        v_numerator = element_R.v_extrapolated_[face.elements_side_[1]][j];
                        denominator = 1.0;
                        break;
                    }

                    const deviceFloat t = barycentric_weights[offset_1D_other + j]/(coordinate - polynomial_nodes[offset_1D_other + j]);
                    p_numerator += t * element_R.p_extrapolated_[face.elements_side_[1]][j];
                    u_numerator += t * element_R.u_extrapolated_[face.elements_side_[1]][j];
                    v_numerator += t * element_R.v_extrapolated_[face.elements_side_[1]][j];
                    denominator += t;
                }
                face.p_[1][i] = p_numerator/denominator;
                face.u_[1][i] = u_numerator/denominator;
                face.v_[1][i] = v_numerator/denominator;
            }
        }
    }
}

__global__
auto SEM::Meshes::project_to_elements(size_t n_elements, const Face2D_t* faces, Element2D_t* elements, const deviceFloat* polynomial_nodes, const deviceFloat* weights, const deviceFloat* barycentric_weights) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t element_index = index; element_index < n_elements; element_index += stride) {
        Element2D_t& element = elements[element_index];

        for (size_t side_index = 0; side_index < element.faces_.size(); ++side_index) {
            // Conforming, forward
            if ((element.faces_[side_index].size() == 1)
                    && (faces[element.faces_[side_index][0]].N_ == element.N_)  
                    && (faces[element.faces_[side_index][0]].nodes_[0] == element.nodes_[side_index]) 
                    && (faces[element.faces_[side_index][0]].nodes_[1] == element.nodes_[(side_index + 1) * !(side_index == (element.faces_.size() - 1))])) {

                const Face2D_t& face = faces[element.faces_[side_index][0]];
                for (int j = 0; j <= faces[element.faces_[side_index][0]].N_; ++j) {
                    element.p_flux_extrapolated_[side_index][j] = face.p_flux_[j] * element.scaling_factor_[side_index][j];
                    element.u_flux_extrapolated_[side_index][j] = face.u_flux_[j] * element.scaling_factor_[side_index][j];
                    element.v_flux_extrapolated_[side_index][j] = face.v_flux_[j] * element.scaling_factor_[side_index][j];
                }
            }
            // Conforming, backwards
            else if ((element.faces_[side_index].size() == 1)
                    && (faces[element.faces_[side_index][0]].N_ == element.N_) 
                    && (faces[element.faces_[side_index][0]].nodes_[1] == element.nodes_[side_index]) 
                    && (faces[element.faces_[side_index][0]].nodes_[0] == element.nodes_[(side_index + 1) * !(side_index == (element.faces_.size() - 1))])) {

                const Face2D_t& face = faces[element.faces_[side_index][0]];
                for (int j = 0; j <= face.N_; ++j) {
                    element.p_flux_extrapolated_[side_index][face.N_ - j] = -face.p_flux_[j] * element.scaling_factor_[side_index][j];
                    element.u_flux_extrapolated_[side_index][face.N_ - j] = -face.u_flux_[j] * element.scaling_factor_[side_index][j];
                    element.v_flux_extrapolated_[side_index][face.N_ - j] = -face.v_flux_[j] * element.scaling_factor_[side_index][j];
                }
            }
            else { // We need to interpolate
                const size_t offset_1D = element.N_ * (element.N_ + 1) /2;

                for (int i = 0; i <= element.N_; ++i) {
                    element.p_flux_extrapolated_[side_index][i] = 0.0;
                    element.u_flux_extrapolated_[side_index][i] = 0.0;
                    element.v_flux_extrapolated_[side_index][i] = 0.0;
                }

                for (size_t face_index = 0; face_index < element.faces_[side_index].size(); ++face_index) {
                    const Face2D_t& face = faces[element.faces_[side_index][face_index]];
                    const size_t offset_1D_other = face.N_ * (face.N_ + 1) /2;

                    // Non-conforming, forward
                    if (element_index == face.elements_[0]) {
                        for (int j = 0; j <= face.N_; ++j) {
                            const deviceFloat coordinate = (polynomial_nodes[offset_1D_other + j] + 1) * face.scale_[0] + 2 * face.offset_[0] - 1;
                            bool found_row = false;
                            
                            for (int i = 0; i <= element.N_; ++i) {
                                if (SEM::Meshes::Mesh2D_t::almost_equal(coordinate, polynomial_nodes[offset_1D + i])) {
                                    element.p_flux_extrapolated_[side_index][i] += weights[offset_1D_other + j]/weights[offset_1D + i] * face.p_flux_[j] * element.scaling_factor_[side_index][i];
                                    element.u_flux_extrapolated_[side_index][i] += weights[offset_1D_other + j]/weights[offset_1D + i] * face.u_flux_[j] * element.scaling_factor_[side_index][i];
                                    element.v_flux_extrapolated_[side_index][i] += weights[offset_1D_other + j]/weights[offset_1D + i] * face.v_flux_[j] * element.scaling_factor_[side_index][i];
                                    found_row = true;
                                    break;
                                }
                            }

                            if (!found_row) {
                                double s = 0.0;
                                for (int i = 0; i <= element.N_; ++i) {
                                    s += barycentric_weights[offset_1D + i]/(coordinate - polynomial_nodes[offset_1D + i]);
                                }
                                for (int i = 0; i <= element.N_; ++i) {
                                    const deviceFloat T = barycentric_weights[offset_1D + i]/((coordinate - polynomial_nodes[offset_1D + i]) * s);

                                    element.p_flux_extrapolated_[side_index][i] += T * weights[offset_1D_other + j]/weights[offset_1D + i] * face.p_flux_[j] * element.scaling_factor_[side_index][i];
                                    element.u_flux_extrapolated_[side_index][i] += T * weights[offset_1D_other + j]/weights[offset_1D + i] * face.u_flux_[j] * element.scaling_factor_[side_index][i];
                                    element.v_flux_extrapolated_[side_index][i] += T * weights[offset_1D_other + j]/weights[offset_1D + i] * face.v_flux_[j] * element.scaling_factor_[side_index][i];
                                }
                            }
                        }
                    }
                    // Non-conforming, backwards
                    else {
                        for (int j = 0; j <= face.N_; ++j) {
                            const deviceFloat coordinate = (polynomial_nodes[offset_1D_other + face.N_ - j] + 1) * face.scale_[1] + 2 * face.offset_[1] - 1;
                            bool found_row = false;
                            
                            for (int i = 0; i <= element.N_; ++i) {
                                if (SEM::Meshes::Mesh2D_t::almost_equal(coordinate, polynomial_nodes[offset_1D + i])) {
                                    element.p_flux_extrapolated_[side_index][i] += -weights[offset_1D_other + j]/weights[offset_1D + i] * face.p_flux_[j] * element.scaling_factor_[side_index][i];
                                    element.u_flux_extrapolated_[side_index][i] += -weights[offset_1D_other + j]/weights[offset_1D + i] * face.u_flux_[j] * element.scaling_factor_[side_index][i];
                                    element.v_flux_extrapolated_[side_index][i] += -weights[offset_1D_other + j]/weights[offset_1D + i] * face.v_flux_[j] * element.scaling_factor_[side_index][i];
                                    found_row = true;
                                    break;
                                }
                            }

                            if (!found_row) {
                                double s = 0.0;
                                for (int i = 0; i <= element.N_; ++i) {
                                    s += barycentric_weights[offset_1D + i]/(coordinate - polynomial_nodes[offset_1D + i]);
                                }
                                for (int i = 0; i <= element.N_; ++i) {
                                    const deviceFloat T = barycentric_weights[offset_1D + i]/((coordinate - polynomial_nodes[offset_1D + i]) * s);

                                    element.p_flux_extrapolated_[side_index][i] += -T * weights[offset_1D_other + j]/weights[offset_1D + i] * face.p_flux_[j] * element.scaling_factor_[side_index][i];
                                    element.u_flux_extrapolated_[side_index][i] += -T * weights[offset_1D_other + j]/weights[offset_1D + i] * face.u_flux_[j] * element.scaling_factor_[side_index][i];
                                    element.v_flux_extrapolated_[side_index][i] += -T * weights[offset_1D_other + j]/weights[offset_1D + i] * face.v_flux_[j] * element.scaling_factor_[side_index][i];
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}

__global__
auto SEM::Meshes::compute_wall_boundaries(size_t n_wall_boundaries, Element2D_t* elements, const size_t* wall_boundaries, const Face2D_t* faces, const deviceFloat* polynomial_nodes, const deviceFloat* weights, const deviceFloat* barycentric_weights) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t boundary_index = index; boundary_index < n_wall_boundaries; boundary_index += stride) {
        const size_t element_index = wall_boundaries[boundary_index];
        Element2D_t& element = elements[element_index];

        if (element.faces_[0].size() == 0) { // Only one neighbour
            const Face2D_t& face = faces[element.faces_[0][0]];
            const int face_side = face.elements_[0] == element_index;
            const Element2D_t& neighbour = elements[face.elements_[face_side]];
            const size_t neighbour_side = face.elements_side_[face_side];
            const Vec2<deviceFloat> normal_inv {face.normal_.x(), face.tangent_.x()};
            const Vec2<deviceFloat> tangent_inv {face.normal_.y(), face.tangent_.y()};

            if (element.N_ == neighbour.N_) { // Conforming
                for (int k = 0; k <= element.N_; ++k) {
                    const Vec2<deviceFloat> neighbour_velocity {neighbour.u_extrapolated_[neighbour_side][neighbour.N_ - k], neighbour.v_extrapolated_[neighbour_side][neighbour.N_ - k]};
                    Vec2<deviceFloat> local_velocity {neighbour_velocity.dot(face.normal_), neighbour_velocity.dot(face.tangent_)};
                    local_velocity.x() = (2 * neighbour.p_extrapolated_[neighbour_side][neighbour.N_ - k] + SEM::Constants::c * local_velocity.x()) / SEM::Constants::c;
                
                    element.p_extrapolated_[0][k] = neighbour.p_extrapolated_[neighbour_side][neighbour.N_ - k];
                    element.u_extrapolated_[0][k] = normal_inv.dot(local_velocity);
                    element.v_extrapolated_[0][k] = tangent_inv.dot(local_velocity);
                }
            }
            else {
                const size_t offset_1D = element.N_ * (element.N_ + 1) /2;
                const size_t offset_1D_neighbour = neighbour.N_ * (neighbour.N_ + 1) /2;

                for (int i = 0; i <= element.N_; ++i) {
                    element.p_extrapolated_[0][i] = 0.0;
                    element.u_extrapolated_[0][i] = 0.0;
                    element.v_extrapolated_[0][i] = 0.0;
                }

                for (int j = 0; j <= neighbour.N_; ++j) {
                    const deviceFloat coordinate = (polynomial_nodes[offset_1D_neighbour + neighbour.N_ - j] + 1) * face.scale_[face_side] + 2 * face.offset_[face_side] - 1;
                    bool found_row = false;
                    
                    for (int i = 0; i <= element.N_; ++i) {
                        if (SEM::Meshes::Mesh2D_t::almost_equal(coordinate, polynomial_nodes[offset_1D + i])) {
                            element.p_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.p_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.u_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.u_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.v_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.v_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            found_row = true;
                            break;
                        }
                    }

                    if (!found_row) {
                        double s = 0.0;
                        for (int i = 0; i <= element.N_; ++i) {
                            s += barycentric_weights[offset_1D + i]/(coordinate - polynomial_nodes[offset_1D + i]);
                        }
                        for (int i = 0; i <= element.N_; ++i) {
                            const deviceFloat T = barycentric_weights[offset_1D + i]/((coordinate - polynomial_nodes[offset_1D + i]) * s);

                            element.p_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.p_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.u_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.u_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.v_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.v_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                        }
                    }
                }

                for (int k = 0; k <= element.N_; ++k) {
                    const Vec2<deviceFloat> neighbour_velocity {element.u_extrapolated_[0][k], element.v_extrapolated_[0][k]};
                    Vec2<deviceFloat> local_velocity {neighbour_velocity.dot(face.normal_), neighbour_velocity.dot(face.tangent_)};
                    local_velocity.x() = (2 * element.p_extrapolated_[0][k] + SEM::Constants::c * local_velocity.x()) / SEM::Constants::c;
                
                    //element.p_extrapolated_[0][k] = element.p_extrapolated_[0][k]; // Does nothing
                    element.u_extrapolated_[0][k] = normal_inv.dot(local_velocity);
                    element.v_extrapolated_[0][k] = tangent_inv.dot(local_velocity);
                }
            }
        }
        else {
            const size_t offset_1D = element.N_ * (element.N_ + 1) /2;

            for (int i = 0; i <= element.N_; ++i) {
                element.p_extrapolated_[0][i] = 0.0;
                element.u_extrapolated_[0][i] = 0.0;
                element.v_extrapolated_[0][i] = 0.0;
            }

            for (size_t face_index = 0; face_index < element.faces_[0].size(); ++face_index) {
                const Face2D_t& face = faces[element.faces_[0][face_index]];
                const int face_side = face.elements_[0] == element_index;
                const Element2D_t& neighbour = elements[face.elements_[face_side]];
                const size_t neighbour_side = face.elements_side_[face_side];
                const Vec2<deviceFloat> normal_inv {face.normal_.x(), face.tangent_.x()};
                const Vec2<deviceFloat> tangent_inv {face.normal_.y(), face.tangent_.y()};
                const size_t offset_1D_neighbour = neighbour.N_ * (neighbour.N_ + 1) /2;

                for (int j = 0; j <= neighbour.N_; ++j) {
                    const deviceFloat coordinate = (polynomial_nodes[offset_1D_neighbour + neighbour.N_ - j] + 1) * face.scale_[face_side] + 2 * face.offset_[face_side] - 1;
                    bool found_row = false;
                    
                    for (int i = 0; i <= element.N_; ++i) {
                        if (SEM::Meshes::Mesh2D_t::almost_equal(coordinate, polynomial_nodes[offset_1D + i])) {
                            element.p_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.p_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.u_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.u_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.v_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.v_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            found_row = true;
                            break;
                        }
                    }

                    if (!found_row) {
                        double s = 0.0;
                        for (int i = 0; i <= element.N_; ++i) {
                            s += barycentric_weights[offset_1D + i]/(coordinate - polynomial_nodes[offset_1D + i]);
                        }
                        for (int i = 0; i <= element.N_; ++i) {
                            const deviceFloat T = barycentric_weights[offset_1D + i]/((coordinate - polynomial_nodes[offset_1D + i]) * s);

                            element.p_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.p_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.u_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.u_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.v_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.v_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                        }
                    }
                }
            }

            const Face2D_t& face = faces[element.faces_[0][0]]; // CHECK this is kinda wrong, but we only use the normal and tangent so let's assume all the faces on a side have the same
            const Vec2<deviceFloat> normal_inv {face.normal_.x(), face.tangent_.x()};
            const Vec2<deviceFloat> tangent_inv {face.normal_.y(), face.tangent_.y()};
            for (int k = 0; k <= element.N_; ++k) {
                const Vec2<deviceFloat> neighbour_velocity {element.u_extrapolated_[0][k], element.v_extrapolated_[0][k]};
                Vec2<deviceFloat> local_velocity {neighbour_velocity.dot(face.normal_), neighbour_velocity.dot(face.tangent_)};
                local_velocity.x() = (2 * element.p_extrapolated_[0][k] + SEM::Constants::c * local_velocity.x()) / SEM::Constants::c;
            
                //element.p_extrapolated_[0][k] = element.p_extrapolated_[0][k]; // Does nothing
                element.u_extrapolated_[0][k] = normal_inv.dot(local_velocity);
                element.v_extrapolated_[0][k] = tangent_inv.dot(local_velocity);
            }
        }
    }
}

__global__
auto SEM::Meshes::compute_symmetry_boundaries(size_t n_symmetry_boundaries, Element2D_t* elements, const size_t* symmetry_boundaries, const Face2D_t* faces, const deviceFloat* polynomial_nodes, const deviceFloat* weights, const deviceFloat* barycentric_weights) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t boundary_index = index; boundary_index < n_symmetry_boundaries; boundary_index += stride) {
        const size_t element_index = symmetry_boundaries[boundary_index];
        Element2D_t& element = elements[element_index];

        if (element.faces_[0].size() == 0) { // Only one neighbour
            const Face2D_t& face = faces[element.faces_[0][0]];
            const int face_side = face.elements_[0] == element_index;
            const Element2D_t& neighbour = elements[face.elements_[face_side]];
            const size_t neighbour_side = face.elements_side_[face_side];
            const Vec2<deviceFloat> normal_inv {face.normal_.x(), face.tangent_.x()};
            const Vec2<deviceFloat> tangent_inv {face.normal_.y(), face.tangent_.y()};

            if (element.N_ == neighbour.N_) { // Conforming
                for (int k = 0; k <= element.N_; ++k) {
                    const Vec2<deviceFloat> neighbour_velocity {neighbour.u_extrapolated_[neighbour_side][neighbour.N_ - k], neighbour.v_extrapolated_[neighbour_side][neighbour.N_ - k]};
                    const Vec2<deviceFloat> local_velocity {-(neighbour_velocity.dot(face.normal_)), neighbour_velocity.dot(face.tangent_)};
                
                    element.p_extrapolated_[0][k] = neighbour.p_extrapolated_[neighbour_side][neighbour.N_ - k];
                    element.u_extrapolated_[0][k] = normal_inv.dot(local_velocity);
                    element.v_extrapolated_[0][k] = tangent_inv.dot(local_velocity);
                }
            }
            else {
                const size_t offset_1D = element.N_ * (element.N_ + 1) /2;
                const size_t offset_1D_neighbour = neighbour.N_ * (neighbour.N_ + 1) /2;

                for (int i = 0; i <= element.N_; ++i) {
                    element.p_extrapolated_[0][i] = 0.0;
                    element.u_extrapolated_[0][i] = 0.0;
                    element.v_extrapolated_[0][i] = 0.0;
                }

                for (int j = 0; j <= neighbour.N_; ++j) {
                    const deviceFloat coordinate = (polynomial_nodes[offset_1D_neighbour + neighbour.N_ - j] + 1) * face.scale_[face_side] + 2 * face.offset_[face_side] - 1;
                    bool found_row = false;
                    
                    for (int i = 0; i <= element.N_; ++i) {
                        if (SEM::Meshes::Mesh2D_t::almost_equal(coordinate, polynomial_nodes[offset_1D + i])) {
                            element.p_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.p_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.u_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.u_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.v_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.v_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            found_row = true;
                            break;
                        }
                    }

                    if (!found_row) {
                        double s = 0.0;
                        for (int i = 0; i <= element.N_; ++i) {
                            s += barycentric_weights[offset_1D + i]/(coordinate - polynomial_nodes[offset_1D + i]);
                        }
                        for (int i = 0; i <= element.N_; ++i) {
                            const deviceFloat T = barycentric_weights[offset_1D + i]/((coordinate - polynomial_nodes[offset_1D + i]) * s);

                            element.p_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.p_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.u_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.u_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.v_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.v_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                        }
                    }
                }

                for (int k = 0; k <= element.N_; ++k) {
                    const Vec2<deviceFloat> neighbour_velocity {element.u_extrapolated_[0][k], element.v_extrapolated_[0][k]};
                    const Vec2<deviceFloat> local_velocity {-(neighbour_velocity.dot(face.normal_)), neighbour_velocity.dot(face.tangent_)};
                
                    //element.p_extrapolated_[0][k] = element.p_extrapolated_[0][k]; // Does nothing
                    element.u_extrapolated_[0][k] = normal_inv.dot(local_velocity);
                    element.v_extrapolated_[0][k] = tangent_inv.dot(local_velocity);
                }
            }
        }
        else {
            const size_t offset_1D = element.N_ * (element.N_ + 1) /2;

            for (int i = 0; i <= element.N_; ++i) {
                element.p_extrapolated_[0][i] = 0.0;
                element.u_extrapolated_[0][i] = 0.0;
                element.v_extrapolated_[0][i] = 0.0;
            }

            for (size_t face_index = 0; face_index < element.faces_[0].size(); ++face_index) {
                const Face2D_t& face = faces[element.faces_[0][face_index]];
                const int face_side = face.elements_[0] == element_index;
                const Element2D_t& neighbour = elements[face.elements_[face_side]];
                const size_t neighbour_side = face.elements_side_[face_side];
                const Vec2<deviceFloat> normal_inv {face.normal_.x(), face.tangent_.x()};
                const Vec2<deviceFloat> tangent_inv {face.normal_.y(), face.tangent_.y()};
                const size_t offset_1D_neighbour = neighbour.N_ * (neighbour.N_ + 1) /2;

                for (int j = 0; j <= neighbour.N_; ++j) {
                    const deviceFloat coordinate = (polynomial_nodes[offset_1D_neighbour + neighbour.N_ - j] + 1) * face.scale_[face_side] + 2 * face.offset_[face_side] - 1;
                    bool found_row = false;
                    
                    for (int i = 0; i <= element.N_; ++i) {
                        if (SEM::Meshes::Mesh2D_t::almost_equal(coordinate, polynomial_nodes[offset_1D + i])) {
                            element.p_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.p_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.u_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.u_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.v_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.v_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            found_row = true;
                            break;
                        }
                    }

                    if (!found_row) {
                        double s = 0.0;
                        for (int i = 0; i <= element.N_; ++i) {
                            s += barycentric_weights[offset_1D + i]/(coordinate - polynomial_nodes[offset_1D + i]);
                        }
                        for (int i = 0; i <= element.N_; ++i) {
                            const deviceFloat T = barycentric_weights[offset_1D + i]/((coordinate - polynomial_nodes[offset_1D + i]) * s);

                            element.p_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.p_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.u_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.u_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.v_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.v_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                        }
                    }
                }
            }

            const Face2D_t& face = faces[element.faces_[0][0]]; // CHECK this is kinda wrong, but we only use the normal and tangent so let's assume all the faces on a side have the same
            const Vec2<deviceFloat> normal_inv {face.normal_.x(), face.tangent_.x()};
            const Vec2<deviceFloat> tangent_inv {face.normal_.y(), face.tangent_.y()};
            for (int k = 0; k <= element.N_; ++k) {
                const Vec2<deviceFloat> neighbour_velocity {element.u_extrapolated_[0][k], element.v_extrapolated_[0][k]};
                const Vec2<deviceFloat> local_velocity {-(neighbour_velocity.dot(face.normal_)), neighbour_velocity.dot(face.tangent_)};
            
                //element.p_extrapolated_[0][k] = element.p_extrapolated_[0][k]; // Does nothing
                element.u_extrapolated_[0][k] = normal_inv.dot(local_velocity);
                element.v_extrapolated_[0][k] = tangent_inv.dot(local_velocity);
            }
        }
    }
}

__global__
auto SEM::Meshes::compute_inflow_boundaries(size_t n_inflow_boundaries, Element2D_t* elements, const size_t* inflow_boundaries, const Face2D_t* faces, deviceFloat t, const Vec2<deviceFloat>* nodes, const deviceFloat* polynomial_nodes) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t boundary_index = index; boundary_index < n_inflow_boundaries; boundary_index += stride) {
        Element2D_t& element = elements[inflow_boundaries[boundary_index]];
        const size_t offset_1D = element.N_ * (element.N_ + 1) /2;;
        const std::array<Vec2<deviceFloat>, 2> points{nodes[element.nodes_[0]], nodes[element.nodes_[1]]};

        for (int k = 0; k <= element.N_; ++k) {
            const deviceFloat interp = (polynomial_nodes[offset_1D + k] + 1)/2;
            const Vec2<deviceFloat> global_coordinates = points[1] * interp + points[0] * (1 - interp);

            const std::array<deviceFloat, 3> state = SEM::Meshes::Mesh2D_t::g(global_coordinates, t);

            element.p_extrapolated_[0][k] = state[0];
            element.u_extrapolated_[0][k] = state[1];
            element.v_extrapolated_[0][k] = state[2];
        }
    }
}

__global__
auto SEM::Meshes::compute_outflow_boundaries(size_t n_outflow_boundaries, Element2D_t* elements, const size_t* outflow_boundaries, const Face2D_t* faces, const deviceFloat* polynomial_nodes, const deviceFloat* weights, const deviceFloat* barycentric_weights) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t boundary_index = index; boundary_index < n_outflow_boundaries; boundary_index += stride) {
        const size_t element_index = outflow_boundaries[boundary_index];
        Element2D_t& element = elements[element_index];

        if (element.faces_[0].size() == 0) { // Only one neighbour
            const Face2D_t& face = faces[element.faces_[0][0]];
            const int face_side = face.elements_[0] == element_index;
            const Element2D_t& neighbour = elements[face.elements_[face_side]];
            const size_t neighbour_side = face.elements_side_[face_side];

            if (element.N_ == neighbour.N_) { // Conforming
                for (int k = 0; k <= element.N_; ++k) {
                    element.p_extrapolated_[0][k] = neighbour.p_extrapolated_[neighbour_side][neighbour.N_ - k];
                    element.u_extrapolated_[0][k] = neighbour.u_extrapolated_[neighbour_side][neighbour.N_ - k];
                    element.v_extrapolated_[0][k] = neighbour.v_extrapolated_[neighbour_side][neighbour.N_ - k];
                }
            }
            else {
                const size_t offset_1D = element.N_ * (element.N_ + 1) /2;
                const size_t offset_1D_neighbour = neighbour.N_ * (neighbour.N_ + 1) /2;

                for (int i = 0; i <= element.N_; ++i) {
                    element.p_extrapolated_[0][i] = 0.0;
                    element.u_extrapolated_[0][i] = 0.0;
                    element.v_extrapolated_[0][i] = 0.0;
                }

                for (int j = 0; j <= neighbour.N_; ++j) {
                    const deviceFloat coordinate = (polynomial_nodes[offset_1D_neighbour + neighbour.N_ - j] + 1) * face.scale_[face_side] + 2 * face.offset_[face_side] - 1;
                    bool found_row = false;
                    
                    for (int i = 0; i <= element.N_; ++i) {
                        if (SEM::Meshes::Mesh2D_t::almost_equal(coordinate, polynomial_nodes[offset_1D + i])) {
                            element.p_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.p_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.u_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.u_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.v_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.v_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            found_row = true;
                            break;
                        }
                    }

                    if (!found_row) {
                        double s = 0.0;
                        for (int i = 0; i <= element.N_; ++i) {
                            s += barycentric_weights[offset_1D + i]/(coordinate - polynomial_nodes[offset_1D + i]);
                        }
                        for (int i = 0; i <= element.N_; ++i) {
                            const deviceFloat T = barycentric_weights[offset_1D + i]/((coordinate - polynomial_nodes[offset_1D + i]) * s);

                            element.p_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.p_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.u_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.u_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.v_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.v_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                        }
                    }
                }
            }
        }
        else {
            const size_t offset_1D = element.N_ * (element.N_ + 1) /2;

            for (int i = 0; i <= element.N_; ++i) {
                element.p_extrapolated_[0][i] = 0.0;
                element.u_extrapolated_[0][i] = 0.0;
                element.v_extrapolated_[0][i] = 0.0;
            }

            for (size_t face_index = 0; face_index < element.faces_[0].size(); ++face_index) {
                const Face2D_t& face = faces[element.faces_[0][face_index]];
                const int face_side = face.elements_[0] == element_index;
                const Element2D_t& neighbour = elements[face.elements_[face_side]];
                const size_t neighbour_side = face.elements_side_[face_side];
                const Vec2<deviceFloat> normal_inv {face.normal_.x(), face.tangent_.x()};
                const Vec2<deviceFloat> tangent_inv {face.normal_.y(), face.tangent_.y()};
                const size_t offset_1D_neighbour = neighbour.N_ * (neighbour.N_ + 1) /2;

                for (int j = 0; j <= neighbour.N_; ++j) {
                    const deviceFloat coordinate = (polynomial_nodes[offset_1D_neighbour + neighbour.N_ - j] + 1) * face.scale_[face_side] + 2 * face.offset_[face_side] - 1;
                    bool found_row = false;
                    
                    for (int i = 0; i <= element.N_; ++i) {
                        if (SEM::Meshes::Mesh2D_t::almost_equal(coordinate, polynomial_nodes[offset_1D + i])) {
                            element.p_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.p_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.u_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.u_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.v_extrapolated_[0][i] += weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.v_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            found_row = true;
                            break;
                        }
                    }

                    if (!found_row) {
                        double s = 0.0;
                        for (int i = 0; i <= element.N_; ++i) {
                            s += barycentric_weights[offset_1D + i]/(coordinate - polynomial_nodes[offset_1D + i]);
                        }
                        for (int i = 0; i <= element.N_; ++i) {
                            const deviceFloat T = barycentric_weights[offset_1D + i]/((coordinate - polynomial_nodes[offset_1D + i]) * s);

                            element.p_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.p_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.u_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.u_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                            element.v_extrapolated_[0][i] += T * weights[offset_1D_neighbour + j]/weights[offset_1D + i] * neighbour.v_extrapolated_[neighbour_side][j] * element.scaling_factor_[0][i];
                        }
                    }
                }
            }
        }
    }
}

__global__
auto SEM::Meshes::local_interfaces(size_t n_local_interfaces, Element2D_t* elements, const size_t* local_interfaces_origin, const size_t* local_interfaces_origin_side, const size_t* local_interfaces_destination) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t interface_index = index; interface_index < n_local_interfaces; interface_index += stride) {
        const Element2D_t& source_element = elements[local_interfaces_origin[interface_index]];
        Element2D_t& destination_element = elements[local_interfaces_destination[interface_index]];
        const size_t element_side = local_interfaces_origin_side[interface_index];

        for (int k = 0; k <= source_element.N_; ++k) {
            destination_element.p_extrapolated_[0][k] = source_element.p_extrapolated_[element_side][k];
            destination_element.u_extrapolated_[0][k] = source_element.u_extrapolated_[element_side][k];
            destination_element.v_extrapolated_[0][k] = source_element.v_extrapolated_[element_side][k];
        }
    }
}

__global__
auto SEM::Meshes::get_MPI_interfaces(size_t n_MPI_interface_elements, const Element2D_t* elements, const size_t* MPI_interfaces_origin, const size_t* MPI_interfaces_origin_side, int maximum_N, deviceFloat* p, deviceFloat* u, deviceFloat* v) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t interface_index = index; interface_index < n_MPI_interface_elements; interface_index += stride) {
        const Element2D_t& source_element = elements[MPI_interfaces_origin[interface_index]];
        const size_t element_side = MPI_interfaces_origin_side[interface_index];
        const size_t boundary_offset = interface_index * (maximum_N + 1);

        for (int k = 0; k <= source_element.N_; ++k) {
            p[boundary_offset + k] = source_element.p_extrapolated_[element_side][k];
            u[boundary_offset + k] = source_element.u_extrapolated_[element_side][k];
            v[boundary_offset + k] = source_element.v_extrapolated_[element_side][k];
        }
    }
}

__global__
auto SEM::Meshes::get_MPI_interfaces_N(size_t n_MPI_interface_elements, const Element2D_t* elements, const size_t* MPI_interfaces_origin, int* N) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t interface_index = index; interface_index < n_MPI_interface_elements; interface_index += stride) {
        N[interface_index] = elements[MPI_interfaces_origin[interface_index]].N_;
    }
}

__global__
auto SEM::Meshes::put_MPI_interfaces(size_t n_MPI_interface_elements, Element2D_t* elements, const size_t* MPI_interfaces_destination, int maximum_N, const deviceFloat* p, const deviceFloat* u, const deviceFloat* v) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t interface_index = index; interface_index < n_MPI_interface_elements; interface_index += stride) {
        Element2D_t& destination_element = elements[MPI_interfaces_destination[interface_index]];
        const size_t boundary_offset = interface_index * (maximum_N + 1);

        for (int k = 0; k <= destination_element.N_; ++k) {
            destination_element.p_extrapolated_[0][k] = p[boundary_offset + k];
            destination_element.u_extrapolated_[0][k] = u[boundary_offset + k];
            destination_element.v_extrapolated_[0][k] = v[boundary_offset + k];
        }
    }
}

__global__
auto SEM::Meshes::put_MPI_interfaces_N(size_t n_MPI_interface_elements, Element2D_t* elements, const size_t* MPI_interfaces_destination, const int* N) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t interface_index = index; interface_index < n_MPI_interface_elements; interface_index += stride) {
        Element2D_t& destination_element = elements[MPI_interfaces_destination[interface_index]];

        if (destination_element.N_ != N[interface_index]) {
            destination_element.resize_boundary_storage(N[interface_index]);
        }
    }
}

__global__
auto SEM::Meshes::put_MPI_interfaces_N_and_rebuild(size_t n_MPI_interface_elements, Element2D_t* elements, Element2D_t* new_elements, const size_t* MPI_interfaces_destination, const int* N) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t interface_index = index; interface_index < n_MPI_interface_elements; interface_index += stride) {
        Element2D_t& destination_element = elements[MPI_interfaces_destination[interface_index]];

        if (destination_element.N_ != N[interface_index]) {
            destination_element.resize_boundary_storage(N[interface_index]);
        }

        new_elements[MPI_interfaces_destination[interface_index]] = std::move(elements[MPI_interfaces_destination[interface_index]]);
    }
}

__global__
auto SEM::Meshes::p_adapt(size_t n_elements, Element2D_t* elements, int N_max, const Vec2<deviceFloat>* nodes, const deviceFloat* polynomial_nodes, const deviceFloat* barycentric_weights) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    
    for (size_t i = index; i < n_elements; i += stride) {
        if (elements[i].would_p_refine(N_max)) {
            Element2D_t new_element(elements[i].N_ + 2, elements[i].split_level_, elements[i].faces_, elements[i].nodes_);

            new_element.interpolate_from(elements[i], polynomial_nodes, barycentric_weights);

            const std::array<Vec2<deviceFloat>, 4> points {nodes[new_element.nodes_[0]],
                                                           nodes[new_element.nodes_[1]],
                                                           nodes[new_element.nodes_[2]],
                                                           nodes[new_element.nodes_[3]]};
            new_element.compute_geometry(points, polynomial_nodes); 

            elements[i] = std::move(new_element);
        }
    }
}

__global__
auto SEM::Meshes::hp_adapt(size_t n_elements, size_t n_faces, size_t n_nodes, Element2D_t* elements, Element2D_t* new_elements, Face2D_t* faces, Face2D_t* new_faces, const size_t* block_offsets, const size_t* nodes_block_offsets, int max_split_level, int N_max, Vec2<deviceFloat>* nodes, const deviceFloat* polynomial_nodes, const deviceFloat* barycentric_weights) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int thread_id = threadIdx.x;
    const int block_id = blockIdx.x;
    const int block_dim = blockDim.x;
    
    for (size_t i = index; i < n_elements; i += stride) {
        Element2D_t& element = elements[i];

        size_t element_index = i + block_offsets[block_id];
        for (size_t j = i - thread_id; j < i; ++j) {
            element_index += 3 * elements[j].would_h_refine(max_split_level);
        }

        // h refinement
        if (element.would_h_refine(max_split_level)) {
            size_t new_node_index = n_nodes + nodes_block_offsets[block_id];
            size_t new_face_index = n_faces + nodes_block_offsets[block_id] + block_offsets[block_id]; // Wow this just happens to work, we need to add 3 faces per splitting element to the number of additional nodes, and the element offset is 3 * n_splitting
            for (size_t j = i - thread_id; j < i; ++j) {
                if (elements[j].would_h_refine(max_split_level)) {
                    ++new_node_index;
                    new_face_index += 4;
                    for (size_t side_index = 0; side_index < elements[j].faces_.size(); ++side_index) {
                        new_node_index += elements[j].additional_nodes_[side_index];
                        new_face_index += elements[j].additional_nodes_[side_index];
                    }
                }
            }
            
            Vec2<deviceFloat> new_center_node {0};
            std::array<size_t, 4> new_nodes {static_cast<size_t>(-1), static_cast<size_t>(-1), static_cast<size_t>(-1), static_cast<size_t>(-1)}; // CHECK this won't work with elements with more than 4 sides
            const std::array<Vec2<deviceFloat>, 4> element_nodes = {nodes[element.nodes_[0]], nodes[element.nodes_[1]], nodes[element.nodes_[2]], nodes[element.nodes_[3]]}; // CHECK this won't work with elements with more than 4 sides
            std::array<size_t, 4> new_faces {static_cast<size_t>(-1), static_cast<size_t>(-1), static_cast<size_t>(-1), static_cast<size_t>(-1)};

            size_t local_node_index = 1;
            size_t local_face_index = 4;
            for (size_t side_index = 0; side_index < element.faces_.size(); ++side_index) {
                new_center_node += element_nodes[side_index];
                const std::array<Vec2<deviceFloat>, 2> side_nodes = {element_nodes[side_index], (side_index < element.faces_.size() - 1) ? element_nodes[side_index + 1] : element_nodes[0]};
                const Vec2<deviceFloat> new_node = (side_nodes[0] + side_nodes[1])/2;

                // Here we check if the new node already exists
                bool found_node = false;
                for (size_t face_index = 0; face_index < element.faces_[side_index].size(); ++face_index) {
                    const Face2D_t& face = faces[element.faces_[side_index][face_index]];
                    if (nodes[face.nodes_[0]] == new_node) {
                        found_node = true;
                        new_nodes[side_index] = face.nodes_[0];
                        break;
                    }
                    if (nodes[face.nodes_[1]] == new_node) {
                        found_node = true;
                        new_nodes[side_index] = face.nodes_[1];
                        break;
                    }
                }

                // Here we check if another element would create the same node, and yield if its index is smaller
                if (!found_node) {
                    for (size_t face_index = 0; face_index < element.faces_[side_index].size(); ++face_index) {
                        const size_t neighbour_face_index = element.faces_[side_index][face_index];
                        const Face2D_t& face = faces[neighbour_face_index];
                        const int face_side = face.elements_[0] == i;
                        const size_t neighbour_side = face.elements_side_[face_side];
                        const size_t neighbour_element_index = face.elements_[face_side];
                        const Element2D_t& neighbour = elements[neighbour_element_index];
                        
                        if (neighbour.would_h_refine(max_split_level)) {
                            const std::array<Vec2<deviceFloat>, 2> neighbour_nodes = {nodes[neighbour.nodes_[neighbour_side]], (neighbour_side < neighbour.faces_.size() - 1) ? nodes[neighbour.nodes_[neighbour_side + 1]] : nodes[neighbour.nodes_[0]]};
                            const Vec2<deviceFloat> neighbour_new_node = (neighbour_nodes[0] + neighbour_nodes[1])/2;

                            if (new_node.almost_equal(neighbour_new_node) && neighbour_element_index < i) {
                                found_node = true;

                                const int neighbour_block_id = neighbour_element_index/block_dim;
                                const int neighbour_thread_id = neighbour_element_index%block_dim;

                                size_t neighbour_new_node_offset = n_nodes + 1 + block_offsets[neighbour_block_id];
                                for (size_t neighbour_side_index = 0; neighbour_side_index < neighbour_side; ++neighbour_side_index) {
                                    neighbour_new_node_offset += neighbour.additional_nodes_[neighbour_side_index];
                                }

                                for (size_t j = neighbour_element_index - neighbour_thread_id; j < neighbour_element_index; ++j) {
                                    if (elements[j].would_h_refine(max_split_level)) {
                                        ++neighbour_new_node_offset;
                                        for (size_t side_index = 0; side_index < elements[j].faces_.size(); ++side_index) {
                                            neighbour_new_node_offset += elements[j].additional_nodes_[side_index];
                                        }
                                    }
                                }


                                new_nodes[side_index] = neighbour_new_node_offset;
                                break;
                            }
                        }
                    }
                }

                if (!found_node) {
                    new_nodes[side_index] = new_node_index + local_node_index;
                    nodes[new_nodes[side_index]] = new_node;
                    ++local_node_index;

                    if (element.faces_[side_index].size() == 1) {
                        const size_t neighbour_face_index = element.faces_[side_index][0];
                        const Face2D_t& face = faces[neighbour_face_index];
                        const int face_side = face.elements_[0] == i;
                        const size_t neighbour_side = face.elements_side_[face_side];
                        const size_t neighbour_element_index = face.elements_[face_side];
                        const Element2D_t& neighbour = elements[neighbour_element_index];

                        // We find the max N, and to do this we need to take care and check if the neighbour would p-adapt
                        const int neighbour_N = neighbour.N_ + 2 * neighbour.would_p_refine(N_max);
                        const int face_N = std::max(element.N_, neighbour_N);

                        const int neighbour_block_id = neighbour_element_index/block_dim;
                        const int neighbour_thread_id = neighbour_element_index%block_dim;
                        size_t neighbour_element_new_index = neighbour_element_index + block_offsets[neighbour_block_id];
                        for (size_t j = neighbour_element_index - neighbour_thread_id; j < neighbour_element_index; ++j) {
                            neighbour_element_new_index += 3 * elements[j].would_h_refine(max_split_level);
                        }

                        std::array<size_t, 2> neighbour_element_new_indices = {neighbour_element_new_index, neighbour_element_new_index};
                        if (neighbour.would_h_refine(max_split_level)) {
                            neighbour_element_new_indices[1] += neighbour_side;
                            if (neighbour_side < neighbour.faces_.size() - 1) {
                                neighbour_element_new_indices[0] += neighbour_side + 1;
                            }
                        }

                        new_faces[neighbour_face_index] = Face2D_t(face_N, {element.nodes_[side_index], new_nodes[side_index]}, {element_index + side_index, neighbour_element_new_indices[0]}, {side_index, neighbour_side});
                        new_faces[new_face_index + local_face_index] = Face2D_t(face_N, {new_nodes[side_index], (side_index < element.faces_.size() - 1) ? element.nodes_[size_index + 1] : element.nodes_[0]}, {(side_index < element.faces_.size() -1 ) ? element_index + side_index : element_index, neighbour_element_new_indices[1]}, {side_index, neighbour_side});
                        faces[neighbour_face_index].N_ = -1;
                        new_faces[side_index] = new_face_index + local_face_index;
                        ++local_face_index;
                    }
                    else { // CHECK This shouldn't happen as is, with elements always splitting in the middle and nodes not moving.
                        fprintf(stderr, "Error: Splitting element %llu creating node and face on side %llu, but there are already %llu faces on that side. Exiting.\n", i, side_index, element.faces_[side_index].size());    
                        exit(53);
                    }

                    
                }
            }

            new_center_node /= element.faces_.size();
            nodes[new_node_index] = new_center_node;

            new_faces[new_face_index]     = Face2D_t{element.N_, {new_nodes[0], new_node_index}, {element_index,     element_index + 1}, {0, 2}};
            new_faces[new_face_index + 1] = Face2D_t{element.N_, {new_nodes[1], new_node_index}, {element_index + 1, element_index + 2}, {1, 3}};
            new_faces[new_face_index + 2] = Face2D_t{element.N_, {new_nodes[2], new_node_index}, {element_index + 2, element_index + 3}, {2, 0}};
            new_faces[new_face_index + 3] = Face2D_t{element.N_, {new_nodes[3], new_node_index}, {element_index + 3, element_index},     {3, 1}};

            // CHECK add order
            // CHECK this won't work with anything other than quadrilaterals
            const std::array<std::array<Vec2<deviceFloat>, 4>, 4> new_elements_nodes {std::array<Vec2<deviceFloat>, 4>{nodes[element.nodes_[0]], nodes[new_nodes[0]], nodes[new_node_index], nodes[new_nodes[3]]},
                                                                                      std::array<Vec2<deviceFloat>, 4>{nodes[new_nodes[0]], nodes[element.nodes_[1]], nodes[new_nodes[1]], nodes[new_node_index]},
                                                                                      std::array<Vec2<deviceFloat>, 4>{nodes[new_node_index], nodes[new_nodes[1]], nodes[element.nodes_[2]], nodes[new_nodes[2]]},
                                                                                      std::array<Vec2<deviceFloat>, 4>{nodes[new_nodes[3]], nodes[new_node_index], nodes[new_nodes[2]], nodes[element.nodes_[3]]}};

            std::array<std:array<cuda_vector<size_t>, 4>, 4> new_elements_faces {{device_vector<deviceFloat>{},  device_vector<deviceFloat>{1}, device_vector<deviceFloat>{1}, device_vector<deviceFloat>{}},
                                                                                 {device_vector<deviceFloat>{},  device_vector<deviceFloat>{},  device_vector<deviceFloat>{1}, device_vector<deviceFloat>{1}},
                                                                                 {device_vector<deviceFloat>{1}, device_vector<deviceFloat>{},  device_vector<deviceFloat>{},  device_vector<deviceFloat>{1}},
                                                                                 {device_vector<deviceFloat>{1}, device_vector<deviceFloat>{1}, device_vector<deviceFloat>{},  device_vector<deviceFloat>{}}};

            // These are the newly created faces
            new_elements_faces[0][1][0] = new_face_index;
            new_elements_faces[0][2][0] = new_face_index + 3;

            new_elements_faces[1][3][0] = new_face_index;
            new_elements_faces[1][2][0] = new_face_index + 1;

            new_elements_faces[2][0][0] = new_face_index + 1;
            new_elements_faces[2][3][0] = new_face_index + 2;

            new_elements_faces[3][1][0] = new_face_index + 2;
            new_elements_faces[3][0][0] = new_face_index + 3;

            if (element.additional_nodes_[0]) {
                new_elements_faces[0][0] = device_vector<deviceFloat>{1};
                new_elements_faces[1][0] = device_vector<deviceFloat>{1};
                new_elements_faces[0][0][0] = element.faces_[0][0]
                new_elements_faces[1][0][0] = new_faces[0];
            }
            else {
                const std::array<std::array<Vec2<deviceFloat>, 2>, 2> element_side_nodes {{new_elements_nodes[0][0], new_elements_nodes[0][1]},
                                                                                          {new_elements_nodes[1][0], new_elements_nodes[1][1]}};
                const std::array<Vec2<deviceFloat>, 2> AB {element_side_nodes[0][1] - element_side_nodes[0][0],
                                                           element_side_nodes[1][1] - element_side_nodes[1][0]};
                const std::array<deviceFloat, 2> AB_dot_inv {1/AB[0].dot(AB[0]),
                                                             1/AB[1].dot(AB[1])};

                std::array<size_t, 2> side_n_faces {0, 0};
                for (size_t face_index = 0; face_index < element.faces_[0].size(); ++face_index) {
                    const Face2D_t& face = faces[element.faces_[0][face_index]];
                    const std::array<Vec2<deviceFloat>, 2> face_nodes {nodes[face.nodes_[0]], nodes[face.nodes_[1]]};
                    const std::array<Vec2<deviceFloat>, 2> AC {face_nodes[0] - element_side_nodes[0][0],
                                                               face_nodes[0] - element_side_nodes[1][0]};
                    const std::array<Vec2<deviceFloat>, 2> AD {face_nodes[1] - element_side_nodes[0][0],
                                                               face_nodes[1] - element_side_nodes[1][0]};

                    const std::array<deviceFloat, 2> C_proj {AC[0].dot(AB[0]) * AB_dot_inv[0],
                                                             AC[1].dot(AB[1]) * AB_dot_inv[1]};
                    const std::array<deviceFloat, 2> D_proj {AD[0].dot(AB[0]) * AB_dot_inv[0],
                                                             AD[1].dot(AB[1]) * AB_dot_inv[1]};

                    if (C_proj[0] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && C_proj[0] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()
                        && D_proj[0] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && D_proj[0] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()) {

                        ++side_n_faces[0];
                    }
                    
                    if (C_proj[1] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && C_proj[1] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()
                        && D_proj[1] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && D_proj[1] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()) {

                        ++side_n_faces[1];
                    }
                }

                new_elements_faces[0][0] = device_vector<deviceFloat>{side_n_faces[0]};
                new_elements_faces[1][0] = device_vector<deviceFloat>{side_n_faces[1]};

                std::array<size_t, 2> new_elements_side_face_index {0, 0};
                for (size_t face_index = 0; face_index < element.faces_[0].size(); ++face_index) {
                    const Face2D_t& face = faces[element.faces_[0][face_index]];
                    const std::array<Vec2<deviceFloat>, 2> face_nodes {nodes[face.nodes_[0]], nodes[face.nodes_[1]]};
                    const std::array<Vec2<deviceFloat>, 2> AC {face_nodes[0] - element_side_nodes[0][0],
                                                               face_nodes[0] - element_side_nodes[1][0]};
                    const std::array<Vec2<deviceFloat>, 2> AD {face_nodes[1] - element_side_nodes[0][0],
                                                               face_nodes[1] - element_side_nodes[1][0]};

                    const std::array<deviceFloat, 2> C_proj {AC[0].dot(AB[0]) * AB_dot_inv[0],
                                                             AC[1].dot(AB[1]) * AB_dot_inv[1]};
                    const std::array<deviceFloat, 2> D_proj {AD[0].dot(AB[0]) * AB_dot_inv[0],
                                                             AD[1].dot(AB[1]) * AB_dot_inv[1]};

                    if (C_proj[0] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && C_proj[0] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()
                        && D_proj[0] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && D_proj[0] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()) {

                        new_elements_faces[1][0][new_elements_side_face_index[0]] = element.faces_[0][face_index];
                        ++new_elements_side_face_index[0];
                    }
                    
                    if (C_proj[1] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && C_proj[1] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()
                        && D_proj[1] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && D_proj[1] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()) {

                        new_elements_faces[1][0][new_elements_side_face_index[1]] = element.faces_[0][face_index];
                        ++new_elements_side_face_index[1];
                    }
                }
            }

            if (element.additional_nodes_[1]) {
                new_elements_faces[1][1] = device_vector<deviceFloat>{1};
                new_elements_faces[2][1] = device_vector<deviceFloat>{1};
                new_elements_faces[1][1][0] = element.faces_[1][0]
                new_elements_faces[2][1][0] = new_faces[1];
            }
            else {
                const std::array<std::array<Vec2<deviceFloat>, 2>, 2> element_side_nodes {{new_elements_nodes[1][1], new_elements_nodes[1][2]},
                                                                                          {new_elements_nodes[2][1], new_elements_nodes[2][2]}};
                const std::array<Vec2<deviceFloat>, 2> AB {element_side_nodes[0][1] - element_side_nodes[0][0],
                                                           element_side_nodes[1][1] - element_side_nodes[1][0]};
                const std::array<deviceFloat, 2> AB_dot_inv {1/AB[0].dot(AB[0]),
                                                             1/AB[1].dot(AB[1])};

                std::array<size_t, 2> side_n_faces {0, 0};
                for (size_t face_index = 0; face_index < element.faces_[1].size(); ++face_index) {
                    const Face2D_t& face = faces[element.faces_[1][face_index]];
                    const std::array<Vec2<deviceFloat>, 2> face_nodes {nodes[face.nodes_[0]], nodes[face.nodes_[1]]};
                    const std::array<Vec2<deviceFloat>, 2> AC {face_nodes[0] - element_side_nodes[0][0],
                                                               face_nodes[0] - element_side_nodes[1][0]};
                    const std::array<Vec2<deviceFloat>, 2> AD {face_nodes[1] - element_side_nodes[0][0],
                                                               face_nodes[1] - element_side_nodes[1][0]};

                    const std::array<deviceFloat, 2> C_proj {AC[0].dot(AB[0]) * AB_dot_inv[0],
                                                             AC[1].dot(AB[1]) * AB_dot_inv[1]};
                    const std::array<deviceFloat, 2> D_proj {AD[0].dot(AB[0]) * AB_dot_inv[0],
                                                             AD[1].dot(AB[1]) * AB_dot_inv[1]};

                    if (C_proj[0] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && C_proj[0] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()
                        && D_proj[0] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && D_proj[0] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()) {

                        ++side_n_faces[0];
                    }
                    
                    if (C_proj[1] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && C_proj[1] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()
                        && D_proj[1] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && D_proj[1] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()) {

                        ++side_n_faces[1];
                    }
                }

                new_elements_faces[1][1] = device_vector<deviceFloat>{side_n_faces[0]};
                new_elements_faces[2][1] = device_vector<deviceFloat>{side_n_faces[1]};

                std::array<size_t, 2> new_elements_side_face_index {0, 0};
                for (size_t face_index = 0; face_index < element.faces_[1].size(); ++face_index) {
                    const Face2D_t& face = faces[element.faces_[1][face_index]];
                    const std::array<Vec2<deviceFloat>, 2> face_nodes {nodes[face.nodes_[0]], nodes[face.nodes_[1]]};
                    const std::array<Vec2<deviceFloat>, 2> AC {face_nodes[0] - element_side_nodes[0][0],
                                                               face_nodes[0] - element_side_nodes[1][0]};
                    const std::array<Vec2<deviceFloat>, 2> AD {face_nodes[1] - element_side_nodes[0][0],
                                                               face_nodes[1] - element_side_nodes[1][0]};

                    const std::array<deviceFloat, 2> C_proj {AC[0].dot(AB[0]) * AB_dot_inv[0],
                                                             AC[1].dot(AB[1]) * AB_dot_inv[1]};
                    const std::array<deviceFloat, 2> D_proj {AD[0].dot(AB[0]) * AB_dot_inv[0],
                                                             AD[1].dot(AB[1]) * AB_dot_inv[1]};

                    if (C_proj[0] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && C_proj[0] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()
                        && D_proj[0] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && D_proj[0] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()) {

                        new_elements_faces[1][1][new_elements_side_face_index[0]] = element.faces_[1][face_index];
                        ++new_elements_side_face_index[0];
                    }
                    
                    if (C_proj[1] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && C_proj[1] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()
                        && D_proj[1] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && D_proj[1] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()) {

                        new_elements_faces[2][1][new_elements_side_face_index[1]] = element.faces_[1][face_index];
                        ++new_elements_side_face_index[1];
                    }
                }
            }

            if (element.additional_nodes_[2]) {
                new_elements_faces[2][2] = device_vector<deviceFloat>{1};
                new_elements_faces[3][2] = device_vector<deviceFloat>{1};
                new_elements_faces[2][2][0] = element.faces_[2][0]
                new_elements_faces[3][2][0] = new_faces[2];
            }
            else {
                const std::array<std::array<Vec2<deviceFloat>, 2>, 2> element_side_nodes {{new_elements_nodes[2][2], new_elements_nodes[2][3]},
                                                                                          {new_elements_nodes[3][2], new_elements_nodes[3][3]}};
                const std::array<Vec2<deviceFloat>, 2> AB {element_side_nodes[0][1] - element_side_nodes[0][0],
                                                           element_side_nodes[1][1] - element_side_nodes[1][0]};
                const std::array<deviceFloat, 2> AB_dot_inv {1/AB[0].dot(AB[0]),
                                                             1/AB[1].dot(AB[1])};

                std::array<size_t, 2> side_n_faces {0, 0};
                for (size_t face_index = 0; face_index < element.faces_[2].size(); ++face_index) {
                    const Face2D_t& face = faces[element.faces_[2][face_index]];
                    const std::array<Vec2<deviceFloat>, 2> face_nodes {nodes[face.nodes_[0]], nodes[face.nodes_[1]]};
                    const std::array<Vec2<deviceFloat>, 2> AC {face_nodes[0] - element_side_nodes[0][0],
                                                               face_nodes[0] - element_side_nodes[1][0]};
                    const std::array<Vec2<deviceFloat>, 2> AD {face_nodes[1] - element_side_nodes[0][0],
                                                               face_nodes[1] - element_side_nodes[1][0]};

                    const std::array<deviceFloat, 2> C_proj {AC[0].dot(AB[0]) * AB_dot_inv[0],
                                                             AC[1].dot(AB[1]) * AB_dot_inv[1]};
                    const std::array<deviceFloat, 2> D_proj {AD[0].dot(AB[0]) * AB_dot_inv[0],
                                                             AD[1].dot(AB[1]) * AB_dot_inv[1]};

                    if (C_proj[0] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && C_proj[0] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()
                        && D_proj[0] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && D_proj[0] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()) {

                        ++side_n_faces[0];
                    }
                    
                    if (C_proj[1] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && C_proj[1] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()
                        && D_proj[1] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && D_proj[1] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()) {

                        ++side_n_faces[1];
                    }
                }

                new_elements_faces[2][2] = device_vector<deviceFloat>{side_n_faces[0]};
                new_elements_faces[3][2] = device_vector<deviceFloat>{side_n_faces[1]};

                std::array<size_t, 2> new_elements_side_face_index {0, 0};
                for (size_t face_index = 0; face_index < element.faces_[2].size(); ++face_index) {
                    const Face2D_t& face = faces[element.faces_[2][face_index]];
                    const std::array<Vec2<deviceFloat>, 2> face_nodes {nodes[face.nodes_[0]], nodes[face.nodes_[1]]};
                    const std::array<Vec2<deviceFloat>, 2> AC {face_nodes[0] - element_side_nodes[0][0],
                                                               face_nodes[0] - element_side_nodes[1][0]};
                    const std::array<Vec2<deviceFloat>, 2> AD {face_nodes[1] - element_side_nodes[0][0],
                                                               face_nodes[1] - element_side_nodes[1][0]};

                    const std::array<deviceFloat, 2> C_proj {AC[0].dot(AB[0]) * AB_dot_inv[0],
                                                             AC[1].dot(AB[1]) * AB_dot_inv[1]};
                    const std::array<deviceFloat, 2> D_proj {AD[0].dot(AB[0]) * AB_dot_inv[0],
                                                             AD[1].dot(AB[1]) * AB_dot_inv[1]};

                    if (C_proj[0] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && C_proj[0] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()
                        && D_proj[0] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && D_proj[0] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()) {

                        new_elements_faces[2][2][new_elements_side_face_index[0]] = element.faces_[2][face_index];
                        ++new_elements_side_face_index[0];
                    }
                    
                    if (C_proj[1] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && C_proj[1] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()
                        && D_proj[1] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && D_proj[1] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()) {

                        new_elements_faces[3][2][new_elements_side_face_index[1]] = element.faces_[2][face_index];
                        ++new_elements_side_face_index[1];
                    }
                }
            }

            if (element.additional_nodes_[3]) {
                new_elements_faces[3][3] = device_vector<deviceFloat>{1};
                new_elements_faces[0][3] = device_vector<deviceFloat>{1};
                new_elements_faces[3][3][0] = element.faces_[3][0]
                new_elements_faces[0][3][0] = new_faces[3];
            }
            else {
                const std::array<std::array<Vec2<deviceFloat>, 2>, 2> element_side_nodes {{new_elements_nodes[3][3], new_elements_nodes[3][0]},
                                                                                          {new_elements_nodes[0][3], new_elements_nodes[0][0]}};
                const std::array<Vec2<deviceFloat>, 2> AB {element_side_nodes[0][1] - element_side_nodes[0][0],
                                                           element_side_nodes[1][1] - element_side_nodes[1][0]};
                const std::array<deviceFloat, 2> AB_dot_inv {1/AB[0].dot(AB[0]),
                                                             1/AB[1].dot(AB[1])};

                std::array<size_t, 2> side_n_faces {0, 0};
                for (size_t face_index = 0; face_index < element.faces_[3].size(); ++face_index) {
                    const Face2D_t& face = faces[element.faces_[3][face_index]];
                    const std::array<Vec2<deviceFloat>, 2> face_nodes {nodes[face.nodes_[0]], nodes[face.nodes_[1]]};
                    const std::array<Vec2<deviceFloat>, 2> AC {face_nodes[0] - element_side_nodes[0][0],
                                                               face_nodes[0] - element_side_nodes[1][0]};
                    const std::array<Vec2<deviceFloat>, 2> AD {face_nodes[1] - element_side_nodes[0][0],
                                                               face_nodes[1] - element_side_nodes[1][0]};

                    const std::array<deviceFloat, 2> C_proj {AC[0].dot(AB[0]) * AB_dot_inv[0],
                                                             AC[1].dot(AB[1]) * AB_dot_inv[1]};
                    const std::array<deviceFloat, 2> D_proj {AD[0].dot(AB[0]) * AB_dot_inv[0],
                                                             AD[1].dot(AB[1]) * AB_dot_inv[1]};

                    if (C_proj[0] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && C_proj[0] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()
                        && D_proj[0] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && D_proj[0] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()) {

                        ++side_n_faces[0];
                    }
                    
                    if (C_proj[1] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && C_proj[1] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()
                        && D_proj[1] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && D_proj[1] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()) {

                        ++side_n_faces[1];
                    }
                }

                new_elements_faces[3][3] = device_vector<deviceFloat>{side_n_faces[0]};
                new_elements_faces[0][3] = device_vector<deviceFloat>{side_n_faces[1]};

                std::array<size_t, 2> new_elements_side_face_index {0, 0};
                for (size_t face_index = 0; face_index < element.faces_[3].size(); ++face_index) {
                    const Face2D_t& face = faces[element.faces_[3][face_index]];
                    const std::array<Vec2<deviceFloat>, 2> face_nodes {nodes[face.nodes_[0]], nodes[face.nodes_[1]]};
                    const std::array<Vec2<deviceFloat>, 2> AC {face_nodes[0] - element_side_nodes[0][0],
                                                               face_nodes[0] - element_side_nodes[1][0]};
                    const std::array<Vec2<deviceFloat>, 2> AD {face_nodes[1] - element_side_nodes[0][0],
                                                               face_nodes[1] - element_side_nodes[1][0]};

                    const std::array<deviceFloat, 2> C_proj {AC[0].dot(AB[0]) * AB_dot_inv[0],
                                                             AC[1].dot(AB[1]) * AB_dot_inv[1]};
                    const std::array<deviceFloat, 2> D_proj {AD[0].dot(AB[0]) * AB_dot_inv[0],
                                                             AD[1].dot(AB[1]) * AB_dot_inv[1]};

                    if (C_proj[0] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && C_proj[0] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()
                        && D_proj[0] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && D_proj[0] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()) {

                        new_elements_faces[3][3][new_elements_side_face_index[0]] = element.faces_[3][face_index];
                        ++new_elements_side_face_index[0];
                    }
                    
                    if (C_proj[1] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && C_proj[1] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()
                        && D_proj[1] + std::numeric_limits<deviceFloat>::epsilon() >= static_cast<deviceFloat>(0) 
                        && D_proj[1] <= static_cast<deviceFloat>(1) + std::numeric_limits<deviceFloat>::epsilon()) {

                        new_elements_faces[0][3][new_elements_side_face_index[1]] = element.faces_[3][face_index];
                        ++new_elements_side_face_index[1];
                    }
                }
            }

            new_elements[element_index]     = Element2D_t{element.N_, element.split_level_ + 1, new_elements_faces[0], {element.nodes_[0], new_nodes[0], new_node_index, new_nodes[3]}};
            new_elements[element_index + 1] = Element2D_t{element.N_, element.split_level_ + 1, new_elements_faces[1], {new_nodes[0], element.nodes_[1], new_nodes[1], new_node_index}};
            new_elements[element_index + 2] = Element2D_t{element.N_, element.split_level_ + 1, new_elements_faces[2], {new_node_index, new_nodes[1], element.nodes_[2], new_nodes[2]}};
            new_elements[element_index + 3] = Element2D_t{element.N_, element.split_level_ + 1, new_elements_faces[3], {new_nodes[3], new_node_index, new_nodes[2], element.nodes_[3]}};

            new_elements[element_index].interpolate_from(new_elements_nodes[0], element_nodes, element, polynomial_nodes, barycentric_weights);
            new_elements[element_index + 1].interpolate_from(new_elements_nodes[1], element_nodes, element, polynomial_nodes, barycentric_weights);
            new_elements[element_index + 2].interpolate_from(new_elements_nodes[2], element_nodes, element, polynomial_nodes, barycentric_weights);
            new_elements[element_index + 3].interpolate_from(new_elements_nodes[3], element_nodes, element, polynomial_nodes, barycentric_weights);
            
            new_elements[element_index].compute_geometry(new_elements_nodes[0], polynomial_nodes);
            new_elements[element_index + 1].compute_geometry(new_elements_nodes[1], polynomial_nodes);
            new_elements[element_index + 2].compute_geometry(new_elements_nodes[2], polynomial_nodes);
            new_elements[element_index + 3].compute_geometry(new_elements_nodes[3], polynomial_nodes); 
            
            new_faces[new_face_index].compute_geometry(new_elements, nodes);
            new_faces[new_face_index + 1].compute_geometry(new_elements, nodes);
            new_faces[new_face_index + 2].compute_geometry(new_elements, nodes);
            new_faces[new_face_index + 3].compute_geometry(new_elements, nodes);
        }
        // p refinement
        else if (element.would_p_refine(N_max)) {
            new_elements[element_index] = Element2D_t{element.N_ + 2, element.split_level_, element.faces_, element.nodes_};

            new_elements[element_index].interpolate_from(element, polynomial_nodes, barycentric_weights);

            const std::array<Vec2<deviceFloat>, 4> points {nodes[new_elements[element_index].nodes_[0]],
                                                           nodes[new_elements[element_index].nodes_[1]],
                                                           nodes[new_elements[element_index].nodes_[2]],
                                                           nodes[new_elements[element_index].nodes_[3]]};
            new_elements[element_index].compute_geometry(points, polynomial_nodes); 
        }
        // move
        else {
            new_elements[element_index] = std::move(element);
        }
    }
}

__global__
auto SEM::Meshes::move_faces(size_t n_faces, Face2D_t* faces, Face2D_t* new_faces, Element2D_t* elements) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t face_index = index; face_index < n_faces; face_index += stride) {

    }
}


__global__
auto SEM::Meshes::adjust_boundaries(size_t n_boundaries, Element2D_t* elements, const size_t* boundaries, const Face2D_t* faces) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t boundary_index = index; boundary_index < n_boundaries; boundary_index += stride) {
        Element2D_t& destination_element = elements[boundaries[boundary_index]];
        int N_max = destination_element.N_;

        for (size_t face_index = 0; face_index < destination_element.faces_[0].size(); ++face_index) {
            const Face2D_t face = faces[destination_element.faces_[0][face_index]];
            const int element_index = face.elements_[0] == boundaries[boundary_index];
            const Element2D_t& source_element = elements[face.elements_[element_index]];

            N_max = std::max(N_max, source_element.N_);
        }

        if (destination_element.N_ != N_max) {
            destination_element.resize_boundary_storage(N_max);
        }
    }
}

__global__
auto SEM::Meshes::rebuild_boundaries(size_t n_boundaries, Element2D_t* elements, Element2D_t* new_elements, const size_t* boundaries, const Face2D_t* faces) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t boundary_index = index; boundary_index < n_boundaries; boundary_index += stride) {
        Element2D_t& destination_element = elements[boundaries[boundary_index]];
        int N_max = destination_element.N_;

        for (size_t face_index = 0; face_index < destination_element.faces_[0].size(); ++face_index) {
            const Face2D_t face = faces[destination_element.faces_[0][face_index]];
            const int element_index = face.elements_[0] == boundaries[boundary_index];
            const Element2D_t& source_element = elements[face.elements_[element_index]];

            N_max = std::max(N_max, source_element.N_);
        }

        if (destination_element.N_ != N_max) {
            destination_element.resize_boundary_storage(N_max);
        }

        new_elements[boundaries[boundary_index]] = std::move(elements[boundaries[boundary_index]]);
    }
}

__global__
auto SEM::Meshes::adjust_interfaces(size_t n_local_interfaces, Element2D_t* elements, const size_t* local_interfaces_origin, const size_t* local_interfaces_destination) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t interface_index = index; interface_index < n_local_interfaces; interface_index += stride) {
        const Element2D_t& source_element = elements[local_interfaces_origin[interface_index]];
        Element2D_t& destination_element = elements[local_interfaces_destination[interface_index]];

        if (destination_element.N_ != source_element.N_) {
            destination_element.resize_boundary_storage(source_element.N_);
        }
    }
}

__global__
auto SEM::Meshes::rebuild_interfaces(size_t n_local_interfaces, Element2D_t* elements, Element2D_t* new_elements, const size_t* local_interfaces_origin, const size_t* local_interfaces_destination) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t interface_index = index; interface_index < n_local_interfaces; interface_index += stride) {
        const Element2D_t& source_element = new_elements[local_interfaces_origin[interface_index]];
        Element2D_t& destination_element = elements[local_interfaces_destination[interface_index]];

        if (destination_element.N_ != source_element.N_) {
            destination_element.resize_boundary_storage(source_element.N_);
        }

        new_elements[local_interfaces_destination[interface_index]] = std::move(elements[local_interfaces_destination[interface_index]]);
    }
}

__global__
auto SEM::Meshes::adjust_faces(size_t n_faces, Face2D_t* faces, const Element2D_t* elements) -> void {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t face_index = index; face_index < n_faces; face_index += stride) {
        Face2D_t& face = faces[face_index];
        const Element2D_t& element_L = elements[face.elements_[0]];
        const Element2D_t& element_R = elements[face.elements_[1]];

        const int N_max = std::max(element_L.N_, element_R.N_);

        if (face.N_ != N_max) {
            face.resize_storage(N_max);
        }
    }
}
