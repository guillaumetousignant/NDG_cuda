#include "meshes/Mesh2D_t.cuh"
#include "polynomials/ChebyshevPolynomial_t.cuh"
#include "polynomials/LegendrePolynomial_t.cuh"
#include "helpers/ProgressBar_t.h"
#include "functions/Utilities.h"
#include "cgnslib.h"
#include <iostream>
#include <fstream>
#include <sstream> 
#include <iomanip>
#include <filesystem>
#include <limits>

namespace fs = std::filesystem;
using SEM::Entities::device_vector;
using SEM::Entities::Vec2;

constexpr int CGIO_MAX_NAME_LENGTH = 33; // Includes the null terminator

SEM::Meshes::Mesh2D_t::Mesh2D_t(std::filesystem::path filename, int initial_N, hipStream_t &stream) :       
        initial_N_(initial_N),        
        stream_(stream) {

    std::string extension = filename.extension().string();
    SEM::to_lower(extension);

    if (extension == ".cgns") {
        read_cgns(filename);
    }
    else if (extension == ".su2") {
        read_su2(filename);
    }
    else {
        std::cerr << "Error: extension '" << extension << "' not recognized. Exiting." << std::endl;
        exit(14);
    }
}

auto SEM::Meshes::Mesh2D_t::read_su2(std::filesystem::path filename) -> void {
    std::cerr << "Error: SU2 meshes not implemented yet. Exiting." << std::endl;
    exit(15);

    /*std::string line;
    std::string token;
    size_t value;

    std::ifstream mesh_file(filename);
    if (!mesh_file.is_open()) {
        std::cerr << "Error: file '" << filename << "' could not be opened. Exiting." << std::endl;
        exit(7);
    }

    do {
        std::getline(mesh_file, line);  
    }
    while (line.empty());
    
    std::istringstream liness(line);
    liness >> token;
    liness >> value;
    if (token != "NDIME=") {
        std::cerr << "Error: first token should be 'NDIME=', found '" << token << "'. Exiting." << std::endl;
        exit(8);
    }

    if (value != 2) {
        std::cerr << "Error: program only works for 2 dimensions, found '" << value << "'. Exiting." << std::endl;
        exit(9);
    }

    //std::vector<Cell_t> farfield;
    //std::vector<Cell_t> wall;
    //std::vector<Cell_t> inlet;
    std::vector<Vec2<deviceFloat>> host_nodes;
    std::vector<std::array<size_t, 4>> host_element_to_nodes;

    while (!mesh_file.eof()) {
        do {
            std::getline(mesh_file, line);  
        }
        while (line.empty() && !mesh_file.eof());

        std::istringstream liness(line);
        liness >> token;
        std::transform(token.begin(), token.end(), token.begin(),
            [](unsigned char c){ return std::toupper(c); });

        if (token == "NPOIN=") {
            liness >> value;
            host_nodes = std::vector<Vec2<deviceFloat>>(value);

            for (size_t i = 0; i < host_nodes.size(); ++i) {
                std::getline(mesh_file, line);
                std::istringstream liness2(line);
                liness2 >> host_nodes[i].pos_[0] >> host_nodes[i].pos_[1];
            }
        }
        else if (token == "NELEM=") {
            liness >> value;
            host_element_to_nodes = std::vector<std::array<size_t, 4>>(value);
            n_elements_ = value;

            for (size_t i = 0; i < host_element_to_nodes.size(); ++i) {
                std::getline(mesh_file, line);
                std::istringstream liness2(line);
                liness2 >> token;

                if (token == "9") {
                    constexpr int n_sides = 4;

                    for (int j = 0; j < n_sides; ++j) {
                        liness2 >> host_element_to_nodes[i][j];
                    }
                }
                /*else if (token == "5") {
                    constexpr int n_sides = 3;

                    for (int j = 0; j < n_sides; ++j) {
                        liness2 >> host_element_to_nodes[i][j];
                    }
                }*/
                /*else {
                    std::cerr << "Error: expected token '9', found '" << token << "'. Exiting." << std::endl;
                    exit(10);
                }
            }
        }
        else if (token == "NMARK=") {
            int n_markers;
            liness >> n_markers;

            n_farfield_ = 0;
            n_wall_ = 0;
            n_inlet_ = 0;

            for (int i = 0; i < n_markers; ++i) {
                std::string type;
                do {
                    std::getline(mesh_file, line);
                    if (!line.empty()) {
                        std::istringstream liness(line);
                        liness >> token;
                        liness >> type;
                    }   
                }
                while (token != "MARKER_TAG=");
                std::transform(type.begin(), type.end(), type.begin(),
                    [](unsigned char c){ return std::tolower(c); });

                if (type == "farfield") {
                    do {
                        std::getline(mesh_file, line);
                        if (!line.empty()) {
                            std::istringstream liness(line);
                            liness >> token;
                            liness >> value;
                        }   
                    }
                    while (token != "MARKER_ELEMS=");

                    n_farfield_ += value;
                    farfield.reserve(n_farfield_);

                    for (size_t j = 0; j < value; ++j) {

                        std::getline(mesh_file, line);
                        std::istringstream liness6(line);

                        liness6 >> token;
                        if (token != "3") {
                            std::cerr << "Error: expected token '3', found '" << token << "'. Exiting." << std::endl;
                            exit(11);
                        }

                        size_t val0, val1;
                        liness6 >> val0 >> val1;
                        farfield.push_back(Cell_t(2));
                        farfield[farfield.size() - 1].nodes_[0] = val0;
                        farfield[farfield.size() - 1].nodes_[1] = val1;
                    }
                }
                else if (type == "wall") {
                    do {
                        std::getline(mesh_file, line);
                        if (!line.empty()) {
                            std::istringstream liness(line);
                            liness >> token;
                            liness >> value;
                        }   
                    }
                    while (token != "MARKER_ELEMS=");

                    n_wall_ += value;
                    wall.reserve(n_wall_);

                    for (size_t j = 0; j < value; ++j) {

                        std::getline(mesh_file, line);
                        std::istringstream liness6(line);

                        liness6 >> token;
                        if (token != "3") {
                            std::cerr << "Error: expected token '3', found '" << token << "'. Exiting." << std::endl;
                            exit(12);
                        }

                        size_t val0, val1;
                        liness6 >> val0 >> val1;
                        wall.push_back(Cell_t(2));
                        wall[wall.size() - 1].nodes_[0] = val0;
                        wall[wall.size() - 1].nodes_[1] = val1;
                    }
                }
                else if (type == "inlet") {
                    do {
                        std::getline(mesh_file, line);
                        if (!line.empty()) {
                            std::istringstream liness(line);
                            liness >> token;
                            liness >> value;
                        }   
                    }
                    while (token != "MARKER_ELEMS=");

                    n_inlet_ += value;
                    inlet.reserve(n_inlet_);

                    for (size_t j = 0; j < value; ++j) {

                        std::getline(mesh_file, line);
                        std::istringstream liness6(line);

                        liness6 >> token;
                        if (token != "3") {
                            std::cerr << "Error: expected token '3', found '" << token << "'. Exiting." << std::endl;
                            exit(12);
                        }

                        size_t val0, val1;
                        liness6 >> val0 >> val1;
                        inlet.push_back(Cell_t(2));
                        inlet[inlet.size() - 1].nodes_[0] = val0;
                        inlet[inlet.size() - 1].nodes_[1] = val1;
                    }
                }
                else {
                    std::cerr << "Error: expected marker tag 'farfield', 'wall' or 'inlet', found '" << type << "'. Exiting." << std::endl;
                    exit(6);
                }
            }
        }
        else {
            if (!mesh_file.eof()) {
                std::cerr << "Error: expected marker 'NPOIN=', 'NELEM=' or 'NMARK=', found '" << token << "'. Exiting." << std::endl;
                exit(13);
            }
        }
    }

    mesh_file.close();

    cells_.insert(std::end(cells_), std::begin(farfield), std::end(farfield));
    cells_.insert(std::end(cells_), std::begin(wall), std::end(wall));
    cells_.insert(std::end(cells_), std::begin(inlet), std::end(inlet));*/
}

auto SEM::Meshes::Mesh2D_t::read_cgns(std::filesystem::path filename) -> void {
    int index_file;
    const int open_error = cg_open(filename.string().c_str(), CG_MODE_READ, &index_file);
    if (open_error != CG_OK) {
        std::cerr << "Error: file '" << filename << "' could not be opened with error '" << cg_get_error() << "'. Exiting." << std::endl;
        exit(16);
    }

    // Getting base information
    int n_bases = 0;
    cg_nbases(index_file, &n_bases);
    if (n_bases != 1) {
        std::cerr << "Error: CGNS mesh has " << n_bases << " base(s), but for now only a single base is supported. Exiting." << std::endl;
        exit(17);
    }
    const int index_base = 1;

    std::array<char, CGIO_MAX_NAME_LENGTH> base_name; // Oh yeah cause it's the 80s still
    int dim = 0;
    int physDim = 0;
    cg_base_read(index_file, index_base, base_name.data(), &dim, &physDim);
    if (dim != 2) {
        std::cerr << "Error: CGNS mesh, base " << index_base << " has " << dim << " dimensions, but the program only supports 2 dimensions. Exiting." << std::endl;
        exit(18);
    }
    if (physDim != 2) {
        std::cerr << "Error: CGNS mesh, base " << index_base << " has " << physDim << " physical dimensions, but the program only supports 2 physical dimensions. Exiting." << std::endl;
        exit(19);
    }

    // Getting zone information
    int n_zones = 0;
    cg_nzones(index_file, index_base, &n_zones);
    if (n_bases != 1) {
        std::cerr << "Error: CGNS mesh, base " << index_base << " has " << n_zones << " zone(s), but for now only a single zone is supported. Exiting." << std::endl;
        exit(20);
    }
    const int index_zone = 1;

    ZoneType_t zone_type = ZoneType_t::ZoneTypeNull;
    cg_zone_type(index_file, index_base, index_zone, &zone_type);
    if (zone_type != ZoneType_t::Unstructured) {
        std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << " is not an unstructured zone. For now only unstructured zones are supported. Exiting." << std::endl;
        exit(21);
    }

    std::array<char, CGIO_MAX_NAME_LENGTH> zone_name; // Oh yeah cause it's the 80s still
    std::array<int, 3> isize{0, 0, 0};
    cg_zone_read(index_file, index_base, index_zone, zone_name.data(), isize.data());
    if (isize[2] != 0) {
        std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << " has " << isize[2] << " boundary vertices, but to be honest I'm not sure how to deal with them. Exiting." << std::endl;
        exit(22);
    }
    const int n_nodes = isize[0];
    const int n_elements = isize[1];

    // Getting nodes
    int n_coords = 0;
    cg_ncoords(index_file, index_base, index_zone, &n_coords);
    if (n_coords != 2) {
        std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << " has " << n_coords << " sets of coordinates, but for now only two are supported. Exiting." << std::endl;
        exit(23);
    }

    std::array<std::array<char, CGIO_MAX_NAME_LENGTH>, 2> coord_names; // Oh yeah cause it's the 80s still
    std::array<DataType_t, 2> coord_data_types {DataType_t::DataTypeNull, DataType_t::DataTypeNull};
    for (int index_coord = 1; index_coord <= n_coords; ++index_coord) {
        cg_coord_info(index_file, index_base, index_zone, index_coord, &coord_data_types[index_coord - 1], coord_names[index_coord - 1].data());
    }

    std::array<std::vector<double>, 2> xy{std::vector<double>(n_nodes), std::vector<double>(n_nodes)};

    for (int index_coord = 1; index_coord <= n_coords; ++index_coord) {
        const int index_coord_start = 1;
        cg_coord_read(index_file, index_base, index_zone, coord_names[index_coord - 1].data(), DataType_t::RealDouble, &index_coord_start, &n_nodes, xy[index_coord - 1].data());
    }
    
    // Getting connectivity
    int n_sections = 0;
    cg_nsections(index_file, index_base, index_zone, &n_sections);

    std::vector<int> section_data_size(n_sections);
    std::vector<std::array<char, CGIO_MAX_NAME_LENGTH>> section_names(n_sections); // Oh yeah cause it's the 80s still
    std::vector<ElementType_t> section_type(n_sections);
    std::vector<std::array<int, 2>> section_ranges(n_sections);
    std::vector<int> section_n_boundaries(n_sections);
    std::vector<int> section_parent_flags(n_sections);
    for (int index_section = 1; index_section <= n_sections; ++index_section) {
        cg_ElementDataSize(index_file, index_base, index_zone, index_section, &section_data_size[index_section - 1]);
        cg_section_read(index_file, index_base, index_zone, index_section, section_names[index_section - 1].data(), &section_type[index_section - 1], &section_ranges[index_section - 1][0], &section_ranges[index_section - 1][1], &section_n_boundaries[index_section - 1], &section_parent_flags[index_section - 1]);
        if (section_n_boundaries[index_section - 1] != 0) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", section " << index_section << " has " << section_n_boundaries[index_section - 1] << " boundary elements, but to be honest I'm not sure how to deal with them. Exiting." << std::endl;
            exit(24);
        }
    }

    std::vector<std::vector<int>> connectivity(n_sections);
    std::vector<std::vector<int>> parent_data(n_sections);
    for (int index_section = 1; index_section <= n_sections; ++index_section) {
        connectivity[index_section - 1] = std::vector<int>(section_data_size[index_section - 1]);
        parent_data[index_section - 1] = std::vector<int>(section_ranges[index_section - 1][1] - section_ranges[index_section - 1][0]);

        cg_elements_read(index_file, index_base, index_zone, index_section, connectivity[index_section - 1].data(), parent_data[index_section - 1].data());
    }

    // Interfaces
    int n_connectivity = 0;
    cg_nconns(index_file, index_base, index_zone, &n_connectivity);

    std::vector<std::array<char, CGIO_MAX_NAME_LENGTH>> connectivity_names(n_connectivity); // Oh yeah cause it's the 80s still
    std::vector<GridLocation_t> connectivity_grid_locations(n_connectivity);
    std::vector<GridConnectivityType_t> connectivity_types(n_connectivity);
    std::vector<PointSetType_t> connectivity_point_set_types(n_connectivity);
    std::vector<int> connectivity_sizes(n_connectivity);
    std::vector<std::array<char, CGIO_MAX_NAME_LENGTH>> connectivity_donor_names(n_connectivity); // Oh yeah cause it's the 80s still
    std::vector<ZoneType_t> connectivity_donor_zone_types(n_connectivity);
    std::vector<PointSetType_t> connectivity_donor_point_set_types(n_connectivity);
    std::vector<DataType_t> connectivity_donor_data_types(n_connectivity);
    std::vector<int> connectivity_donor_sizes(n_connectivity);
    for (int index_connectivity = 1; index_connectivity <= n_connectivity; ++index_connectivity) {
        cg_conn_info(index_file, index_base, index_zone, index_connectivity, connectivity_names[index_connectivity - 1].data(),
            &connectivity_grid_locations[index_connectivity - 1], &connectivity_types[index_connectivity - 1],
            &connectivity_point_set_types[index_connectivity - 1], &connectivity_sizes[index_connectivity - 1], connectivity_donor_names[index_connectivity - 1].data(),
            &connectivity_donor_zone_types[index_connectivity - 1], &connectivity_donor_point_set_types[index_connectivity - 1],
            &connectivity_donor_data_types[index_connectivity - 1], &connectivity_donor_sizes[index_connectivity - 1]);

        if (connectivity_donor_zone_types[index_connectivity - 1] != ZoneType_t::Unstructured) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", connectivity " << index_connectivity << " has a donor zone type that isn't unstructured. For now only unstructured zones are supported. Exiting." << std::endl;
            exit(25);
        }
        if (connectivity_point_set_types[index_connectivity - 1] != PointSetType_t::PointList) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", connectivity " << index_connectivity << " has a point set type that isn't PointList. For now only PointList point set types are supported. Exiting." << std::endl;
            exit(26);
        }
        if (connectivity_donor_point_set_types[index_connectivity - 1] != PointSetType_t::PointListDonor) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", connectivity " << index_connectivity << " has a donor point set type that isn't PointListDonor. For now only PointListDonor point set types are supported. Exiting." << std::endl;
            exit(27);
        }

        if (connectivity_grid_locations[index_connectivity - 1] != GridLocation_t::FaceCenter) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", connectivity " << index_connectivity << " has a grid location that isn't FaceCenter. For now only FaceCenter grid locations are supported. Exiting." << std::endl;
            exit(28);
        }

        if (connectivity_types[index_connectivity - 1] != GridConnectivityType_t::Abutting1to1) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", connectivity " << index_connectivity << " has a grid connectivity type that isn't Abutting1to1. For now only Abutting1to1 grid connectivity types are supported. Exiting." << std::endl;
            exit(29);
        }
    }

    std::vector<std::vector<int>> interface_elements(n_connectivity);
    std::vector<std::vector<int>> interface_donor_elements(n_connectivity);
    for (int index_connectivity = 1; index_connectivity <= n_connectivity; ++index_connectivity) {
        interface_elements[index_connectivity - 1] = std::vector<int>(connectivity_sizes[index_connectivity - 1]);
        interface_donor_elements[index_connectivity - 1] = std::vector<int>(connectivity_donor_sizes[index_connectivity - 1]);
        cg_conn_read(index_file, index_base, index_zone, index_connectivity, interface_elements[index_connectivity - 1].data(),
            DataType_t::Integer, interface_donor_elements[index_connectivity - 1].data());
    }

    // Boundary conditions
    int n_boundaries = 0;
    cg_nbocos(index_file, index_base, index_zone, &n_boundaries);

    std::vector<std::array<char, CGIO_MAX_NAME_LENGTH>> boundary_names(n_boundaries); // Oh yeah cause it's the 80s still
    std::vector<BCType_t> boundary_types(n_boundaries);
    std::vector<PointSetType_t> boundary_point_set_types(n_boundaries);
    std::vector<int> boundary_sizes(n_boundaries);
    std::vector<int> boundary_normal_indices(n_boundaries);
    std::vector<int> boundary_normal_list_sizes(n_boundaries);
    std::vector<DataType_t> boundary_normal_data_types(n_boundaries);
    std::vector<int> boundary_n_datasets(n_boundaries);
    std::vector<GridLocation_t> boundary_grid_locations(n_boundaries);
    for (int index_boundary = 1; index_boundary <= n_boundaries; ++index_boundary) {
        cg_boco_info(index_file, index_base, index_zone, index_boundary, boundary_names[index_boundary - 1].data(),
            &boundary_types[index_boundary - 1], &boundary_point_set_types[index_boundary - 1], &boundary_sizes[index_boundary - 1],
            &boundary_normal_indices[index_boundary - 1], &boundary_normal_list_sizes[index_boundary - 1],
            &boundary_normal_data_types[index_boundary - 1], &boundary_n_datasets[index_boundary - 1]);

        if (boundary_point_set_types[index_boundary - 1] != PointSetType_t::PointList) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", boundary " << index_boundary << " has a point set type that isn't PointList. For now only PointList point set types are supported. Exiting." << std::endl;
            exit(30);
        }

        cg_boco_gridlocation_read(index_file, index_base, index_zone, index_boundary, &boundary_grid_locations[index_boundary - 1]);

        if (boundary_grid_locations[index_boundary - 1] != GridLocation_t::EdgeCenter) {
            std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", boundary " << index_boundary << " has a grid location that isn't EdgeCenter. For now only EdgeCenter grid locations are supported. Exiting." << std::endl;
            exit(31);
        }
    }

    std::vector<std::vector<int>> boundary_elements(n_boundaries);
    std::vector<std::vector<int>> boundary_normals(n_boundaries);
    for (int index_boundary = 1; index_boundary <= n_boundaries; ++index_boundary) {
        boundary_elements[index_boundary - 1] = std::vector<int>(boundary_sizes[index_boundary - 1]);
        boundary_normals[index_boundary - 1] = std::vector<int>(boundary_normal_list_sizes[index_boundary - 1]);
        cg_boco_read(index_file, index_base, index_zone, index_boundary, boundary_elements[index_boundary - 1].data(), boundary_normals[index_boundary - 1].data());
    }

    cg_close(index_file);

    // Putting nodes in the format used by the mesh
    std::vector<Vec2<deviceFloat>> host_nodes(n_nodes);
    for (int i = 0; i < n_nodes; ++i) {
        host_nodes[i].x() = xy[0][i];
        host_nodes[i].y() = xy[1][i];
    }

    // Figuring out which sections are the domain and which are ghost cells
    std::vector<bool> section_is_domain(n_sections);
    int n_elements_domain = 0;
    int n_elements_ghost = 0;
    for (int i = 0; i < n_sections; ++i) {
        switch (section_type[i]) {
            case ElementType_t::BAR_2:
                section_is_domain[i] = false;
                n_elements_ghost += section_ranges[i][1] - section_ranges[i][0] + 1;
                break;

            case ElementType_t::QUAD_4:
                section_is_domain[i] = true;
                n_elements_domain += section_ranges[i][1] - section_ranges[i][0] + 1;
                break;

            default:
                std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << ", section " << i << " has an unknown element type. For now only BAR_2 and QUAD_4 are implemented, for boundaries and domain respectively. Exiting." << std::endl;
                exit(32);
        }
    }

    if (n_elements_domain + n_elements_ghost != n_elements) {
        std::cerr << "Error: CGNS mesh, base " << index_base << ", zone " << index_zone << " has " << n_elements << " elements but the sum of its sections is " << n_elements_domain + n_elements_ghost << " elements. Exiting." << std::endl;
        exit(33);
    }

    // Putting connectivity data in the format used by the mesh
    std::vector<SEM::Entities::Element2D_t> host_elements(n_elements);
    std::vector<size_t> section_start_indices(n_sections);
    size_t element_domain_index = 0;
    size_t element_ghost_index = n_elements_domain;
    for (int i = 0; i < n_sections; ++i) {
        if (section_is_domain[i]) {
            section_start_indices[i] = element_domain_index;
            for (int j = 0; j < section_ranges[i][1] - section_ranges[i][0] + 1; ++j) {
                host_elements[section_start_indices[i] + j].N_ = initial_N_;
                host_elements[section_start_indices[i] + j].nodes_ = {static_cast<size_t>(connectivity[i][4 * j] - 1),
                                                                      static_cast<size_t>(connectivity[i][4 * j + 1] - 1),
                                                                      static_cast<size_t>(connectivity[i][4 * j + 2] - 1),
                                                                      static_cast<size_t>(connectivity[i][4 * j + 3] - 1)};
            }
            element_domain_index += section_ranges[i][1] - section_ranges[i][0] + 1;
        }
        else {
            section_start_indices[i] = element_ghost_index;
            for (int j = 0; j < section_ranges[i][1] - section_ranges[i][0] + 1; ++j) {
                host_elements[section_start_indices[i] + j].N_ = 0;
                host_elements[section_start_indices[i] + j].nodes_ = {static_cast<size_t>(connectivity[i][2 * j] - 1),
                                                                      static_cast<size_t>(connectivity[i][2 * j + 1] - 1),
                                                                      static_cast<size_t>(connectivity[i][2 * j + 1] - 1),
                                                                      static_cast<size_t>(connectivity[i][2 * j] - 1)};
            }
            element_ghost_index += section_ranges[i][1] - section_ranges[i][0] + 1;
        }
    }

    // Computing nodes to elements
    const std::vector<std::vector<size_t>> node_to_element = build_node_to_element(n_nodes, host_elements);

    // Computing element to elements
    const std::vector<std::vector<size_t>> element_to_element = build_element_to_element(host_elements, node_to_element);

    // Computing faces and filling element faces
    auto [host_faces, node_to_face] = build_faces(n_nodes, host_elements);
}

auto SEM::Meshes::Mesh2D_t::build_node_to_element(size_t n_nodes, const std::vector<SEM::Entities::Element2D_t>& elements) -> std::vector<std::vector<size_t>> {
    std::vector<std::vector<size_t>> node_to_element(n_nodes);

    for (size_t j = 0; j < elements.size(); ++j) {
        for (auto node_index: elements[j].nodes_) {
            if (std::find(node_to_element[node_index].begin(), node_to_element[node_index].end(), j) == node_to_element[node_index].end()) { // This will be slower, but is needed because boundaries have 4 sides and not 2. Remove when variable geometry elements are added.
                node_to_element[node_index].push_back(j);
            }
        }
    }

    return node_to_element;
}

auto SEM::Meshes::Mesh2D_t::build_element_to_element(const std::vector<SEM::Entities::Element2D_t>& elements, const std::vector<std::vector<size_t>>& node_to_element) -> std::vector<std::vector<size_t>> {
    std::vector<std::vector<size_t>> element_to_element(elements.size());

    for (size_t i = 0; i < elements.size(); ++i) {
        const SEM::Entities::Element2D_t& element = elements[i];
        element_to_element[i] = std::vector<size_t>(element.nodes_.size());

        for (size_t j = 0; j < element.nodes_.size(); ++j) {
            const size_t node_index = element.nodes_[j];
            const size_t node_index_next = (j < element.nodes_.size() - 1) ? element.nodes_[j + 1] : element.nodes_[0];

            for (auto element_index : node_to_element[node_index]) {
                if (element_index != i) {
                    const SEM::Entities::Element2D_t& element_neighbor = elements[element_index];

                    auto it = find(element_neighbor.nodes_.begin(), element_neighbor.nodes_.end(), node_index);
                    if (it != element_neighbor.nodes_.end()) {
                        const size_t node_element_index = it - element_neighbor.nodes_.begin();
                        
                        for (size_t node_element_index_next = 0; node_element_index_next < node_element_index; ++node_element_index_next) {
                            if (element_neighbor.nodes_[node_element_index_next] == node_index_next) {
                                element_to_element[i][j] = element_index;
                                goto endloop; // I hate this too don't worry
                            }
                        }

                        for (size_t node_element_index_next = node_element_index + 1; node_element_index_next < element_neighbor.nodes_.size(); ++node_element_index_next) {
                            if (element_neighbor.nodes_[node_element_index_next] == node_index_next) {
                                element_to_element[i][j] = element_index;
                                goto endloop; // I hate this too don't worry
                            }
                        }
                    }
                }
            }
            endloop: ;
        }
    }
 
    return element_to_element;
}

auto SEM::Meshes::Mesh2D_t::build_faces(size_t n_nodes, std::vector<SEM::Entities::Element2D_t>& elements) -> std::pair<std::vector<SEM::Entities::Face2D_t>, std::vector<std::vector<size_t>>> {
    size_t total_edges = 0;
    for (const auto& element: elements) {
        total_edges += element.nodes_.size();
    }

    std::vector<SEM::Entities::Face2D_t> faces;
    faces.reserve(total_edges/2); // This is not exact

    std::vector<std::vector<size_t>> node_to_face(n_nodes);

    for (size_t i = 0; i < elements.size(); ++i) {
        for (size_t j = 0; j < elements[i].nodes_.size(); ++j) {
            std::array<size_t, 2> nodes{elements[i].nodes_[j], (j < elements[i].nodes_.size() - 1) ? elements[i].nodes_[j + 1] : elements[i].nodes_[0]};
            bool found = false;
            for (auto face_index: node_to_face[nodes[0]]) {
                if (faces[face_index].nodes_[0] == nodes[1] || faces[face_index].nodes_[1] == nodes[1] ) {
                    found = true;
                    faces[face_index].elements_[1] = i;
                    elements[i].faces_[j] = face_index;
                    break;
                }
            }

            if (!found) {
                elements[i].faces_[j] = faces.size();
                node_to_face[nodes[0]].push_back(faces.size());
                node_to_face[nodes[1]].push_back(faces.size());
                faces.push_back(SEM::Entities::Face2D_t({nodes[0], nodes[1]}, {i, static_cast<size_t>(-1)}));
            }
        }
    }

    return std::make_pair(faces, node_to_face);
}

auto SEM::Meshes::Mesh2D_t::set_initial_conditions(const deviceFloat* nodes) -> void {

}

auto SEM::Meshes::Mesh2D_t::print() -> void {
    
}

auto SEM::Meshes::Mesh2D_t::write_file_data(size_t N_interpolation_points, size_t N_elements, deviceFloat time, int rank, const std::vector<deviceFloat>& coordinates, const std::vector<deviceFloat>& velocity, const std::vector<deviceFloat>& du_dx, const std::vector<deviceFloat>& intermediate, const std::vector<deviceFloat>& x_L, const std::vector<deviceFloat>& x_R, const std::vector<int>& N, const std::vector<deviceFloat>& sigma, const bool* refine, const bool* coarsen, const std::vector<deviceFloat>& error) -> void {
    
}

auto SEM::Meshes::Mesh2D_t::write_data(deviceFloat time, size_t N_interpolation_points, const deviceFloat* interpolation_matrices) -> void {

}

template auto SEM::Meshes::Mesh2D_t::solve(const deviceFloat delta_t, const std::vector<deviceFloat> output_times, const SEM::Entities::NDG_t<SEM::Polynomials::ChebyshevPolynomial_t> &NDG, deviceFloat viscosity) -> void; // Get with the times c++, it's crazy I have to do this
template auto SEM::Meshes::Mesh2D_t::solve(const deviceFloat delta_t, const std::vector<deviceFloat> output_times, const SEM::Entities::NDG_t<SEM::Polynomials::LegendrePolynomial_t> &NDG, deviceFloat viscosity) -> void;

template<typename Polynomial>
auto SEM::Meshes::Mesh2D_t::solve(const deviceFloat CFL, const std::vector<deviceFloat> output_times, const SEM::Entities::NDG_t<Polynomial> &NDG, deviceFloat viscosity) -> void {
    
}

auto SEM::Meshes::Mesh2D_t::get_delta_t(const deviceFloat CFL) -> deviceFloat {   
    return 0.0;
}

auto SEM::Meshes::Mesh2D_t::adapt(int N_max, const deviceFloat* nodes, const deviceFloat* barycentric_weights) -> void {
    
}

auto SEM::Meshes::Mesh2D_t::boundary_conditions() -> void {
    
}
