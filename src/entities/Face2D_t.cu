#include "hip/hip_runtime.h"
#include "entities/Face2D_t.cuh"
#include <utility>

__device__ 
SEM::Entities::Face2D_t::Face2D_t(int N, std::array<size_t, 2> nodes, std::array<size_t, 2> elements, std::array<size_t, 2> elements_side) : 
        N_{N},
        nodes_{nodes},
        elements_{elements},
        elements_side_{elements_side},
        normal_{0, 0},
        tangent_{0, 0},
        length_{0},
        offset_{0.0, 0.0},
        scale_{0.0, 0.0},
        p_{N_ + 1, N_ + 1},
        u_{N_ + 1, N_ + 1},
        v_{N_ + 1, N_ + 1},
        p_flux_{N_ + 1},
        u_flux_{N_ + 1},
        v_flux_{N_ + 1} {}

__host__ __device__
SEM::Entities::Face2D_t::Face2D_t() :
        N_{0},
        nodes_{0, 0},
        elements_{0, 0},
        elements_side_{0, 0},
        normal_{0, 0},
        tangent_{0, 0},
        length_{0},
        offset_{0.0, 0.0},
        scale_{0.0, 0.0},
        p_{},
        u_{},
        v_{} {}

__device__
auto SEM::Entities::Face2D_t::allocate_storage() -> void {
    p_ = {cuda_vector<deviceFloat>(N_ + 1), cuda_vector<deviceFloat>(N_ + 1)};
    u_ = {cuda_vector<deviceFloat>(N_ + 1), cuda_vector<deviceFloat>(N_ + 1)};
    v_ = {cuda_vector<deviceFloat>(N_ + 1), cuda_vector<deviceFloat>(N_ + 1)};
    p_flux_ = cuda_vector<deviceFloat>(N_ + 1);
    u_flux_ = cuda_vector<deviceFloat>(N_ + 1);
    v_flux_ = cuda_vector<deviceFloat>(N_ + 1);
}

__device__
auto SEM::Entities::Face2D_t::resize_storage(int N) -> void {
    N_ = N;
    p_ = {cuda_vector<deviceFloat>(N_ + 1), cuda_vector<deviceFloat>(N_ + 1)};
    u_ = {cuda_vector<deviceFloat>(N_ + 1), cuda_vector<deviceFloat>(N_ + 1)};
    v_ = {cuda_vector<deviceFloat>(N_ + 1), cuda_vector<deviceFloat>(N_ + 1)};
    p_flux_ = cuda_vector<deviceFloat>(N_ + 1);
    u_flux_ = cuda_vector<deviceFloat>(N_ + 1);
    v_flux_ = cuda_vector<deviceFloat>(N_ + 1);
}

__device__
auto SEM::Entities::Face2D_t::compute_geometry(const SEM::Entities::Element2D_t* elements, const SEM::Entities::Vec2<deviceFloat>* nodes) -> void {
    tangent_ = nodes[nodes_[1]] - nodes[nodes_[0]]; 
    length_ = tangent_.magnitude();
    tangent_ /= length_; // CHECK should be normalized or not?
    normal_ = SEM::Entities::Vec2<deviceFloat>(tangent_.y(), -tangent_.x());         

    const SEM::Entities::Vec2<deviceFloat> center = (nodes[nodes_[0]] + nodes[nodes_[1]])/2;
    const SEM::Entities::Vec2<deviceFloat> delta = center - elements[elements_[0]].center_; // CHECK doesn't work with ghost cells
    const deviceFloat sign = std::copysign(static_cast<deviceFloat>(1.0), normal_.dot(delta));
    normal_ *= sign;
    tangent_ *= sign;

    offset_ = {0.0, 0.0}; // CHECK change
    scale_ = {1.0, 1.0};
}
