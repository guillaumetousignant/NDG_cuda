#include "hip/hip_runtime.h"
#include "entities/Face2D_t.cuh"
#include <utility>

__device__ 
SEM::Entities::Face2D_t::Face2D_t(int N, std::array<size_t, 2> nodes, std::array<size_t, 2> elements, std::array<size_t, 2> elements_side) : 
        N_{N},
        nodes_{nodes},
        elements_{elements},
        elements_side_{elements_side},
        normal_{0, 0},
        tangent_{0, 0},
        length_{0},
        offset_{0.0, 0.0},
        scale_{0.0, 0.0},
        refine_{false},
        p_{N_ + 1, N_ + 1},
        u_{N_ + 1, N_ + 1},
        v_{N_ + 1, N_ + 1},
        p_flux_{N_ + 1},
        u_flux_{N_ + 1},
        v_flux_{N_ + 1} {}

__host__ __device__
SEM::Entities::Face2D_t::Face2D_t() :
        N_{0},
        nodes_{0, 0},
        elements_{0, 0},
        elements_side_{0, 0},
        normal_{0, 0},
        tangent_{0, 0},
        length_{0},
        offset_{0.0, 0.0},
        scale_{0.0, 0.0},
        refine_{false},
        p_{},
        u_{},
        v_{} {}

__device__
auto SEM::Entities::Face2D_t::allocate_storage() -> void {
    p_ = {cuda_vector<deviceFloat>(N_ + 1), cuda_vector<deviceFloat>(N_ + 1)};
    u_ = {cuda_vector<deviceFloat>(N_ + 1), cuda_vector<deviceFloat>(N_ + 1)};
    v_ = {cuda_vector<deviceFloat>(N_ + 1), cuda_vector<deviceFloat>(N_ + 1)};
    p_flux_ = cuda_vector<deviceFloat>(N_ + 1);
    u_flux_ = cuda_vector<deviceFloat>(N_ + 1);
    v_flux_ = cuda_vector<deviceFloat>(N_ + 1);
}

__device__
auto SEM::Entities::Face2D_t::resize_storage(int N) -> void {
    N_ = N;
    p_ = {cuda_vector<deviceFloat>(N_ + 1), cuda_vector<deviceFloat>(N_ + 1)};
    u_ = {cuda_vector<deviceFloat>(N_ + 1), cuda_vector<deviceFloat>(N_ + 1)};
    v_ = {cuda_vector<deviceFloat>(N_ + 1), cuda_vector<deviceFloat>(N_ + 1)};
    p_flux_ = cuda_vector<deviceFloat>(N_ + 1);
    u_flux_ = cuda_vector<deviceFloat>(N_ + 1);
    v_flux_ = cuda_vector<deviceFloat>(N_ + 1);
}

__device__
auto SEM::Entities::Face2D_t::compute_geometry(const std::array<SEM::Entities::Vec2<deviceFloat>, 2>& elements_centres, const std::array<SEM::Entities::Vec2<deviceFloat>, 2>& nodes, const std::array<std::array<SEM::Entities::Vec2<deviceFloat>, 2>, 2>& element_nodes) -> void {
    const std::array<SEM::Entities::Vec2<deviceFloat>, 2> points {nodes[nodes_[0]], nodes[nodes_[1]]};
    
    tangent_ = nodes[1] - nodes[0]; 
    length_ = tangent_.magnitude();
    tangent_ /= length_; // CHECK should be normalized or not?
    normal_ = SEM::Entities::Vec2<deviceFloat>(tangent_.y(), -tangent_.x());     

    const SEM::Entities::Vec2<deviceFloat> center = (nodes[0] + nodes[1])/2;
    const SEM::Entities::Vec2<deviceFloat> delta = center - elements_centres[0]; // CHECK doesn't work with ghost cells
    const deviceFloat sign = std::copysign(static_cast<deviceFloat>(1), normal_.dot(delta));
    normal_ *= sign;
    tangent_ *= sign;

    const std::array<SEM::Entities::Vec2<deviceFloat>, 2> elements_delta {
        element_nodes[0][1] - element_nodes[0][0], 
        element_nodes[1][1] - element_nodes[1][0]
    };

    const SEM::Entities::Vec2<deviceFloat> face_delta = nodes[1] - nodes[0];

    const std::array<SEM::Entities::Vec2<deviceFloat>, 2> edge_delta {
        nodes[0] - element_nodes[0][0], 
        nodes[1] - element_nodes[1][0]
    };

    offset_ = {edge_delta[0].magnitude()/elements_delta[0].magnitude(), edge_delta[1].magnitude()/elements_delta[1].magnitude()};
    scale_ = {face_delta.magnitude()/elements_delta[0].magnitude(), face_delta.magnitude()/elements_delta[1].magnitude()};
}
