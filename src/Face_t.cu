#include "hip/hip_runtime.h"
#include "Face_t.cuh"
#include <utility>

__device__ 
Face_t::Face_t(size_t element_L, size_t element_R) : elements_{element_L, element_R} {};

__device__
Face_t::Face_t(const Face_t& other) :
        elements_{other.elements_[0], other.elements_[1]},
        flux_{other.flux_},
        derivative_flux_{other.derivative_flux_} {}

__device__
Face_t::Face_t(Face_t&& other) :
        elements_{other.elements_[0], other.elements_[1]},
        flux_{other.flux_},
        derivative_flux_{other.derivative_flux_} {}

__device__
Face_t& Face_t::operator=(const Face_t& other) {
    elements_[0] = other.elements_[0];
    elements_[1] = other.elements_[1];
    flux_ = other.flux_;
    derivative_flux_ = other.derivative_flux_;

    return *this;
}

__device__
Face_t& Face_t::operator=(Face_t&& other) {
    elements_[0] = other.elements_[0];
    elements_[1] = other.elements_[1];
    flux_ = other.flux_;
    derivative_flux_ = other.derivative_flux_;

    return *this;
}

__host__
Face_t::Face_t() {}

__host__ __device__
Face_t::~Face_t() {}

__global__
void SEM::build_faces(size_t N_faces, Face_t* faces) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_faces; i += stride) {
        const size_t neighbour_L = i;
        const size_t neighbour_R = (i < N_faces - 1) ? i + 1 : 0; // Last face links last element to first element
        faces[i] = Face_t(neighbour_L, neighbour_R);
    }
}

__global__
void SEM::copy_faces(size_t N_faces, const Face_t* faces, Face_t* new_faces) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_faces; i += stride) {
        new_faces[i] = std::move(faces[i]);
    }
}