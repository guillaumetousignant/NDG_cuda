#include "hip/hip_runtime.h"
#include "Mesh_t.cuh"
#include "ChebyshevPolynomial_t.cuh"
#include "LegendrePolynomial_t.cuh"
#include "ProgressBar_t.h"
#include <iostream>
#include <fstream>
#include <sstream> 
#include <iomanip>
#include <filesystem>
#include <limits>

namespace fs = std::filesystem;

SEM::Mesh_t::Mesh_t(size_t N_elements, int initial_N, deviceFloat x_min, deviceFloat x_max, hipStream_t &stream) : 
        N_elements_global_(N_elements),        
        stream_(stream) {

    int global_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &global_rank);
    int global_size;
    MPI_Comm_size(MPI_COMM_WORLD, &global_size);

    N_elements_per_process_ = (N_elements_global_ + global_size - 1)/global_size;
    N_elements_ = (global_rank == global_size - 1) ? N_elements_per_process_ + N_elements_global_ - N_elements_per_process_ * global_size : N_elements_per_process_;
    if (N_elements_ == N_elements_global_) {
        N_local_boundaries_ = 2;
        N_MPI_boundaries_ = 0;
    }
    else {
        N_local_boundaries_ = 0;
        N_MPI_boundaries_ = 2;
    }

    N_faces_ = N_elements_ + N_local_boundaries_ + N_MPI_boundaries_ - 1; 
    global_element_offset_ = global_rank * N_elements_per_process_;
    initial_N_ = initial_N;
    elements_numBlocks_ = (N_elements_ + elements_blockSize_ - 1) / elements_blockSize_;
    faces_numBlocks_ = (N_faces_ + faces_blockSize_ - 1) / faces_blockSize_;
    boundaries_numBlocks_ = (N_local_boundaries_ + N_MPI_boundaries_ + boundaries_blockSize_ - 1) / boundaries_blockSize_;

    host_delta_t_array_ = std::vector<deviceFloat>(elements_numBlocks_);
    host_refine_array_ = std::vector<unsigned long>(elements_numBlocks_);
    host_boundary_phi_L_ = std::vector<deviceFloat>(N_MPI_boundaries_);
    host_boundary_phi_R_ = std::vector<deviceFloat>(N_MPI_boundaries_);
    host_boundary_phi_prime_L_ = std::vector<deviceFloat>(N_MPI_boundaries_);
    host_boundary_phi_prime_R_ = std::vector<deviceFloat>(N_MPI_boundaries_);
    host_MPI_boundary_to_element_ = std::vector<size_t>(N_MPI_boundaries_);
    host_MPI_boundary_from_element_ = std::vector<size_t>(N_MPI_boundaries_);
    send_buffers_ = std::vector<std::array<double, 4>>(N_MPI_boundaries_);
    receive_buffers_ = std::vector<std::array<double, 4>>(N_MPI_boundaries_);
    requests_ = std::vector<MPI_Request>(N_MPI_boundaries_*2);
    statuses_ = std::vector<MPI_Status>(N_MPI_boundaries_*2);

    hipMalloc(&elements_, (N_elements_ + N_local_boundaries_ + N_MPI_boundaries_) * sizeof(Element_t));
    hipMalloc(&faces_, N_faces_ * sizeof(Face_t));
    hipMalloc(&local_boundary_to_element_, N_local_boundaries_ * sizeof(size_t));
    hipMalloc(&MPI_boundary_to_element_, N_MPI_boundaries_ * sizeof(size_t));
    hipMalloc(&MPI_boundary_from_element_, N_MPI_boundaries_ * sizeof(size_t));
    hipMalloc(&device_delta_t_array_, elements_numBlocks_ * sizeof(deviceFloat));
    hipMalloc(&device_refine_array_, elements_numBlocks_ * sizeof(unsigned long));
    hipMalloc(&device_boundary_phi_L_, N_MPI_boundaries_ * sizeof(deviceFloat));
    hipMalloc(&device_boundary_phi_R_, N_MPI_boundaries_ * sizeof(deviceFloat));
    hipMalloc(&device_boundary_phi_prime_L_, N_MPI_boundaries_ * sizeof(deviceFloat));
    hipMalloc(&device_boundary_phi_prime_R_, N_MPI_boundaries_ * sizeof(deviceFloat));

    const deviceFloat delta_x = (x_max - x_min)/N_elements_global_;
    const deviceFloat x_min_local = x_min + delta_x * global_rank * N_elements_per_process_;
    const deviceFloat x_max_local = x_min_local + N_elements_ * delta_x;

    SEM::build_elements<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, initial_N_, elements_, x_min_local, x_max_local);
    SEM::build_boundaries<<<boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(N_elements_, N_elements_global_, N_local_boundaries_, N_MPI_boundaries_, elements_, global_element_offset_, local_boundary_to_element_, MPI_boundary_to_element_, MPI_boundary_from_element_);
    SEM::build_faces<<<faces_numBlocks_, faces_blockSize_, 0, stream_>>>(N_faces_, faces_);

    hipMemcpy(host_MPI_boundary_to_element_.data(), MPI_boundary_to_element_, N_MPI_boundaries_ * sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(host_MPI_boundary_from_element_.data(), MPI_boundary_from_element_, N_MPI_boundaries_ * sizeof(size_t), hipMemcpyDeviceToHost);
}

SEM::Mesh_t::~Mesh_t() {
    SEM::free_elements<<<elements_numBlocks_, elements_blockSize_>>>(N_elements_ + N_local_boundaries_ + N_MPI_boundaries_, elements_);
    hipFree(elements_);
    hipFree(faces_);
    hipFree(local_boundary_to_element_);
    hipFree(MPI_boundary_to_element_);
    hipFree(MPI_boundary_from_element_);
    hipFree(device_delta_t_array_);
    hipFree(device_refine_array_);
    hipFree(device_boundary_phi_L_);
    hipFree(device_boundary_phi_R_);
    hipFree(device_boundary_phi_prime_L_);
    hipFree(device_boundary_phi_prime_R_);
}

void SEM::Mesh_t::set_initial_conditions(const deviceFloat* nodes) {
    SEM::initial_conditions<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, elements_, nodes);
}

void SEM::Mesh_t::print() {
    std::vector<Face_t> host_faces(N_faces_);
    std::vector<Element_t> host_elements(N_elements_ + N_local_boundaries_ + N_MPI_boundaries_);
    std::vector<size_t> host_local_boundary_to_element(N_local_boundaries_);

    hipMemcpy(host_faces.data(), faces_, N_faces_ * sizeof(Face_t), hipMemcpyDeviceToHost);
    hipMemcpy(host_elements.data(), elements_, (N_elements_ + N_local_boundaries_ + N_MPI_boundaries_) * sizeof(Element_t), hipMemcpyDeviceToHost);
    hipMemcpy(host_local_boundary_to_element.data(), local_boundary_to_element_, N_local_boundaries_ * sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(host_MPI_boundary_to_element_.data(), MPI_boundary_to_element_, N_MPI_boundaries_ * sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(host_MPI_boundary_from_element_.data(), MPI_boundary_from_element_, N_MPI_boundaries_ * sizeof(size_t), hipMemcpyDeviceToHost);

    // Invalidate GPU pointers, or else they will be deleted on the CPU, where they point to random stuff
    for (size_t i = 0; i < N_elements_ + N_local_boundaries_ + N_MPI_boundaries_; ++i) {
        host_elements[i].phi_ = nullptr;
        host_elements[i].q_ = nullptr;
        host_elements[i].ux_ = nullptr;
        host_elements[i].phi_prime_ = nullptr;
        host_elements[i].intermediate_ = nullptr;
    }

    std::cout << "N elements global: " << N_elements_global_ << std::endl;
    std::cout << "N elements local: " << N_elements_ << std::endl;
    std::cout << "N faces: " << N_faces_ << std::endl;
    std::cout << "N local boundaries: " << N_local_boundaries_ << std::endl;
    std::cout << "N MPI boundaries: " << N_MPI_boundaries_ << std::endl;
    std::cout << "Global element offset: " << global_element_offset_ << std::endl;
    std::cout << "Number of elements per process: " << N_elements_per_process_ << std::endl;
    std::cout << "Initial N: " << initial_N_ << std::endl;

    std::cout << std::endl << "Phi interpolated: " << std::endl;
    for (size_t i = 0; i < N_elements_ + N_local_boundaries_ + N_MPI_boundaries_; ++i) {
        std::cout << '\t' << "Element " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_elements[i].phi_L_ << " ";
        std::cout << host_elements[i].phi_R_;
        std::cout << std::endl;
    }

    std::cout << std::endl << "Phi prime interpolated: " << std::endl;
    for (size_t i = 0; i < N_elements_ + N_local_boundaries_ + N_MPI_boundaries_; ++i) {
        std::cout << '\t' << "Element " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_elements[i].phi_prime_L_ << " ";
        std::cout << host_elements[i].phi_prime_R_;
        std::cout << std::endl;
    }

    std::cout << std::endl << "x: " << std::endl;
    for (size_t i = 0; i < N_elements_ + N_local_boundaries_ + N_MPI_boundaries_; ++i) {
        std::cout << '\t' << "Element " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_elements[i].x_[0] << " ";
        std::cout << host_elements[i].x_[1];
        std::cout << std::endl;
    }

    std::cout << std::endl << "Neighbouring faces: " << std::endl;
    for (size_t i = 0; i < N_elements_ + N_local_boundaries_ + N_MPI_boundaries_; ++i) {
        std::cout << '\t' << "Element " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_elements[i].faces_[0] << " ";
        std::cout << host_elements[i].faces_[1];
        std::cout << std::endl;
    }

    std::cout << std::endl << "N: " << std::endl;
    for (size_t i = 0; i < N_elements_ + N_local_boundaries_ + N_MPI_boundaries_; ++i) {
        std::cout << '\t' << "Element " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_elements[i].N_;
        std::cout << std::endl;
    }

    std::cout << std::endl << "delta x: " << std::endl;
    for (size_t i = 0; i < N_elements_ + N_local_boundaries_ + N_MPI_boundaries_; ++i) {
        std::cout << '\t' << "Element " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_elements[i].delta_x_;
        std::cout << std::endl;
    }

    std::cout << std::endl << "Fluxes: " << std::endl;
    for (size_t i = 0; i < N_faces_; ++i) {
        std::cout << '\t' << "Face " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_faces[i].flux_ << std::endl;
    }

    std::cout << std::endl << "Derivative fluxes: " << std::endl;
    for (size_t i = 0; i < N_faces_; ++i) {
        std::cout << '\t' << "Face " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_faces[i].derivative_flux_ << std::endl;
    }

    std::cout << std::endl << "Non linear fluxes: " << std::endl;
    for (size_t i = 0; i < N_faces_; ++i) {
        std::cout << '\t' << "Face " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_faces[i].nl_flux_ << std::endl;
    }

    std::cout << std::endl << "Elements: " << std::endl;
    for (size_t i = 0; i < N_faces_; ++i) {
        std::cout << '\t' << "Face " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_faces[i].elements_[0] << " ";
        std::cout << host_faces[i].elements_[1] << std::endl;
    }

    std::cout << std::endl << "Local boundaries elements: " << std::endl;
    for (size_t i = 0; i < N_local_boundaries_; ++i) {
        std::cout << '\t' << "Local boundary " << i << ": ";
        std::cout << '\t';
        std::cout << host_local_boundary_to_element[i] << std::endl;
    }

    std::cout << std::endl << "MPI boundaries to elements: " << std::endl;
    for (size_t i = 0; i < N_MPI_boundaries_; ++i) {
        std::cout << '\t' << "MPI boundary " << i << ": ";
        std::cout << '\t';
        std::cout << host_MPI_boundary_to_element_[N_local_boundaries_ + i] << std::endl;
    }

    std::cout << std::endl << "MPI boundaries from elements: " << std::endl;
    for (size_t i = 0; i < N_MPI_boundaries_; ++i) {
        std::cout << '\t' << "MPI boundary " << i << ": ";
        std::cout << '\t';
        std::cout << host_MPI_boundary_from_element_[N_local_boundaries_ + i] << std::endl;
    }
}

void SEM::Mesh_t::write_file_data(size_t N_interpolation_points, size_t N_elements, deviceFloat time, int rank, const std::vector<deviceFloat>& coordinates, const std::vector<deviceFloat>& velocity, const std::vector<deviceFloat>& du_dx, const std::vector<deviceFloat>& intermediate, const std::vector<deviceFloat>& x_L, const std::vector<deviceFloat>& x_R, const std::vector<int>& N, const std::vector<deviceFloat>& sigma, const bool* refine, const bool* coarsen, const std::vector<deviceFloat>& error) {
    fs::path save_dir = fs::current_path() / "data";
    fs::create_directory(save_dir);

    std::stringstream ss;
    std::ofstream file;
    ss << "output_t" << std::setprecision(9) << std::fixed << time << "_proc" << std::setfill('0') << std::setw(6) << rank << ".dat";
    file.open(save_dir / ss.str());

    file << "TITLE = \"Velocity at t= " << time << "\"" << std::endl;
    file << "VARIABLES = \"X\", \"U_x\", \"U_x_prime\", \"intermediate\"" << std::endl;

    for (size_t i = 0; i < N_elements; ++i) {
        file << "ZONE T= \"Zone " << i + 1 << "\",  I= " << N_interpolation_points << ",  J= 1,  DATAPACKING = POINT, SOLUTIONTIME = " << time << std::endl;

        for (size_t j = 0; j < N_interpolation_points; ++j) {
            file       << std::setw(12) << coordinates[i*N_interpolation_points + j] 
                << " " << std::setw(12) << velocity[i*N_interpolation_points + j]
                << " " << std::setw(12) << du_dx[i*N_interpolation_points + j]
                << " " << std::setw(12) << intermediate[i*N_interpolation_points + j] << std::endl;
        }
    }

    file.close();

    std::stringstream ss_element;
    std::ofstream file_element;
    ss_element << "output_element_t" << std::setprecision(9) << std::fixed << time << "_proc" << std::setfill('0') << std::setw(6) << rank << ".dat";
    file_element.open(save_dir / ss_element.str());

    file_element << "TITLE = \"Element values at t= " << time << "\"" << std::endl
                 << "VARIABLES = \"X\", \"X_L\", \"X_R\", \"N\", \"sigma\", \"refine\", \"coarsen\", \"error\"" << std::endl
                 << "ZONE T= \"Zone     1\",  I= " << N_elements << ",  J= 1,  DATAPACKING = POINT, SOLUTIONTIME = " << time << std::endl;

    for (size_t j = 0; j < N_elements; ++j) {
        file_element << std::setw(12) << (x_L[j] + x_R[j]) * 0.5
              << " " << std::setw(12) << x_L[j]
              << " " << std::setw(12) << x_R[j]
              << " " << std::setw(12) << N[j]
              << " " << std::setw(12) << sigma[j]
              << " " << std::setw(12) << refine[j]
              << " " << std::setw(12) << coarsen[j]
              << " " << std::setw(12) << error[j] << std::endl;
    }

    file_element.close();
}

void SEM::Mesh_t::write_data(deviceFloat time, size_t N_interpolation_points, const deviceFloat* interpolation_matrices) {
    deviceFloat* x;
    deviceFloat* phi;
    deviceFloat* phi_prime;
    deviceFloat* intermediate;
    deviceFloat* x_L;
    deviceFloat* x_R;
    int* N;
    deviceFloat* sigma;
    bool* refine;
    bool* coarsen;
    deviceFloat* error;
    std::vector<deviceFloat> host_x(N_elements_ * N_interpolation_points);
    std::vector<deviceFloat> host_phi(N_elements_ * N_interpolation_points);
    std::vector<deviceFloat> host_phi_prime(N_elements_ * N_interpolation_points);
    std::vector<deviceFloat> host_intermediate(N_elements_ * N_interpolation_points);
    std::vector<deviceFloat> host_x_L(N_elements_);
    std::vector<deviceFloat> host_x_R(N_elements_);
    std::vector<int> host_N(N_elements_);
    std::vector<deviceFloat> host_sigma(N_elements_);
    bool* host_refine = new bool[N_elements_]; // Vectors of bools can be messed-up by some implementations
    bool* host_coarsen = new bool[N_elements_]; // Like they won't be an array of bools but packed in integers, in which case getting them from Cuda will fail.
    std::vector<deviceFloat> host_error(N_elements_);
    hipMalloc(&x, N_elements_ * N_interpolation_points * sizeof(deviceFloat));
    hipMalloc(&phi, N_elements_ * N_interpolation_points * sizeof(deviceFloat));
    hipMalloc(&phi_prime, N_elements_ * N_interpolation_points * sizeof(deviceFloat));
    hipMalloc(&intermediate, N_elements_ * N_interpolation_points * sizeof(deviceFloat));
    hipMalloc(&x_L, N_elements_ * sizeof(deviceFloat));
    hipMalloc(&x_R, N_elements_ * sizeof(deviceFloat));
    hipMalloc(&N, N_elements_ * sizeof(int));
    hipMalloc(&sigma, N_elements_ * sizeof(deviceFloat));
    hipMalloc(&refine, N_elements_ * sizeof(bool));
    hipMalloc(&coarsen, N_elements_ * sizeof(bool));
    hipMalloc(&error, N_elements_ * sizeof(deviceFloat));

    SEM::get_solution<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, N_interpolation_points, elements_, interpolation_matrices, x, phi, phi_prime, intermediate, x_L, x_R, N, sigma, refine, coarsen, error);
    
    hipMemcpy(host_x.data(), x , N_elements_ * N_interpolation_points * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_phi.data(), phi, N_elements_ * N_interpolation_points * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_phi_prime.data(), phi_prime, N_elements_ * N_interpolation_points * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_intermediate.data(), intermediate, N_elements_ * N_interpolation_points * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_x_L.data(), x_L, N_elements_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_x_R.data(), x_R, N_elements_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_N.data(), N, N_elements_ * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(host_sigma.data(), sigma, N_elements_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_refine, refine, N_elements_ * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(host_coarsen, coarsen, N_elements_ * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(host_error.data(), error, N_elements_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);

    int global_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &global_rank);
    
    write_file_data(N_interpolation_points, N_elements_, time, global_rank, host_x, host_phi, host_phi_prime, host_intermediate, host_x_L, host_x_R, host_N, host_sigma, host_refine, host_coarsen, host_error);

    delete[] host_refine;
    delete[] host_coarsen;
    hipFree(x);
    hipFree(phi);
    hipFree(phi_prime);
    hipFree(intermediate);
    hipFree(x_L);
    hipFree(x_R);
    hipFree(N);
    hipFree(sigma);
    hipFree(refine);
    hipFree(coarsen);
    hipFree(error);
}

template void SEM::Mesh_t::solve(const deviceFloat delta_t, const std::vector<deviceFloat> output_times, const NDG_t<ChebyshevPolynomial_t> &NDG, deviceFloat viscosity); // Get with the times c++, it's crazy I have to do this
template void SEM::Mesh_t::solve(const deviceFloat delta_t, const std::vector<deviceFloat> output_times, const NDG_t<LegendrePolynomial_t> &NDG, deviceFloat viscosity);

template<typename Polynomial>
void SEM::Mesh_t::solve(const deviceFloat CFL, const std::vector<deviceFloat> output_times, const NDG_t<Polynomial> &NDG, deviceFloat viscosity) {
    int global_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &global_rank);
    deviceFloat time = 0.0;
    const deviceFloat t_end = output_times.back();
    ProgressBar_t bar;
    size_t timestep = 0;

    deviceFloat delta_t = get_delta_t(CFL);
    write_data(time, NDG.N_interpolation_points_, NDG.interpolation_matrices_);
    if (global_rank == 0) {
        bar.update(0.0);
    }
    
    while (time < t_end) {
        ++timestep;
        delta_t = get_delta_t(CFL);
        if (time + delta_t > t_end) {
            delta_t = t_end - time;
        }

        // Kinda algorithm 62
        deviceFloat t = time;
        SEM::interpolate_to_boundaries<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, elements_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_, NDG.lagrange_interpolant_derivative_left_, NDG.lagrange_interpolant_derivative_right_);
        boundary_conditions();
        SEM::calculate_fluxes<<<faces_numBlocks_, faces_blockSize_, 0, stream_>>>(N_faces_, faces_, elements_);
        SEM::compute_dg_derivative<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(viscosity, N_elements_, elements_, faces_, NDG.weights_, NDG.derivative_matrices_hat_, NDG.g_hat_derivative_matrices_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        SEM::interpolate_q_to_boundaries<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, elements_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_, NDG.lagrange_interpolant_derivative_left_, NDG.lagrange_interpolant_derivative_right_);
        boundary_conditions();
        SEM::calculate_q_fluxes<<<faces_numBlocks_, faces_blockSize_, 0, stream_>>>(N_faces_, faces_, elements_);
        SEM::compute_dg_derivative2<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(viscosity, N_elements_, elements_, faces_, NDG.weights_, NDG.derivative_matrices_hat_, NDG.g_hat_derivative_matrices_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        SEM::rk3_first_step<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, elements_, delta_t, 1.0f/3.0f);

        t = time + 0.33333333333f * delta_t;
        SEM::interpolate_to_boundaries<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, elements_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_, NDG.lagrange_interpolant_derivative_left_, NDG.lagrange_interpolant_derivative_right_);
        boundary_conditions();
        SEM::calculate_fluxes<<<faces_numBlocks_, faces_blockSize_, 0, stream_>>>(N_faces_, faces_, elements_);
        SEM::compute_dg_derivative<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(viscosity, N_elements_, elements_, faces_, NDG.weights_, NDG.derivative_matrices_hat_, NDG.g_hat_derivative_matrices_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        SEM::interpolate_q_to_boundaries<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, elements_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_, NDG.lagrange_interpolant_derivative_left_, NDG.lagrange_interpolant_derivative_right_);
        boundary_conditions();
        SEM::calculate_q_fluxes<<<faces_numBlocks_, faces_blockSize_, 0, stream_>>>(N_faces_, faces_, elements_);
        SEM::compute_dg_derivative2<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(viscosity, N_elements_, elements_, faces_, NDG.weights_, NDG.derivative_matrices_hat_, NDG.g_hat_derivative_matrices_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        SEM::rk3_step<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, elements_, delta_t, -5.0f/9.0f, 15.0f/16.0f);

        t = time + 0.75f * delta_t;
        SEM::interpolate_to_boundaries<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, elements_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_, NDG.lagrange_interpolant_derivative_left_, NDG.lagrange_interpolant_derivative_right_);
        boundary_conditions();
        SEM::calculate_fluxes<<<faces_numBlocks_, faces_blockSize_, 0, stream_>>>(N_faces_, faces_, elements_);
        SEM::compute_dg_derivative<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(viscosity, N_elements_, elements_, faces_, NDG.weights_, NDG.derivative_matrices_hat_, NDG.g_hat_derivative_matrices_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        SEM::interpolate_q_to_boundaries<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, elements_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_, NDG.lagrange_interpolant_derivative_left_, NDG.lagrange_interpolant_derivative_right_);
        boundary_conditions();
        SEM::calculate_q_fluxes<<<faces_numBlocks_, faces_blockSize_, 0, stream_>>>(N_faces_, faces_, elements_);
        SEM::compute_dg_derivative2<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(viscosity, N_elements_, elements_, faces_, NDG.weights_, NDG.derivative_matrices_hat_, NDG.g_hat_derivative_matrices_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        SEM::rk3_step<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, elements_, delta_t, -153.0f/128.0f, 8.0f/15.0f);
              
        time += delta_t;
        if (global_rank == 0) {
            bar.update(time/t_end);
        }
        for (auto const& e : std::as_const(output_times)) {
            if ((time >= e) && (time < e + delta_t)) {
                SEM::estimate_error<Polynomial><<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, elements_, NDG.nodes_, NDG.weights_);
                write_data(time, NDG.N_interpolation_points_, NDG.interpolation_matrices_);
                break;
            }
        }

        if (timestep % 100 == 0) {
            SEM::estimate_error<Polynomial><<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, elements_, NDG.nodes_, NDG.weights_);
            adapt(NDG.N_max_, NDG.nodes_, NDG.barycentric_weights_);
        }
    }
    if (global_rank == 0) {
        std::cout << std::endl;
    }

    bool did_write = false;
    for (auto const& e : std::as_const(output_times)) {
        if ((time >= e) && (time < e + delta_t)) {
            did_write = true;
            break;
        }
    }

    if (!did_write) {
        SEM::estimate_error<Polynomial><<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, elements_, NDG.nodes_, NDG.weights_);
        write_data(time, NDG.N_interpolation_points_, NDG.interpolation_matrices_);
    }
}

deviceFloat SEM::Mesh_t::get_delta_t(const deviceFloat CFL) {   
    SEM::reduce_delta_t<elements_blockSize_/2><<<elements_numBlocks_, elements_blockSize_/2, 0, stream_>>>(CFL, N_elements_, elements_, device_delta_t_array_);
    hipMemcpy(host_delta_t_array_.data(), device_delta_t_array_, elements_numBlocks_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);

    double delta_t_min_local = std::numeric_limits<double>::infinity();
    for (int i = 0; i < elements_numBlocks_; ++i) {
        delta_t_min_local = min(delta_t_min_local, host_delta_t_array_[i]);
    }

    double delta_t_min;
    MPI_Allreduce(&delta_t_min_local, &delta_t_min, 1, MPI_DOUBLE, MPI_MIN, MPI_COMM_WORLD);

    return delta_t_min;
}

void SEM::Mesh_t::adapt(int N_max, const deviceFloat* nodes, const deviceFloat* barycentric_weights) {
    // CHECK needs to rebuild boundaries
    SEM::reduce_refine<elements_blockSize_/2><<<elements_numBlocks_, elements_blockSize_/2, 0, stream_>>>(N_elements_, elements_, device_refine_array_);
    hipMemcpy(host_refine_array_.data(), device_refine_array_, elements_numBlocks_ * sizeof(unsigned long), hipMemcpyDeviceToHost);

    unsigned long additional_elements = 0;
    for (int i = 0; i < elements_numBlocks_; ++i) {
        additional_elements += host_refine_array_[i];
        host_refine_array_[i] = additional_elements - host_refine_array_[i]; // Current block offset
    }

    int global_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &global_rank);
    int global_size;
    MPI_Comm_size(MPI_COMM_WORLD, &global_size);

    std::vector<unsigned long> additional_elements_global(global_size);
    MPI_Allgather(&additional_elements, 1, MPI_UNSIGNED_LONG, additional_elements_global.data(), 1, MPI_UNSIGNED_LONG, MPI_COMM_WORLD);

    size_t global_element_offset_current = global_element_offset_;
    for (int i = 0; i < global_rank; ++i) {
        global_element_offset_current += additional_elements_global[i];
    }
    for (int i = 0; i < global_size; ++i) {
        N_elements_global_ += additional_elements_global[i];
    }
    const size_t global_element_offset_end_current = global_element_offset_current + N_elements_ + additional_elements - 1;

    const size_t N_elements_per_process_old = N_elements_per_process_;
    N_elements_per_process_ = (N_elements_global_ + global_size - 1)/global_size;
    global_element_offset_ = global_rank * N_elements_per_process_;
    const size_t global_element_offset_end = min(global_element_offset_ + N_elements_per_process_ - 1, N_elements_global_ - 1);

    if ((additional_elements == 0) && (global_element_offset_ == global_element_offset_current) && (global_element_offset_end == global_element_offset_end_current)) {
        SEM::p_adapt<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, elements_, N_max, nodes, barycentric_weights);
        return;
    }

    hipMemcpy(device_refine_array_, host_refine_array_.data(), elements_numBlocks_ * sizeof(unsigned long), hipMemcpyHostToDevice);

    Element_t* new_elements;

    // CHECK N_faces = N_elements only for periodic BC.
    hipMalloc(&new_elements, (N_elements_ + additional_elements) * sizeof(Element_t));

    //SEM::copy_faces<<<faces_numBlocks_, faces_blockSize_, 0, stream_>>>(N_faces_, faces_, new_faces);
    //SEM::copy_boundaries<<boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>(N_elements_, N_elements_global_, N_local_boundaries_, N_MPI_boundaries_, additional_elements, elements_, new_elements, new_faces, global_element_offset_, local_boundary_to_element_, MPI_boundary_to_element_, MPI_boundary_from_element_);                                                                            
    SEM::hp_adapt<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, elements_, new_elements, device_refine_array_, N_max, nodes, barycentric_weights);

    SEM::free_elements<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_ + N_local_boundaries_ + N_MPI_boundaries_, elements_);
    hipFree(elements_);
    
    N_elements_ = (global_rank == global_size - 1) ? N_elements_per_process_ + N_elements_global_ - N_elements_per_process_ * global_size : N_elements_per_process_;
    N_faces_ = N_elements_ + N_local_boundaries_ + N_MPI_boundaries_ - 1; 
    elements_numBlocks_ = (N_elements_ + elements_blockSize_ - 1) / elements_blockSize_;
    faces_numBlocks_ = (N_faces_ + faces_blockSize_ - 1) / faces_blockSize_;
    boundaries_numBlocks_ = (N_local_boundaries_ + N_MPI_boundaries_ + boundaries_blockSize_ - 1) / boundaries_blockSize_;

    hipFree(faces_);
    hipMalloc(&faces_, N_faces_ * sizeof(Face_t));
    SEM::build_faces<<<faces_numBlocks_, faces_blockSize_, 0, stream_>>>(N_faces_, faces_);

    hipMalloc(&elements_, (N_elements_ + N_local_boundaries_ + N_MPI_boundaries_) * sizeof(Element_t));

    const size_t N_elements_send_left = std::max(global_element_offset_ - global_element_offset_current, 0ULL);
    const size_t N_elements_recv_left = std::max(global_element_offset_current - global_element_offset_, 0ULL);
    const size_t N_elements_send_right = std::max(global_element_offset_end_current - global_element_offset_end, 0ULL);
    const size_t N_elements_recv_right = std::max(global_element_offset_end - global_element_offset_end_current, 0ULL);

    if (N_elements_send_left + N_elements_recv_left + N_elements_send_right + N_elements_recv_right > 0) {
        std::vector<Element_t> elements_send_left(N_elements_send_left);
        std::vector<Element_t> elements_recv_left(N_elements_recv_left);
        std::vector<Element_t> elements_send_right(N_elements_send_right);
        std::vector<Element_t> elements_recv_right(N_elements_recv_right);

        hipMemcpy(elements_send_left.data(), new_elements, N_elements_send_left * sizeof(Element_t), hipMemcpyDeviceToHost);
        hipMemcpy(elements_send_right.data(), new_elements + N_elements_ - N_elements_send_right, N_elements_send_right * sizeof(Element_t), hipMemcpyDeviceToHost);

        for (auto& element: elements_send_left) {
            element.phi_ = nullptr; // Those are GPU pointers, deleting them would delete random memory
            element.q_ = nullptr;
            element.ux_ = nullptr;
            element.phi_prime_ = nullptr;
            element.intermediate_ = nullptr;
        }

        for (auto& element: elements_send_right) {
            element.phi_ = nullptr; // Those are GPU pointers, deleting them would delete random memory
            element.q_ = nullptr;
            element.ux_ = nullptr;
            element.phi_prime_ = nullptr;
            element.intermediate_ = nullptr;
        }

        std::vector<std::vector<deviceFloat>> phi_arrays_send_left(N_elements_send_left);
        std::vector<std::vector<deviceFloat>> phi_arrays_recv_left(N_elements_recv_left);
        std::vector<deviceFloat*> phi_arrays_send_left_host(N_elements_send_left);
        std::vector<deviceFloat*> phi_arrays_recv_left_host(N_elements_recv_left);
        for (int i = 0; i < N_elements_send_left; ++i) {
            phi_arrays_send_left[i] = std::vector<deviceFloat>(elements_send_left[i].N_ + 1);
            hipMalloc(&phi_arrays_send_left_host[i], (elements_send_left[i].N_ + 1) * sizeof(deviceFloat));
        }
        deviceFloat** phi_arrays_send_left_device;
        deviceFloat** phi_arrays_recv_left_device;
        hipMalloc(&phi_arrays_send_left_device, N_elements_send_left * sizeof(deviceFloat*)); 
        hipMalloc(&phi_arrays_recv_left_device, N_elements_recv_left * sizeof(deviceFloat*)); 
        hipMemcpy(phi_arrays_send_left_device, phi_arrays_send_left_host.data(), N_elements_send_left * sizeof(deviceFloat*), hipMemcpyHostToDevice);

        std::vector<std::vector<deviceFloat>> phi_arrays_send_right(N_elements_send_right);
        std::vector<std::vector<deviceFloat>> phi_arrays_recv_right(N_elements_recv_right);
        std::vector<deviceFloat*> phi_arrays_send_right_host(N_elements_send_right);
        std::vector<deviceFloat*> phi_arrays_recv_right_host(N_elements_recv_right);
        for (int i = 0; i < N_elements_send_right; ++i) {
            phi_arrays_send_right[i] = std::vector<deviceFloat>(elements_send_right[i].N_ + 1);
            hipMalloc(&phi_arrays_send_right_host[i], (elements_send_right[i].N_ + 1) * sizeof(deviceFloat));
        }
        deviceFloat** phi_arrays_send_right_device;
        deviceFloat** phi_arrays_recv_right_device;
        hipMalloc(&phi_arrays_send_right_device, N_elements_send_right * sizeof(deviceFloat*)); 
        hipMalloc(&phi_arrays_recv_right_device, N_elements_recv_right * sizeof(deviceFloat*)); 
        hipMemcpy(phi_arrays_send_right_device, phi_arrays_send_right_host.data(), N_elements_send_right * sizeof(deviceFloat*), hipMemcpyHostToDevice);

        SEM::get_phi<<<boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(N_elements_send_left, new_elements, phi_arrays_send_left_device);
        SEM::get_phi<<<boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(N_elements_send_right, new_elements + N_elements_ - N_elements_send_right, phi_arrays_send_right_device);

        for (int i = 0; i < N_elements_send_left; ++i) {
            hipMemcpy(phi_arrays_send_left[i].data(), phi_arrays_send_left_host[i], (elements_send_left[i].N_ + 1) * sizeof(deviceFloat), hipMemcpyDeviceToHost);
        }

        for (int i = 0; i < N_elements_send_right; ++i) {
            hipMemcpy(phi_arrays_send_right[i].data(), phi_arrays_send_right_host[i], (elements_send_right[i].N_ + 1) * sizeof(deviceFloat), hipMemcpyDeviceToHost);
        }

        for (int i = 0; i < N_elements_send_left; ++i) {
            hipFree(phi_arrays_send_left_host[i]);
        }
        hipFree(phi_arrays_send_left_device);

        for (int i = 0; i < N_elements_send_right; ++i) {
            hipFree(phi_arrays_send_right_host[i]);
        }
        hipFree(phi_arrays_send_right_device);

        std::vector<int> left_origins(N_elements_recv_left);
        std::vector<int> right_origins(N_elements_recv_right);
        for (int i = 0; i < N_elements_recv_left; ++i) {
            const int index = global_element_offset_ + i;
            int process_end_index = -1;
            for (int rank = 0; rank < global_rank; ++rank) { 
                process_end_index += N_elements_per_process_old + additional_elements_global[rank];
                if (process_end_index >= index) {
                    left_origins[i] = rank;
                    break;
                }
            }
        }
        for (int i = 0; i < N_elements_recv_right; ++i) {
            const int index = global_element_offset_end_current + i + 1;
            int process_start_index = 0;
            for (int rank = 1; rank < global_size; ++rank) { 
                process_start_index += N_elements_per_process_old + additional_elements_global[rank - 1];
                if (process_start_index >= index) {
                    right_origins[i] = rank;
                    break;
                }
            }
        }
        
        std::vector<MPI_Request> adaptivity_requests(3 * (N_elements_send_left + N_elements_recv_left + N_elements_send_right + N_elements_recv_right));
        std::vector<MPI_Status> adaptivity_statuses(3 * (N_elements_send_left + N_elements_recv_left));
        constexpr MPI_Datatype data_type = (sizeof(deviceFloat) == sizeof(float)) ? MPI_FLOAT : MPI_DOUBLE;

        for (int i = 0; i < N_elements_send_left; ++i) {
            const int index = global_element_offset_current + i;
            const int destination = index/N_elements_per_process_;

            MPI_Isend(&elements_send_left[i].N_, 1, MPI_INT, destination, 3 * index, MPI_COMM_WORLD, &adaptivity_requests[i + 3 * N_elements_recv_left + 3 * N_elements_recv_right]);
        }

        for (int i = 0; i < N_elements_send_right; ++i) {
            const int index = global_element_offset_end + 1 + i;
            const int destination = index/N_elements_per_process_;

            MPI_Isend(&elements_send_right[i].N_, 1, MPI_INT, destination, 3 * index, MPI_COMM_WORLD, &adaptivity_requests[i + 4 * N_elements_recv_left + 3 * N_elements_recv_right]);
        }

        for (int i = 0; i < N_elements_recv_left; ++i) {
            const int index = global_element_offset_ + i;

            MPI_Irecv(&elements_recv_left[i].N_, 1, MPI_INT, left_origins[i], 3 * index, MPI_COMM_WORLD, &adaptivity_requests[i]);
        }

        for (int i = 0; i < N_elements_recv_right; ++i) {
            const int index = global_element_offset_end_current + i + 1;

            MPI_Irecv(&elements_recv_right[i].N_, 1, MPI_INT, right_origins[i], 3 * index, MPI_COMM_WORLD, &adaptivity_requests[i + N_elements_recv_left]);
        }

        MPI_Waitall(N_elements_send_right + N_elements_recv_right, adaptivity_requests.data(), adaptivity_statuses.data());

        for (int i = 0; i < N_elements_recv_left; ++i) {
            hipMalloc(&phi_arrays_recv_left_host[i], (elements_recv_left[i].N_ + 1) * sizeof(deviceFloat));
            phi_arrays_recv_left[i] = std::vector<deviceFloat>(elements_recv_left[i].N_ + 1);
        }
        hipMemcpy(phi_arrays_recv_left_device, phi_arrays_recv_left_host.data(), N_elements_recv_left * sizeof(deviceFloat*), hipMemcpyHostToDevice);

        for (int i = 0; i < N_elements_recv_right; ++i) {
            hipMalloc(&phi_arrays_recv_right_host[i], (elements_recv_right[i].N_ + 1) * sizeof(deviceFloat));
            phi_arrays_recv_right[i] = std::vector<deviceFloat>(elements_recv_right[i].N_ + 1);
        }
        hipMemcpy(phi_arrays_recv_right_device, phi_arrays_recv_right_host.data(), N_elements_recv_right * sizeof(deviceFloat*), hipMemcpyHostToDevice);

        for (int i = 0; i < N_elements_send_left; ++i) {
            const int index = global_element_offset_current + i;
            const int destination = index/N_elements_per_process_;

            MPI_Isend(&elements_send_left[i].x_[0], 2, data_type, destination, 3 * index + 1, MPI_COMM_WORLD, &adaptivity_requests[i + 4 * N_elements_recv_left + 4 * N_elements_recv_right]);
            MPI_Isend(phi_arrays_send_left[i].data(), elements_send_left[i].N_ + 1, data_type, destination, 3 * index + 2, MPI_COMM_WORLD, &adaptivity_requests[i + 5 * N_elements_recv_left + 5 * N_elements_recv_right]);
        }

        for (int i = 0; i < N_elements_send_right; ++i) {
            const int index = global_element_offset_end + 1 + i;
            const int destination = index/N_elements_per_process_;

            MPI_Isend(&elements_send_right[i].x_[0], 2, data_type, destination, 3 * index + 1, MPI_COMM_WORLD, &adaptivity_requests[i + 5 * N_elements_recv_left + 4 * N_elements_recv_right]);
            MPI_Isend(phi_arrays_send_right[i].data(), elements_send_right[i].N_ + 1, data_type, destination, 3 * index + 2, MPI_COMM_WORLD, &adaptivity_requests[i + 6 * N_elements_recv_left + 5 * N_elements_recv_right]);
        }

        for (int i = 0; i < N_elements_recv_left; ++i) {
            const int index = global_element_offset_ + i;

            MPI_Irecv(&elements_recv_left[i].x_[0], 2, data_type, left_origins[i], 3 * index + 1, MPI_COMM_WORLD, &adaptivity_requests[i + N_elements_recv_left + N_elements_recv_right]);
            MPI_Irecv(phi_arrays_recv_left[i].data(), elements_recv_left[i].N_ + 1, data_type, left_origins[i], 3 * index + 2, MPI_COMM_WORLD, &adaptivity_requests[i + 2 * N_elements_recv_left + 2 * N_elements_recv_right]);
        }

        for (int i = 0; i < N_elements_recv_right; ++i) {
            const int index = global_element_offset_end_current + i + 1;

            MPI_Irecv(&elements_recv_right[i].x_[0], 2, data_type, right_origins[i], 3 * index + 1, MPI_COMM_WORLD, &adaptivity_requests[i + 2 * N_elements_recv_left + N_elements_recv_right]);
            MPI_Irecv(phi_arrays_recv_right[i].data(), elements_recv_right[i].N_ + 1, data_type, right_origins[i], 3 * index + 2, MPI_COMM_WORLD, &adaptivity_requests[i + 3 * N_elements_recv_left + 2 * N_elements_recv_right]);
        }

        MPI_Waitall(2 * N_elements_send_right + 2 * N_elements_recv_right, adaptivity_requests.data() + N_elements_send_right + N_elements_recv_right, adaptivity_statuses.data() + N_elements_send_left + N_elements_recv_left);

        for (int i = 0; i < N_elements_recv_left; ++i) {
            elements_recv_left[i].delta_x_ = elements_recv_left[i].x_[1] - elements_recv_left[i].x_[0];
        }

        for (int i = 0; i < N_elements_recv_right; ++i) {
            elements_recv_right[i].delta_x_ = elements_recv_right[i].x_[1] - elements_recv_right[i].x_[0];
        }

        for (int i = 0; i < N_elements_recv_left; ++i) {
            hipMemcpy(phi_arrays_recv_left_host[i], phi_arrays_recv_left[i].data(), (elements_recv_left[i].N_ + 1) * sizeof(deviceFloat), hipMemcpyHostToDevice);
        }

        for (int i = 0; i < N_elements_recv_right; ++i) {
            hipMemcpy(phi_arrays_recv_right_host[i], phi_arrays_recv_right[i].data(), (elements_recv_right[i].N_ + 1) * sizeof(deviceFloat), hipMemcpyHostToDevice);
        }

        hipMemcpy(elements_, elements_recv_left.data(), N_elements_recv_left * sizeof(Element_t), hipMemcpyHostToDevice);
        hipMemcpy(elements_ + N_elements_ - N_elements_recv_right, elements_recv_right.data(), N_elements_recv_right * sizeof(Element_t), hipMemcpyHostToDevice);
        SEM::put_phi<<<boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(N_elements_recv_left, elements_, phi_arrays_recv_left_device);
        SEM::put_phi<<<boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(N_elements_recv_right, elements_ + N_elements_ - N_elements_recv_right, phi_arrays_recv_right_device);

        for (int i = 0; i < N_elements_recv_left; ++i) {
            hipFree(phi_arrays_recv_left_host[i]);
        }
        hipFree(phi_arrays_recv_left_device);

        for (int i = 0; i < N_elements_recv_right; ++i) {
            hipFree(phi_arrays_recv_right_host[i]);
        }
        hipFree(phi_arrays_recv_right_device);
    }

    SEM::move_elements<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_ - N_elements_send_left - N_elements_send_right, new_elements + N_elements_send_left, elements_ + N_elements_recv_left);

    SEM::free_elements<<<elements_numBlocks_, elements_blockSize_, 0, stream_>>>(N_elements_, new_elements);
    hipFree(new_elements);

    host_delta_t_array_ = std::vector<deviceFloat>(elements_numBlocks_);
    host_refine_array_ = std::vector<unsigned long>(elements_numBlocks_);
    host_boundary_phi_L_ = std::vector<deviceFloat>(N_MPI_boundaries_);
    host_boundary_phi_R_ = std::vector<deviceFloat>(N_MPI_boundaries_);
    host_boundary_phi_prime_L_ = std::vector<deviceFloat>(N_MPI_boundaries_);
    host_boundary_phi_prime_R_ = std::vector<deviceFloat>(N_MPI_boundaries_);
    host_MPI_boundary_to_element_ = std::vector<size_t>(N_MPI_boundaries_);
    host_MPI_boundary_from_element_ = std::vector<size_t>(N_MPI_boundaries_);
    send_buffers_ = std::vector<std::array<double, 4>>(N_MPI_boundaries_);
    receive_buffers_ = std::vector<std::array<double, 4>>(N_MPI_boundaries_);
    requests_ = std::vector<MPI_Request>(N_MPI_boundaries_*2);
    statuses_ = std::vector<MPI_Status>(N_MPI_boundaries_*2);

    hipFree(local_boundary_to_element_);
    hipFree(MPI_boundary_to_element_);
    hipFree(MPI_boundary_from_element_);
    hipFree(device_delta_t_array_);
    hipFree(device_refine_array_);
    hipFree(device_boundary_phi_L_);
    hipFree(device_boundary_phi_R_);
    hipFree(device_boundary_phi_prime_L_);
    hipFree(device_boundary_phi_prime_R_);
    hipMalloc(&local_boundary_to_element_, N_local_boundaries_ * sizeof(deviceFloat));
    hipMalloc(&MPI_boundary_to_element_, N_MPI_boundaries_ * sizeof(size_t));
    hipMalloc(&MPI_boundary_from_element_, N_MPI_boundaries_ * sizeof(size_t));
    hipMalloc(&device_delta_t_array_, elements_numBlocks_ * sizeof(deviceFloat));
    hipMalloc(&device_refine_array_, elements_numBlocks_ * sizeof(unsigned long));
    hipMalloc(&device_boundary_phi_L_, N_MPI_boundaries_ * sizeof(deviceFloat));
    hipMalloc(&device_boundary_phi_R_, N_MPI_boundaries_ * sizeof(deviceFloat));
    hipMalloc(&device_boundary_phi_prime_L_, N_MPI_boundaries_ * sizeof(deviceFloat));
    hipMalloc(&device_boundary_phi_prime_R_, N_MPI_boundaries_ * sizeof(deviceFloat));

    SEM::build_boundaries<<<boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(N_elements_, N_elements_global_, N_local_boundaries_, N_MPI_boundaries_, elements_, global_element_offset_, local_boundary_to_element_, MPI_boundary_to_element_, MPI_boundary_from_element_);

    hipMemcpy(host_MPI_boundary_to_element_.data(), MPI_boundary_to_element_, N_MPI_boundaries_ * sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(host_MPI_boundary_from_element_.data(), MPI_boundary_from_element_, N_MPI_boundaries_ * sizeof(size_t), hipMemcpyDeviceToHost);
}

void SEM::Mesh_t::boundary_conditions() {
    SEM::local_boundaries<<<boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(N_elements_, N_local_boundaries_, elements_, local_boundary_to_element_);
    SEM::get_MPI_boundaries<<<boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(N_elements_, N_local_boundaries_, N_MPI_boundaries_, elements_, faces_, device_boundary_phi_L_, device_boundary_phi_R_, device_boundary_phi_prime_L_, device_boundary_phi_prime_R_);
    
    hipMemcpy(host_boundary_phi_L_.data(), device_boundary_phi_L_, N_MPI_boundaries_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_boundary_phi_R_.data(), device_boundary_phi_R_, N_MPI_boundaries_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_boundary_phi_prime_L_.data(), device_boundary_phi_prime_L_, N_MPI_boundaries_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);
    hipMemcpy(host_boundary_phi_prime_R_.data(), device_boundary_phi_prime_R_, N_MPI_boundaries_ * sizeof(deviceFloat), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < N_MPI_boundaries_; ++i) {
        send_buffers_[i] = {host_boundary_phi_L_[i],
                            host_boundary_phi_R_[i],
                            host_boundary_phi_prime_L_[i],
                            host_boundary_phi_prime_R_[i]};
        const int destination = host_MPI_boundary_to_element_[i]/N_elements_per_process_;

        MPI_Irecv(&receive_buffers_[i][0], 4, MPI_DOUBLE, destination, host_MPI_boundary_from_element_[i], MPI_COMM_WORLD, &requests_[i]);
        MPI_Isend(&send_buffers_[i][0], 4, MPI_DOUBLE, destination, host_MPI_boundary_to_element_[i], MPI_COMM_WORLD, &requests_[i + N_MPI_boundaries_]);
    }

    MPI_Waitall(N_MPI_boundaries_, requests_.data(), statuses_.data()); // CHECK maybe MPI barrier?

    for (size_t i = 0; i < N_MPI_boundaries_; ++i) {
        host_boundary_phi_L_[i] = receive_buffers_[i][0];
        host_boundary_phi_R_[i] = receive_buffers_[i][1];
        host_boundary_phi_prime_L_[i] = receive_buffers_[i][2];
        host_boundary_phi_prime_R_[i] = receive_buffers_[i][3];
    }

    hipMemcpy(device_boundary_phi_L_, host_boundary_phi_L_.data(), N_MPI_boundaries_ * sizeof(deviceFloat), hipMemcpyHostToDevice);
    hipMemcpy(device_boundary_phi_R_, host_boundary_phi_R_.data(), N_MPI_boundaries_ * sizeof(deviceFloat), hipMemcpyHostToDevice);
    hipMemcpy(device_boundary_phi_prime_L_, host_boundary_phi_prime_L_.data(), N_MPI_boundaries_ * sizeof(deviceFloat), hipMemcpyHostToDevice);
    hipMemcpy(device_boundary_phi_prime_R_, host_boundary_phi_prime_R_.data(), N_MPI_boundaries_ * sizeof(deviceFloat), hipMemcpyHostToDevice);

    SEM::put_MPI_boundaries<<<boundaries_numBlocks_, boundaries_blockSize_, 0, stream_>>>(N_elements_, N_local_boundaries_, N_MPI_boundaries_, elements_, device_boundary_phi_L_, device_boundary_phi_R_, device_boundary_phi_prime_L_, device_boundary_phi_prime_R_);
}

__global__
void SEM::rk3_first_step(size_t N_elements, Element_t* elements, deviceFloat delta_t, deviceFloat g) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_elements; i += stride) {
        for (int j = 0; j <= elements[i].N_; ++j){
            elements[i].intermediate_[j] = elements[i].phi_prime_[j];
            elements[i].phi_[j] += g * delta_t * elements[i].intermediate_[j];
        }
    }
}

__global__
void SEM::rk3_step(size_t N_elements, Element_t* elements, deviceFloat delta_t, deviceFloat a, deviceFloat g) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_elements; i += stride) {
        for (int j = 0; j <= elements[i].N_; ++j){
            elements[i].intermediate_[j] = a * elements[i].intermediate_[j] + elements[i].phi_prime_[j];
            elements[i].phi_[j] += g * delta_t * elements[i].intermediate_[j];
        }
    }
}

__global__
void SEM::calculate_fluxes(size_t N_faces, Face_t* faces, const Element_t* elements) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_faces; i += stride) {
        deviceFloat u;
        const deviceFloat u_left = elements[faces[i].elements_[0]].phi_R_;
        const deviceFloat u_right = elements[faces[i].elements_[1]].phi_L_;

        if (u_left < 0.0f && u_right > 0.0f) { // In expansion fan
            u = 0.5f * (u_left + u_right);
        }
        else if (u_left >= u_right) { // Shock
            if (u_left > 0.0f) {
                u = u_left;
            }
            else {
                u = u_right;
            }
        }
        else { // Expansion fan
            if (u_left > 0.0f) {
                u = u_left;
            }
            else  {
                u = u_right;
            }
        }
    
        faces[i].flux_ = u_right;
        faces[i].nl_flux_ = 0.5f * u * u;
    }
}

__global__
void SEM::calculate_q_fluxes(size_t N_faces, Face_t* faces, const Element_t* elements) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_faces; i += stride) {
        const deviceFloat u_prime_left = elements[faces[i].elements_[0]].phi_prime_R_;

        faces[i].derivative_flux_ = u_prime_left;
    }
}

__device__
void SEM::matrix_vector_multiply(int N, const deviceFloat* matrix, const deviceFloat* vector, deviceFloat* result) {
    for (int i = 0; i <= N; ++i) {
        result[i] = 0.0f;
        for (int j = 0; j <= N; ++j) {
            result[i] +=  matrix[i * (N + 1) + j] * vector[j];
        }
    }
}

// Algorithm 19
__device__
void SEM::matrix_vector_derivative(int N, const deviceFloat* derivative_matrices_hat, const deviceFloat* phi, deviceFloat* phi_prime) {
    // s = 0, e = N (p.55 says N - 1)
    for (int i = 0; i <= N; ++i) {
        phi_prime[i] = 0.0f;
        for (int j = 0; j <= N; ++j) {
            phi_prime[i] += derivative_matrices_hat[i * (N + 1) + j] * phi[j] * phi[j] * 0.5;
        }
    }
}

// Algorithm 60 (not really anymore)
__global__
void SEM::compute_dg_derivative(deviceFloat viscosity, size_t N_elements, Element_t* elements, const Face_t* faces, const deviceFloat* weights, const deviceFloat* derivative_matrices_hat, const deviceFloat* g_hat_derivative_matrices, const deviceFloat* lagrange_interpolant_left, const deviceFloat* lagrange_interpolant_right) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_elements; i += stride) {
        const size_t offset_1D = elements[i].N_ * (elements[i].N_ + 1) /2; // CHECK cache?
        const size_t offset_2D = elements[i].N_ * (elements[i].N_ + 1) * (2 * elements[i].N_ + 1) /6;

        const deviceFloat flux_L = faces[elements[i].faces_[0]].flux_;
        const deviceFloat flux_R = faces[elements[i].faces_[1]].flux_;

        SEM::matrix_vector_multiply(elements[i].N_, derivative_matrices_hat + offset_2D, elements[i].phi_, elements[i].q_);
        for (int j = 0; j <= elements[i].N_; ++j) {
            elements[i].q_[j] = -elements[i].q_[j] - (flux_R * lagrange_interpolant_right[offset_1D + j]
                                                     - flux_L * lagrange_interpolant_left[offset_1D + j]) / weights[offset_1D + j];
            elements[i].q_[j] *= 2.0f/elements[i].delta_x_;
        }
    }
}

// Algorithm 60 (not really anymore)
__global__
void SEM::compute_dg_derivative2(deviceFloat viscosity, size_t N_elements, Element_t* elements, const Face_t* faces, const deviceFloat* weights, const deviceFloat* derivative_matrices_hat, const deviceFloat* g_hat_derivative_matrices, const deviceFloat* lagrange_interpolant_left, const deviceFloat* lagrange_interpolant_right) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_elements; i += stride) {
        const size_t offset_1D = elements[i].N_ * (elements[i].N_ + 1) /2; // CHECK cache?
        const size_t offset_2D = elements[i].N_ * (elements[i].N_ + 1) * (2 * elements[i].N_ + 1) /6;

        const deviceFloat derivative_flux_L = faces[elements[i].faces_[0]].derivative_flux_;
        const deviceFloat derivative_flux_R = faces[elements[i].faces_[1]].derivative_flux_;
        const deviceFloat nl_flux_L = faces[elements[i].faces_[0]].nl_flux_;
        const deviceFloat nl_flux_R = faces[elements[i].faces_[1]].nl_flux_;
        
        SEM::matrix_vector_derivative(elements[i].N_, derivative_matrices_hat + offset_2D, elements[i].phi_, elements[i].ux_);
        SEM::matrix_vector_multiply(elements[i].N_, derivative_matrices_hat + offset_2D, elements[i].q_, elements[i].phi_prime_);
        
        for (int j = 0; j <= elements[i].N_; ++j) {
            elements[i].phi_prime_[j] = -elements[i].phi_prime_[j] * viscosity
                                        - (derivative_flux_R * lagrange_interpolant_right[offset_1D + j]
                                           - derivative_flux_L * lagrange_interpolant_left[offset_1D + j]) * viscosity /weights[offset_1D + j]
                                        - elements[i].ux_[j]
                                        + (nl_flux_L * lagrange_interpolant_left[offset_1D + j] 
                                            - nl_flux_R * lagrange_interpolant_right[offset_1D + j]) / weights[offset_1D + j];

            elements[i].phi_prime_[j] *= 2.0f/elements[i].delta_x_;
        }
    }
}
