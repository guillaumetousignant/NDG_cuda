#include "hip/hip_runtime.h"
#include "Mesh_t.cuh"
#include "ChebyshevPolynomial_t.cuh"
#include "LegendrePolynomial_t.cuh"
#include <iostream>
#include <fstream>
#include <sstream> 
#include <iomanip>
#include <filesystem>

namespace fs = std::filesystem;

constexpr int elements_blockSize = 32; // For when we'll have multiple elements
constexpr int faces_blockSize = 32; // Same number of faces as elements for periodic BC

Mesh_t::Mesh_t(int N_elements, int initial_N, float x_min, float x_max) : N_elements_(N_elements), N_faces_(N_elements), initial_N_(initial_N) {
    // CHECK N_faces = N_elements only for periodic BC.
    hipMalloc(&elements_, N_elements_ * sizeof(Element_t));
    hipMalloc(&faces_, N_faces_ * sizeof(Face_t));

    const int elements_numBlocks = (N_elements_ + elements_blockSize - 1) / elements_blockSize;
    const int faces_numBlocks = (N_faces_ + faces_blockSize - 1) / faces_blockSize;
    SEM::build_elements<<<elements_numBlocks, elements_blockSize>>>(N_elements_, initial_N_, elements_, x_min, x_max);
    SEM::build_faces<<<faces_numBlocks, faces_blockSize>>>(N_faces_, faces_); // CHECK
}

Mesh_t::~Mesh_t() {
    if (elements_ != nullptr){
        hipFree(elements_);
    }

    if (faces_ != nullptr){
        hipFree(faces_);
    }
}

void Mesh_t::set_initial_conditions(const float* nodes) {
    const int elements_numBlocks = (N_elements_ + elements_blockSize - 1) / elements_blockSize;
    SEM::initial_conditions<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, nodes);
}

void Mesh_t::print() {
    // CHECK find better solution for multiple elements. This only works if all elements have the same N.
    float* phi;
    float* phi_prime;
    float* host_phi = new float[(initial_N_ + 1) * N_elements_];
    float* host_phi_prime = new float[(initial_N_ + 1) * N_elements_];
    Face_t* host_faces = new Face_t[N_faces_];
    Element_t* host_elements = new Element_t[N_elements_];
    hipMalloc(&phi, (initial_N_ + 1) * N_elements_ * sizeof(float));
    hipMalloc(&phi_prime, (initial_N_ + 1) * N_elements_ * sizeof(float));

    const int elements_numBlocks = (N_elements_ + elements_blockSize - 1) / elements_blockSize;
    SEM::get_elements_data<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, phi, phi_prime);
    
    hipDeviceSynchronize();
    hipMemcpy(host_phi, phi, (initial_N_ + 1) * N_elements_ * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(host_phi_prime, phi_prime, (initial_N_ + 1) * N_elements_ * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(host_faces, faces_, N_faces_ * sizeof(Face_t), hipMemcpyDeviceToHost);
    hipMemcpy(host_elements, elements_, N_elements_ * sizeof(Element_t), hipMemcpyDeviceToHost);

    // Invalidate GPU pointers, or else they will be deleted on the CPU, where they point to random stuff
    for (int i = 0; i < N_elements_; ++i) {
        host_elements[i].phi_ = nullptr;
        host_elements[i].phi_prime_ = nullptr;
        host_elements[i].intermediate_ = nullptr;
    }

    std::cout << std::endl << "Phi: " << std::endl;
    for (int i = 0; i < N_elements_; ++i) {
        const int element_offset = i * (initial_N_ + 1);
        std::cout << '\t' << "Element " << i << ": ";
        std::cout << '\t' << '\t';
        for (int j = 0; j <= initial_N_; ++j) {
            std::cout << host_phi[element_offset + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << std::endl << "Phi prime: " << std::endl;
    for (int i = 0; i < N_elements_; ++i) {
        const int element_offset = i * (initial_N_ + 1);
        std::cout << '\t' << "Element " << i << ": ";
        std::cout << '\t' << '\t';
        for (int j = 0; j <= initial_N_; ++j) {
            std::cout << host_phi_prime[element_offset + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << std::endl << "Phi interpolated: " << std::endl;
    for (int i = 0; i < N_elements_; ++i) {
        std::cout << '\t' << "Element " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_elements[i].phi_L_ << " ";
        std::cout << host_elements[i].phi_R_;
        std::cout << std::endl;
    }

    std::cout << std::endl << "x: " << std::endl;
    for (int i = 0; i < N_elements_; ++i) {
        std::cout << '\t' << "Element " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_elements[i].x_[0] << " ";
        std::cout << host_elements[i].x_[1];
        std::cout << std::endl;
    }

    std::cout << std::endl << "Neighbouring elements: " << std::endl;
    for (int i = 0; i < N_elements_; ++i) {
        std::cout << '\t' << "Element " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_elements[i].neighbours_[0] << " ";
        std::cout << host_elements[i].neighbours_[1];
        std::cout << std::endl;
    }

    std::cout << std::endl << "Neighbouring faces: " << std::endl;
    for (int i = 0; i < N_elements_; ++i) {
        std::cout << '\t' << "Element " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_elements[i].faces_[0] << " ";
        std::cout << host_elements[i].faces_[1];
        std::cout << std::endl;
    }

    std::cout << std::endl << "N: " << std::endl;
    for (int i = 0; i < N_elements_; ++i) {
        std::cout << '\t' << "Element " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_elements[i].N_;
        std::cout << std::endl;
    }

    std::cout << std::endl << "delta x: " << std::endl;
    for (int i = 0; i < N_elements_; ++i) {
        std::cout << '\t' << "Element " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_elements[i].delta_x_;
        std::cout << std::endl;
    }

    std::cout << std::endl << "Fluxes: " << std::endl;
    for (int i = 0; i < N_faces_; ++i) {
        std::cout << '\t' << "Face " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_faces[i].flux_ << std::endl;
    }

    std::cout << std::endl << "Elements: " << std::endl;
    for (int i = 0; i < N_faces_; ++i) {
        std::cout << '\t' << "Face " << i << ": ";
        std::cout << '\t' << '\t';
        std::cout << host_faces[i].elements_[0] << " ";
        std::cout << host_faces[i].elements_[1] << std::endl;
    }

    delete[] host_phi;
    delete[] host_phi_prime;
    delete[] host_faces;
    delete[] host_elements;

    hipFree(phi);
    hipFree(phi_prime);
}

void Mesh_t::write_file_data(int N_points, float time, const float* velocity, const float* coordinates) {
    std::stringstream ss;
    std::ofstream file;

    fs::path save_dir = fs::current_path() / "data";
    fs::create_directory(save_dir);

    ss << "output_t" << std::setprecision(4) << std::fixed << time << ".dat";
    file.open(save_dir / ss.str());

    file << "TITLE = \"Velocity  at t= " << time << "\"" << std::endl;
    file << "VARIABLES = \"X\", \"U_x\"" << std::endl;
    file << "ZONE T= \"Zone     1\",  I= " << N_points << ",  J= 1,  DATAPACKING = POINT, SOLUTIONTIME = " << time << std::endl;

    for (int i = 0; i < N_points; ++i) {
        file << std::setw(12) << coordinates[i] << " " << std::setw(12) << velocity[i] << std::endl;
    }

    file.close();
}

void Mesh_t::write_data(float time, int N_interpolation_points, const float* interpolation_matrices) {
    // CHECK find better solution for multiple elements
    float* phi;
    float* x;
    float* host_phi = new float[N_elements_ * N_interpolation_points];
    float* host_x = new float[N_elements_ * N_interpolation_points];
    hipMalloc(&phi, N_elements_ * N_interpolation_points * sizeof(float));
    hipMalloc(&x, N_elements_ * N_interpolation_points * sizeof(float));

    const int elements_numBlocks = (N_elements_ + elements_blockSize - 1) / elements_blockSize;
    SEM::get_solution<<<elements_numBlocks, elements_blockSize>>>(N_elements_, N_interpolation_points, elements_, interpolation_matrices, phi, x);
    
    hipDeviceSynchronize();
    hipMemcpy(host_phi, phi, N_elements_ * N_interpolation_points * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(host_x, x , N_elements_ * N_interpolation_points * sizeof(float), hipMemcpyDeviceToHost);

    write_file_data(N_elements_ * N_interpolation_points, time, host_phi, host_x);

    delete[] host_phi;
    delete[] host_x;
    hipFree(phi);
    hipFree(x);
}

template void Mesh_t::solve(const float delta_t, const std::vector<float> output_times, const NDG_t<ChebyshevPolynomial_t> &NDG); // Get with the times c++, it's crazy I have to do this
template void Mesh_t::solve(const float delta_t, const std::vector<float> output_times, const NDG_t<LegendrePolynomial_t> &NDG);

template<typename Polynomial>
void Mesh_t::solve(const float delta_t, const std::vector<float> output_times, const NDG_t<Polynomial> &NDG) {
    const int elements_numBlocks = (N_elements_ + elements_blockSize - 1) / elements_blockSize;
    const int faces_numBlocks = (N_faces_ + faces_blockSize - 1) / faces_blockSize;
    float time = 0.0;
    const float t_end = output_times.back();

    write_data(time, NDG.N_interpolation_points_, NDG.interpolation_matrices_);

    while (time < t_end) {
        // Kinda algorithm 62
        float t = time;
        SEM::interpolate_to_boundaries<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        SEM::calculate_fluxes<<<faces_numBlocks, faces_blockSize>>>(N_faces_, faces_, elements_);
        SEM::compute_dg_derivative<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, faces_, NDG.weights_, NDG.derivative_matrices_hat_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        SEM::rk3_step<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, delta_t, 0.0f, 1.0f/3.0f);

        t = time + 0.33333333333f * delta_t;
        SEM::interpolate_to_boundaries<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        SEM::calculate_fluxes<<<faces_numBlocks, faces_blockSize>>>(N_faces_, faces_, elements_);
        SEM::compute_dg_derivative<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, faces_, NDG.weights_, NDG.derivative_matrices_hat_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        SEM::rk3_step<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, delta_t, -5.0f/9.0f, 15.0f/16.0f);

        t = time + 0.75f * delta_t;
        SEM::interpolate_to_boundaries<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        SEM::calculate_fluxes<<<faces_numBlocks, faces_blockSize>>>(N_faces_, faces_, elements_);
        SEM::compute_dg_derivative<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, faces_, NDG.weights_, NDG.derivative_matrices_hat_, NDG.lagrange_interpolant_left_, NDG.lagrange_interpolant_right_);
        SEM::rk3_step<<<elements_numBlocks, elements_blockSize>>>(N_elements_, elements_, delta_t, -153.0f/128.0f, 8.0f/15.0f);
              
        time += delta_t;
        for (auto const& e : std::as_const(output_times)) {
            if ((time >= e) && (time < e + delta_t)) {
                write_data(time, NDG.N_interpolation_points_, NDG.interpolation_matrices_);
                break;
            }
        }
    }

    bool did_write = false;
    for (auto const& e : std::as_const(output_times)) {
        if ((time >= e) && (time < e + delta_t)) {
            did_write = true;
            break;
        }
    }

    if (!did_write) {
        write_data(time, NDG.N_interpolation_points_, NDG.interpolation_matrices_);
    }
}

__global__
void SEM::rk3_step(int N_elements, Element_t* elements, float delta_t, float a, float g) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N_elements; i += stride) {
        for (int j = 0; j <= elements[i].N_; ++j){
            elements[i].intermediate_[j] = a * elements[i].intermediate_[j] + elements[i].phi_prime_[j];
            elements[i].phi_[j] += g * delta_t * elements[i].intermediate_[j];
        }
    }
}

__global__
void SEM::calculate_fluxes(int N_faces, Face_t* faces, const Element_t* elements) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N_faces; i += stride) {
        float u;
        const float u_left = elements[faces[i].elements_[0]].phi_R_;
        const float u_right = elements[faces[i].elements_[1]].phi_L_;

        if (u_left < 0.0f && u_right > 0.0f) { // In expansion fan
            u = 0.5f * (u_left + u_right);
        }
        else if (u_left > u_right) { // Shock
            if (u_left > 0.0f) {
                u = u_left;
            }
            else if (u_left < u_right) {
                u = u_right;
            }
            else { // ADDED
                u = 0.5f * (u_left + u_right);
            }
        }
        else if (u_left < u_right) { // Expansion fan
            if (u_left > 0.0f) {
                u = u_left;
            }
            else if (u_left < 0.0f) {
                u = u_right;
            }
            else { // ADDED
                u = 0.5f * (u_left + u_right);
            }
        }
        else { // ADDED
            u = 0.5f * (u_left + u_right);
        }

        faces[i].flux_ = 0.5f * u * u;
    }
}

// Algorithm 19
__device__
void SEM::matrix_vector_derivative(int N, const float* derivative_matrices_hat, const float* phi, float* phi_prime) {
    // s = 0, e = N (p.55 says N - 1)
    const int offset_2D = N * (N + 1) * (2 * N + 1) /6;

    for (int i = 0; i <= N; ++i) {
        phi_prime[i] = 0.0f;
        for (int j = 0; j <= N; ++j) {
            phi_prime[i] += derivative_matrices_hat[offset_2D + i * (N + 1) + j] * phi[j] * phi[j] * 0.5f; // phi not squared in textbook, squared for Burger's
        }
    }
}

// Algorithm 60 (not really anymore)
__global__
void SEM::compute_dg_derivative(int N_elements, Element_t* elements, const Face_t* faces, const float* weights, const float* derivative_matrices_hat, const float* lagrange_interpolant_left, const float* lagrange_interpolant_right) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N_elements; i += stride) {
        const int offset_1D = elements[i].N_ * (elements[i].N_ + 1) /2; // CHECK cache?

        const float flux_L = faces[elements[i].faces_[0]].flux_;
        const float flux_R = faces[elements[i].faces_[1]].flux_;

        SEM::matrix_vector_derivative(elements[i].N_, derivative_matrices_hat, elements[i].phi_, elements[i].phi_prime_);

        for (int j = 0; j <= elements[i].N_; ++j) {
            elements[i].phi_prime_[j] += (flux_L * lagrange_interpolant_left[offset_1D + j] - flux_R * lagrange_interpolant_right[offset_1D + j]) / weights[offset_1D + j];
            elements[i].phi_prime_[j] *= 2.0f/elements[i].delta_x_;
        }
    }
}