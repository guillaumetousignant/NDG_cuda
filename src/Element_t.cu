#include "hip/hip_runtime.h"
#include "Element_t.cuh"
#include "ChebyshevPolynomial_t.cuh"
#include "LegendrePolynomial_t.cuh"
#include <cmath>
#include <thrust/swap.h>

constexpr deviceFloat pi = 3.14159265358979323846;

__device__ 
SEM::Element_t::Element_t(int N, size_t face_L, size_t face_R, deviceFloat x_L, deviceFloat x_R) : 
        N_(N),
        faces_{face_L, face_R},
        x_{x_L, x_R},
        delta_x_(x_[1] - x_[0]),
        phi_(new deviceFloat[N_ + 1]),
        phi_prime_(new deviceFloat[N_ + 1]),
        intermediate_(new deviceFloat[N_ + 1]),
        sigma_(0.0),
        refine_(false),
        coarsen_(false),
        error_(0.0) {}

__device__
SEM::Element_t::Element_t(const SEM::Element_t& other) :
        N_(other.N_),
        faces_{other.faces_[0], other.faces_[1]},
        x_{other.x_[0], other.x_[1]},
        delta_x_(other.delta_x_),
        phi_(new deviceFloat[N_ + 1]),
        phi_prime_(new deviceFloat[N_ + 1]),
        intermediate_(new deviceFloat[N_ + 1]),
        sigma_(other.sigma_),
        refine_(other.refine_),
        coarsen_(other.coarsen_),
        error_(other.error_) {

    for (int i = 0; i <= N_; ++i) {
        phi_[i] = other.phi_[i];
        phi_prime_[i] = other.phi_prime_[i];
        intermediate_[i] = other.intermediate_[i];
    }
}

__device__
SEM::Element_t::Element_t(SEM::Element_t&& other) :
        N_(other.N_),
        faces_{other.faces_[0], other.faces_[1]},
        x_{other.x_[0], other.x_[1]},
        delta_x_(other.delta_x_),
        phi_(other.phi_),
        phi_prime_(other.phi_prime_),
        intermediate_(other.intermediate_),
        sigma_(other.sigma_),
        refine_(other.refine_),
        coarsen_(other.coarsen_),
        error_(other.error_) {
    
    other.phi_ = nullptr;
    other.phi_prime_ = nullptr;
    other.intermediate_ = nullptr;
}

__device__
SEM::Element_t& SEM::Element_t::operator=(const SEM::Element_t& other) {
    if (N_ != other.N_) {
        delete[] phi_;
        delete[] phi_prime_;
        delete[] intermediate_;

        N_ = other.N_;
        phi_ = new deviceFloat[N_];
        phi_prime_ = new deviceFloat[N_];
        intermediate_ = new deviceFloat[N_];
    }

    faces_[0] = other.faces_[0];
    faces_[1] = other.faces_[1];
    x_[0] = other.x_[0];
    x_[1] = other.x_[1];
    delta_x_ = other.delta_x_;
    sigma_ = other.sigma_;
    refine_ = other.refine_;
    coarsen_ = other.coarsen_;
    error_ = other.error_;

    for (int i = 0; i <= N_; ++i) {
        phi_[i] = other.phi_[i];
        phi_prime_[i] = other.phi_prime_[i];
        intermediate_[i] = other.intermediate_[i];
    }
    
    return *this;
}

__device__
SEM::Element_t& SEM::Element_t::operator=(SEM::Element_t&& other) {
    N_ = other.N_;
    faces_[0] = other.faces_[0];
    faces_[1] = other.faces_[1];
    x_[0] = other.x_[0];
    x_[1] = other.x_[1];
    delta_x_ = other.delta_x_;
    sigma_ = other.sigma_;
    refine_ = other.refine_;
    coarsen_ = other.coarsen_;
    error_ = other.error_;

    thrust::swap(phi_, other.phi_);
    thrust::swap(phi_prime_, other.phi_prime_);
    thrust::swap(intermediate_, other.intermediate_);
    
    return *this;
}

__host__ __device__
SEM::Element_t::Element_t() :
        N_(0),
        faces_{0, 0},
        x_{0.0, 0.0},
        delta_x_(0.0),
        phi_(nullptr),
        phi_prime_(nullptr),
        intermediate_(nullptr),
        sigma_(0.0),
        refine_(false),
        coarsen_(false),
        error_(0.0) {};

__host__ __device__
SEM::Element_t::~Element_t() {
    delete [] phi_;
    delete [] phi_prime_;
    delete [] intermediate_;
}

// Algorithm 61
__device__
void SEM::Element_t::interpolate_to_boundaries(const deviceFloat* lagrange_interpolant_left, const deviceFloat* lagrange_interpolant_right, const deviceFloat* lagrange_interpolant_derivative_left, const deviceFloat* lagrange_interpolant_derivative_right) {
    const int offset_1D = N_ * (N_ + 1) /2;
    phi_L_ = 0.0;
    phi_R_ = 0.0;
    phi_prime_L_ = 0.0;
    phi_prime_R_ = 0.0;

    for (int j = 0; j <= N_; ++j) {
        phi_L_ += lagrange_interpolant_left[offset_1D + j] * phi_[j];
        phi_R_ += lagrange_interpolant_right[offset_1D + j] * phi_[j];
    }

    for (int j = 0; j <= N_; ++j) {
        phi_prime_L_ += lagrange_interpolant_derivative_left[offset_1D + j] * (phi_L_ - phi_[j]);
        phi_prime_R_ += lagrange_interpolant_derivative_right[offset_1D + j] * (phi_R_ - phi_[j]);
    }
}

template __device__ void SEM::Element_t::estimate_error<SEM::ChebyshevPolynomial_t>(const deviceFloat* nodes, const deviceFloat* weights);
template __device__ void SEM::Element_t::estimate_error<SEM::LegendrePolynomial_t>(const deviceFloat* nodes, const deviceFloat* weights);

template<typename Polynomial>
__device__
void SEM::Element_t::estimate_error<Polynomial>(const deviceFloat* nodes, const deviceFloat* weights) {
    const int offset_1D = N_ * (N_ + 1) /2;

    for (int k = 0; k <= N_; ++k) {
        intermediate_[k] = 0.0;
        for (int i = 0; i <= N_; ++i) {
            deviceFloat L_N;
            Polynomial::polynomial(k, nodes[offset_1D + i], L_N);

            intermediate_[k] += (2 * k + 1) * 0.5 * phi_[i] * L_N * weights[offset_1D + i];
        }
        intermediate_[k] = std::abs(intermediate_[k]);
    }

    constexpr deviceFloat tolerance_min = 1e25;     // Refine above this
    constexpr deviceFloat tolerance_max = 1e-25;    // Coarsen below this

    const deviceFloat C = exponential_decay();

    // sum of error
    error_ = std::sqrt(C * C * 0.5/sigma_) * std::exp(-sigma_ * (N_ + 1));

    if(error_ > tolerance_min) {	// need refine
        refine_ = true;
        coarsen_ = false;
    }
    else if(error_ <= tolerance_max ) {	// need coarsen
        refine_ = false;
        coarsen_ = true;
    }
    else {	// if error in between then do nothing
        refine_ = false;
        coarsen_ = false;
    }
}

__device__
deviceFloat SEM::Element_t::exponential_decay() {
    const int n_points_least_squares = min(N_, 4); // Number of points to use for thew least squares reduction, but don't go above N.

    deviceFloat x_avg = 0.0;
    deviceFloat y_avg = 0.0;

    for (int i = 0; i < n_points_least_squares; ++i) {
        x_avg += N_ - i;
        y_avg += std::log(intermediate_[N_ - i]);
    }

    x_avg /= n_points_least_squares;
    y_avg /= n_points_least_squares;

    deviceFloat numerator = 0.0;
    deviceFloat denominator = 0.0;

    for (int i = 0; i < n_points_least_squares; ++i) {
        numerator += (N_ - i - x_avg) * (std::log(intermediate_[N_ - i]) - y_avg);
        denominator += (N_ - i - x_avg) * (N_ - i - x_avg);
    }

    sigma_ = numerator/denominator;

    const deviceFloat C = std::exp(y_avg - sigma_ * x_avg);
    sigma_ = std::abs(sigma_);
    return C;
}

__device__
void SEM::Element_t::interpolate_from(const SEM::Element_t& other, const deviceFloat* nodes, const deviceFloat* barycentric_weights) {
    const int offset = N_ * (N_ + 1) /2;
    const int offset_other = other.N_ * (other.N_ + 1) /2;

    for (int i = 0; i <= N_; ++i) {
        const deviceFloat x = (x_[1] - x_[0]) * (nodes[offset + i] + 1) * 0.5 + x_[0];
        const deviceFloat node = (2 * x - other.x_[0] - other.x_[1])/(other.x_[1] - other.x_[0]);
        deviceFloat numerator = 0.0;
        deviceFloat denominator = 0.0;
        for (int j = 0; j <= other.N_; ++j) {
            if (SEM::almost_equal2(node, nodes[offset_other + j])) {
                numerator = other.phi_[j];
                denominator = 1.0;
                break;
            }
            const deviceFloat t = barycentric_weights[offset_other + j]/(node - nodes[offset_other + j]);
            numerator += t * other.phi_[j];
            denominator += t;
        }
        phi_[i] = numerator/denominator;
    }
}

__global__
void SEM::build_elements(size_t N_elements, int N, SEM::Element_t* elements, deviceFloat x_min, deviceFloat x_max) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N_elements; i += stride) {
        const size_t face_L = i;
        const size_t face_R = i + 1;
        const deviceFloat delta_x = (x_max - x_min)/N_elements;
        const deviceFloat element_x_min = x_min + i * delta_x;
        const deviceFloat element_x_max = x_min + (i + 1) * delta_x;

        // Those are uninitialised because they are created via hipMalloc, so they need to be set if we don't want the move constructor to delete random memory.
        elements[i].phi_ = nullptr;
        elements[i].phi_prime_ = nullptr;
        elements[i].intermediate_ = nullptr;

        elements[i] = SEM::Element_t(N, face_L, face_R, element_x_min, element_x_max);
    }
}

__global__
void SEM::build_boundaries(size_t N_elements, size_t N_elements_global, size_t N_local_boundaries, size_t N_MPI_boundaries, Element_t* elements, deviceFloat x_min, deviceFloat x_max, size_t global_element_offset, size_t* local_boundary_to_element, size_t* MPI_boundary_to_element, size_t* MPI_boundary_from_element) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N_local_boundaries; i += stride) {
        const deviceFloat delta_x = (x_max - x_min)/N_elements;
        size_t face_L;
        size_t face_R;
        deviceFloat element_x_min;
        deviceFloat element_x_max;

        if (i == 0) { // CHECK this is hardcoded for 1D
            face_L = 0;
            face_R = 0;
            element_x_min = x_min - delta_x;
            element_x_max = x_min;
            local_boundary_to_element[i] = N_elements - 1;
        }
        else if (i == 1) {
            face_L = N_elements + N_local_boundaries + N_MPI_boundaries - 2;
            face_R = N_elements + N_local_boundaries + N_MPI_boundaries - 2;
            element_x_min = x_max;
            element_x_max = x_max + delta_x;
            local_boundary_to_element[i] = 0;
        }

        // Those are uninitialised because they are created via hipMalloc, so they need to be set if we don't want the move constructor to delete random memory.
        elements[N_elements + i].phi_ = nullptr;
        elements[N_elements + i].phi_prime_ = nullptr;
        elements[N_elements + i].intermediate_ = nullptr;

        elements[N_elements + i] = SEM::Element_t(-1, face_L, face_R, element_x_min, element_x_max);
    }

    for (int i = index; i < N_MPI_boundaries; i += stride) {
        const deviceFloat delta_x = (x_max - x_min)/N_elements;
        size_t face_L;
        size_t face_R;
        deviceFloat element_x_min;
        deviceFloat element_x_max;

        if (i == 0) { // CHECK this is hardcoded for 1D
            face_L = 0;
            face_R = 0;
            element_x_min = x_min - delta_x;
            element_x_max = x_min;
            MPI_boundary_to_element[i] = (global_element_offset == 0) ? N_elements_global - 1 : global_element_offset - 1;
            MPI_boundary_from_element[i] = global_element_offset;
        }
        else if (i == 1) {
            face_L = N_elements + N_local_boundaries + N_MPI_boundaries - 2;
            face_R = N_elements + N_local_boundaries + N_MPI_boundaries - 2;
            element_x_min = x_max;
            element_x_max = x_max + delta_x;
            MPI_boundary_to_element[i] = (global_element_offset + N_elements == N_elements_global) ? 0 : global_element_offset + N_elements;
            MPI_boundary_from_element[i] = global_element_offset + N_elements - 1;
        }

        // Those are uninitialised because they are created via hipMalloc, so they need to be set if we don't want the move constructor to delete random memory.
        elements[N_elements + N_local_boundaries + i].phi_ = nullptr;
        elements[N_elements + N_local_boundaries + i].phi_prime_ = nullptr;
        elements[N_elements + N_local_boundaries + i].intermediate_ = nullptr;

        elements[N_elements + N_local_boundaries + i] = SEM::Element_t(-1, face_L, face_R, element_x_min, element_x_max);
    }
}

__global__
void SEM::free_elements(size_t N_elements, SEM::Element_t* elements) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N_elements; i += stride) {
        delete[] elements[i].phi_;
        delete[] elements[i].phi_prime_;
        delete[] elements[i].intermediate_;
        elements[i].phi_ = nullptr;
        elements[i].phi_prime_ = nullptr;
        elements[i].intermediate_ = nullptr;
    }
}

template __global__ void SEM::estimate_error<SEM::ChebyshevPolynomial_t>(size_t N_elements, SEM::Element_t* elements, const deviceFloat* nodes, const deviceFloat* weights);
template __global__ void SEM::estimate_error<SEM::LegendrePolynomial_t>(size_t N_elements, SEM::Element_t* elements, const deviceFloat* nodes, const deviceFloat* weights);

template<typename Polynomial>
__global__
void SEM::estimate_error<Polynomial>(size_t N_elements, SEM::Element_t* elements, const deviceFloat* nodes, const deviceFloat* weights) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_elements; i += stride) {
        elements[i].estimate_error<Polynomial>(nodes, weights);
    }
}

__device__
deviceFloat SEM::g(deviceFloat x) {
    //return (x < -0.2f || x > 0.2f) ? 0.2f : 0.8f;
    return -std::sin(pi * x);
}


__global__
void SEM::initial_conditions(size_t N_elements, SEM::Element_t* elements, const deviceFloat* nodes) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_elements; i += stride) {
        const size_t offset = elements[i].N_ * (elements[i].N_ + 1) /2;
        for (int j = 0; j <= elements[i].N_; ++j) {
            const deviceFloat x = (0.5 + nodes[offset + j]/2.0f) * (elements[i].x_[1] - elements[i].x_[0]) + elements[i].x_[0];
            elements[i].phi_[j] = SEM::g(x);
        }
    }
}

// Basically useless, find better solution when multiple elements.
__global__
void SEM::get_elements_data(size_t N_elements, const SEM::Element_t* elements, deviceFloat* phi, deviceFloat* phi_prime) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_elements; i += stride) {
        const size_t element_offset = i * (elements[i].N_ + 1);
        for (int j = 0; j <= elements[i].N_; ++j) {
            phi[element_offset + j] = elements[i].phi_[j];
            phi_prime[element_offset + j] = elements[i].phi_prime_[j];
        }
    }
}

// Basically useless, find better solution when multiple elements.
__global__
void SEM::get_phi(size_t N_elements, const SEM::Element_t* elements, deviceFloat* phi) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_elements; i += stride) {
        for (int j = 0; j <= elements[i].N_; ++j) {
            phi[j] = elements[i].phi_[j];
        }
    }
}

__global__
void SEM::get_solution(size_t N_elements, size_t N_interpolation_points, const SEM::Element_t* elements, const deviceFloat* interpolation_matrices, deviceFloat* x, deviceFloat* phi, deviceFloat* phi_prime, deviceFloat* intermediate, deviceFloat* x_L, deviceFloat* x_R, int* N, deviceFloat* sigma, bool* refine, bool* coarsen, deviceFloat* error) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_elements; i += stride) {
        const size_t offset_interp_1D = i * N_interpolation_points;
        const size_t offset_interp = elements[i].N_ * (elements[i].N_ + 1) * N_interpolation_points/2;
        const size_t step = N_interpolation_points/(elements[i].N_ + 1);

        for (size_t j = 0; j < N_interpolation_points; ++j) {
            phi[offset_interp_1D + j] = 0.0f;
            phi_prime[offset_interp_1D + j] = 0.0f;
            for (int k = 0; k <= elements[i].N_; ++k) {
                phi[offset_interp_1D + j] += interpolation_matrices[offset_interp + j * (elements[i].N_ + 1) + k] * elements[i].phi_[k];
                phi_prime[offset_interp_1D + j] += interpolation_matrices[offset_interp + j * (elements[i].N_ + 1) + k] * elements[i].phi_prime_[k]; 
            }
            intermediate[offset_interp_1D + j] = elements[i].intermediate_[min(static_cast<int>(j/step), elements[i].N_)];
            x[offset_interp_1D + j] = j * (elements[i].x_[1] - elements[i].x_[0]) / (N_interpolation_points - 1) + elements[i].x_[0];
        }

        x_L[i] = elements[i].x_[0];
        x_R[i] = elements[i].x_[1];
        N[i] = elements[i].N_;
        sigma[i] = elements[i].sigma_;
        refine[i] = elements[i].refine_;
        coarsen[i] = elements[i].coarsen_;
        error[i] = elements[i].error_;
    }
}

__global__
void SEM::interpolate_to_boundaries(size_t N_elements, SEM::Element_t* elements, const deviceFloat* lagrange_interpolant_left, const deviceFloat* lagrange_interpolant_right, const deviceFloat* lagrange_interpolant_derivative_left, const deviceFloat* lagrange_interpolant_derivative_right) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_elements; i += stride) {
        elements[i].interpolate_to_boundaries(lagrange_interpolant_left, lagrange_interpolant_right, lagrange_interpolant_derivative_left, lagrange_interpolant_derivative_right);
    }
}

__global__
void SEM::adapt(unsigned long N_elements, SEM::Element_t* elements, SEM::Element_t* new_elements, SEM::Face_t* new_faces, const unsigned long* block_offsets, int N_max, const deviceFloat* nodes, const deviceFloat* barycentric_weights) {
    const unsigned long index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned long stride = blockDim.x * gridDim.x;
    const int thread_id = threadIdx.x;
    const int block_id = blockIdx.x;
    
    for (unsigned long i = index; i < N_elements; i += stride) {
        if (elements[i].refine_ && elements[i].sigma_ < 1.0) {
            unsigned long offset = 0;
            for (unsigned long j = i - thread_id; j < i; ++j) {
                offset += elements[j].refine_ * (elements[j].sigma_ < 1.0);
            }
            unsigned long new_index = N_elements + block_offsets[block_id] + offset;
            
            // Those are uninitialised because they are created via hipMalloc, so they need to be set if we don't want the move constructor to delete random memory.
            new_elements[i].phi_ = nullptr;
            new_elements[i].phi_prime_ = nullptr;
            new_elements[i].intermediate_ = nullptr;
            new_elements[new_index].phi_ = nullptr;
            new_elements[new_index].phi_prime_ = nullptr;
            new_elements[new_index].intermediate_ = nullptr;

            new_elements[i] = SEM::Element_t(elements[i].N_, elements[i].faces_[0], new_index, elements[i].x_[0], (elements[i].x_[0] + elements[i].x_[1]) * 0.5);
            new_elements[new_index] = SEM::Element_t(elements[i].N_, new_index, elements[i].faces_[1], (elements[i].x_[0] + elements[i].x_[1]) * 0.5, elements[i].x_[1]);
            new_elements[i].interpolate_from(elements[i], nodes, barycentric_weights);
            new_elements[new_index].interpolate_from(elements[i], nodes, barycentric_weights);
            
            new_faces[new_index] = SEM::Face_t(i, new_index);
            new_faces[elements[i].faces_[1]].elements_[0] = new_index;
        }
        else if (elements[i].refine_ && elements[i].N_ < N_max) {
            new_elements[i].phi_ = nullptr;
            new_elements[i].phi_prime_ = nullptr;
            new_elements[i].intermediate_ = nullptr;

            new_elements[i] = SEM::Element_t(min(elements[i].N_ + 2, N_max), elements[i].faces_[0], elements[i].faces_[1], elements[i].x_[0], elements[i].x_[1]);
            new_elements[i].interpolate_from(elements[i], nodes, barycentric_weights);
        }
        else {
            // Those are uninitialised because they are created via hipMalloc, so they need to be set if we don't want the move constructor to delete random memory.
            new_elements[i].phi_ = nullptr;
            new_elements[i].phi_prime_ = nullptr;
            new_elements[i].intermediate_ = nullptr;
            
            new_elements[i] = std::move(elements[i]);
        }
    }
}

__global__
void SEM::p_adapt(unsigned long N_elements, SEM::Element_t* elements, int N_max, const deviceFloat* nodes, const deviceFloat* barycentric_weights) {
    const unsigned long index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned long stride = blockDim.x * gridDim.x;
    
    for (unsigned long i = index; i < N_elements; i += stride) {
        if (elements[i].refine_ && elements[i].sigma_ >= 1.0 && elements[i].N_ < N_max) {
            SEM::Element_t new_element(min(elements[i].N_ + 2, N_max), elements[i].faces_[0], elements[i].faces_[1], elements[i].x_[0], elements[i].x_[1]);
            new_element.interpolate_from(elements[i], nodes, barycentric_weights);
            elements[i] = std::move(new_element);
        }
    }
}

// From cppreference.com
__device__
bool SEM::almost_equal2(deviceFloat x, deviceFloat y) {
    constexpr int ulp = 2; // ULP
    // the machine epsilon has to be scaled to the magnitude of the values used
    // and multiplied by the desired precision in ULPs (units in the last place)
    return std::abs(x-y) <= FLT_EPSILON * std::abs(x+y) * ulp // CHECK change this to double equivalent if using double instead of float
        // unless the result is subnormal
        || std::abs(x-y) < FLT_MIN; // CHECK change this to 64F if using double instead of float
}

__global__
void SEM::local_boundaries(size_t N_elements, size_t N_local_boundaries, Element_t* elements, const size_t* local_boundary_to_element) {
    const unsigned long index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned long stride = blockDim.x * gridDim.x;
    
    for (unsigned long i = index; i < N_local_boundaries; i += stride) {
        elements[N_elements + i].phi_L_ = elements[local_boundary_to_element[i]].phi_L_;
        elements[N_elements + i].phi_R_ = elements[local_boundary_to_element[i]].phi_R_;
        elements[N_elements + i].phi_prime_L_ = elements[local_boundary_to_element[i]].phi_prime_L_;
        elements[N_elements + i].phi_prime_R_ = elements[local_boundary_to_element[i]].phi_prime_R_;
    }
}

__global__
void SEM::get_MPI_boundaries(size_t N_elements, size_t N_local_boundaries, size_t N_MPI_boundaries, const Element_t* elements, const Face_t* faces, deviceFloat* phi_L, deviceFloat* phi_R, deviceFloat* phi_prime_L, deviceFloat* phi_prime_R) {
    const unsigned long index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned long stride = blockDim.x * gridDim.x;
    
    for (unsigned long i = index; i < N_MPI_boundaries; i += stride) {
        const Element_t& boundary_element = elements[N_elements + N_local_boundaries + i];
        const Face_t& boundary_face = faces[boundary_element.faces_[0]];
        const Element_t& domain_element = elements[boundary_face.elements_[boundary_face.elements_[0] == N_elements + N_local_boundaries + i]];
        phi_L[i] = domain_element.phi_L_;
        phi_R[i] = domain_element.phi_R_;
        phi_prime_L[i] = domain_element.phi_prime_L_;
        phi_prime_R[i] = domain_element.phi_prime_R_;
    }
}

__global__
void SEM::put_MPI_boundaries(size_t N_elements, size_t N_local_boundaries, size_t N_MPI_boundaries, Element_t* elements, const deviceFloat* phi_L, const deviceFloat* phi_R, const deviceFloat* phi_prime_L, const deviceFloat* phi_prime_R) {
    const unsigned long index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned long stride = blockDim.x * gridDim.x;
    
    for (unsigned long i = index; i < N_MPI_boundaries; i += stride) {
        elements[N_elements + N_local_boundaries + i].phi_L_ = phi_L[i];
        elements[N_elements + N_local_boundaries + i].phi_R_ = phi_R[i];
        elements[N_elements + N_local_boundaries + i].phi_prime_L_ = phi_prime_L[i];
        elements[N_elements + N_local_boundaries + i].phi_prime_R_ = phi_prime_R[i];
    }
}
